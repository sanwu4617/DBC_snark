#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <random>
#include "gmp.h"
#include "gpuec256.h"
#include "openssl/sha.h"
#include "cuda_common.h"
#include "sha256.cuh"
#include <sys/time.h>

#define N_DATA (1024)
#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}
#define TIME_TEST

mpz_t n;
const UINT64 h_Gx[4]={0x59F2815B16F81798L,0x029BFCDB2DCE28D9L,0x55A06295CE870B07L,0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4]={0x9C47D08FFB10D4B8L,0xFD17B448A6855419L,0x5DA4FBFC0E1108A8L,0x483ADA7726A3C465L};
const UINT64 h_Gz[4]={0x1L,0x0L,0x0L,0x0L};

const UINT64 h_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};
// const UINT64 h_R2modN[4]={0x896cf21467d7d140L,0x741496c20e7cf878L,0xe697f5e45bcd07c6L,0x9d671cd581c69bc5L};




char Vx[] =     "4e37ee0ff806bc2a90adb4a9fbc2bcac4853e688f96074c27d4f8504067bb821" ;
char Vy[] =     "35d0be4c081d5886d3b537be233a0523a03e065281f0f6fb7824d7d3407428cb" ;

char Ax[] =     "ccd393ca0432f633be28af8f9418e1b7a6c04a561470ad31eedae90014213b2c" ;
char Ay[] =     "777eb3cc1e68c7810c299273e98f0843975fe13d3666666f2d82c3195ccb282d" ;

char Sx[] =     "b978a8f312d3dc589ea2ec1a13b6297ce143977d0580abd4fdf3e3f0ac757bd6" ;
char Sy[] =     "fccf48f6190ff1ace5627770148e0ef4775f55995bd2beb9e34e4f3e06462953" ;

char T1x[] =    "77afdb45531e1f777e7d82c82697085f025f54d96469a23981689cb9bc69d30c" ;
char T1y[] =    "e34b9d20e2ae413f112816452def92605bee2ddeee5be074f054665eda44dba1" ;

char T2x[] =    "c7dc07138ebdc0192a14d8aacc2f8463aa1fcf7bcbe65c526e71ed7ebca46d88" ;
char T2y[] =    "261cca6c0a4fd2158b65ab8974959ae64332f3cb2a049c1df3388ca8a066e963" ;

char Taux[] =   "efe10e0a6a89d6e46ec64f85734cee91acd094295b9dabb6424ad826e37d65ce" ;
char Tprime[] = "2c05837d7bcdfcd3e82ff90e444512966ba22009ce6a2f119e45c739035af098" ;
char Mu[] =     "e8e4a791ba0a5edeca104fae41b6273a3edee5a0a07f868506f3c8c4fa7c5588" ;


char ipCommitx[] = "1bd12210b9eb1f0ad612157f5fb7f72ca52ec1267c66f39b252926434aff6b8f";
char ipCommity[] = "b550cb1997688b1db545b80b12489b9618872ba47797b0e5e9f849b0a50762fc";

char ipA[] = "24593a5ee28c5581536637414bfebec9d25df0423047b38cc019382b142b8f21";
char ipB[] = "6ac01e46063d4e5b4577e85127fa7bdec2b308eea7df253403561129ab9c477c";

char ipUx[] = "d77df5c1a44024ad4388a28096ec6a2ef7c9dc00d29aeba38d6f1865307b6013";
char ipUy[] = "abf3f071be36929429d48dd92bd7d885c9506ceb01fb8779f37e5a3a51bc2d9e";

char ippramPx[] = "ed0f725abe71e10cab9ed861cfa40529e3a97e4da3a4b05325376d8433fe6788";
char ippramPy[] = "62db7f7a5b12c8f1dcdf7a124c92eb7b7b1228420aaf407ce3da33608c68f313";

char ipProvex[] = "ba56003e29bc5aa060964ab93388a2d4c377d18dd4fbe2efba5979c789fdcfab";
char ipProvey[] = "f0e405322e2a5286ef619a89913ce1ea395551585c815eb4785716adb7892d94";

char ipLsRs[][65] ={
    "71d1601a4bb8cee3e75b64b75029c10994ac3c8a7f6905c4c1b6c8265fb62d72",
    "a261c85c27389f72a6f6000d799d04b700a807df4d43e5878c1d27a449fbe8ad",
    "ba8ce28517d3e5ca7343b9405c511595eec77f9792ae3524a586310cd8b38a1b",
    "d94658ecb9b19636ad2d2e84a60c2a5700d95af483ec55314c805f893c23af03",
    "f0ac8b23fa9e54e638a1ee92ab610a1d35a34c6dc7fa5bae3283c46fb9601e79",
    "25c5adacdcfab9c10804e035b4e045103a75587329f7e64fc7bc7cdcb72e720b",
    "657669461f195d18030e5fa6407c0d9721d54624ff6a68eb5dd70c786161d3e9",
    "5b3c0d719647b7a04fd77daa98bba1bc69b26c3e3b2f7f548ed1d3d3aaa8fa07",
    "2064cbc107bc3e6814dfb0564b560c543140c90abf120d5925d747013fa774d2",
    "8e8f096b264b70105ed867c5fbe0751e8550a78a1e768342eefa750226ed9c60",
    "6550c8ffeb8dc7d47bf2dac0f1e7abd77338d0f6648c85bd0e349425eb5f7b26",
    "9298c66f731f550af9cd296ae541f4aab3d3ef378f82a4dc971a3130c947ae02",
    "7384ce47988daa2e14b603fddd404ef78a41fe0be027f52d02d9053c86636a23",
    "24e9c96f60757dd9dfe9cf64b7a414674afda445025e60c64ffee66799270250",
    "b2bbbe121ae465507e87081c5bf023e00d008ba8b9935a8b6247db115aff03dd",
    "f6b16da6046bf26a297ff754822344388ecd006321c49a00aaa6d78aab38029c",
    "4dd04e33fe49bca8076deb7d814da16494f100e6e00261a905e976ffa3e5a75b",
    "6c4fd632f33eaf4a7cba515897f75e61f2f8fa2adf414bda9085c9cc2cb8d37b",
    "182609eefde301e907150754ce5dd326f967e822e610798303d65223eb02cd2b",
    "f42de008ab86d51c052c8d88a8be2327d132a786220357ae111c7c8317ebb286"
};

char BL[32][65] ={
    "f1ff061b2e1945b6a9336170508f4012119f19c48a38ced70cd8d54b3290ad3f",
    "b5b5f5a7a71b6d809d8712c2ff56da24df49975cf0c1d5e0646d243715e38695",
    "a6c74ece1bf58c5f3aed7e51474271c5f380e36c0b6a3eeab57878aa39fc8b6",
    "a71a69da2b930ae1b1c16b72faa3fb4e415f1631d18523753dd9627f1eb10ee3",
    "c0ec984ab4c2b3b6b4d9e50b51dff2de225f7f19122403b4e63b958da931c739",
    "96d0c6fdc9eadc373fa1cfa0ea8a6311a3897615a99713fd990a78f419263701",
    "6b2e7b1fd39e0a58ee996da1c9d2287ff1ad86551e2726ad477c965c550247c",
    "f610efd48903cb4f30f3b217cda1045da05046fadcdae5c7d7b65e210d5323d9",
    "296db58f81293713628653a59ea3a74dafce9edde965739eace2db7456380afa",
    "6f987eb53e1c24da036e9358c40910f73845be58a97efa2298fc5102cf7f9b45",
    "25f64c6d215f61eae535a2f5e23c6f59c671ecc87f8be39215252cdb6dd4ef5",
    "daf86b0c7c39ccbb3b145647a07071cde86ed6fa75256ef35db636ddcf52936e",
    "dac3892a935c75c1a42b8e336c60588f12324a754745360069a63dd848efa92",
    "31c1561f5299c144350de78d9b7198d36ab597d719001a60b9208796cc11824e",
    "7f4a97adebf4ad2399279ee525a07f0d7be16644a464e8e6aabe06b1b1c7e970",
    "27ff06127072cd41d205c1e08da564b3380670f7ef2c4376d3f8441d1e8c4b35",
    "24efb2cbee5a5d6c21132923adaa90bafa73cc70c6d04b962197e5a8f13dcda6",
    "a2fa5e1a7a62f0a96a63af4960b3258ad199c9c5da2a376f9be18a32690ad5fc",
    "cab9a2edaf106ccd731eb1dbc682fdf5fb729bf1bbdc4370f5573a002866eb00",
    "8ea2ddc69638b969fb1cf2172970155aa51b7237f6cc24e7c4530294ce350ad4",
    "2566ce58589c97c4635ea6c6824b37a654e9d8e3da61b32b4e3e8691198b9f9b",
    "f7cf09f81683d71eb1d027f1667f887e8fe03f50f6474cbcb401f3edeacfa1d9",
    "3601e02bb005dfa3c1c80e501229554264c9db97af18904fc1e93abf29f6f1e9",
    "6b60ef35c86887e6a5d5be3f863728bd794594445d72c6c9d1905d3315d8d592",
    "337e0df7c9567c18ff873e0ebeb3b442ed521e81ea38727c05fde6fe794d881b",
    "c5578548b7b3afdd7fbe480e4e427b948cb6547fc07c685dec18bdb0ade4a76e",
    "ec02c55789de1dfd9939e94e51552f089d02ade587835fb58ad53c941f2964ba",
    "f2a5a507ad656499515fe4bc45360dcf0f4b7e1d6f00683b6cf7b32a739e4061",
    "86d247a0f5e847bae9baa4d0f173dd5a25a0d25cb5c91772513d30d1722fa5da",
    "8678f9f06068bf2dfd8ee8015f62aa0d30bfb5cb311e6b0245d93e94b4382dbb",
    "adfe0f06f3bb4d420e2501577f829eab8b6cf342af526fce37ee6d28d39e2002",
    "bc7716e40d651b4764125bbfeb8975f4386a836dc49d95e66b9576db217f22b5"
};
char BR[32][65] ={
    "b76f4d3278cfa4da06b5ca64b80b3762981e7ab1f60bf6540f8f7da23382213a",
    "d9074b02449df795bd83414726abce8a00ef74a01c34244749580074d7b19937",
    "247cc73bd5d1c1045864192af37d4b1c06d05bb9449bc883033ac97459bdaab9",
    "d188fcc2ad8c36da679302cf74363a4d1bfc5d6d0e5b8da2bdc69971d30998f9",
    "57aece4bf7b0f50ac7158a5287415bf1f78ab8735585a9cbf0a7d87284173454",
    "cb8ed282a12f8eaad5ab10e855f2eac75bae80cc130bea00d2b7680ec1966f51",
    "3272267027ef2f0872932db583b1a1510d8c24044ead47265fc9576b131b8dfd",
    "6ae31da34585809dc6b57181a28df38af88a44f51fddea01745555899c03c564",
    "12ceceba85721f3866e448ed3eacc4a474cd00296716cb9cf89b3f06f98daf32",
    "7a4404c0b601dafa95ff5b66cd41cbc19e6f09af6fe5fa18712f9822a2a6efc4",
    "5351ed1521e3946df2a1817ffab799e506f64431d641931e88478a74f6b55e06",
    "101d26aefbc67129895aa17dc59f2b1480f1b2e19938f84c6778b61b0399c1d1",
    "129c3d2b4018f228536c5b01249345037ab8910342572f26311b3caa309a42d1",
    "809af26522029703353ba23ab10628ff5c08e767eb37b1e0941d6fde1e31756c",
    "ceaa2788d83564b163999e5ca2fe33472eafc80abc8578ca093f23b0e9bedf47",
    "eeb680e0e918e3a321c96c25e4b185a02c6bc4ac50db154740a22a54d7496897",
    "a41fc0b5659acbeb24997ab639a1e94d68af0689ff017a97631196b7151ec067",
    "69447842f5c4a5b96b49c5442aae203068cea7be0fb0b99d25825e53d4c05ec4",
    "fba6dd9a1121d3157ca8d32ac3b0df75904097dd88533b87157f6ccb8311479f",
    "ce2f65936c3bc67d5df1b26f3a38400364cef25192f0faebdb6e9acc4e29fd9",
    "2482bac6150922d09afb80cf3d4fbb2bde4b6ccee667f1342c90823aa1df6e0e",
    "7d176056e407c91ab0bab80c00b03adde5f110dcae19e702d9cd0e4bfa523386",
    "955ad43f5eb343eb7ffe76f82d3f070cc1a0a105862e0f13e9c0a4b4e990032e",
    "c478af54c2e119010e6d0b9755e8411425a4de705043ec06ab1b89ee43680be8",
    "f90906f6bac11998c6b9030eccd831d44fa53f6a6c96971573b7d3ca582ab3dc",
    "ac1b87cef65f876fa15e2a0d8ba340c5fb250484fdd09ee5c08114c4bfcfa354",
    "27f6a98d7336fbbe4b769a48ceeeb7462385fb2fa4e492ebb3cd29b317d299cb",
    "e60b1b47e548a2748b05d18e7b0d27c5cef22e93808985e52a2d0c6e5d668c0b",
    "e63807eaddb3349fcabc42856f6e4255efe931ec26b5c7b143ca003e26db150e",
    "1437a65cd01f407bfc151e8af2f4528922555bffbc751d731a3b073478fd0ff",
    "8567b7b886a20092677cdc72bb4d63bd21a13eba903b8d2c814e5f19a10e6ad9",
    "34a9707e2e22f7330363b7957668ed722d48779fdab481477d2e7f7bde5fcd53"
};



BPSetupParams h_params[N_DATA];
initParamRandom h_ranParams[N_DATA];
std::string SEED="gyy hello world";
std::string SEEDU="gyy innerproduct";
BPProve h_prove[N_DATA];
UINT64 h_bLR[256*N_DATA];

Jpoint h_ipcommit[N_DATA];

Jpoint h_lsrs[10*N_DATA*2];
Jpoint *d_lsrs;

BPSetupParams *d_params;
initParamRandom *d_ranParams;

BPProve *d_prove;
UINT64 *d_bLR;

Jpoint *d_hprime;
Jpoint *d_ipcommit;
Jpoint* d_tmpJ;
Jpoint *d_tmpgphp;

UINT64 h_xyz[4*4*N_DATA],*d_xyz; //x,y,z,(y-1)^-1


void str2uint64(char *s,UINT64* x){
    std::string tmps(s);
    int len = tmps.size();
    // std::cout<<"len="<<len<<std::endl;
    // std::cout<<"size="<<tmps.size()<<std::endl;
    // std::cout<<"tmps="<<tmps<<std::endl;
    // std::cout<<"s="<<s<<std::endl;
    // std::cout<<tmps<<std::endl;
    // std::cout<<"0="<<tmps.substr(tmps.size()-16,16).c_str()<<std::endl;
    // std::cout<<"1="<<tmps.substr(tmps.size()-32,16).c_str()<<std::endl;
    // std::cout<<"2="<<tmps.substr(tmps.size()-48,16).c_str()<<std::endl;
    // std::cout<<"3="<<tmps.substr(0,16-(64-tmps.size())).c_str()<<std::endl;
    if(len<16){
        x[0]=strtoull(tmps.c_str(),NULL,16);
        x[1]=0;
        x[2]=0;
        x[3]=0;
        return ;
    }
    if(len<32){
        x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
        x[1]=strtoull(tmps.substr(0,16-(32-tmps.size())).c_str(),NULL,16);
        printf("%s\n",tmps.substr(0,16-(32-tmps.size())).c_str());
        x[2]=0;
        x[3]=0;
        return ;
    }
    if(len<48){
        x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
        x[1]=strtoull(tmps.substr(tmps.size()-32,16).c_str(),NULL,16);
        x[2]=strtoull(tmps.substr(0,16-(48-tmps.size())).c_str(),NULL,16);
        x[3]=0;
        return ;
    }
    x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
    x[1]=strtoull(tmps.substr(tmps.size()-32,16).c_str(),NULL,16);
    x[2]=strtoull(tmps.substr(tmps.size()-48,16).c_str(),NULL,16);
    x[3]=strtoull(tmps.substr(0,16-(64-tmps.size())).c_str(),NULL,16);
    return ;
    

    // h_mybig_print(x);
    // std::cout<<std::endl;
}
void setJpoint(char* x,char* y,Jpoint* jp){
    str2uint64(x,jp->x);
    str2uint64(y,jp->y);
    str2uint64("1",jp->z);
}

void sha256(char *string, char *outputBuffer)
{
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    SHA256_Update(&sha256, string, strlen(string));
    SHA256_Final(hash, &sha256);
    int i = 0;
    for(i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(outputBuffer + (i * 2), "%02x", hash[i]);
    }
    outputBuffer[64] = 0;
}
void uint642str(UINT64* x,char *s){
    int cur=0;
    for(int i=3;i>=0;i--){
        for(int j=0;j<16;j++){
            // std::cout<<((x[i]>>((15-j)*4))&0xf==0)<<std::endl;
            // printf("%d\n",(x[i]>>((15-j)*4))&0xf);
            // printf("%d\n",(int)(x[i]>>((15-j)*4))&0xf == (int)0);
            if(cur==0 && (((x[i]>>((15-j)*4))&0xf) ==0)){
                continue;
            }
            sprintf(s + cur, "%x", (x[i]>>((15-j)*4))&0xf);
            cur++;
            // sprintf(s + (3-i)*16+j*2, "%02x", (x[i]>>((7-j)*8))&0xff);
            // printf("%x\n",(x[i]>>((7-j)*8))&0xff);
        }
        
    }
    s[cur]=0;
}
void uint642byte(UINT64 *x,char *h){
    for(int i=0;i<4;i++){
        for(int j=0;j<8;j++){
            h[i*8+j]=(x[3-i]>>((7-j)*8))&0xff;
        }
    }
}
__device__  void d_uint642byte(UINT64 *x, unsigned char *h){
    for(int i=0;i<4;i++){
        for(int j=0;j<8;j++){
            h[i*8+j]=(x[3-i]>>((7-j)*8))&0xff;
        }
    }
}
void uint642bin(UINT64* x,unsigned char *s){
    for(int i=3;i>=0;i--){
        for(int j=0;j<8;j++){
            s[(3-i)*8+j] = (x[i]>>((7-j)*8))&0xff;
            // printf("%02x\n",(x[i]>>((7-j)*8))&0xff);
        }
        
    }
    s[64]=0;
}
void HashBP(Jpoint* A,Jpoint* S,UINT64 *o1,UINT64 *o2){
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    // unsigned char tmp[32];

    char tmp[65];
    char o1str[65];
    uint642str(A->x,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(A->y,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(S->x,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(S->y,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    SHA256_Final(hash, &sha256);
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(o1str + (i * 2), "%02x", hash[i]);
    }
    o1str[64] = 0;
    str2uint64(o1str,o1);
    // printf("tmp=%s\n",o1str);
    // h_mybig_print(o1);
    if(o2==NULL) return;
    SHA256_CTX sha2562;
    SHA256_Init(&sha2562);
    // unsigned char tmp[32];

    uint642str(A->x,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(A->y,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(S->x,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(S->y,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    SHA256_Update(&sha2562, o1str, strlen(o1str));
    SHA256_Final(hash, &sha2562);
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(tmp + (i * 2), "%02x", hash[i]);
    }
    str2uint64(tmp,o2);
    // printf("tmp=%s\n",tmp);
    // h_mybig_print(o2);
    // o2[64] = 0;
}

//不转成str
void HashBP_V2(Jpoint* A,Jpoint* S,UINT64 *o1,UINT64 *o2){
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    // unsigned char tmp[32];

    char tmp[65];
    char o1str[65];
    uint642byte(A->x,tmp);
    SHA256_Update(&sha256, tmp, 32);
    uint642byte(A->y,tmp);
    SHA256_Update(&sha256, tmp, 32);
    uint642byte(S->x,tmp);
    SHA256_Update(&sha256, tmp, 32);
    uint642byte(S->y,tmp);
    SHA256_Update(&sha256, tmp, 32);
    SHA256_Final(hash, &sha256);
    // for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    // {
    //     sprintf(o1str + (i * 2), "%02x", hash[i]);
    // }
    // for(int i=0;i<32;i++){
    //     printf("%x ",hash[i]);
    // }
    // printf("\n");
    // o1str[64] = 0;
    // str2uint64(o1str,o1);
    o1[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
    o1[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
            |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
    o1[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
            |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
    o1[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
            |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
    // printf("tmp=%s\n",o1str);
    // printf("myO1=\n");
    // h_mybig_print(o1);
    if(o2==NULL) return;
    SHA256_CTX sha2562;
    SHA256_Init(&sha2562);
    // unsigned char tmp[32];

    uint642byte(A->x,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(A->y,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(S->x,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(S->y,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(o1,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    SHA256_Final(hash, &sha2562);
    // for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    // {
    //     sprintf(tmp + (i * 2), "%02x", hash[i]);
    // }
    // str2uint64(tmp,o2);
    o2[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
    o2[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
            |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
    o2[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
            |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
    o2[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
            |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
    
    // printf("myO2=\n");
    // h_mybig_print(o2);
    // printf("tmp=%s\n",tmp);
    // h_mybig_print(o2);
    // o2[64] = 0;
}

void cudaInit(){
    hipMalloc(&d_params,sizeof(BPSetupParams)*N_DATA);
    hipMalloc(&d_prove,sizeof(BPProve)*N_DATA);
    hipMalloc(&d_tmpJ,sizeof(Jpoint)*5*N_DATA);
    hipMalloc(&d_xyz,sizeof(UINT64)*16*N_DATA);
    hipMalloc(&d_hprime,sizeof(Jpoint)*h_params[0].N*N_DATA);
    hipMalloc(&d_ipcommit,sizeof(Jpoint)*N_DATA);
    hipMalloc(&d_bLR,sizeof(UINT64)*256*N_DATA);
    hipMalloc(&d_lsrs,sizeof(Jpoint)*10*N_DATA*2);
    hipMalloc(&d_tmpgphp,sizeof(Jpoint)*2*N_DATA);
}

void init(){
    for(int j=0;j<N_DATA;j++){
        setJpoint(Vx,Vy,&h_prove[j].V);
        setJpoint(Ax,Ay,&h_prove[j].A); 
        setJpoint(Sx,Sy,&h_prove[j].S);
        setJpoint(T1x,T1y,&h_prove[j].T1);
        setJpoint(T2x,T2y,&h_prove[j].T2);
        setJpoint(ipCommitx,ipCommity,&h_ipcommit[j]);

        for(int i=0;i<5;i++){
            setJpoint(ipLsRs[i*2],ipLsRs[i*2+1],&h_lsrs[j*5+i]);
            
        }
        for(int i=0;i<5;i++){
            setJpoint(ipLsRs[10+i*2],ipLsRs[10+i*2+1],&h_lsrs[N_DATA*5+j*5+i]);
            
        }

        setJpoint(ipUx,ipUy,&h_params[j].ipU);
        setJpoint(ippramPx,ippramPy,&h_params[j].ipP);

        str2uint64(Taux,h_prove[j].Taux);
        str2uint64(Mu,h_prove[j].Mu);
        str2uint64(Tprime,h_prove[j].Tprime);

        for(int i=0;i<32;i++){
            str2uint64(BL[i],&h_bLR[256*j+4*i]); 
            str2uint64(BR[i],&h_bLR[256*j+(32+i)*4]); 
        }

        str2uint64(ipA,h_params[j].ipA);
        str2uint64(ipB,h_params[j].ipB);
    }
    
}
    

void calInvy(UINT64 *yinv,char *stry){
    mpz_init(n);
    mpz_set_str(n,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141",16);
    mpz_t bny,bnyinv;
    mpz_init(bny);
    mpz_set_str(bny,stry,16);
    mpz_invert(bnyinv,bny,n);
    mpz_mod(bnyinv,bnyinv,n);
    char tmp[65]={0};
    mpz_get_str(tmp,16,bnyinv);
    str2uint64(tmp,yinv);
}

void inline JpointCpyFromXYZ(Jpoint *jp,const UINT64 *x,const UINT64 *y,const UINT64 *z){
    jp->x[0] = x[0];
    jp->x[1] = x[1];
    jp->x[2] = x[2];
    jp->x[3] = x[3];

    jp->y[0] = y[0];
    jp->y[1] = y[1];
    jp->y[2] = y[2];
    jp->y[3] = y[3];

    jp->z[0] = z[0];
    jp->z[1] = z[1];
    jp->z[2] = z[2];
    jp->z[3] = z[3];
}

void inline JpointCpy(Jpoint *des,Jpoint *src){
    des->x[0]=src->x[0];
    des->x[1]=src->x[1];
    des->x[2]=src->x[2];
    des->x[3]=src->x[3];

    des->y[0]=src->y[0];
    des->y[1]=src->y[1];
    des->y[2]=src->y[2];
    des->y[3]=src->y[3];

    des->z[0]=src->z[0];
    des->z[1]=src->z[1];
    des->z[2]=src->z[2];
    des->z[3]=src->z[3];
}

void sha256(const std::string &srcStr, std::string &encodedHexStr)  
{  
    // 调用sha256哈希    
    unsigned char mdStr[33] = {0};  
    SHA256((const unsigned char *)srcStr.c_str(), srcStr.length(), mdStr);  
  
    // 哈希后的字符串    
    // 哈希后的十六进制串 32字节    
    char buf[65] = {0};  
    char tmp[3] = {0};  
    for (int i = 0; i < 32; i++)  
    {  
        sprintf(tmp, "%02x", mdStr[i]);  
        strcat(buf, tmp);  
    }  
    buf[64] = '\0';   
    encodedHexStr = std::string(buf);  
} 
int check_quadratic_residue(mpz_t num){
    mpz_t t1,t2,d;
    mpz_init(t1);
    mpz_init(t2);
    mpz_init(d);
    mpz_set_str(d,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F",16);

    mpz_sub_ui(t1,d,1);
    mpz_tdiv_q_ui(t2,t1,2);
    // gmp_printf("%#Zx\n",t1);
    mpz_powm(t2,num,t2,d);
    // gmp_printf("%#Zx\n",t2);
    if(mpz_cmp_ui(t2,1)==0){
        // gmp_printf("right\n");
        return 1;
    }
    if(mpz_cmp(t2,t1)==0){
        // gmp_printf("not\n");
        return -1;
    }
    return 0;
}

void mapToGroup(const std::string &s,Jpoint* jp){
    std::string tmphex;
    sha256(s,tmphex);
    // std::cout<<tmphex<<std::endl;
    mpz_t hexr,d;
    mpz_t t1,t2;
    mpz_t rx,ry;
    mpz_init(hexr);
    mpz_init(d);
    mpz_set_str(d,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F",16);
    mpz_init(t1);
    mpz_init(t2);
    mpz_init(rx);
    mpz_init(ry);
    mpz_set_str(hexr,tmphex.c_str(),16);


    for(int i=0;i<2048;i+=1){
        mpz_add_ui(rx,hexr,i);
        mpz_powm_ui(hexr,rx,3,d);
        mpz_add_ui(hexr,hexr,7);
        mpz_mod(hexr,hexr,d);


        // mpz_sub_ui(t1,d,1);
        // mpz_tdiv_q_ui(t2,t1,2);
        // gmp_printf("%#Zx\n",t1);
        // mpz_powm(t2,hexr,t2,d);
        if(check_quadratic_residue(hexr)==1){
            // gmp_printf("right\n");
            break;
        }
        if(check_quadratic_residue(hexr)==-1){
            // gmp_printf("not\n");
        }
        // gmp_printf("%#Zx\n",t2);
    }
    //这里hexr就是满足条件的二次剩余，现在要解二次剩余,rx里存的是x值
    //由于secp256k1曲线素数的特殊性，所以直接采用Tonelli-Shanks算法
    mpz_add_ui(t1,d,1);

    mpz_tdiv_qr_ui(t1,t2,t1,4);
    mpz_powm(ry,hexr,t1,d);

    // gmp_printf("%#Zx\n",rx);
    // gmp_printf("%#Zx\n",ry);
    char jx[65]={0};
    char jy[65]={0};
    mpz_get_str(jx,16,rx);
    mpz_get_str(jy,16,ry);
    str2uint64(jx,jp->x);
    str2uint64(jy,jp->y);
    jp->z[0]=0x1L;
    // h_print_pointJ(jp);
    //验证

    // mpz_powm_ui(t1,rx,3,d);
    // mpz_add_ui(t1,t1,7);
    // mpz_mod(t1,t1,d);
    // mpz_powm_ui(t2,ry,2,d);
    // mpz_mod(t2,t2,d);
    // if(mpz_cmp(t1,t2)==0){
    //     gmp_printf("x and y right\n");
    // }else{
    //     gmp_printf("WRONG!!!\n");
    // }
    
    //下面是Cipolla算法第一步找a^2-n为非二次剩余（由于最终采用了别的方法，所以注释掉）
    /*
    while(1){
        mpz_add_ui(t2,hexr,i);
        mpz_powm_ui(t1,t2,2,d);
        mpz_sub(t1,t1,hexr);
        mpz_mod(t1,t1,d);
        if(check_quadratic_residue(t1)==-1){
            break;
        }
        i++;
    }
    gmp_printf("i=%d\n",i);
    gmp_printf("%#Zx\n",t2);
    gmp_printf("%#Zx\n",hexr);
    */


}
void gen_random_uint64(std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> &engine,UINT64 s[4]){
    // std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine(clock());
    for(int i=0;i<4;i++){
        s[i] = engine();
    }
    if(s[0]==0xFFFFFFFFFFFFFFFF&&s[1]==0xFFFFFFFFFFFFFFFF&&
        s[2]==0xFFFFFFFFFFFFFFFF&&s[3]>0xFFFFFFFEFFFFFC2F){
            s[3]-=0xFFFFFFFEFFFFFC2F;
    }
}

void init_random_param(){
    for(int j=0;j<N_DATA;j++){
        std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine(19970504);
        // std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine;
        gen_random_uint64(engine,h_ranParams[j].gamma);
        gen_random_uint64(engine,h_ranParams[j].alpha);
        gen_random_uint64(engine,h_ranParams[j].rho);
        gen_random_uint64(engine,h_ranParams[j].tau1);
        gen_random_uint64(engine,h_ranParams[j].tau2);
        
        

        for(int i=0;i<32;i++){

            gen_random_uint64(engine,&(h_ranParams[j].SL[i*4]));
            gen_random_uint64(engine,&(h_ranParams[j].SR[i*4]));         
        }
    }
    
    // h_mybig_print(ranParams.gamma);
    // h_mybig_print(ranParams.alpha);
}
void initPoint(){
    for(int i=0;i<N_DATA;i++){
        JpointCpyFromXYZ(&h_params[i].G,h_Gx,h_Gy,h_Gz);
        mapToGroup(SEED,&h_params[i].H);
        // printf("param[%d].H=\n",i);
        // h_print_pointJ(&h_params[i].H);
        h_params[i].N=32;

        for(int j=0;j<h_params[i].N;j++){
        
            char tmp[3];
            sprintf(tmp,"%u",j);
            mapToGroup(SEED+"h"+tmp,&h_params[i].Hh[j]);
            mapToGroup(SEED+"g"+tmp,&h_params[i].Gg[j]);
        }
        // mapToGroup(SEEDU,&h_params[i].ipU);
    }

    // JpointCpyFromXYZ(&h_params.G,h_Gx,h_Gy,h_Gz);
    // mapToGroup(SEED,&h_params.H);
    // printf("param.H=\n");
    // h_print_pointJ(&h_params.H);
    // h_params.N=32;
}

 void Jpoint2Apoint(Jpoint *A,Jpoint *ret){
    #ifdef __CUDA_ARCH__
		const UINT64 *R2=dc_R2;
        const UINT64 *ONE=dc_ONE;
	#else
		const UINT64 *R2=h_R2;
        const UINT64 *ONE=h_ONE;
	#endif
    UINT64 t1[4],t2[4];
    dh_mybig_moninv(A->z,t1);
    dh_mybig_monmult_64(t1,t1,t2);
    dh_mybig_monmult_64(t1,t2,t1);
    dh_mybig_monmult_64(A->x,R2,ret->x);
    dh_mybig_monmult_64(A->y,R2,ret->y);
    dh_mybig_monmult_64(ret->x,t2,ret->x);
    dh_mybig_monmult_64(ret->y,t1,ret->y);

    dh_mybig_monmult_64(ret->x,ONE,ret->x);
    dh_mybig_monmult_64(ret->y,ONE,ret->y);
    ret->z[0]=1;
    ret->z[1]=0;
    ret->z[2]=0;
    ret->z[3]=0;
}

void compute_al_ar(int v,int *al,UINT64 *ar,int n){
    for(int i=0;i<n;i++){
        al[i] = (v>>i) &0x1;
        if(al[i]){
            ar[i*4]=0;
            ar[i*4+1]=0;
            ar[i*4+2]=0;
            ar[i*4+3]=0;
        }else{
            ar[i*4  ]=0xBFD25E8CD0364141-1;
            ar[i*4+1]=0xBAAEDCE6AF48A03B;
            ar[i*4+2]=0xFFFFFFFFFFFFFFFE;
            ar[i*4+3]=0xFFFFFFFFFFFFFFFF;
            //  
        }
        
    }
}

void updateGen(Jpoint *hprime,Jpoint *Hh,UINT64 *y,int N){
    UINT64 yinv[4],expy[4],mony[4];
    JpointCpy(&hprime[0],&Hh[0]);
    dh_mybig_monmult_64_modN(y,h_R2modN,mony);
    // h_mybig_print(mony);
    dh_mybig_copy(expy,mony);
    // h_mybig_print(expy);
    dh_mybig_moninv_modN(expy,yinv);
    
    dh_mybig_monmult_64(hprime[0].x,h_R2,hprime[0].x);
    dh_mybig_monmult_64(hprime[0].y,h_R2,hprime[0].y);
    dh_mybig_monmult_64(hprime[0].z,h_R2,hprime[0].z);

    // printf("yinv=\n");
    // h_mybig_print(yinv);
    // dh_mybig_monmult_64(expy,h_R2,expy);
    for(int i=1;i<N;i++){
        dh_mybig_moninv_modN(expy,yinv);
        dh_mybig_monmult_64(Hh[i].x,h_R2,hprime[i].x);
        dh_mybig_monmult_64(Hh[i].y,h_R2,hprime[i].y);
        dh_mybig_monmult_64(Hh[i].z,h_R2,hprime[i].z);
        dh_point_mult_finalversion(&hprime[i],yinv,&hprime[i]);
        // dh_mybig_monmult_64(hprime[i].x,h_ONE,hprime[i].x);
        // dh_mybig_monmult_64(hprime[i].y,h_ONE,hprime[i].y);
        // dh_mybig_monmult_64(hprime[i].z,h_ONE,hprime[i].z);
        dh_mybig_monmult_64_modN(expy,mony,expy);
    }
}

void Jpoint_to_mon(Jpoint *jp){
    dh_mybig_monmult_64(jp->x,h_R2,jp->x);
    dh_mybig_monmult_64(jp->y,h_R2,jp->y);
    dh_mybig_monmult_64(jp->z,h_R2,jp->z);
}
void Jpoint_from_mon(Jpoint *jp){
    dh_mybig_monmult_64(jp->x,h_ONE,jp->x);
    dh_mybig_monmult_64(jp->y,h_ONE,jp->y);
    dh_mybig_monmult_64(jp->z,h_ONE,jp->z);
}

void BPProve_to_mon(){
    for(int i=0;i<N_DATA;i++){
        Jpoint_to_mon(&h_prove[i].V);
        Jpoint_to_mon(&h_prove[i].A);
        Jpoint_to_mon(&h_prove[i].S);
        Jpoint_to_mon(&h_prove[i].T1);
        Jpoint_to_mon(&h_prove[i].T2);
    }
    
   
}
void xyz_to_monN(){
    for(int i=0;i<N_DATA;i++){
        UINT64 *curhxyz = h_xyz+i*16;
        dh_mybig_monmult_64_modN(curhxyz  ,h_R2modN,curhxyz  );
        dh_mybig_monmult_64_modN(curhxyz+4,h_R2modN,curhxyz+4);
        dh_mybig_monmult_64_modN(curhxyz+8,h_R2modN,curhxyz+8);
    }
    
}
void param_to_mon(){
    for(int j=0;j<N_DATA;j++){
        for(int i=0;i<h_params[j].N;i++){
            Jpoint_to_mon(&h_params[j].Gg[i]);
        }
        Jpoint_to_mon(&h_ipcommit[j]);
        Jpoint_to_mon(&h_params[j].ipU);
        Jpoint_to_mon(&h_params[j].ipP);
        dh_mybig_monmult_64_modN(h_params[j].ipA,h_R2modN,h_params[j].ipA);
        dh_mybig_monmult_64_modN(h_params[j].ipB,h_R2modN,h_params[j].ipB);
        
    }
    
    
    
}
void blr_to_mon(){
    for(int j=0;j<N_DATA;j++){
        UINT64 *curbLR = h_bLR+4*64*j;
        for(int i=0;i<h_params[j].N*2;i++){
            dh_mybig_monmult_64_modN(curbLR+4*i,h_R2modN,curbLR+4*i);
        }
    }
    // for(int i=0;i<64;i++){
    //     dh_mybig_monmult_64_modN(h_bLR+4*i,h_R2modN,h_bLR+4*i);
    // }
    
}
void lsrs_to_mon(){
    for(int i=0;i<N_DATA;i++){
        for(int j=0;j<10;j++){
            dh_mybig_monmult_64(h_lsrs[i*10+j].x,h_R2,h_lsrs[N_DATA*10+i*10+j].x);
            dh_mybig_monmult_64(h_lsrs[i*10+j].y,h_R2,h_lsrs[N_DATA*10+i*10+j].y);
            dh_mybig_monmult_64(h_lsrs[i*10+j].z,h_R2,h_lsrs[N_DATA*10+i*10+j].z);           
        }
        
    }
    
}
void var2mon(){
    BPProve_to_mon();
    xyz_to_monN();
    param_to_mon();
    blr_to_mon();
    lsrs_to_mon();
    
}
__global__ void kernel_commitG1(Jpoint* res,BPSetupParams *param,BPProve *prove,int N){
    int tx = threadIdx.x;
    int idx = tx+blockIdx.x*blockDim.x;
    
    if(idx<N){
        Jpoint *curres = res+idx;
        UINT64 *curx = prove[idx].Tprime;
        UINT64 *curr = prove[idx].Taux ;
        Jpoint *curh = &param[idx].H;

        Jpoint tmp;
        Jpoint tmp2;
        dh_point_mult_finalversion(curh,curr,&tmp);
        d_base_point_mul(&tmp2,curx);
        // printf("tmp2=\n");
        // d_mybig_print(tmp2.x);
        // d_mybig_print(tmp2.y);
        // d_mybig_print(tmp2.z);
        // dh_mybig_copy(res->x,tmp2.x);
        // dh_mybig_copy(res->y,tmp2.y);
        // dh_mybig_copy(res->z,tmp2.z);
        dh_ellipticAdd_JJ(&tmp,&tmp2,curres);
        // printf("g^t'*htaux=\n");
        // d_mybig_print(tmp2.x);
        // d_mybig_print(tmp2.y);
        // d_mybig_print(tmp2.z);


    }
}

__global__ void kernel_rhs65(Jpoint* res,UINT64* d_xyz,BPProve *prove,BPSetupParams *param){
    int tx = threadIdx.x;
    int idx = tx + blockDim.x*blockIdx.x;
    int bx = blockIdx.x;

    int n = param[bx].N;
    if(tx>=4) return;
    __shared__ Jpoint sh_tmp[6];
    Jpoint *VT1T2 = &prove[bx].V;
    Jpoint* cur = (Jpoint*)VT1T2+tx;
    UINT64* curk = ((UINT64*)sh_tmp) + tx*4;
    UINT64 *xyz = d_xyz+bx*16;
    Jpoint *curres = res+bx;

    UINT64 x[4],y[4],z[4];
    dh_mybig_copy(z,xyz+8);
    dh_mybig_copy(y,xyz+4);
    dh_mybig_copy(x,xyz);
    if(tx==0){
        UINT64 t1[4],z3[4],yn[4];
        dh_mybig_monmult_64_modN(z,z,curk);   //z^2
        dh_mybig_monmult_64_modN(z,curk,z3);  //z^3
        dh_mybig_modsub_64_modN(z,curk,z);    //z-z^2
        // dh_mybig_monmult_64_modN(z3,dc_ONE,z3);
        // printf("z3=\n");
        // d_mybig_print(z3);
        // dh_mybig_monmult_64_modN(z,dc_ONE,z);
        // printf("z-z2=\n");
        // d_mybig_print(z);
        t1[3] = t1[2] = t1[1] = 0;
        t1[0] = 0xffffffff; 
        dh_mybig_monmult_64_modN(t1,dc_R2modN,t1);
        // dh_mybig_monmult_64_modN(t1,dc_ONE,t1);
        // printf("t1=\n");
        // d_mybig_print(t1);
        // dh_mybig_monmult_64_modN(y,dc_ONE,y);
        // printf("y=\n");
        // d_mybig_print(y);
        dh_mybig_modexp_ui32_modN(y,n,yn);
        
        dh_mybig_modsub_64_modN(yn,dc_mon_ONE_modN,yn);
        // dh_mybig_monmult_64_modN(yn,dc_ONE,yn);
        // printf("y32-1=\n");
        // d_mybig_print(yn);

        dh_mybig_copy(y,xyz+12); //y = (y-1)^-1
        // dh_mybig_monmult_64_modN(y,dc_ONE,y);
        // printf("y-1^-1=\n");
        // d_mybig_print(y);

        dh_mybig_monmult_64_modN(yn,y,yn);
        // dh_mybig_monmult_64_modN(yn,dc_ONE,yn);
        // printf("yn/y=\n");
        // d_mybig_print(yn);

        dh_mybig_monmult_64_modN(yn,z,yn);
        dh_mybig_monmult_64_modN(z3,t1,z3);
        dh_mybig_modsub_64_modN(yn,z3,yn);
        dh_mybig_monmult_64_modN(yn,dc_ONE,sh_tmp[1].x);
        // printf("delta=\n");
        // d_mybig_print(sh_tmp[1].x);

        dh_mybig_copy(sh_tmp[0].y,x);
        dh_mybig_monmult_64_modN(x,x,x);
        dh_mybig_copy(sh_tmp[0].z,x);
    }
    
    
    if(tx==3){
        d_base_point_mul(&sh_tmp[5],sh_tmp[1].x);
    }else{
        dh_mybig_monmult_64_modN(curk,dc_ONE,curk);
        dh_point_mult_finalversion(cur,curk,&sh_tmp[tx+2]);
    }
    // for(int i=1;i<4;i<<2){
    
    // }
    for(int i=1;i<4;i<<=1){
        if(tx%(i*2)==0)
            dh_ellipticAdd_JJ(&sh_tmp[tx+2],&sh_tmp[tx+2+i],&sh_tmp[tx+2]);
    }
    if(tx==0){
        dh_mybig_copy(curres->x,sh_tmp[2].x);
        dh_mybig_copy(curres->y,sh_tmp[2].y);
        dh_mybig_copy(curres->z,sh_tmp[2].z);
        // printf("rhs65\n");
        // dh_mybig_monmult_64(sh_tmp[2].x,dc_ONE,sh_tmp[2].x);
        // dh_mybig_monmult_64(sh_tmp[2].y,dc_ONE,sh_tmp[2].y);
        // dh_mybig_monmult_64(sh_tmp[2].z,dc_ONE,sh_tmp[2].z);
        // d_mybig_print(sh_tmp[2].x);
        // d_mybig_print(sh_tmp[2].y);
        // d_mybig_print(sh_tmp[2].z);
    }




}
// __global__ void kernel_calP(Jpoint* res,Jpoint *Gg,UINT64* xyz,Jpoint *hprime,Jpoint *A,Jpoint *S,int n)
__global__ void kernel_calP(Jpoint* res,BPSetupParams *params,UINT64* xyz,Jpoint *hprime,BPProve *prove,int n){
    int tx = threadIdx.x;
    int idx = tx + blockDim.x*blockIdx.x;
    int bx = blockIdx.x;
    __shared__ Jpoint sh_jp[64];//x,z,-z,z2

    UINT64 x[4],z[4],mz[4],z2[4];

    Jpoint *Gg = params[bx].Gg;
    UINT64 *curxyz = xyz+bx*16;
    Jpoint *curhprime = hprime+bx*32;
    Jpoint *A = &(prove[bx].A);
    Jpoint *S = &(prove[bx].S);
    Jpoint *curres = res+bx;

    dh_mybig_copy(x,curxyz);
    dh_mybig_copy(z,curxyz+8);
    dh_mybig_neg_modN(z,mz);
    dh_mybig_monmult_64_modN(mz,dc_ONE,mz);
    dh_mybig_monmult_64_modN(z,z,z2);

    dh_point_mult_finalversion(&Gg[tx],mz,&sh_jp[tx]);

    UINT64 t1[4],t2[4];
    dh_mybig_copy(t1,curxyz+4);
    dh_mybig_copy(t2,dc_mon_TWO_modN);
    dh_mybig_modexp_ui32_modN(t1,tx,t1);
    dh_mybig_modexp_ui32_modN(t2,tx,t2);
    dh_mybig_monmult_64_modN(t1,z,t1);
    dh_mybig_monmult_64_modN(t2,z2,t2);

    
    dh_mybig_modadd_64_modN(t1,t2,t1);
    dh_mybig_monmult_64_modN(t1,dc_ONE,t1);

    dh_point_mult_finalversion(&curhprime[tx],t1,&sh_jp[tx+32]);

    for(int i=1;i<=32;i<<=1){
        if(tx%(i)==0)
            dh_ellipticAdd_JJ(&sh_jp[tx*2],&sh_jp[tx*2+i],&sh_jp[tx*2]);
        __syncthreads();
    }
        // __syncthreads();


    if(tx==0){
        dh_ellipticAdd_JJ(&sh_jp[0],A,&sh_jp[0]);
        dh_mybig_monmult_64_modN(x,dc_ONE,x);
        dh_point_mult_finalversion(S,x,&sh_jp[1]);
        dh_ellipticAdd_JJ(&sh_jp[0],&sh_jp[1],&sh_jp[0]);

        dh_mybig_monmult_64(sh_jp[0].x,dc_ONE,curres->x);
        dh_mybig_monmult_64(sh_jp[0].y,dc_ONE,curres->y);
        dh_mybig_monmult_64(sh_jp[0].z,dc_ONE,curres->z);
        // printf("66P\n");
        // d_mybig_print(res->x);
        // d_mybig_print(res->y);
        // d_mybig_print(res->z);

    }
    


    
}
// __global__ void kernel_calP67(Jpoint *res,Jpoint *commit,Jpoint *h,UINT64 *mu)
__global__ void kernel_calP67(Jpoint *res,Jpoint *commit,BPSetupParams *params,BPProve *prove){
    int tx = threadIdx.x;
    int idx = tx + blockDim.x*blockIdx.x;
    int bx = blockIdx.x;

    Jpoint *curres = res+bx;
    Jpoint *h = &params[bx].H;
    UINT64 *mu = prove[bx].Mu;
    Jpoint *curcommit = commit + bx;

    if(tx==0){
        dh_point_mult_finalversion(h,mu,curres);
        dh_ellipticAdd_JJ(curres,curcommit,curres);

        // printf("67Ph^ucommit\n");
        dh_mybig_monmult_64(curres->x,dc_ONE,curres->x);
        dh_mybig_monmult_64(curres->y,dc_ONE,curres->y);
        dh_mybig_monmult_64(curres->z,dc_ONE,curres->z); 
        // d_mybig_print(res->x);
        // d_mybig_print(res->y);
        // d_mybig_print(res->z);
    }
}
__global__ void kernel_calip68(UINT64 *blr,int n){
    int tx = threadIdx.x;
    int idx = tx + blockDim.x*blockIdx.x;
    int bx = blockIdx.x;

    UINT64 *curblr = blr+bx*64*4;;

    dh_mybig_monmult_64_modN(&curblr[tx*4],&curblr[4*(tx+n)],&curblr[4*tx]);
    for(int i=n/2;i>0;i>>=1){
        if(tx<i)
            dh_mybig_modadd_64_modN(&curblr[tx*4],&curblr[(tx+i)*4],&curblr[tx*4]);
        __syncthreads();
    }
    // if(tx==0){
    //     dh_mybig_monmult_64_modN(&blr[4*(tx)],dc_ONE,&blr[4*(tx)]);
    //     printf("blr[%d] t'\n",tx);
    //     d_mybig_print(&blr[4*(tx)]);
    // }
    
}
void updateGen2(Jpoint *hprime,Jpoint *Hh,UINT64 *y,int N){
    UINT64 yinv[4],expy[4],mony[4];
    JpointCpy(&hprime[0],&Hh[0]);
    dh_mybig_monmult_64_modN(y,h_R2modN,mony);
    // h_mybig_print(mony);
    dh_mybig_copy(expy,mony);
    // h_mybig_print(expy);
    dh_mybig_moninv_modN(expy,yinv);
    
    dh_mybig_monmult_64(hprime[0].x,h_R2,hprime[0].x);
    dh_mybig_monmult_64(hprime[0].y,h_R2,hprime[0].y);
    dh_mybig_monmult_64(hprime[0].z,h_R2,hprime[0].z);

    // printf("yinv=\n");
    // h_mybig_print(yinv);
    // dh_mybig_monmult_64(expy,h_R2,expy);
    for(int i=1;i<N;i++){
        dh_mybig_moninv_modN(expy,yinv);
        dh_mybig_monmult_64(Hh[i].x,h_R2,hprime[i].x);
        dh_mybig_monmult_64(Hh[i].y,h_R2,hprime[i].y);
        dh_mybig_monmult_64(Hh[i].z,h_R2,hprime[i].z);
        dh_point_mult_finalversion(&hprime[i],yinv,&hprime[i]);
        // dh_mybig_monmult_64(hprime[i].x,h_ONE,hprime[i].x);
        // dh_mybig_monmult_64(hprime[i].y,h_ONE,hprime[i].y);
        // dh_mybig_monmult_64(hprime[i].z,h_ONE,hprime[i].z);
        dh_mybig_monmult_64_modN(expy,mony,expy);
    }
}
// __global__ void kernel_updateGen2(Jpoint *hprime,BPSetupParams *params,UINT64 *xyz,int N){
//     int tx = threadIdx.x;
//     int bx = blockIdx.x;
//     int idx = bx*blockDim.x+tx;

//     Jpoint *Hh = params[idx].Hh;
//     UINT64 *y = xyz+idx*16+4;
//     Jpoint *curhprime = hprime+idx*N;

//     UINT64 yinv[4],expy[4],mony[4];
//     // dh_mybig_copy(curhprime[0].x,Hh[0].x);
//     // dh_mybig_copy(curhprime[0].y,Hh[0].y);
//     // dh_mybig_copy(curhprime[0].z,Hh[0].z);
    
//     // dh_mybig_monmult_64_modN(y,dc_R2modN,mony);
//     dh_mybig_copy(mony,y);
//     dh_mybig_copy(expy,mony);
//     dh_mybig_moninv_modN(expy,yinv);
//     dh_mybig_monmult_64(Hh[0].x,dc_R2,curhprime[0].x);
//     dh_mybig_monmult_64(Hh[0].y,dc_R2,curhprime[0].y);
//     dh_mybig_monmult_64(Hh[0].z,dc_R2,curhprime[0].z);

//     for(int i=1;i<N;i++){
//         dh_mybig_moninv_modN(expy,yinv);
//         dh_mybig_monmult_64(Hh[i].x,dc_R2,curhprime[i].x);
//         dh_mybig_monmult_64(Hh[i].y,dc_R2,curhprime[i].y);
//         dh_mybig_monmult_64(Hh[i].z,dc_R2,curhprime[i].z);
//         dh_point_mult_finalversion(&curhprime[i],yinv,&curhprime[i]);
//         // dh_mybig_monmult_64(hprime[i].x,h_ONE,hprime[i].x);
//         // dh_mybig_monmult_64(hprime[i].y,h_ONE,hprime[i].y);
//         // dh_mybig_monmult_64(hprime[i].z,h_ONE,hprime[i].z);
//         dh_mybig_monmult_64_modN(expy,mony,expy);
//     }

// }
__global__ void kernel_updateGen(Jpoint *hprime,BPSetupParams *params,UINT64 *xyz,int N,int totalN){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int idx = bx*blockDim.x+tx;
    
    int curp = idx/N;
    if (curp>=totalN) return ;

    int laneid = idx%N;
    Jpoint *Hh = params[curp].Hh;
    UINT64 *y = xyz+curp*16+4;
    Jpoint *curhprime = hprime+idx;

    UINT64 yinv[4],expy[4],mony[4];
    // dh_mybig_copy(curhprime[0].x,Hh[0].x);
    // dh_mybig_copy(curhprime[0].y,Hh[0].y);
    // dh_mybig_copy(curhprime[0].z,Hh[0].z);
    
    // dh_mybig_monmult_64_modN(y,dc_R2modN,mony);
    dh_mybig_copy(mony,y);
    dh_mybig_copy(expy,mony);
    // dh_mybig_moninv_modN(expy,yinv);
    dh_mybig_monmult_64(Hh[laneid].x,dc_R2,curhprime->x);
    dh_mybig_monmult_64(Hh[laneid].y,dc_R2,curhprime->y);
    dh_mybig_monmult_64(Hh[laneid].z,dc_R2,curhprime->z);

    dh_mybig_modexp_ui32_modN(mony,(unsigned int)laneid,mony);
    dh_mybig_moninv_modN(mony,yinv);
    dh_point_mult_finalversion(curhprime,yinv,curhprime);

    // for(int i=1;i<N;i++){
    //     dh_mybig_moninv_modN(expy,yinv);
    //     dh_mybig_monmult_64(Hh[i].x,dc_R2,curhprime[i].x);
    //     dh_mybig_monmult_64(Hh[i].y,dc_R2,curhprime[i].y);
    //     dh_mybig_monmult_64(Hh[i].z,dc_R2,curhprime[i].z);
    //     dh_point_mult_finalversion(&curhprime[i],yinv,&curhprime[i]);
    //     // dh_mybig_monmult_64(hprime[i].x,h_ONE,hprime[i].x);
    //     // dh_mybig_monmult_64(hprime[i].y,h_ONE,hprime[i].y);
    //     // dh_mybig_monmult_64(hprime[i].z,h_ONE,hprime[i].z);
    //     dh_mybig_monmult_64_modN(expy,mony,expy);
    // }

}
__device__ void device_hashBP(Jpoint *A,Jpoint *B,UINT64 *x,UINT64 *y){
    
    
    
    unsigned char tmp[32];
    unsigned char hash[32];
    // char tmp2[65];
    // char o1str[65];
    MYSHA256_CTX ctx;
    sha256_init(&ctx);
    d_uint642byte(A->x,tmp);
    sha256_update(&ctx, tmp, 32);
    d_uint642byte(A->y,tmp);
    sha256_update(&ctx, tmp, 32);
    d_uint642byte(B->x,tmp);
    sha256_update(&ctx, tmp, 32);
    d_uint642byte(B->y,tmp);
    sha256_update(&ctx, tmp, 32);
    sha256_final(&ctx, hash);
    // tmp2[64] = '\0';
    // printf("str = %s\n",tmp2);
    // d_mybig_print(x);
    // for(int i=0;i<32;i++){
    //     printf("%x ",hash[i]);
    // }
    // printf("\n");

    x[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
        |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
    x[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
            |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
    x[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
            |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
    x[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
            |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;

    // d_mybig_print(x);

    if(y==NULL) return ;
    MYSHA256_CTX ctx2;
    sha256_init(&ctx2);
    d_uint642byte(A->x,tmp);
    sha256_update(&ctx2, tmp, 32);
    d_uint642byte(A->y,tmp);
    sha256_update(&ctx2, tmp, 32);
    d_uint642byte(B->x,tmp);
    sha256_update(&ctx2, tmp, 32);
    d_uint642byte(B->y,tmp);
    sha256_update(&ctx2, tmp, 32);
    sha256_update(&ctx2, hash, 32);
    sha256_final(&ctx2, hash);
    y[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
        |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
    y[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
            |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
    y[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
            |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
    y[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
            |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
    // d_mybig_print(y);

}
__global__ void kernel_ipverify(Jpoint *tmpgphp,Jpoint* LsRs,BPSetupParams *params,Jpoint *hprime,int N,int logN,int totalN){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int idx = bx*blockDim.x+tx;
    extern __shared__ Jpoint sh[];
    extern __shared__ UINT64 shuint64[];

    int NperGroup = N*2;
    int curp = idx/NperGroup;
    int laneid = idx%NperGroup;
    if(curp>=totalN) return ;
    int groupPerBlock = blockDim.x/NperGroup;
   
    Jpoint *curLs = LsRs+curp*logN;
    Jpoint *curRs = LsRs+(totalN+curp)*logN;
    UINT64 *curx = (UINT64*)(&shuint64[blockDim.x*3*4]);
    UINT64 *curxinv = curx+logN*groupPerBlock*4;
    Jpoint *Gg = params[curp].Gg;
    Jpoint *curhprime = hprime+curp*N;
    UINT64 *ipA = params[curp].ipA;
    UINT64 *ipB = params[curp].ipB;
    Jpoint *ipU = &params[curp].ipU;
    Jpoint *ipP = &params[curp].ipP;
    // Jpoint *cursh = sh+(tx/N)*N;
    Jpoint *curmonLs = LsRs+N_DATA*logN*2+curp*logN;
    Jpoint *curmonRs = LsRs+N_DATA*logN*2+(totalN+curp)*logN;;
    if(tx<logN*groupPerBlock){
        device_hashBP(&curLs[tx],&curRs[tx],curx+tx*4,NULL);
        dh_mybig_monmult_64_modN(curx+tx*4,dc_R2modN,curxinv+tx*4);
        dh_mybig_moninv_modN(curxinv+tx*4,curxinv+tx*4);
        // if(bx==444&&tx==0){
        //     printf("lsrs invx=\n");
        //     d_mybig_print(curxinv);
        //     printf("lsrs invx[1]=\n");
        //     d_mybig_print(curxinv+(1)*4);
        //     printf("lsrs invx[2]=\n");
        //     d_mybig_print(curxinv+(2)*4);
        //     printf("lsrs invx[3]=\n");
        //     d_mybig_print(curxinv+(3)*4);
        //     printf("lsrs invx[4]=\n");
        //     d_mybig_print(curxinv+(4)*4);
        //     printf("lsrs invx[1][0]=\n");
        //     d_mybig_print(curxinv+(5)*4);
        //     printf("lsrs invx[1][1]=\n");
        //     d_mybig_print(curxinv+(6)*4);
        // }
        
    }
    __syncthreads();
    Jpoint *tmpshJ = sh+logN*groupPerBlock;
    if(tx<logN*groupPerBlock*2){
        dh_mybig_monmult_64_modN(curx+tx*4,dc_R2modN,shuint64+tx*4); 
        dh_mybig_monmult_64_modN(shuint64+tx*4,shuint64+tx*4,shuint64+tx*4);
        dh_mybig_monmult_64_modN(shuint64+tx*4,dc_ONE,shuint64+tx*4);
        
        if(tx<logN*groupPerBlock)
            dh_point_mult_finalversion(&curmonLs[tx],shuint64+tx*4,tmpshJ+tx);
        else
            dh_point_mult_finalversion(&curmonRs[tx-logN*groupPerBlock],shuint64+tx*4,tmpshJ+tx);
        
        
    }
    __syncthreads();
    int curid = tx%10;
    for(int i=8;i>=1;i/=2){
        if((tx<logN*groupPerBlock*2)&&(curid+i<10)&&(curid<i)){
            dh_ellipticAdd_JJ(&tmpshJ[tx],&tmpshJ[tx+i],&tmpshJ[tx]);
        }
        __syncthreads();
                
    }
    if(tx<logN*groupPerBlock*2&&curid==0){
        dh_ellipticAdd_JJ(&tmpshJ[tx],ipP,&tmpshJ[tx]);
    }
    
    // if(bx==0&&tx==0){
        
    //     printf("Ls[0]^x0^2=\n");
    //     Jpoint tmp;
    //     // d_mybig_print(curLs[0].x); 
    //     // d_mybig_print(curLs[0].y); 
    //     // d_mybig_print(curLs[0].z); 
        
    //     dh_mybig_monmult_64(tmpshJ[0].x,dc_ONE,tmp.x);
    //     dh_mybig_monmult_64(tmpshJ[0].y,dc_ONE,tmp.y);
    //     dh_mybig_monmult_64(tmpshJ[0].z,dc_ONE,tmp.z);
    //     d_mybig_print(tmp.x); 
    //     d_mybig_print(tmp.y); 
    //     d_mybig_print(tmp.z); 
    // }
    
    
    curx=curx+tx/NperGroup*logN*4;
    curxinv = curxinv+tx/NperGroup*logN*4;
    if(laneid<N){
        dh_mybig_copy(sh[tx].x,Gg[laneid].x);
        dh_mybig_copy(sh[tx].y,Gg[laneid].y);
        dh_mybig_copy(sh[tx].z,Gg[laneid].z);
    }else{
        dh_mybig_copy(sh[tx].x,curhprime[laneid].x);
        dh_mybig_copy(sh[tx].y,curhprime[laneid].y);
        dh_mybig_copy(sh[tx].z,curhprime[laneid].z);
    }
    
    
    // __syncthreads();
    for(int k=logN-1;k>=0;k--){
        if(((laneid>>k)&0x1)==1){
            // if(bx==0&&tx==0){
            //     printf("1111\n");
            // }
            if(laneid<N)
                dh_point_mult_finalversion(&sh[tx],curx+(logN-1-k)*4,&sh[tx]);
            else
                dh_point_mult_finalversion(&sh[tx],curxinv+(logN-1-k)*4,&sh[tx]);
            
        }else{
            // if(bx==0&&tx==32){
            //     d_mybig_print(curxinv+(logN-1-k)*4);
            //     printf("gx0x1x2[0]=\n");
            //     Jpoint tmp;
            //     dh_mybig_monmult_64(sh[32].x,dc_ONE,tmp.x);
            //     dh_mybig_monmult_64(sh[32].y,dc_ONE,tmp.y);
            //     dh_mybig_monmult_64(sh[32].z,dc_ONE,tmp.z);
            //     d_mybig_print(tmp.x); 
            //     d_mybig_print(tmp.y); 
            //     d_mybig_print(tmp.z); 
            // }
            if(laneid<N)
                dh_point_mult_finalversion(&sh[tx],curxinv+(logN-1-k)*4,&sh[tx]);
            else
            dh_point_mult_finalversion(&sh[tx],curx+(logN-1-k)*4,&sh[tx]);
            
            
        }
    }

    
    // __syncthreads();
    
    for(int i=N/2;i>=1;i/=2){
        if((laneid%N)<i){
            dh_ellipticAdd_JJ(&sh[tx],&sh[tx+i],&sh[tx]);
        }
        
    }
    __syncthreads();
    
    // if(bx==0&tx==0){
    //     printf("gx0x1x2[0][0]=\n");
    //     Jpoint tmp;
    //     dh_mybig_monmult_64(sh[0].x,dc_ONE,tmp.x);
    //     dh_mybig_monmult_64(sh[0].y,dc_ONE,tmp.y);
    //     dh_mybig_monmult_64(sh[0].z,dc_ONE,tmp.z);
    //     d_mybig_print(tmp.x); 
    //     d_mybig_print(tmp.y); 
    //     d_mybig_print(tmp.z); 
        
    //     printf("gx0x1x2[1][0]=\n");
    //     dh_mybig_monmult_64(sh[64].x,dc_ONE,tmp.x);
    //     dh_mybig_monmult_64(sh[64].y,dc_ONE,tmp.y);
    //     dh_mybig_monmult_64(sh[64].z,dc_ONE,tmp.z);
    //     d_mybig_print(tmp.x); 
    //     d_mybig_print(tmp.y); 
    //     d_mybig_print(tmp.z); 

    //     printf("hx0x1x2[0][0]=\n");
    //     dh_mybig_monmult_64(sh[32].x,dc_ONE,tmp.x);
    //     dh_mybig_monmult_64(sh[32].y,dc_ONE,tmp.y);
    //     dh_mybig_monmult_64(sh[32].z,dc_ONE,tmp.z);
    //     d_mybig_print(tmp.x); 
    //     d_mybig_print(tmp.y); 
    //     d_mybig_print(tmp.z); 

    //     printf("hx0x1x2[1][0]=\n");
    //     dh_mybig_monmult_64(sh[96].x,dc_ONE,tmp.x);
    //     dh_mybig_monmult_64(sh[96].y,dc_ONE,tmp.y);
    //     dh_mybig_monmult_64(sh[96].z,dc_ONE,tmp.z);
    //     d_mybig_print(tmp.x); 
    //     d_mybig_print(tmp.y); 
    //     d_mybig_print(tmp.z); 
    // }
    if(laneid<2){
        dh_mybig_copy(tmpgphp[curp*2+laneid].x,sh[tx+laneid*N].x);
        dh_mybig_copy(tmpgphp[curp*2+laneid].y,sh[tx+laneid*N].y);
        dh_mybig_copy(tmpgphp[curp*2+laneid].z,sh[tx+laneid*N].z);
        // dh_mybig_copy(tmpgphp[curp*2].x,sh[tx].x);
        // dh_mybig_copy(tmpgphp[curp*2].y,sh[tx].y);
        // dh_mybig_copy(tmpgphp[curp*2].z,sh[tx].z);
        
    }
    
    return ;
    if(laneid==0){
        dh_mybig_monmult_64_modN(ipA,ipB,curx+8);
        dh_mybig_monmult_64_modN(ipA,dc_ONE,curx);
        dh_mybig_monmult_64_modN(ipB,dc_ONE,curx+4);
        dh_mybig_monmult_64_modN(curx+8,dc_ONE,curx+8);
        dh_point_mult_finalversion(&sh[tx],curx,&sh[tx]);
        dh_point_mult_finalversion(&sh[tx+N],curx+4,&sh[tx+1]);
        dh_point_mult_finalversion(ipU,curx+8,&sh[tx+2]);
        dh_ellipticAdd_JJ(&sh[tx],&sh[tx+1],&sh[tx]);
        dh_ellipticAdd_JJ(&sh[tx],&sh[tx+2],&sh[tx]);
    }
    

    // __syncthreads();
    // if(tx==0&&bx==423){
    //     printf("g^a*h^b*U^ab[0]=\n");
    //     Jpoint tmp;
    //     dh_mybig_monmult_64(sh[0].x,dc_ONE,tmp.x);
    //     dh_mybig_monmult_64(sh[0].y,dc_ONE,tmp.y);
    //     dh_mybig_monmult_64(sh[0].z,dc_ONE,tmp.z);
    //     d_mybig_print(tmp.x); 
    //     d_mybig_print(tmp.y); 
    //     d_mybig_print(tmp.z); 

    //     // printf("g^a*h^b*U^ab[1]=\n");
        
    //     // dh_mybig_monmult_64(sh[64].x,dc_ONE,tmp.x);
    //     // dh_mybig_monmult_64(sh[64].y,dc_ONE,tmp.y);
    //     // dh_mybig_monmult_64(sh[64].z,dc_ONE,tmp.z);
    //     // d_mybig_print(tmp.x); 
    //     // d_mybig_print(tmp.y); 
    //     // d_mybig_print(tmp.z); 
    // }

}

void gpu_commitG1(){ 
    int nT = 256;
    int nB = (N_DATA+nT-1)/nT;
    kernel_commitG1<<<nB,nT>>>(d_tmpJ,d_params,d_prove,N_DATA);
}

void gpu_rhs65(){
    int nT = 32;
    int nB = N_DATA;
    kernel_rhs65<<<nB,nT>>>(d_tmpJ+N_DATA,d_xyz,d_prove,d_params);
}

void gpu_calP(){
    int nT = 32;
    int nB = N_DATA;
    int n = h_params[0].N;
    // kernel_calP<<<nB,nT>>>(d_tmpJ+2,d_params->Gg,d_xyz,d_hprime,&d_prove->A,&d_prove->S,n);
    kernel_calP<<<nB,nT>>>(d_tmpJ+2*N_DATA,d_params,d_xyz,d_hprime,d_prove,n);
}

void gpu_calP67(){
    int nT = 32;
    int nB = N_DATA;
    kernel_calP67<<<nB,nT>>>(d_tmpJ+3*N_DATA,d_ipcommit,d_params,d_prove);
    // kernel_calP67<<<1,32>>>(d_tmpJ+3,d_ipcommit,&d_params->H,d_prove->Mu);
}

void gpu_calip68(){
    int nT = 32;
    int nB = N_DATA;
    int n = h_params[0].N;
    // printf("n=%d\n",n);
    kernel_calip68<<<nB,nT>>>(d_bLR,n);
}

void gpu_updateGen(){
    int nT = 256;
    int n = h_params[0].N;
    int nB = ((N_DATA+(nT/n))-1)/(nT/n);
    
    kernel_updateGen<<<nB,nT>>>(d_hprime,d_params,d_xyz,n,N_DATA);
}
// void gpu_updateGen2(){
//     int nT = 32;
//     int nB = (N_DATA+32-1)/nT;
//     int n = h_params[0].N;
//     kernel_updateGen2<<<nB,nT>>>(d_hprime,d_params,d_xyz,n);
// }

void gpu_ipverify(){
    int n = h_params[0].N;
    int nT = 256;
    int nB = (N_DATA+(nT/n/2)-1)/(nT/n/2);
    int logn = log2((double)n);
    // printf("n=%d\n",n);
    // printf("logn=%d\n",logn);
    int sm_size = nT*3*4+(logn*nT/n/2*4*2);
    kernel_ipverify<<<nB,nT,sizeof(UINT64)*sm_size>>>(d_tmpgphp,d_lsrs,d_params,d_hprime,n,logn,N_DATA);
    // printf("sm_size=%d\n",sm_size);
}

size_t SMwithipverify(int bs){
    int logn =5;
    int sm_size=bs*3*4+(logn*bs/32/2*4*2);
    return sizeof(UINT64)*sm_size;
}

int main(){
    hipSetDevice(0);

    int nB,nT;
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,kernel_commitG1);
    printf("kernel_commitG1 nB=%d,nT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,kernel_updateGen);
    printf("kernel_updateGen nB=%d,nT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,kernel_ipverify);
    printf("kernel_ipverify nB=%d,nT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSizeVariableSMem(&nB,&nT,kernel_ipverify,SMwithipverify);
    printf("kernel_ipverify nB=%d,nT=%d\n",nB,nT);
    printf("byte sm = %d\n",SMwithipverify(256));
    init();

    // CUDA_CHECK_ERROR();
    int al[32]={0};
    UINT64 ar[32*4]={0};
    int value = rand();
    compute_al_ar(value,al,ar,32);

    printf("value=%d\n",value);

    printf("N_DATA = %d\n",N_DATA);
    // for(int i=31;i>=0;i--){
    //     printf("%d",al[i]);
    // }
    // printf("\n");
    // for(int i=31;i>=0;i--){
    //     printf("%d",ar[i]);
    // }
    // printf("\n");
    
    // std::string s = "gyy hello world";
    initPoint();
    // JpointCpyFromXYZ(&h_params.G,h_Gx,h_Gy,h_Gz);
    // mapToGroup(SEED,&h_params.H);
    // printf("param.H=\n");
    // h_print_pointJ(&h_params.H);
    // h_params.N=32;
    // cout<<SEED
    // for(int i=0;i<h_params.N;i++){
        
    //     char tmp[3];
    //     sprintf(tmp,"%u",i);
    //     mapToGroup(SEED+"h"+tmp,&h_params.Hh[i]);
    //     mapToGroup(SEED+"g"+tmp,&h_params.Gg[i]);
    // }
        

 
    init_random_param();
    // printf("ipU=\n");
    // h_print_pointJ(&h_params[0].ipU);
    // printf("Gg[0]=\n");
    // h_print_pointJ(&h_params[0].Gg[0]);
    // printf("Gg[1]=\n");
    // h_print_pointJ(&h_params[0].Gg[1]);
    // printf("Hh[0]=\n");
    // h_print_pointJ(&h_params[0].Hh[0]);

    
    // printf("rho=\n");
    // h_mybig_print(h_ranParams[0].rho);
    // h_mybig_print(h_ranParams[1].rho);
    // printf("gamma=\n");
    // h_mybig_print(h_ranParams[0].gamma);
    // h_mybig_print(h_ranParams[1].gamma);
    // printf("alpha=\n");
    // h_mybig_print(h_ranParams[0].alpha);
    // h_mybig_print(h_ranParams[1].alpha);
    // printf("tau1=\n");
    // h_mybig_print(h_ranParams[0].tau1);
    // h_mybig_print(h_ranParams[1].tau1);
    // printf("tau2=\n");
    // h_mybig_print(h_ranParams[0].tau2);
    // h_mybig_print(h_ranParams[1].tau2);

    
    Jpoint* hprime = (Jpoint*)malloc(sizeof(Jpoint)*h_params[0].N*N_DATA);
    unsigned char sd[32];
    UINT64 myz[4];
    UINT64 myy[4];
    // HashBP_V2(&h_prove.A,&h_prove.S,myy,myz);
    // UINT64 z[4];
    UINT64 ym1[4*N_DATA];


    cudaInit();

    struct timeval s1,e1,s2,e2,s3,e3,s4,e4,s5,e5,s6,e6,s7,e7,s8,e8,s9,e9,s10,e10;
    gettimeofday(&s1,NULL);

    for(int i=0;i<N_DATA;i++){
        UINT64 *curhxyz = h_xyz+i*16;
        HashBP_V2(&h_prove[i].A,&h_prove[i].S,curhxyz+4,curhxyz+8);
        HashBP_V2(&h_prove[i].T1,&h_prove[i].T2,curhxyz,NULL);

        dh_mybig_modsub_64_ui32_modN(curhxyz+4,1,ym1);
        dh_mybig_moninv_modN(ym1,curhxyz+12);
        // printf("xyz for data[%d]\n",i);
        // h_mybig_print(curhxyz);
        // h_mybig_print(curhxyz+4);
        // h_mybig_print(curhxyz+8);
        // h_mybig_print(curhxyz+12);
        
        
    }
    gettimeofday(&e1,NULL);
    // HashBP_V2(&h_prove.A,&h_prove.S,h_xyz+4,h_xyz+8);
    // HashBP_V2(&h_prove.T1,&h_prove.T2,h_xyz,NULL);
    // dh_mybig_modsub_64_ui32_modN(h_xyz+4,1,ym1);
    // dh_mybig_moninv_modN(ym1,h_xyz+12);
    // dh_mybig_monmult_64_modN(h_xyz+12,h_ONE,h_xyz+12);
    // printf("x,y,z,(y-1)^-1=\n");
    // h_mybig_print(h_xyz);
    // h_mybig_print(h_xyz+4);
    // h_mybig_print(h_xyz+8);
    // h_mybig_print(h_xyz+12);
    
    // gettimeofday(&s2,NULL);
    // for(int i=0;i<N_DATA;i++){
    //     UINT64 *curhxyz = h_xyz+i*16;
    //     Jpoint *curhprime = hprime+i*h_params[0].N;
    //     updateGen(curhprime,h_params[i].Hh,curhxyz+4,h_params[i].N);
    // }
    // gettimeofday(&e2,NULL);

    // Jpoint tmpJp;
    // printf("hprime[0]=\n");
    // JpointCpy(&tmpJp,&hprime[0]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1]=\n");
    // JpointCpy(&tmpJp,&hprime[1]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[31]=\n");
    // JpointCpy(&tmpJp,&hprime[31]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1][0]=\n");
    // JpointCpy(&tmpJp,&hprime[32]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1][1]=\n");
    // JpointCpy(&tmpJp,&hprime[33]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1][31]=\n");
    // JpointCpy(&tmpJp,&hprime[63]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    gettimeofday(&s3,NULL);
    var2mon();
    // BPProve_to_mon();
    for(int i=0;i<N_DATA;i++){
        Jpoint_to_mon(&h_params[i].H);
    }
    gettimeofday(&e3,NULL);

    // printf("taux=\n");
    // h_mybig_print(h_prove.Taux);
    // printf("tprime=\n");
    // h_mybig_print(h_prove.Tprime);

    
    gettimeofday(&s4,NULL);
    CUDA_SAFE_CALL(hipMemcpy(d_prove,&h_prove,sizeof(BPProve)*N_DATA,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_params,&h_params,sizeof(BPSetupParams)*N_DATA,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_xyz,h_xyz,sizeof(UINT64)*16*N_DATA,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_hprime,hprime,sizeof(Jpoint)*h_params[0].N*N_DATA,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_ipcommit,&h_ipcommit,sizeof(Jpoint)*N_DATA,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_bLR,h_bLR,sizeof(UINT64)*256*N_DATA,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_lsrs,h_lsrs,sizeof(Jpoint)*10*N_DATA*2,hipMemcpyHostToDevice));
    gettimeofday(&e4,NULL);
    


    gettimeofday(&s2,NULL);

    gpu_updateGen();

    hipDeviceSynchronize();
    gettimeofday(&e2,NULL);


    // CUDA_SAFE_CALL(hipMemcpy(hprime,d_hprime,sizeof(Jpoint)*h_params[0].N*N_DATA,hipMemcpyDeviceToHost));
    // Jpoint tmpJp;
    // printf("hprime[0]=\n");
    // JpointCpy(&tmpJp,&hprime[0]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1]=\n");
    // JpointCpy(&tmpJp,&hprime[1]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[31]=\n");
    // JpointCpy(&tmpJp,&hprime[31]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1][0]=\n");
    // JpointCpy(&tmpJp,&hprime[32]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1][1]=\n");
    // JpointCpy(&tmpJp,&hprime[33]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[1][31]=\n");
    // JpointCpy(&tmpJp,&hprime[63]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);
    // printf("hprime[5][0]=\n");
    // JpointCpy(&tmpJp,&hprime[32*4]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[5][1]=\n");
    // JpointCpy(&tmpJp,&hprime[32*4+1]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);

    // printf("hprime[5][31]=\n");
    // JpointCpy(&tmpJp,&hprime[32*4+31]);
    // Jpoint_from_mon(&tmpJp);
    // Jpoint2Apoint(&tmpJp,&tmpJp);
    // h_print_pointJ(&tmpJp);


    gettimeofday(&s5,NULL);

    gpu_commitG1();

    hipDeviceSynchronize();
    gettimeofday(&e5,NULL);

    // Jpoint hres[N_DATA];
    // CUDA_SAFE_CALL(hipMemcpy(hres,d_tmpJ,sizeof(Jpoint)*N_DATA,hipMemcpyDeviceToHost)) ;
    // for(int i=0;i<N_DATA;i++){
    //     printf("G^t' * H^taux [%d]=\n",i);
    //     Jpoint_from_mon(&hres[i]);
    //     Jpoint2Apoint(&hres[i],&hres[i]);
    //     h_print_pointJ(&hres[i]);
    // }

    
    gettimeofday(&s6,NULL);   

    gpu_rhs65();

    hipDeviceSynchronize();
    gettimeofday(&e6,NULL);

    
    // Jpoint lh65;
    // CUDA_SAFE_CALL(hipMemcpy(&hres,d_tmpJ+N_DATA,sizeof(Jpoint)*N_DATA,hipMemcpyDeviceToHost)) ;
    // for(int i=0;i<N_DATA;i++){
    //     printf("lh65 [%d]=\n",i);
    //     Jpoint_from_mon(&hres[i]);
    //     Jpoint2Apoint(&hres[i],&hres[i]);
    //     h_print_pointJ(&hres[i]);
    // }
    // printf("lh65=\n");
    // h_print_pointJ(&lh65);
    // Jpoint_from_mon(&lh65);

    // printf("lh65=\n");
    // h_print_pointJ(&lh65);
    
    gettimeofday(&s7,NULL);

    gpu_calP();

    hipDeviceSynchronize();
    gettimeofday(&e7,NULL);

    // CUDA_SAFE_CALL(hipMemcpy(&hres,d_tmpJ+N_DATA*2,sizeof(Jpoint)*N_DATA,hipMemcpyDeviceToHost)) ;
    // for(int i=0;i<N_DATA;i++){
    //     printf("66P [%d]=\n",i);
    //     // Jpoint_from_mon(&hres[i]);
    //     Jpoint2Apoint(&hres[i],&hres[i]);
    //     h_print_pointJ(&hres[i]);
    // }

    
    gettimeofday(&s8,NULL);

    gpu_calP67();

    hipDeviceSynchronize();
    gettimeofday(&e8,NULL);

    // Jpoint hres[N_DATA];
    // CUDA_SAFE_CALL(hipMemcpy(&hres,d_tmpJ+N_DATA*3,sizeof(Jpoint)*N_DATA,hipMemcpyDeviceToHost)) ;
    // for(int i=0;i<N_DATA;i++){
    //     printf("67P [%d]=\n",i);
    //     // Jpoint_from_mon(&hres[i]);
    //     Jpoint2Apoint(&hres[i],&hres[i]);
    //     h_print_pointJ(&hres[i]);
    // }
    hipDeviceSynchronize();
    gettimeofday(&s10,NULL);
    gpu_ipverify();
    hipDeviceSynchronize();
    gettimeofday(&e10,NULL);
    
    gettimeofday(&s9,NULL);

    gpu_calip68();

    hipDeviceSynchronize();
    gettimeofday(&e9,NULL);

    UINT64 htpres[256*N_DATA];
    CUDA_SAFE_CALL(hipMemcpy(&htpres,d_bLR,sizeof(UINT64)*N_DATA*256,hipMemcpyDeviceToHost)) ;
    // for(int i=0;i<N_DATA;i++){
    //     printf("68ip [%d]=\n",i);
    //     // Jpoint_from_mon(&hres[i]);
    //     dh_mybig_monmult_64_modN(&htpres[i*256],h_ONE,&htpres[i*256]);
    //     h_mybig_print(&htpres[i*256]);
    // }

    

    
    
    
    long long time_use;
    time_use=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("t1 is %llu\n",time_use);
    time_use=(e2.tv_sec-s2.tv_sec)*1000000+(e2.tv_usec-s2.tv_usec);//微秒
    printf("t2 is %llu\n",time_use);
    time_use=(e3.tv_sec-s3.tv_sec)*1000000+(e3.tv_usec-s3.tv_usec);//微秒
    printf("t3 is %llu\n",time_use);
    time_use=(e4.tv_sec-s4.tv_sec)*1000000+(e4.tv_usec-s4.tv_usec);//微秒
    printf("t4 is %llu\n",time_use);
    time_use=(e5.tv_sec-s5.tv_sec)*1000000+(e5.tv_usec-s5.tv_usec);//微秒
    printf("t5 is %llu\n",time_use);
    time_use=(e6.tv_sec-s6.tv_sec)*1000000+(e6.tv_usec-s6.tv_usec);//微秒
    printf("t6 is %llu\n",time_use);
    time_use=(e7.tv_sec-s7.tv_sec)*1000000+(e7.tv_usec-s7.tv_usec);//微秒
    printf("t7 is %llu\n",time_use);
    time_use=(e8.tv_sec-s8.tv_sec)*1000000+(e8.tv_usec-s8.tv_usec);//微秒
    printf("t8 is %llu\n",time_use);
    time_use=(e9.tv_sec-s9.tv_sec)*1000000+(e9.tv_usec-s9.tv_usec);//微秒
    printf("t9 is %llu\n",time_use);
    time_use=(e10.tv_sec-s10.tv_sec)*1000000+(e10.tv_usec-s10.tv_usec);//微秒
    printf("t10 is %llu\n",time_use);
    
    time_use=(e9.tv_sec-s1.tv_sec)*1000000+(e9.tv_usec-s1.tv_usec);//微秒
    printf("all is %llu\n",time_use);
    return 0;
    

    std::cout<<"Hello world!"<<std::endl;
}
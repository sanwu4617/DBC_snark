#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ag_gpuec256.h"
// typedef unsigned long long UINT64; //定义64位字类型
// typedef long long INT64;
// typedef unsigned int UINT32;
// // 仿射点构造
// typedef struct Affine_point{
// 	UINT64 x[8];
// 	UINT64 y[8];
// }Apoint;

// // 射影点构造
// typedef struct Jacobi_point{
// 	UINT64 x[8];
// 	UINT64 y[8];
// 	UINT64 z[8];
// }Jpoint;


// 在tesla C2050上目前这组参数测得效率最高，不可修改。
#define PARAL 64
#define BLOCKNUM (14*8)
#define BLOCKSIZE 32
#define THREADNUM (BLOCKNUM*BLOCKSIZE)


// 定义__global__类型的变量，存放16比特表。
#define d_BIN_WINDOW_16 16 //16比特表
#define d_ROWS_16 32
#define d_COLS_16 (1L<<d_BIN_WINDOW_16)


#define HANDLE_ERROR( err ) { if (err != hipSuccess) { \
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), __FILE__, __LINE__ );\
	  exit( EXIT_FAILURE ); }  \
}

//版本为512比特曲线规模，可自行修改为256比特版本
//该版本曲线所在素域整数为特殊素数
//h_ONE是host用的域元素1
//dc_ONE是gpu卡用的域元素1
//h_mon_ONE和dc_mon_ONE相等，分别定义host和gpu卡所用的蒙哥马利域上的1
//h_p和dc_p是512比特素数
//Pa0到Pa7是将素数分开定义为宏定义

// const UINT64 h_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// const UINT64 h_mon_ONE[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000L,0x0000000000000000L,0x0000000000000001L,0x0000000100000000L,0x0000000000000000L};
// const UINT64 h_p[8]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0x0L,0x0L,0x0L,0x0L};

// __constant__ UINT64 dc_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// __constant__ UINT64 dc_mon_ONE[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000L,0x0000000000000000L,0x0000000000000001L,0x0000000100000000L,0x0000000000000000L};
// __constant__ UINT64 dc_p[8]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0x0L,0x0L,0x0L,0x0L};


// const UINT64 h_ONE[4]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// const UINT64 h_mon_ONE[4]={0x1000003d1L,0x0L,0x0L,0x0L};
// const UINT64 h_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};

// __constant__ UINT64 dc_ONE[4]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// __constant__ UINT64 dc_mon_ONE[4]={0x1000003d1L,0x0L,0x0L,0x0L};
// __constant__ UINT64 dc_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};

__constant__ UINT64 dc_mon_inv_two[4]={0x0L,0x0L,0x0L,0x8000000000000000L};
const UINT64 h_mon_inv_two[4]={0x0L,0x0L,0x0L,0x8000000000000000L};

#define Pa0 0xFFFFFFFEFFFFFC2FLL //-1
#define Pa1 0xFFFFFFFFFFFFFFFFLL 
#define Pa2 0xFFFFFFFFFFFFFFFFLL //-1
#define Pa3 0xFFFFFFFFFFFFFFFFLL 
// #define Pa4 0x0 //-1
// #define Pa5 0x0 //-2
// #define Pa6 0x0 
// #define Pa7 0x0 //-1


/////////////////GPU大整数运算函数开始////////////////////////////////////////////////////

// #define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];(a)[4]=(b)[4];(a)[5]=(b)[5];(a)[6]=(b)[6];(a)[7]=(b)[7];}
#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}

//实现整大整数模加函数
__device__ __host__ void dh_mybig_modadd_64(const UINT64 *x, const UINT64 *y, UINT64 *z)//可以z=x+y, x=x+y, 不能实现y=x+y！
{
	int i;
	UINT64 f,g;
		
	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
	#else
		const UINT64 *Pa=h_p;
	#endif	
	
	z[0] = x[0] + y[0]; f = z[0] < y[0];
	z[1] = x[1] + f; g = z[1] < f; z[1] += y[1]; g += z[1] < y[1];
	z[2] = x[2] + g; f = z[2] < g; z[2] += y[2]; f += z[2] < y[2];
	z[3] = x[3] + f; g = z[3] < f; z[3] += y[3]; g += z[3] < y[3];
	
	// z[4] = x[4] + g; f = z[4] < g; z[4] += y[4]; f += z[4] < y[4];
	// z[5] = x[5] + f; g = z[5] < f; z[5] += y[5]; g += z[5] < y[5];
	// z[6] = x[6] + g; f = z[6] < g; z[6] += y[6]; f += z[6] < y[6];
	// z[7] = x[7] + f; g = z[7] < f; z[7] += y[7]; g += z[7] < y[7];
		
	if(g==0)
	{
		for(i=3;i>=0;i--)
		{
			if(z[i]!=Pa[i])
			{
				g=(z[i]>Pa[i]);
				break;
			}
			else if(i==0)//全相等,即t=P, 其实此时赋值为0即可
			{
				g=1;
			}
		}
	}
	
	if(g)//x+y可能等于模数
	{
		f = z[0] < Pa0; z[0] -= Pa0;
		g = z[1] < f; z[1] -= f; g += z[1] < Pa1; z[1] -= Pa1;
		f = z[2] < g; z[2] -= g; f += z[2] < Pa2; z[2] -= Pa2;
		z[3] -= f; z[3] -= Pa3;		                                                  
		// g = z[3] < f; z[3] -= f; g += z[3] < Pa3; z[3] -= Pa3;		
		// f = z[4] < g; z[4] -= g; f += z[4] < Pa4; z[4] -= Pa4;		                                                  
		// g = z[5] < f; z[5] -= f; g += z[5] < Pa5; z[5] -= Pa5;
		// f = z[6] < g; z[6] -= g; f += z[6] < Pa6; z[6] -= Pa6;				
		// z[7] -= f; z[7] -= Pa7;
	}
}

//实现整大整数模减函数
__device__ __host__ void dh_mybig_modsub_64(const UINT64 *x, const UINT64 *y, UINT64 *z)//可以z=x-y, x=x-y, 不能实现y=x-y！
{
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(x[0]<y[0]); z[0]=x[0]-y[0];
	g=(x[1]<f); z[1]=x[1]-f; g+=(z[1]<y[1]); z[1]-=y[1];
	f=(x[2]<g); z[2]=x[2]-g; f+=(z[2]<y[2]); z[2]-=y[2];
	g=(x[3]<f); z[3]=x[3]-f; g+=(z[3]<y[3]); z[3]-=y[3];
	
	// f=(x[4]<g); z[4]=x[4]-g; f+=(z[4]<y[4]); z[4]-=y[4];
	// g=(x[5]<f); z[5]=x[5]-f; g+=(z[5]<y[5]); z[5]-=y[5];
	// f=(x[6]<g); z[6]=x[6]-g; f+=(z[6]<y[6]); z[6]-=y[6];
	// g=(x[7]<f); z[7]=x[7]-f; g+=(z[7]<y[7]); z[7]-=y[7];
	//进位为g;

	if(g)//只判断进位，速度应该快一点
	{
		z[0]+=Pa0; f=(z[0]<Pa0);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa1; f+=(z[1]<Pa1);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa2; f+=(z[2]<Pa2);
		z[3]+=f; z[3]+=Pa3;		
		// z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		// z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		// z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		// z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		// z[7]+=f; z[7]+=Pa7;		
	}	
}

//主机汇编语言乘法，取64位乘法高位
static inline UINT64 h_Hi64(const UINT64 x, const UINT64 y)
{
   UINT64 z;   
   __asm__ __volatile__ (
	  "movq  %0,%%rax\n"   
	  "mulq %1\n"
	  "movq %%rdx,%2\n"
	  : 
	  : "m"(x),"m"(y),"m"(z)
	  : "rax","rdx","memory"
	  );      	
	return z;
}

//实现蒙哥马利乘法C=a*b
/////////////////////////////////////////
//Montgomery模乘
//C=A*B*2^-512 mod P
//采用CIOS算法
//////////////////////////////////////////
__device__ __host__  void dh_mybig_monmult_64(const UINT64 *Aa, const UINT64 *Ba, UINT64 *Ca)//pass, c=a*b
{
	UINT64 t[4+2]={0};//8个64比特整字,限定512
	
	//minv*P[0] mod 2^wordlen = -1. 因为该程序字长是64比特，P[0]=0xffffffffffffffff=-1，所以minv=1, minv*P[0]=-1 mod 2^64=-1
	
	//如果是其他P，则需要重新计算设置。这个值最好是传入，或者定义为全局变量
	// UINT64 minv=1;//minv值需要跟域特征P的最低位P[0]乘积模2^64=-1(因为P[0]是64比特字，如果是32比特字，就找到模2^32=-1那个数， 即得到域特征素数的最低字的模2^wordlen的逆的负值。
	UINT64 minv = 0xd838091dd2253531;
	UINT64 m;	
	UINT64 c,s,cin;
	int i,j;
	
	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_p;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_p;
	#endif	
	
	for(i=0;i<4;i++)
	{
		c=0;
		m=Ba[i];
		
		s=m*Aa[0]; c+=t[0];	cin=(c<t[0]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[0])+cin; 	t[0]=s;
		s=m*Aa[1]; c+=t[1];	cin=(c<t[1]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[1])+cin; 	t[1]=s;
		s=m*Aa[2]; c+=t[2];	cin=(c<t[2]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[2])+cin; 	t[2]=s;
		s=m*Aa[3]; c+=t[3];	cin=(c<t[3]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[3])+cin; 	t[3]=s;
		// s=m*Aa[4]; c+=t[4];	cin=(c<t[4]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[4])+cin; 	t[4]=s;
		// s=m*Aa[5]; c+=t[5];	cin=(c<t[5]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[5])+cin; 	t[5]=s;
		// s=m*Aa[6]; c+=t[6];	cin=(c<t[6]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[6])+cin; 	t[6]=s;
		// s=m*Aa[7]; c+=t[7];	cin=(c<t[7]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[7])+cin; 	t[7]=s;		
								
		s=t[4]+c;
		c=(s<c);
		t[4]=s;
		t[4+1]=c;
				
		m=minv*t[0];//特殊p，minv=1，等价于m=t[0]
		c=h_Hi64(m,Pa0);
		s=m*Pa0+t[0];   //因为对于这个域素数，Pa0=-1, minv=1，所以s=0;		
		c+=(s<t[0]);
		
		s=m*Pa1; c+=t[1]; cin=(c<t[1]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa1)+cin; t[0]=s;		
		s=m*Pa2; c+=t[2]; cin=(c<t[2]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa2)+cin; t[1]=s;
		s=m*Pa3; c+=t[3]; cin=(c<t[3]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa3)+cin; t[2]=s;		
	    // s=m*Pa4; c+=t[4]; cin=(c<t[4]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa4)+cin; t[3]=s;		
		// s=m*Pa5; c+=t[5]; cin=(c<t[5]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa5)+cin; t[4]=s;
		// s=m*Pa6; c+=t[6]; cin=(c<t[6]); s+=c;	cin+=(s<c); c=h_Hi64(m,Pa6)+cin; t[5]=s;		
		// s=m*Pa7; c+=t[7]; cin=(c<t[7]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa7)+cin; t[6]=s;
		
		s=t[4]+c;
		c=(s<c);
		t[4-1]=s;
		t[4]=t[4+1]+c;
	}
	
	j=(t[4]!=0);
	if(j==0)
	{
		for(i=4-1;i>=0;i--)
		{
			if(t[i]!=Pa[i])
			{				
				j=(t[i]>Pa[i]);
				break;
			}
			else if(i==0)//全相等,即t=P, 其实此时赋值为0即可
			{
				j=1;
			}
		}
	}
	
	//减法	
	if(j)
	{
		cin=1;	
		for(i=0;i<4;i++)
		{
			m=cin+(~Pa[i]);
			cin=(m<cin);
			m+=t[i];
			cin+=(m<t[i]);
			Ca[i]=m;
		}
	}
	else
	{
		for(i=0;i<4;i++)
		{
			Ca[i]=t[i];
		}
	}
	
	#ifdef __CUDA_ARCH__	
		#undef h_Hi64 
	#endif	
}

__device__ __host__  void dh_mybig_monmult_64_modN(const UINT64 *Aa, const UINT64 *Ba ,UINT64 *Ca)//pass, c=a*b
{
	UINT64 t[4+2]={0};//8个64比特整字,限定512
	
	//minv*P[0] mod 2^wordlen = -1. 因为该程序字长是64比特，P[0]=0xffffffffffffffff=-1，所以minv=1, minv*P[0]=-1 mod 2^64=-1
	
	//如果是其他P，则需要重新计算设置。这个值最好是传入，或者定义为全局变量
	// UINT64 minv=1;//minv值需要跟域特征P的最低位P[0]乘积模2^64=-1(因为P[0]是64比特字，如果是32比特字，就找到模2^32=-1那个数， 即得到域特征素数的最低字的模2^wordlen的逆的负值。
	UINT64 minv = 0x4b0dff665588b13f;
	UINT64 m;	
	UINT64 c,s,cin;
	int i,j;
	
	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_N;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_N;
	#endif	
	// const UINT64 *Pa=P; 
	
	for(i=0;i<4;i++)
	{
		c=0;
		m=Ba[i];
		
		s=m*Aa[0]; c+=t[0];	cin=(c<t[0]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[0])+cin; 	t[0]=s;
		s=m*Aa[1]; c+=t[1];	cin=(c<t[1]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[1])+cin; 	t[1]=s;
		s=m*Aa[2]; c+=t[2];	cin=(c<t[2]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[2])+cin; 	t[2]=s;
		s=m*Aa[3]; c+=t[3];	cin=(c<t[3]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[3])+cin; 	t[3]=s;
		// s=m*Aa[4]; c+=t[4];	cin=(c<t[4]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[4])+cin; 	t[4]=s;
		// s=m*Aa[5]; c+=t[5];	cin=(c<t[5]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[5])+cin; 	t[5]=s;
		// s=m*Aa[6]; c+=t[6];	cin=(c<t[6]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[6])+cin; 	t[6]=s;
		// s=m*Aa[7]; c+=t[7];	cin=(c<t[7]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[7])+cin; 	t[7]=s;		
								
		s=t[4]+c;
		c=(s<c);
		t[4]=s;
		t[4+1]=c;
				
		m=minv*t[0];//特殊p，minv=1，等价于m=t[0]
		c=h_Hi64(m,Pa[0]);
		s=m*Pa[0]+t[0];   //因为对于这个域素数，Pa0=-1, minv=1，所以s=0;		
		c+=(s<t[0]);
		
		s=m*Pa[1]; c+=t[1]; cin=(c<t[1]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa[1])+cin; t[0]=s;		
		s=m*Pa[2]; c+=t[2]; cin=(c<t[2]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa[2])+cin; t[1]=s;
		s=m*Pa[3]; c+=t[3]; cin=(c<t[3]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa[3])+cin; t[2]=s;		
	    // s=m*Pa4; c+=t[4]; cin=(c<t[4]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa4)+cin; t[3]=s;		
		// s=m*Pa5; c+=t[5]; cin=(c<t[5]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa5)+cin; t[4]=s;
		// s=m*Pa6; c+=t[6]; cin=(c<t[6]); s+=c;	cin+=(s<c); c=h_Hi64(m,Pa6)+cin; t[5]=s;		
		// s=m*Pa7; c+=t[7]; cin=(c<t[7]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa7)+cin; t[6]=s;
		
		s=t[4]+c;
		c=(s<c);
		t[4-1]=s;
		t[4]=t[4+1]+c;
	}
	
	j=(t[4]!=0);
	if(j==0)
	{
		for(i=4-1;i>=0;i--)
		{
			if(t[i]!=Pa[i])
			{				
				j=(t[i]>Pa[i]);
				break;
			}
			else if(i==0)//全相等,即t=P, 其实此时赋值为0即可
			{
				j=1;
			}
		}
	}
	
	//减法	
	if(j)
	{
		cin=1;	
		for(i=0;i<4;i++)
		{
			m=cin+(~Pa[i]);
			cin=(m<cin);
			m+=t[i];
			cin+=(m<t[i]);
			Ca[i]=m;
		}
	}
	else
	{
		for(i=0;i<4;i++)
		{
			Ca[i]=t[i];
		}
	}
	
	#ifdef __CUDA_ARCH__	
		#undef h_Hi64 
	#endif	
}



//////////////////////////////////////test inv//////////////////////////////////////
//A!=0 return 0
//A=0 return 1
__device__ __host__ int dh_mybig_iszero_64(const UINT64 *A)
{
	int i;
	for(i=0;i<4;i++)
	{
		if(A[i]!=0)	return 0;
	}
	return 1;	
}


//比较大小
//return 1	A>B
//return 0	A=B
//return -1 A<B
__device__ __host__ int dh_mybig_compare_64(const UINT64 *A, const UINT64 *B)
{
	int i;
	int flag=0;

	for(i=3;i>=0;i--)
	{
		if(A[i]>B[i])
		{
			flag=1;
			break;
		}
		else if(A[i]<B[i])
		{
			flag=-1;
			break;
		}
	}
	return (flag);
}

//C=A/2
__device__ __host__ void dh_mybig_half_64(const UINT64 *A, UINT64 *C)
{
	int i;
	UINT64 c,c1;

	c=0;
	for(i=4-1;i>=0;i--)
	{
		c1=A[i]&0x1;
		C[i]=c<<(63)|A[i]>>1;
		c=c1;
	}
}

//C=2A
__device__ __host__ UINT64 dh_mybig_double_64(const UINT64 *A, UINT64 *C)//pass
{
	int i;
	UINT64 c,c1;

	c=0;
	for(i=0;i<4;i++)
	{
		c1=A[i]>>63;
		C[i]=(C[i]<<1)|c;
		c=c1;
	}
	return c;
}

//C=A-B
__device__ __host__ void dh_mybig_sub_64(const UINT64 *A, const UINT64 *B, UINT64 *C)
{
	int i;
	UINT64 c,l,h;

	c=1;
	for(i=0;i<4;i++)
	{
		l=(~B[i])+c;
		h=(l<c);
		l+=A[i];
		c=h+(l<A[i]);
		C[i]=l;	
	}
}

//C=A+B
__device__ __host__ UINT64 dh_mybig_add_64(const UINT64 *A, const UINT64 *B, UINT64 *C)
{
	int i;
	UINT64 c,l,h;
	
	c=0;
	for(i=0;i<4;i++)
	{
		l=A[i]+c;
		h=(l<c);
		l+=B[i];
		c=h+(l<B[i]);
		C[i]=l;
	}
	return c;
}

/////////////////////////////////////////
//乘2模
//输入:		A P
//输出:		C
//C=2*A mod P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moddouble_64(const UINT64 *A, const UINT64 *P, UINT64 *C)
{
	int i,sub_en=0;
	UINT64 cin,c,temp64;

	//移位
	cin=(A[0]>>63)&0x1;//全体左移一位,相当于乘2
	C[0]=A[0]<<1;
	for(i=1;i<4;i++)
	{
		c=(A[i]>>63)&0x1;
		C[i]=(A[i]<<1)|cin;
		cin=c;
	}

	//比较大小
	if(cin==1)//最后还有一个进位为1,表明这么多字节已经存不下2*A，因此2*A肯定比p大
	{
		sub_en=1;
	}
	else
	{
		for(i=3;i>=0;i--)
		{
			if(C[i]!=P[i])
			{
				if(C[i]>P[i]) sub_en=1;
				break;
			}
			else if(i==0) sub_en=1;//就是全相等
		}
	}
	
	//减法
	if(sub_en)//这里应该写为if(sub_en!=0)即可避免问题
	{
		cin=1;
		for(i=0;i<4;i++)
		{			
			temp64=(~P[i])+cin;
			c=(temp64<cin);
			temp64+=C[i];
			cin=c+(temp64<C[i]);
			C[i]=temp64;
		}
	}
}


__device__ __host__  int dh_mybig_iszero(const UINT64 * A)
{
	// if( A[0]|A[1]|A[2]|A[3]|A[4]|A[5]|A[6]|A[7])
	// 	return 0;
	// else return 1;
	if( A[0]|A[1]|A[2]|A[3])
		return 0;
	else return 1;
}

void h_mybig_print(const UINT64 *a)
{
	int i;
	unsigned char *t=(unsigned char*) a;
	for(i=32-1;i>=0;i--) if(*(t+i)) break;//把0都跳过去
	if(i<0) printf("0");
	else
	{
		printf("%x",*(t+i)&0xff);//第一个0不打印
		for(i=i-1;i>=0;i--)printf("%02x",*(t+i)&0xff);
	}
	printf("\n");	
}

/////////////////////////////////////////
//程序功能: 计算模逆C=A^-512 * 2^512 mod P
//输入:		A C l(l为数的比特长度512)
//输出:		C (C=A^-1 * 2^512 mod P)
//说明: A<P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moninv(const UINT64 *A,UINT64 *C)//test
{
	int i,k;
	UINT64 U[4],V[4],R[4],S[4];
	int z,cp,cs,cr,sh;
	/*
	#ifdef __CUDA_ARCH__	
		UINT64 *P=dc_p;	
	#else
		const UINT64 *P=h_p;
	#endif	
	//*/

	/************************算法********************************
	Preferences: E. Savas, C.K.Koc, The Montgomery Modular
	Inverse-Revisited, IEEE TRANSACTIONS ON COMPUTERS,
	VOL. 49, NO.7, JULY 2000

	--Phase I
	U=P V=A R=0 S=1
	k=0
	while(V>0)
		if U is even then U=U/2 S=2S
		else if V is even then V=V/2 R=2R
		else if U>V then U=(U-V)/2 R=R+S S=2S
		else if V>=U then V=(V-U)/2 S=S+R R=2R
		k=k+1
	if R>=P then R=R-P
	R=P-R

	--Phase II
  R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P 
  = A^-1 * 2^l mod P
	
	return R
	*************************************************************/

	////0. initial
	//U=P V=A R=0 S=1
	// U[0]=0xffffffffffffffffL; //U=P，P的值根据需要去更换
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = Pa0;
	U[1] = Pa1;
	U[2] = Pa2;
	U[3] = Pa3;


	for(i=0;i<4;i++) V[i]=A[i];
	for(i=0;i<4;i++) R[i]=0;
	for(i=1;i<4;i++) S[i]=0;
	S[0]=0x1UL; 

	////1. phase I
	//get R=A^-1*2^k mod P
	k=0;
	z=dh_mybig_iszero_64(V);
	while(z==0)
	{	
		//printf("here k=%d  ",k);//getchar();
		cp=dh_mybig_compare_64(U,V);
		if((U[0]&0x1)==0)
		{
			dh_mybig_half_64(U,U);
			cs=dh_mybig_double_64(S,S);
		}
		else if((V[0]&0x1)==0)
		{			
			dh_mybig_half_64(V,V);
			cr=dh_mybig_double_64(R,R);
		}
		else if(cp==1)
		{
			dh_mybig_sub_64(U,V,U);
			dh_mybig_half_64(U,U);
			cr=dh_mybig_add_64(R,S,R);
			cs=dh_mybig_double_64(S,S);
		}
		else if(cp<1) //if(cp==0 || cp==-1)
		{
			dh_mybig_sub_64(V,U,V);
			dh_mybig_half_64(V,V);
			cs=dh_mybig_add_64(S,R,S);
			cr=dh_mybig_double_64(R,R);
		}
		k++;
		z=dh_mybig_iszero_64(V);
		//if(cr==1) printf("\nr\n");
		//if(cs==1) printf("\ns\n");		
	}
	
	// U[0]=0xffffffffffffffffL;
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = Pa0;
	U[1] = Pa1;
	U[2] = Pa2;
	U[3] = Pa3;

	cp=dh_mybig_compare_64(R,U);
	if(cp==1 || cp==0 || cr==1)
	{
		dh_mybig_sub_64(R,U,R);
	}
	dh_mybig_sub_64(U,R,R);
	////////////////////
	//the result of phase I is R
	//R=A^-1 * 2^k mod P

    
	////2. phaseII
	//get R=A^-1 * 2^l mod P
 	// R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P = A^-1 * 2^l mod P
 	 if(k>256)
	 {	
		// printf("case 1\n");
	    for(i=0;i<4;i++) V[i]=0;
	    V[(int)((512-k)/64)]=(((UINT64)1)<<((int)((512-k)%64)));
	    dh_mybig_monmult_64(R,V,C);
	 }	
	 else if(k==256) 
	 {
			printf("case 2\n");
			// V[0]=0x0000000000000001L;
			// V[1]=0x0000000100000000L;
			// V[2]=0x0000000000000000L;
			// V[3]=0x0000000100000000L;
			// V[4]=0x0000000000000000L;
			// V[5]=0x0000000000000001L;
			// V[6]=0x0000000100000000L;
			// V[7]=0x0000000000000000L;
			// dh_mybig_monmult_64(R,V,C);		
			for(i=0;i<4;i++) C[i]=R[i];	
		}
		else if(k<256) 
		{
		  printf("case 3\n");
		  for(i=1;i<=(256-k);i++)
			{		
				dh_mybig_moddouble_64(R,U,R);			
			}
		  for(i=0;i<4;i++) C[i]=R[i];
		}

}

/////////////////////////////////////////
//程序功能: 计算模逆C=A^-512 * 2^512 mod P 可以指定P
//输入:		A C l(l为数的比特长度512)
//输出:		C (C=A^-1 * 2^512 mod P)
//说明: A<P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moninv_modN(const UINT64 *A,UINT64 *C)//test
{
	int i,k;
	UINT64 U[4],V[4],R[4],S[4];
	int z,cp,cs,cr,sh;
	/*
	#ifdef __CUDA_ARCH__	
		UINT64 *P=dc_p;	
	#else
		const UINT64 *P=h_p;
	#endif	
	//*/

	/************************算法********************************
	Preferences: E. Savas, C.K.Koc, The Montgomery Modular
	Inverse-Revisited, IEEE TRANSACTIONS ON COMPUTERS,
	VOL. 49, NO.7, JULY 2000

	--Phase I
	U=P V=A R=0 S=1
	k=0
	while(V>0)
		if U is even then U=U/2 S=2S
		else if V is even then V=V/2 R=2R
		else if U>V then U=(U-V)/2 R=R+S S=2S
		else if V>=U then V=(V-U)/2 S=S+R R=2R
		k=k+1
	if R>=P then R=R-P
	R=P-R

	--Phase II
  R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P 
  = A^-1 * 2^l mod P
	
	return R
	*************************************************************/

	////0. initial
	//U=P V=A R=0 S=1
	// U[0]=0xffffffffffffffffL; //U=P，P的值根据需要去更换
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = 0xBFD25E8CD0364141L;
	U[1] = 0xBAAEDCE6AF48A03BL;
	U[2] = 0xFFFFFFFFFFFFFFFEL;
	U[3] = 0xFFFFFFFFFFFFFFFFL;


	for(i=0;i<4;i++) V[i]=A[i];
	for(i=0;i<4;i++) R[i]=0;
	for(i=1;i<4;i++) S[i]=0;
	S[0]=0x1UL; 

	////1. phase I
	//get R=A^-1*2^k mod P
	k=0;
	z=dh_mybig_iszero_64(V);
	while(z==0)
	{	
		//printf("here k=%d  ",k);//getchar();
		cp=dh_mybig_compare_64(U,V);
		if((U[0]&0x1)==0)
		{
			dh_mybig_half_64(U,U);
			cs=dh_mybig_double_64(S,S);
		}
		else if((V[0]&0x1)==0)
		{			
			dh_mybig_half_64(V,V);
			cr=dh_mybig_double_64(R,R);
		}
		else if(cp==1)
		{
			dh_mybig_sub_64(U,V,U);
			dh_mybig_half_64(U,U);
			cr=dh_mybig_add_64(R,S,R);
			cs=dh_mybig_double_64(S,S);
		}
		else if(cp<1) //if(cp==0 || cp==-1)
		{
			dh_mybig_sub_64(V,U,V);
			dh_mybig_half_64(V,V);
			cs=dh_mybig_add_64(S,R,S);
			cr=dh_mybig_double_64(R,R);
		}
		k++;
		z=dh_mybig_iszero_64(V);
		//if(cr==1) printf("\nr\n");
		//if(cs==1) printf("\ns\n");		
	}
	
	// U[0]=0xffffffffffffffffL;
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = 0xBFD25E8CD0364141L;
	U[1] = 0xBAAEDCE6AF48A03BL;
	U[2] = 0xFFFFFFFFFFFFFFFEL;
	U[3] = 0xFFFFFFFFFFFFFFFFL;

	cp=dh_mybig_compare_64(R,U);
	if(cp==1 || cp==0 || cr==1)
	{
		dh_mybig_sub_64(R,U,R);
	}
	dh_mybig_sub_64(U,R,R);
	////////////////////
	//the result of phase I is R
	//R=A^-1 * 2^k mod P

    
	////2. phaseII
	//get R=A^-1 * 2^l mod P
 	// R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P = A^-1 * 2^l mod P
 	 if(k>256)
	 {	
		// printf("case 1\n");
	    for(i=0;i<4;i++) V[i]=0;
	    V[(int)((512-k)/64)]=(((UINT64)1)<<((int)((512-k)%64)));
	    dh_mybig_monmult_64_modN(R,V,C);
	 }	
	 else if(k==256) 
	 {
			printf("case 2\n");
			// V[0]=0x0000000000000001L;
			// V[1]=0x0000000100000000L;
			// V[2]=0x0000000000000000L;
			// V[3]=0x0000000100000000L;
			// V[4]=0x0000000000000000L;
			// V[5]=0x0000000000000001L;
			// V[6]=0x0000000100000000L;
			// V[7]=0x0000000000000000L;
			// dh_mybig_monmult_64(R,V,C);		
			for(i=0;i<4;i++) C[i]=R[i];	
		}
		else if(k<256) 
		{
		  printf("case 3\n");
		  for(i=1;i<=(256-k);i++)
			{		
				dh_mybig_moddouble_64(R,U,R);			
			}
		  for(i=0;i<4;i++) C[i]=R[i];
		}

}

//GAO: mod exp
//C = A^k mod P
__device__ __host__ void dh_mybig_modexp(UINT64* a,UINT64 *k,UINT64* c){
	//gyy
	int i,j;
	UINT64 tbn[4];
	// Jpoint t2;
	

	// find first 1
	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if((k[i]>>j)&0x01!=0) break;
	}
	
	dh_mybig_copy(tbn,a);

	j--;


		
		for(;j>=0;j--){
				// printf("double\n");
			dh_mybig_monmult_64(tbn,tbn,tbn);
			// ppoint_double(pt1,pt1);
			if((k[i]>>j)&0x01==1){
					// printf("add\n");
				dh_mybig_monmult_64(tbn,a,tbn);
				// dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				// ppoint_double(pt1,pt1);
				dh_mybig_monmult_64(tbn,tbn,tbn);
				if((k[i]>>j)&0x01==1){
						// printf("add\n");
					// dh_ellipticAdd_JJ(pt1,&tp,pt1);
					dh_mybig_monmult_64(tbn,a,tbn);
				}
			}
		}

		dh_mybig_copy(c,tbn);
	
	// printf("j=%d,i=%d\n",j,i);
	
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}


//////////////////////////////////////test inv end//////////////////////////////////////

/////////////////GPU大整数运算函数结束////////////////////////////////////////////////////


/////////////////GPU点加和倍点函数开始，没有写全的点加函数，可以利用上述大数运算，自己根据点加公式补充///////////////////////////////////////
//GAO:未修改
__device__ __host__  void dh_setzero_J(Jpoint *pt)
{
	int i;
	for(i=0;i<8;i++) pt->z[i]=0UL;
}
//GAO:未修改
__device__ __host__  void dh_setzero_A(Apoint *pt)
{
	int i;
	for(i=0;i<8;i++) pt->x[i]=0UL;	
}

__device__ __host__  int dh_iszero_J(const Jpoint *pt)
{
	// if((pt->z[0]|pt->z[1]|pt->z[2]|pt->z[3]|pt->z[4]|pt->z[5]|pt->z[6]|pt->z[7]) == 0UL)	return 1;
	// return 0;	
	if((pt->z[0]|pt->z[1]|pt->z[2]|pt->z[3]) == 0UL)	return 1;
	return 0;
}


__device__ __host__  int dh_iszero_A(const Apoint *pt)
{	
	// if((pt->x[0]|pt->x[1]|pt->x[2]|pt->x[3]|pt->x[4]|pt->x[5]|pt->x[6]|pt->x[7]) == 0UL)	return 1;
	// return 0;
	if((pt->x[0]|pt->x[1]|pt->x[2]|pt->x[3]) == 0UL)	return 1;
	return 0;
}

//GAO:未修改
//将射影点坐标转换为仿射坐标
__device__ __host__  void dh_normlize_J(Jpoint *pt)
{	
	UINT64 tmp[8],invtmp[8];
	
	if(dh_iszero_J(pt))	return;
	
	tmp[0]=0x1L; tmp[1]=0x0L; tmp[2]=0x0L; tmp[3]=0x0L; tmp[4]=0x0L; tmp[5]=0x0L; tmp[6]=0x0L; tmp[7]=0x0L;
	
	
	dh_mybig_monmult_64(pt->z,tmp, invtmp);
	dh_mybig_moninv(invtmp, invtmp);
	//invMod(invtmp,pt->z,mod);//invtmp=1/z
	
	//squareMod(tmp,invtmp,mod);//tmp=1/zz
	dh_mybig_monmult_64(invtmp,invtmp, tmp);
	
	//productMod(pt->x,pt->x,tmp,mod);//x/zz
	dh_mybig_monmult_64(pt->x,tmp, pt->x);		
	
	//productMod(tmp,tmp,invtmp,mod);//1/zzz
	dh_mybig_monmult_64(tmp,invtmp, tmp);
	
	//productMod(pt->y,pt->y,tmp,mod);//y/zzz
	dh_mybig_monmult_64(pt->y,tmp, pt->y);
	
	//z=mon_one
	pt->z[0]=0x0000000000000001L;
	pt->z[1]=0x0000000100000000L;
	pt->z[2]=0x0000000000000000L;
	pt->z[3]=0x0000000100000000L;
	pt->z[4]=0x0000000000000000L;
	pt->z[5]=0x0000000000000001L;
	pt->z[6]=0x0000000100000000L;
	pt->z[7]=0x0000000000000000L;	
	
}
//GAO:未修改
__device__ void d_mon2normal_J(Jpoint *pt)
{
	dh_mybig_monmult_64(pt->x, dc_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, dc_ONE, pt->y);
	dh_mybig_monmult_64(pt->z, dc_ONE, pt->z);	
}
//GAO:未修改
__device__ __host__  void dh_mon2normal_A(Apoint *pt)
{
	#ifdef __CUDA_ARCH__
	dh_mybig_monmult_64(pt->x, dc_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, dc_ONE, pt->y);
	#else
	dh_mybig_monmult_64(pt->x, h_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, h_ONE, pt->y);
	#endif
}

void h_print_pointA(const Apoint *pt)
{
	if(dh_iszero_A(pt))
	{ 
		printf("(Infinity)\n");
		return;
	}	
	printf("x: ");h_mybig_print(pt->x);
	printf("y: ");h_mybig_print(pt->y);
	
}

void h_print_pointJ(const Jpoint *pt)
{
	/*
	if(dh_iszero_J(pt))
	{ 
		printf("(Infinity)\n");
		return;
	}*/
	printf("x: ");h_mybig_print(pt->x);
	printf("y: ");h_mybig_print(pt->y);
	printf("z: ");h_mybig_print(pt->z);
}

__device__ __host__ void ppoint_double(Jpoint *pt1,Jpoint* pt2){
	UINT64 u1[4],u2[4],u3[4];

	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
	#else
		const UINT64 *Pa=h_p;
	#endif	

	//这里不判断是否为无穷远
	//secp256k1中a为0，如果需要用其他的曲线需要修改
	dh_mybig_moddouble_64(pt1->y,Pa,u1); 	//u1=2y
	dh_mybig_monmult_64(pt1->z,u1,pt2->z);		//z=u1*z=2yz

	dh_mybig_monmult_64(pt1->x,pt1->x,u2);	//u2=x^2
	dh_mybig_moddouble_64(u2,Pa,u3);		//u3=2*u2=2x^2
	dh_mybig_modadd_64(u3,u2,u3);			//u3=u3+u2 = 3x^2 = lambda_1
	dh_mybig_monmult_64(u1,pt1->y,u1);		//u1 = u1*y = 2y^2
	dh_mybig_monmult_64(u1,pt1->x,u2);		//u2 = u1*x = 2xy^2 
	dh_mybig_moddouble_64(u2,Pa,u2);		//u2 = 2*u1 = 4xy^2= lambda_2
	dh_mybig_monmult_64(u3,u3,pt1->x);		//pt1x = lambda_1^2
	dh_mybig_moddouble_64(u2,Pa,pt1->y);		//pt1y = 2*u2 = 2*lambda_2
	dh_mybig_modsub_64(pt1->x,pt1->y,pt2->x);	//x = pt1x-pt1y = lambda_1^2-2*labmda_2

	dh_mybig_monmult_64(u1,u1,u1);			//u1 = u1*u1 = 4y^4;
	dh_mybig_moddouble_64(u1,Pa,u1);		//u1 = 2u1 = 8y^4 = lambda_3

	dh_mybig_modsub_64(u2,pt2->x,u2);		//u2 = u2-pt1x = lambda2-pt1x
	dh_mybig_monmult_64(u3,u2,pt1->y);		//pt1y = u2*u3 = Lambda_1 * (lambda2-pt1x)
	dh_mybig_modsub_64(pt1->y,u1,pt2->y);	//y = pt1y - labmda_3;
}

__device__ __host__ void dh_ellipticAdd_JJ(Jpoint *pt1, Jpoint* pt2,Jpoint* pt3){
	UINT64 u1[4],u2[4],u3[4],u4[4];
	//u1=z2^2

	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
		const UINT64 *mon_inv_two = dc_mon_inv_two;

	#else
		const UINT64 *Pa=h_p;
		const UINT64 *mon_inv_two = h_mon_inv_two;

	#endif	

	dh_mybig_monmult_64(pt2->z,pt2->z,u1);

	//u2=z1^2
	dh_mybig_monmult_64(pt1->z,pt1->z,u2);

	//u3 = u1*x1=x1*z2^2=lam1
	dh_mybig_monmult_64(pt1->x,u1,u3);

	//u4 = u2*x2 = x2*z1^2=lam2
	dh_mybig_monmult_64(pt2->x,u2,u4);

	//u1 = z2^3
	dh_mybig_monmult_64(u1,pt2->z,u1);

	//u2 = z1^3
	dh_mybig_monmult_64(u2,pt1->z,u2);

	//u1 = u1*y1 = lam4
	dh_mybig_monmult_64(u1,pt1->y,u1);

	//u2 = u2*y2 = lam5
	dh_mybig_monmult_64(u2,pt2->y,u2);

	//pt3y = u3-u4 = lam3
	dh_mybig_modsub_64(u3,u4,pt3->y);

	//u3+=u4 = lam1+lam2 = lam7
	dh_mybig_modadd_64(u3,u4,u3);

	//U4=u1-u2=lam4-lam5=lam6
	dh_mybig_modsub_64(u1,u2,u4);

	//u1+=u2 = lam4+lam5=lam8
	dh_mybig_modadd_64(u1,u2,u1);

	//pt3z = z1*z2*lam3
	dh_mybig_monmult_64(pt1->z,pt2->z,pt3->z);
	dh_mybig_monmult_64(pt3->z,pt3->y,pt3->z);

	//pt3x = u4^2 = lam6^2
	dh_mybig_monmult_64(u4,u4,pt3->x);
	//u2 = pt3y^2 = lam3^2
	dh_mybig_monmult_64(pt3->y,pt3->y,u2);

	//pt3y = pt3y*u2 = lam3^3
	dh_mybig_monmult_64(pt3->y,u2,pt3->y);

	//u2 = u2*u3 = lam7*lam3^2
	dh_mybig_monmult_64(u2,u3,u2);

	//pt3x -= u2 = lam6^2-lam7*lam3^2
	dh_mybig_modsub_64(pt3->x,u2,pt3->x);

	//u3 = 2pt3x
	dh_mybig_moddouble_64(pt3->x,Pa,u3);
	//u2-=u3=lam7lam3^2-2pt3x=lam9
	dh_mybig_modsub_64(u2,u3,u2);

	//u1 *= pt3y = lam8*lam3^3
	dh_mybig_monmult_64(u1,pt3->y,u1);

	//pt3y = u2*u4 = lam9*lam6
	dh_mybig_monmult_64(u2,u4,pt3->y);

	//pt3y-=u1
	dh_mybig_modsub_64(pt3->y,u1,pt3->y);

	//pt3y/=2
	dh_mybig_monmult_64(pt3->y,mon_inv_two,pt3->y);
	// dh_mybig_half_64(pt3->y,pt3->y);

}
//射影点pt1加等放射点pt2, pt1+=pt2
__device__ __host__ void dh_ellipticSumEqual_AJ(Jpoint *pt1, Apoint* pt2)//pt1,pt2必须保证非无穷远点，在函数中无判断
{
	UINT64 u1[4],u2[4];
	if(dh_iszero_A(pt2))	return;	
	if(dh_iszero_J(pt1))
	{
		dh_mybig_copy(pt1->x, pt2->x);
		dh_mybig_copy(pt1->y, pt2->y);
		//Z赋值为mon_ONE，这个值需要根据p重新设置
		//0x1000003d1L,0x0L,0x0L,0x0L
		pt1->z[0]=0x1000003d1L;		pt1->z[1]=0x0L;		pt1->z[2]=0x0L;		pt1->z[3]=0x0L;
		// pt1->z[4]=0x0000000000000000L;		pt1->z[5]=0x0000000000000001L;		pt1->z[6]=0x0000000100000000L;		pt1->z[7]=0x0000000000000000L;				
		return;
	}	
	//可在程序中判断，禁止加仿射坐标的无穷远点
	
	
	//3.计算u1=(pt1->z)^2.
	dh_mybig_monmult_64(pt1->z, pt1->z, u1);
	
	//4.计算u2=(pt1->z)*u1.
	dh_mybig_monmult_64(pt1->z, u1, u2);
	
	//5.计算u1=(pt2->x)*u1.
	dh_mybig_monmult_64(pt2->x, u1, u1);
	
	//6.计算u2=(pt2->y)*u2.
	dh_mybig_monmult_64(pt2->y, u2, u2);
	
	//7.计算u1=u1-pt1->x.		
	dh_mybig_modsub_64(u1, pt1->x,u1);
	
	//8.计算u2=u2-pt1->y.	
	dh_mybig_modsub_64(u2,pt1->y,u2);
	
	///*	
	//9.判断等点,需要调用二倍点程序.
	if(dh_mybig_iszero(u1))
	{
		if(dh_mybig_iszero(u2))
		{
			//GAO:这里自己添加了二倍点函数
			ppoint_double(pt1,pt1);//y坐标也相同，返回二倍点
			printf("here! use ppoint double!\n");
			
			return;
		}
		else//正负点相加，返回无穷远点
		{
			dh_setzero_J(pt1);
			return ;
		}
	}
	//*/
	//10.pt1->z=pt1->z*u1.
	dh_mybig_monmult_64(pt1->z, u1, pt1->z);
	
	//11.计算pt2->x=u1^2.
	dh_mybig_monmult_64(u1, u1, pt2->x);
	
	//12.计算pt2->y=pt2->x*u1.
	dh_mybig_monmult_64(u1, pt2->x, pt2->y);
	
	//13.计算pt2->x=pt1->x*pt2->x.
	dh_mybig_monmult_64(pt1->x, pt2->x, pt2->x);
	
	//14.计算u1=2*pt2->x.
	dh_mybig_modadd_64(pt2->x,pt2->x,u1);
	
	//15.x1=u2^2.
	dh_mybig_monmult_64(u2, u2, pt1->x);
	
	//16.x1=pt2->x
	dh_mybig_modsub_64(pt1->x,u1,pt1->x);
	
	//17.x1=x1-pt2->y
	dh_mybig_modsub_64(pt1->x,pt2->y,pt1->x);
	
	//18.计算pt2->x=pt2->x-x1.
	dh_mybig_modsub_64(pt2->x,pt1->x,pt2->x);
	
	//19.pt2->x=pt2->x*u2
	dh_mybig_monmult_64(pt2->x, u2, pt2->x);
	
	//20.pt2->y=pt2->y*y1
	dh_mybig_monmult_64(pt2->y, pt1->y, pt2->y);
	
	//21.y1=pt2->x-pt2->y
	dh_mybig_modsub_64(pt2->x,pt2->y,pt1->y);
}

__device__ __host__ void dh_point_mult_inplace(Jpoint* pt1,UINT64 *k){
	//gyy
	int i,j;
	Jpoint tp;
	Jpoint t2;
	//找到第一个1的位置

	//testcode
	// dh_mybig_copy(tp.x,pt1->x);
	// dh_mybig_copy(tp.y,pt1->y);
	// dh_mybig_copy(tp.z,pt1->z);



	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);

	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if((k[i]>>j)&0x01!=0) break;
	}
	
	dh_mybig_copy(tp.x,pt1->x);
	dh_mybig_copy(tp.y,pt1->y);
	dh_mybig_copy(tp.z,pt1->z);
	j--;
	// printf("j=%d,i=%d\n",j,i);
	for(;j>=0;j--){
			// printf("double\n");
		ppoint_double(pt1,pt1);
		if((k[i]>>j)&0x01==1){
				// printf("add\n");
			dh_ellipticAdd_JJ(pt1,&tp,pt1);
		}
	}
	i--;
	for(;i>=0;i--){
		for(j=63;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt1,pt1);
			if((k[i]>>j)&0x01==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
	}
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

__device__ __host__ void dh_point_mult_finalversion(Jpoint* pt1,UINT64 *k,Jpoint* pt2){
	//gyy
	int i,j;
	Jpoint tp;
	// Jpoint t2;
	
	//找到第一个1的位置

	//testcode
	// dh_mybig_copy(tp.x,pt1->x);
	// dh_mybig_copy(tp.y,pt1->y);
	// dh_mybig_copy(tp.z,pt1->z);



	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);

	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if((k[i]>>j)&0x01!=0) break;
	}
	
	dh_mybig_copy(tp.x,pt1->x);
	dh_mybig_copy(tp.y,pt1->y);
	dh_mybig_copy(tp.z,pt1->z);
	j--;

	if(pt1==pt2){
		
		for(;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt1,pt1);
			if((k[i]>>j)&0x01==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				ppoint_double(pt1,pt1);
				if((k[i]>>j)&0x01==1){
						// printf("add\n");
					dh_ellipticAdd_JJ(pt1,&tp,pt1);
				}
			}
		}
	}else{
		
		// h_print_pointJ(pt1);
		// h_print_pointJ(&tp);
		for(;j>=0;j--){
				// printf("double\n");
			ppoint_double(&tp,&tp);
			if((k[i]>>j)&0x01==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(&tp,pt1,&tp);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				ppoint_double(&tp,&tp);
				if((k[i]>>j)&0x01==1){
						// printf("add\n");
					dh_ellipticAdd_JJ(&tp,pt1,&tp);
				}
			}
		}
		// h_print_pointJ(&tp);
		dh_mybig_copy(pt2->x,tp.x);
		dh_mybig_copy(pt2->y,tp.y);
		dh_mybig_copy(pt2->z,tp.z);
	}
	// printf("j=%d,i=%d\n",j,i);
	
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

__device__ __host__ void dh_point_mult_uint32(Jpoint* pt1, int k,Jpoint* pt2){
	int i;
	Jpoint tp;
	for(i=31;i>=0;i--){
		if((k>>i)&0x01!=0) break;
	}
	dh_mybig_copy(tp.x,pt1->x);
	dh_mybig_copy(tp.y,pt1->y);
	dh_mybig_copy(tp.z,pt1->z);
	i--;

	if(pt1==pt2){
		for(;i>=0;i--){
			// printf("double\n");
			ppoint_double(pt1,pt1);
			if((k>>i)&0x01==1){
				// printf("add\n");
				dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
	}else{
		for(;i>=0;i--){
			// printf("double\n");
			ppoint_double(&tp,&tp);
			if((k>>i)&0x01==1){
				// printf("add\n");
				dh_ellipticAdd_JJ(&tp,pt1,&tp);
			}
		}
		dh_mybig_copy(pt2->x,tp.x);
		dh_mybig_copy(pt2->y,tp.y);
		dh_mybig_copy(pt2->z,tp.z);
	}

}


__device__ __host__ void dh_point_mult_outofplace(Jpoint* pt1,UINT64 *k,Jpoint* pt2){
	//gyy
	int i,j;

	//找到第一个1的位置

	//testcode
	dh_mybig_copy(pt2->x,pt1->x);
	dh_mybig_copy(pt2->y,pt1->y);
	dh_mybig_copy(pt2->z,pt1->z);



	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if((k[i]>>j)&0x01!=0) break;
	}
	

	j--;
	// printf("j=%d,i=%d\n",j,i);
	for(;j>=0;j--){
			// printf("double\n");
		ppoint_double(pt2,pt2);
		if((k[i]>>j)&0x01==1){
				// printf("add\n");
			dh_ellipticAdd_JJ(pt2,pt1,pt2);
		}
	}
	i--;
	for(;i>=0;i--){
		for(j=63;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt2,pt2);
			if((k[i]>>j)&0x01==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(pt2,pt1,pt2);
			}
		}
	}
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

//这个函数因为他自己提供的那个点加函数会更改pt2，所以不能用，除非重写之前的函数
__device__ __host__ void dh_apoint_mult(Jpoint* pt1,Apoint* pt2,UINT64 *k){
	
	dh_mybig_copy(pt1->x,pt2->x);
	dh_mybig_copy(pt1->y,pt2->y);
	dh_mybig_copy(pt1->z,dc_mon_ONE);
	
	ppoint_double(pt1,pt1);
	dh_ellipticSumEqual_AJ(pt1,pt2);
	ppoint_double(pt1,pt1);
	// dh_ellipticSumEqual_AJ(pt1,pt2);
	return;
	//gyy
	int i,j;
	// Jpoint tp;

	//找到第一个1的位置

	//testcode

	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if((k[i]>>j)&0x01!=0) break;
	}
	
	dh_mybig_copy(pt1->x,pt2->x);
	dh_mybig_copy(pt1->y,pt2->y);
	dh_mybig_copy(pt1->z,dc_mon_ONE);
	j--;
	
	printf("j=%d,i=%d\n",j,i);
	// return;
	for(;j>=0;j--){
			// printf("double\n");
		ppoint_double(pt1,pt1);
		if((k[i]>>j)&0x01==1){
				// printf("add\n");
			dh_ellipticSumEqual_AJ(pt1,pt2);
		}
	}
	i--;
	for(;i>=0;i--){
		for(j=63;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt1,pt1);
			if((k[i]>>j)&0x01==1){
					// printf("add\n");
					dh_ellipticSumEqual_AJ(pt1,pt2);
			}
		}
	}
	printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

//GAO:未修改
//批量求逆
__device__ void d_multi_inverse(UINT64 *x)
{
	int i,j;
	UINT64 invlj[PARAL*8];//寄存器每个小核可以分4KB字节，4096=64*256,可以有256个512比特数
	UINT64 lmd[8];
	
	dh_mybig_copy(invlj,x);//	for(i=0;i<8;i++) invlj[0][i]=x[0][i];
		
	for(i=1;i<PARAL;i++)
	{			    					
		dh_mybig_monmult_64(invlj+(i-1)*8,x+i*8,invlj+i*8);
	}
		
	dh_mybig_monmult_64(invlj+(PARAL-1)*8,dc_ONE,lmd);//z=Z mod P //先去掉蒙哥马利
	dh_mybig_moninv(lmd,lmd); //求逆的时候又戴上2^n了
		
	for(i=PARAL-1;i>0;i--)
	{
		dh_mybig_monmult_64(invlj+(i-1)*8,lmd,invlj+i*8);
		dh_mybig_monmult_64(x+i*8,lmd,lmd);
		dh_mybig_copy(x+i*8,invlj+i*8);
		//for(j=0;j<8;j++) x[i*8+j]=invlj[i*8+j];	
	}	
	dh_mybig_copy(x,lmd);
	//for(j=0;j<8;j++) x[j]=lmd[j];	
}
//GAO:未修改
//批量正规化射影点
__global__ void d_multi_normlize_J(Jpoint *A, int n)
{	
	//参数n必须为PARAL的整数倍
	
	int i,j,k;
	UINT64 tmp[8];
	int mytid=threadIdx.x+blockDim.x*blockIdx.x;
	int threadnum=blockDim.x*gridDim.x;
	UINT64 z[8*PARAL];//此处z用作伪并行求逆
			
	for(i=mytid*PARAL;i<n;i+=threadnum*PARAL) //n个按PARAL个连续的一段，每个小核做一段，循环做下去
	{
		for(j=0;j<PARAL;j++)
		{
			for(k=0;k<8;k++)
				z[j*8+k]=A[i+j].z[k];
		}
				
		d_multi_inverse(z);
		
		for(j=0;j<PARAL;j++)
		{
			dh_mybig_monmult_64(z+j*8,z+j*8, tmp);
			dh_mybig_monmult_64(A[i+j].x,tmp, A[i+j].x);		
			dh_mybig_monmult_64(tmp,z+j*8, tmp);
			dh_mybig_monmult_64(A[i+j].y,tmp, A[i+j].y);

			//z初始化为mon_ONE，这个mon_ONE跟字长有关
			A[i+j].z[0]=0x0000000000000001L;	A[i+j].z[1]=0x0000000100000000L;	A[i+j].z[2]=0x0000000000000000L;	A[i+j].z[3]=0x0000000100000000L;
			A[i+j].z[4]=0x0000000000000000L;	A[i+j].z[5]=0x0000000000000001L;	A[i+j].z[6]=0x0000000100000000L;	A[i+j].z[7]=0x0000000000000000L;	
			
			d_mon2normal_J(A+i+j);
		}
	}
	
}

/////////////////GPU点加和倍点函数结束////////////////////////////////////////////////////

__global__ void d_get_para(UINT64 *para)
{
	int i;
	UINT64 *pt64;
	
	if(threadIdx.x==0 && blockIdx.x==blockDim.x-1)
	{
		pt64=para+  0; for(i=0;i<8;i++) pt64[i]=dc_p[i];
		pt64=para+ 56; for(i=0;i<8;i++) pt64[i]=dc_ONE[i];
		pt64=para+ 64; for(i=0;i<8;i++) pt64[i]=dc_mon_ONE[i];						
	}	
}

void h_print_para()
{
	int groups=11;
	UINT64 *testdata=(UINT64 *)malloc(groups*8*8);	
	UINT64 *d_testdata;
	
	HANDLE_ERROR( hipMalloc((void**)&d_testdata, groups*8*8) );
	
	d_get_para<<<BLOCKNUM,BLOCKSIZE>>>(d_testdata);
	
	HANDLE_ERROR( hipMemcpy( testdata, d_testdata,	64*groups, hipMemcpyDeviceToHost));
	printf("\n__const__ UINT64 testdc_p[8]=");h_mybig_print(testdata);
	printf("\n__const__ UINT64 testdc_ONE[8]=");h_mybig_print(testdata+56);
	printf("\n__const__ UINT64 testdc_mon_ONE[8]=");h_mybig_print(testdata+64);
	
	free(testdata);
	HANDLE_ERROR(hipFree(d_testdata));
}



int h_get_gpu_info()
{
	  hipDeviceProp_t  prop;

    int count;
    HANDLE_ERROR( hipGetDeviceCount( &count ) );
    
    for (int i=0; i< count; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
    }

	return 1;
}

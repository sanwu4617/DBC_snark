#include "hip/hip_runtime.h"
#include<stdio.h>
#include "gpuec.h"
#include "cuda_common.h"
// typedef unsigned long long UINT64; //定义64位字类型
// typedef long long INT64;

#define N_BIGNUM 8
const UINT64 h_p[8]={0xffffffffffffffffL,0xfffffffeffffffffL,0xffffffffffffffffL,0xfffffffeffffffffL,0xffffffffffffffffL,0xfffffffffffffffeL,0xfffffffeffffffffL,0xffffffffffffffffL};
const UINT64 h_r[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000,0x0000000000000000L ,0x0000000000000001L,0x100000000L,0x0L };
__constant__ UINT64 h_R2[8]={0x0000000d00000003L,0x0000000400000000L,0x0000000000000014L,0x0000000e00000001,0x000000000000000dL ,0x000000140000000aL,0x0000000700000000L,0x18L };
__constant__ UINT64 h_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
//18 0000000700000000 000000140000000a 000000000000000d 0000000e00000001 0000000000000014 0000000400000000 0000000d00000003
void __global__ testadd(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_modadd_64(a+tx*8,b+tx*8,c+tx*8);
}
void __global__ testsub(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_modsub_64(a+tx*8,b+tx*8,c+tx*8);
}
void __global__ testmul(UINT64* a,UINT64 *b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*8,h_R2,a+tx*8);
    // dh_mybig_monmult_64(a+tx*8,h_R2,a+tx*8);
    dh_mybig_monmult_64(a+tx*8,a+tx*8,c+tx*8);
    // dh_mybig_monmult_64(c+tx*8,h_ONE,c+tx*8);
}

void print_big_arr(UINT64* nums,int n){
    for(int i=0;i<n;i++){
        h_mybig_print(nums+i*8);
        printf("\n");
    }
}

void init_big(UINT64 *nums){
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<8;j++){
            nums[i*8+j]=i;
        }
        // nums[i*8] = h_p[0]-1;
    }
    // nums[0]=25;
    // nums[15]=35;
}
void init_big2(UINT64 *nums){
    for(int i=0;i<N_BIGNUM;i++){
        // for(int j=0;j<8;j++)
        nums[i*8] = i;
    }
    // nums[0]=25;
    // nums[15]=35;
}

int main(){
    // const int N = 8;
    // h_print_para();
    UINT64 *h_nums1 = (UINT64*)malloc(sizeof(UINT64)*8*N_BIGNUM);
    UINT64 *h_nums2 = (UINT64*)malloc(sizeof(UINT64)*8*N_BIGNUM);
    UINT64 *h_nums3 = (UINT64*)malloc(sizeof(UINT64)*8*N_BIGNUM);
    init_big2(h_nums1);
    init_big2(h_nums2);

    UINT64 *d_nums1;
    UINT64 *d_nums2;
    UINT64 *d_nums3;

    CUDA_SAFE_CALL(hipMalloc((void**)&d_nums1,sizeof(UINT64)*8*N_BIGNUM));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nums2,sizeof(UINT64)*8*N_BIGNUM));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nums3,sizeof(UINT64)*8*N_BIGNUM));

    CUDA_SAFE_CALL(hipMemcpy(d_nums1,h_nums1,sizeof(UINT64)*8*N_BIGNUM,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_nums2,h_nums2,sizeof(UINT64)*8*N_BIGNUM,hipMemcpyHostToDevice));

    testmul<<<1,N_BIGNUM>>>(d_nums1,d_nums2,d_nums3);

    CUDA_SAFE_CALL(hipMemcpy(h_nums3,d_nums3,sizeof(UINT64)*8*N_BIGNUM,hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_nums1,d_nums1,sizeof(UINT64)*8*N_BIGNUM,hipMemcpyDeviceToHost));
    print_big_arr(h_nums3,N_BIGNUM);
    // print_big_arr(h_nums1,N_BIGNUM);
    // printf("\n");
    // h_mybig_print(h_nums+8);
}
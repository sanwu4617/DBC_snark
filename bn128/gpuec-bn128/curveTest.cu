#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include<random>
#include <cstring>
#include <vector>
#include "gpuec256.h"
#include "cuda_common.h"
using namespace std;
#define DEBUG
#define CONST_SCALE 10
const int N_BIGNUM = 1 << CONST_SCALE; // maxsize = 2^15, for not run out of memory.
const int BATCHSIZE = 1 << CONST_SCALE; // 
const int N_POINT = BATCHSIZE;
#define CONST_THREAD_PER_BLOCK 256
const int N_THREAD_PER_BLOCK = CONST_THREAD_PER_BLOCK;
const int N_BLOCK = ((BATCHSIZE+N_THREAD_PER_BLOCK-1)/N_THREAD_PER_BLOCK);
#define CONST_BIGNUM (1 << CONST_SCALE)
#define CONST_BATCHSIZE CONST_BIGNUM
#define CONST_BLOCK ((CONST_BATCHSIZE + CONST_THREAD_PER_BLOCK - 1) / CONST_THREAD_PER_BLOCK)

const UINT64 h_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};

const UINT64 h_Gx[4]={0x59F2815B16F81798L,0x029BFCDB2DCE28D9L,0x55A06295CE870B07L,0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4]={0x9C47D08FFB10D4B8L,0xFD17B448A6855419L,0x5DA4FBFC0E1108A8L,0x483ADA7726A3C465L};

const UINT64 testGx[4]={0x252931db128244c9L,0x80ec2e92027d7e6eL,0x32c5ee6d51cb1e89L,0xb89bd74c7352f570L};
const UINT64 testGy[4]={0xd8cbce4f20d0d9e4L,0x5b636389add7cc6eL,0xccd07463f61e7fbeL,0x13fae72c0d3c849bL};
const UINT64 testGz[4]={0xbc5c645f1b1c297dL,0x0ba1469cd0bdd88aL,0x40bad30e143dcdceL,0x4bba49beb75cce43L};

Jpoint testG = {{0x252931db128244c9L,0x80ec2e92027d7e6eL,0x32c5ee6d51cb1e89L,0xb89bd74c7352f570L}, {0xd8cbce4f20d0d9e4L,0x5b636389add7cc6eL,0xccd07463f61e7fbeL,0x13fae72c0d3c849bL}, {0xbc5c645f1b1c297dL,0x0ba1469cd0bdd88aL,0x40bad30e143dcdceL,0x4bba49beb75cce43L}};
__constant__ Jpoint testG1 = {{0xe0fab25ad59c562, 0xd0cc9fb31a35f436, 0xc5e610bafb19c056, 0x217289b9513973aa}, {0x75f74a17ffc3e93c, 0x9db49310f34754b5, 0x521011f2ca8e700e, 0x63bece3335bdbb7}, {0xd35d438dc58f0d9d, 0xa78eb28f5c70b3d, 0x666ea36f7879462c, 0xe0a77c19a07df2f}};
__constant__ Jpoint testG2 = {{0xb97158ae823a5071, 0xec3d6bb634b9f59a, 0x72216c72e40ace02, 0x4dd15b3c5d503a9}, {0xf5b36c9edbc93646, 0xd3afebebf560e7b0, 0x23872b3d1e2cba9d, 0x1e2e96a27a126dd4}, {0x9ea13e394583ef20, 0x1a16ec863a8d9d4e, 0xf4642279979430f9, 0x567cb2990c1d68d}};


//#define PRECOMPUTE // precompute test.
#define CUDA_VISIBLE_DEVICES 1

// 32 dbcs for test. to long dbc are very slow to compile.
const int SMALL_DBC_NUM = 32;

unsigned long long px[] = {0xc1291ea73f54ce04ll, 0xbaae63f186512829ll, 0x849ffc311521657all, 0xdb05b8be3bc3ea12ll, 0xec734eb90e60d146ll, 0x47e3a925f4c2f8d3ll, 0xe404a3c34bfc4331ll, 0xa09f73b213ee39abll, 0xdb13ac478ea2128ell, 0x2c53e4e8f64e797dll, 0x39b7a582468b2b26ll, 0x8a7cc04c32f86ce1ll, 0xcd358ec4bf05d09ell, 0x6b2044c68221ce4ll, 0x87a85903221d0748ll, 0x46f323835084b5a9ll, 0x46f97386edeaf803ll, 0xa49b5c87eb315db7ll, 0xe3081f1ad7d6d04all, 0x83059f0ea01a4aeell, 0xee6fba127fec7397ll, 0xaba6a4a20bad9c85ll, 0x7644585b6c958d2cll, 0x84938de81702b2a3ll, 0x1a3ad24da633218ell, 0xec8c6b7a2e2d7fd4ll, 0x5893289ae2b2d9bll, 0xe5ef285c943653aell, 0xa04bd5faef99b5cell, 0x978380dff860c313ll, 0xa993bd51570b133ell, 0xb6d827a6ffe8c506ll, 0xf3fc602ee2294187ll, 0x885972979b942caell, 0xfe36bc732ebf4a28ll, 0x44524be479e97992ll, 0xf46d14dc6fdec9fll, 0x8548cb4caa31b9d7ll, 0xeefb2bfedba2f870ll, 0xee2cf31377c30c02ll, 0x8abcb93f4f13ae56ll, 0x4b03bbfee680553ell, 0x17a359c48f5f772ell, 0x4b8590bc7ad10993ll, 0xd81ef93a1c4bc230ll, 0x3fe5e94a6137a639ll, 0xdf6937fecfe2ad57ll, 0x70c0b2bac6f886f1ll, 0x9f04236a79cf01f6ll, 0x13e1640167007df8ll, 0xf780f2e0d9a1ae0fll, 0xbf7d094c158291cfll, 0x212b875aa42d2b3cll, 0x446ebb199ab8c39fll, 0xdcaf114ec3d14cfdll, 0x2ba87b8b8d3f1f59ll, 0x9bcd364ea3325d90ll, 0x3a47a15979d76791ll, 0x587c8cc21bc825dell, 0x89c87f72dfc370fcll, 0xb0d283bbfa0e29b3ll, 0x26308aa1138b7509ll, 0xaf529fa55238314ll, 0x39816c256e503a7ell, 0x5f22c397ef9fe0all, 0xb9e5bdb683e15a95ll, 0xaaa3fb7ade2d4d88ll, 0x4d7e7ee5951e2398ll, 0x5470acdb8695857ll, 0x5166d1ad1ad03d79ll, 0xf3305c87d6eb6a66ll, 0xc176900a101104fll, 0xce06d34209f8948all, 0xd82193ffdfe2c9e5ll, 0x931075c79ce0d306ll, 0x775fb75a7642d179ll, 0x888cda266c2aad54ll, 0x40fc82795775f1fall, 0x497bad018e821b36ll, 0xa29b1ff55c8448c9ll, 0x69af3b5510579da2ll, 0x2f1fbea9ba3d7984ll, 0x774fd33a1752595cll, 0x56a3a84b5041c8ebll, 0x7cde84ea0461ba1cll, 0x603181bf3283c6d6ll, 0x62f2c535e53dcee8ll, 0x2451e9cc1fdf3d7ll, 0xda232c0afcdc6597ll, 0xeb4ba40875b6a4dall, 0x2d5f9584192f072bll, 0x3198177f0b01bcdcll, 0xacd3d97303bbd12cll, 0x8db5b517cbe942e0ll, 0x51d09eb6d2933d2ll, 0xe6d9c03b36990772ll, 0xb964889dbf78ebd5ll, 0xb9811d6fc0356d18ll, 0xa2410b8e143e5130ll, 0xcc38ce20d9807de7ll};

unsigned long long py[] = {0x8ceceeb28411ba32ll, 0x3e781792a382e71ell, 0xd3fa02dae181fc95ll, 0x2e304597764f60a6ll, 0xfce1486fcc18d692ll, 0x1375c5b6e87f2c3all, 0xb48702fc8a96ba27ll, 0x57e69125d0114f2cll, 0x362d1b3393fa7588ll, 0x78796df78872500dll, 0x9138dfc1bdc15a3dll, 0xa167f3d2414d0205ll, 0x1b9b9a5f34b8afecll, 0x583a0f67a99a3788ll, 0xc4acd28dffe1410cll, 0xf07e21417abbcc3cll, 0x3ade3c45dfbc863fll, 0x9eda6f139f31b018ll, 0xe0adb41adfca8622ll, 0x15ea15417084f16bll, 0xb3b2c55417193756ll, 0x13712c30fdc9e718ll, 0x2952caab2f05eab1ll, 0x358afc9cbd7e5192ll, 0xd536b8e7f4c507d6ll, 0x66c5a5ff171507e0ll, 0x725b2f3fb86bac7bll, 0x9e36795fd946b087ll, 0x357fcb8b68e5b458ll, 0x7766f17d2f3054d7ll, 0x8343812a93422063ll, 0xa1f100198fe5c71dll, 0x68620cb7296393a2ll, 0xd95ea31298a2c21dll, 0x77d391aa12784ca4ll, 0x8fcf1ba16eaf7c5bll, 0xc68cfe2925c1ac51ll, 0x2352bcdd153d436cll, 0xcfeeeb13fb787307ll, 0xac675e68a3bff56bll, 0xd621070cf7495c20ll, 0x31c021898ac69039ll, 0x8f0c823687669c4all, 0x144b3504c3329402ll, 0x10a0a577850a8ac9ll, 0x2a0026f19fabf26ll, 0x8b726c9485375803ll, 0xf2e492176f575e7ell, 0xbba9a542f2b6cf79ll, 0x117cb4563c3d2f96ll, 0x65b88205addecb52ll, 0x615c0a66a7d6a4dell, 0x35f4afe2701217e2ll, 0x15e4a785245be7e3ll, 0x2993bd9756922e8ell, 0x5c2ff49d4f54fda0ll, 0xa82c16071118f14fll, 0x800ca14fc42286e7ll, 0x1e5b6da6f7f886e3ll, 0x37997aecc99fc704ll, 0x6991710fc91f47e3ll, 0x9877646ac9334869ll, 0x61ac8c5c317aba27ll, 0x69b0ede62479a906ll, 0xf1d3bf52ace992d5ll, 0x1c9980570a29208all, 0x53784402059629d6ll, 0x9698f219e1c2443all, 0xf6f31dc5852bdbbbll, 0x261e0d1867a702f8ll, 0xd166e64d5d8829b9ll, 0xc77dd21b6cbdd1b6ll, 0xe69accceb5e2b4e9ll, 0x69baa5bbe515be9dll, 0xb616554bece011cdll, 0xa34be20aa7d9c90cll, 0x5e7cc1b8428e709cll, 0x5c58a6ac2c829e11ll, 0x9a3dce73b43a903ll, 0xa58d08ec4c58bd52ll, 0x7f7751124655e13ll, 0x5424aa4fdbf8b18dll, 0xc982cbae45ccce95ll, 0x9398e8c27258250dll, 0xf17129bfa5b1b510ll, 0x41fe283dfb4327afll, 0x5f2f33e7714c97dall, 0x559cf7242bb8fe8ell, 0x17b3a3bb46dd3e91ll, 0x1af7d0a9f1b04817ll, 0x6d23e0dd841a2672ll, 0x8f6d1a2693629455ll, 0xff20d2520292748dll, 0x24d2beca6756a36dll, 0x7644c94f54b6cb43ll, 0x8b5d5592833e959fll, 0xb0c016d4371e5f9ell, 0xa0acac44e0ca9561ll, 0x9d27f8c2c840e7ell, 0x73b4245bfb56fc89ll};

unsigned long long px3[] = {0x4d3c296b3e8b6988ll, 0xddcc9ca08396d1cbll, 0x7ad77e63364511dfll, 0xa0ac877d59fe43all, 0x4a87562b177eae72ll, 0xf52831b49c6a6112ll, 0xf0b0a46021a488d2ll, 0xd4575222d4cda88bll, 0x9fd2acb1e587292bll, 0xd8c007515b734f4ll, 0x86d785c545c96c0dll, 0x81b2826c2c4d2dd8ll, 0x94001024d35f4e03ll, 0xa7f67326231808c0ll, 0x49e61768c043f864ll, 0xa6bb07c8044cb754ll, 0x3bc5fce63b5c6eb7ll, 0xe975105a35c835b0ll, 0x5449ac7731bf26a6ll, 0x8bb21922e7870e9cll, 0xbf51fc11ed6b7becll, 0x90c2a7dd970b6a7all, 0x87b1d7c096ab0fd1ll, 0xef4efa6172ed7f00ll, 0x70bfa5991f8040b5ll, 0xf63d2f69ab92e571ll, 0x30f4f32599299452ll, 0xc55ac8e19c98b574ll, 0x1e7bd7e1f2a7f00all, 0x9bcc5a675c011ab3ll, 0x1b0ac2245d3f3e60ll, 0x6f91233e31a13e1cll, 0x15c3bbf222eb03b3ll, 0x39c2af9d552948efll, 0x51a6d8ded9469e3bll, 0x162d2cf1cfd90c8ell, 0xc3241eee21e04335ll, 0x1f7ef0ec35bb3a16ll, 0x2082ddcd712cea20ll, 0xea6dd878e1cf83bell, 0xe20708512d57af51ll, 0x53d8c5e73bfd68e7ll, 0xbc06c96708a2d80dll, 0xe5303f14e2c4ed2ll, 0xf2baa21ec43fbf3cll, 0xc66aaf40605089d8ll, 0xb98898d342d1a70all, 0x79eaa5df944cdcc3ll, 0xa4ae33d8a9392101ll, 0xd2bd0d1e685b2895ll, 0xf5dee131682af06all, 0xee9d971db32100a9ll, 0x979d6bc8cba8f21all, 0x627a73d284d1e077ll, 0x3791b8bed2f990ecll, 0xf1b664cf858bf41ll, 0x8bf00b84d5ef3043ll, 0x5290e9b8024f8d15ll, 0x586a32e199bc60afll, 0xb0011ca2bfc88320ll, 0xd246570ad39d881dll, 0xa8d58a8fb5195e07ll, 0x2d81c5167f40a226ll, 0x4d5f609d233f8b61ll, 0xe9c3989a896b89d7ll, 0xeb4f7588bf903347ll, 0x833013f909b93343ll, 0x9a573acb50f4a9ebll, 0x66d01a5810f8d137ll, 0xaa9f929069bcc2dell, 0xa6adb6c784046febll, 0x773fb2dd1fca11d2ll, 0xf8504ef7bbb60baell, 0xb8ff3fd004c40d6dll, 0xa2709481fa5be1f4ll, 0xe3b67f87945b1f0dll, 0xb8c2b550fe024263ll, 0x162da9f5da318c7ell, 0x43bd85c11f0e6f40ll, 0x576e83352bbff70bll, 0x6f532e35310c746ll, 0xde275c4ae8b7c108ll, 0x366542ca116d445dll, 0x192377a9f89ad8f5ll, 0xee8f3b67d22d5d99ll, 0x579cb13009210d66ll, 0xf3873431e3850df3ll, 0x7c69e10852936f22ll, 0x2b3a9bd371dfaba7ll, 0x6be9676d1bbce350ll, 0x7e6ee26c8b0dc356ll, 0x1b3261a0e40f0779ll, 0xa9706717ed456ab8ll, 0x5d765dabbeb674c6ll, 0x7d739283e4f43033ll, 0x8e305790a447736all, 0xe352f5943cf487fbll, 0x64398b5b8cbe1143ll, 0xfc2cb7207d937bd2ll, 0x7753cf89671c8cdell};

unsigned long long py3[] = {0xcacdf749cb8d5f19ll, 0x3a26b17a6f13f006ll, 0x1cde4777eaf1fa11ll, 0xa4414ed881c0e56dll, 0x3b571c5fa5508c0ell, 0x7cdc1849f76f2ec2ll, 0x1a468db56c8568e2ll, 0x8c57ec98bf38fe59ll, 0xd10b947bbcaf4bf5ll, 0x4e64eb670fae53bell, 0x430c61252c4de283ll, 0xab68117b7ff30299ll, 0x26a8f75f7d79ee30ll, 0x1053de0301a4bccbll, 0xbb50c1849212a0f1ll, 0x423862d46548c4bell, 0xc31f86616d9390ecll, 0xfa83bcfd2bc8718fll, 0x22d6538e7ca2b142ll, 0x21ebff421deb94b7ll, 0x8a5128224b2e1e5bll, 0xb518a254c60bd352ll, 0x8c10a7ec1dd632cfll, 0xb8b94fd1a20f561ell, 0xcbf7274e4320fde9ll, 0x7f825189b430ad19ll, 0x7fc5b0de13e6ae42ll, 0xb61ac93784c46e70ll, 0x86e902c618d1d0e2ll, 0xc7ab820d0f5ddfa6ll, 0x2927e90be13fcc69ll, 0x5a7a8539a069fa70ll, 0x8fa0db2c14690950ll, 0xee3365a358ca1d1bll, 0xd2107bae0e0d2c93ll, 0xb19305443f66057ll, 0x505575ab92a3dbd0ll, 0xfc0d0110bab28c74ll, 0x64b5014ed56a5165ll, 0x210a5c2d815d6c7ell, 0x71c2a24bc399717ll, 0x2f7c82a1be3ed40all, 0xa0fe03461542dc9all, 0x9c8175bc3b7aac92ll, 0x52a5d275fa70f38fll, 0x7f280e75ef818498ll, 0x782e3358a54ea194ll, 0xb286ede5b6c69b77ll, 0xb2fae70f5b4cb3e6ll, 0x985669303148fc90ll, 0xba4f880f241bfd5ll, 0x5134a1168ffb9c2bll, 0x88aaa0f440c2fc84ll, 0x732a815a26af6096ll, 0x8ada0e8b66cc7c48ll, 0x85d80c3e63934b97ll, 0xc276948f76dc160cll, 0x83354992d9f8f4ecll, 0xdb89f215877d3fd2ll, 0x607cf4a51e6d4beall, 0x7bc812a269cdb273ll, 0x5e91c14838a6da32ll, 0xeea1beca0c82783cll, 0xb402c8733b869ce8ll, 0x6bb6539a3a626f9fll, 0x65c3c28b9fb0dd7ell, 0xfd31658429a1fd8cll, 0x283fb6056d1dbd54ll, 0x4d1fd3ab9ce96b92ll, 0x59307fca2f3e66d2ll, 0x47899b04717f8830ll, 0x2c04c9c438a97ccell, 0x41a43e034990ccb2ll, 0xac5c62a314ae7fc1ll, 0x2bc9c15f5ce5f21dll, 0x4351c7f4bc543587ll, 0xe714a0633eb54391ll, 0x49a981d8f96d8c83ll, 0xcc039db192259e41ll, 0xb0365ab40ef69a54ll, 0xf7c0aa189d06d5a4ll, 0xe3c9ef6dbcbe0cb3ll, 0x5053f2e2d58472e7ll, 0xeed3a254c43ae758ll, 0x783efea48bc01e51ll, 0xfd450216e34bb240ll, 0xaa59875b491dfe0dll, 0x18c4e66c88aa43a4ll, 0x21ea3d478bc5b59bll, 0x50b61cc4607a4882ll, 0x7c3a24521c8e8bb8ll, 0xdcc23b4a6fae5af9ll, 0x34f723ffe88c492ll, 0x7a1049306af2c39cll, 0xb8fafb10e9e11d70ll, 0x45e280142165ae6ell, 0xd1bc897d2caf1fb5ll, 0x58c41c04b69ddcfdll, 0x3869b0569cc204c5ll, 0xfc0be1c06a7ecb4ell};


unsigned long long dx[] = {0x3e9b58b481a0c1e7ll, 0xd2790f58ad702ac3ll, 0xf3df624ee861f1all, 0x7422dbc25514e317ll, 0xb6c363056b6d570ll, 0x64b8ed462321d1b1ll, 0xfaf242de5b1e7585ll, 0xd4c307f48c33830fll, 0xd418348214808c65ll, 0xf8932fa155f832ebll, 0xd9145a22f78c225cll, 0xac037fc3f8423648ll, 0xa28bafa272ad91a5ll, 0x9795cc3f52dd1fdfll, 0x617bf6ea64012f39ll, 0x15c959cd69b96173ll, 0x79938dacb05340f0ll, 0xb73c17845e5f0655ll, 0x3caed57f4bd41d80ll, 0xa1ce5385c6053c48ll, 0x2970d3e74b38c8a2ll, 0x981e9824714f72f9ll, 0x8d5deba7fe2831cfll, 0x94fc5ff28dd55157ll, 0x9b7d882b71f43546ll, 0x7c6c74ebad9a3f32ll, 0xbebd6b634e1a88eell, 0xeb0ac31cf931b374ll, 0x7861226eaee648bll, 0x3862d67ed71d09f5ll, 0x1505d4e24a270420ll, 0xf8b0088e5648acecll, 0x90c1545404bc4a0ll, 0xe297c7d92264f5fbll, 0x9ab771c2882e07a5ll, 0x30a271fd9414e288ll, 0xc47f35f595e118b5ll, 0x552c5a97b9aa7516ll, 0x144f974df1b2ad09ll, 0x427e96ea27f5846bll, 0x78820a6c772f2688ll, 0x47a88dfd3542f024ll, 0x40c3d261f513e1a8ll, 0xd62b6192ae7bdab9ll, 0x77f9db302dcd5c28ll, 0xba5881b5c7ddd140ll, 0xfe44a6e288bf3851ll, 0x4d1f56e7962c2dell, 0x3c0ca396a6f16a01ll, 0xdea3647470db11b2ll, 0xb1dfb8febcc7541bll, 0x91d5f4ac42502ad1ll, 0x43a3f5dc4363be16ll, 0x60a7186206c08f8bll, 0x4ad258e55da97160ll, 0x510df6e5c3232ec7ll, 0xa2da54efe96f9514ll, 0xe3c1b95dad759e1bll, 0xd03e5645dbea5195ll, 0x2b64bdbb8c1cd0a6ll, 0x3d64efeb284b8ad2ll, 0xed645de57f9311ccll, 0xa9caeb58742da602ll, 0xabb9772059774479ll, 0xa23cae275137c4d1ll, 0x826d0433243e9a9ell, 0x95eba298e227b592ll, 0x3a2ed739fb98c7e9ll, 0x5cfebb35d7f90e1bll, 0xf732aff4ff5a4fb7ll, 0x748112df101e87fcll, 0x834fdc11baab3838ll, 0xd75c9e46d0aee654ll, 0x1745092a6abf8770ll, 0x1362d90dcded199ell, 0x863dfc5e91a7a9b5ll, 0xd18f3cd686e412fll, 0xe04fe216bcc01378ll, 0xb3a1cd3bdce4c117ll, 0x2283efec24582825ll, 0xf744c2c74567e2f5ll, 0x97afc09b7b282e43ll, 0xd198c777f05886bbll, 0x6b23e4c0f51dadb5ll, 0xdae050c76c1eddb3ll, 0xdaab67905a7d0659ll, 0x2b174248a0584645ll, 0xd34e1a4d0bd960b1ll, 0x808d3cc6dd83fc59ll, 0xbf91a51648dd0960ll, 0xf38567e92f48df68ll, 0xacb571f8dd66500ll, 0xecd41f0b86138682ll, 0x21e873127c645bb6ll, 0xb5a0e535ee29b6b1ll, 0xeaeeab56d06c2c0ell, 0x929db043d8f2cdfall, 0xc0d2a2608be97e5ll, 0x426dc138552c4432ll, 0x868a971759b755ccll};

unsigned long long dy[] = {0x3542486d3e8384e3ll, 0x13d051ad6b464dedll, 0x5b42fc37a6293dbdll, 0x6e4b2cbd7e92276fll, 0x9fef0e9c8b15264fll, 0xabb70e6bee9cb2ll, 0xee25466956363179ll, 0xfd11ac5d51563753ll, 0x58b1fc3af2489a29ll, 0xa82aa19eaac8748ll, 0x51f02a42fec69c5fll, 0x725f87d6edcef39ll, 0xb51e0df1a07df532ll, 0x8e08134f4074abd4ll, 0xbfdfdba93252fa31ll, 0x37c5c7692f917865ll, 0xe0ed9cb3d636f6b3ll, 0x5b83a93f64405366ll, 0xf06a4f823a1989d7ll, 0xb9ff31ce194773bfll, 0x1d98a6d47f5270c2ll, 0xb43486e2eb7e136bll, 0x64d788eb8413f93dll, 0x9bb30eb73e93d3b2ll, 0xf309ee64e7d4f92ll, 0xb8dde885b32312all, 0x21118ded95ac6f29ll, 0xc983dab25faec20bll, 0x7b74ac6b8d452c1bll, 0xcaccbe26c46e1ea2ll, 0xfeab89e32a652279ll, 0x81b537578dece23dll, 0x714ece790da7d5ebll, 0x771bbd884024ee48ll, 0x6881642b2437ea69ll, 0x72a353465dbfc234ll, 0x2e7739c8b3284985ll, 0x518b6ce43f74b9fdll, 0x1d11b310f142340cll, 0xb8b5f8a7047cc78bll, 0xc98112a0eccee6aell, 0xd4a12b5796e5b7a0ll, 0x24d6431fd4b6ea05ll, 0xafa380a9e9478faall, 0x98a35a0f382f2211ll, 0x608d3b1db228e853ll, 0xec7ada4b8c2e0277ll, 0x59b28e0ee1d21e1cll, 0xc0a93100d57028call, 0xb2ee8e366d7a78d3ll, 0x409e61311cac105bll, 0x152478a51a53cec0ll, 0xa6a22e64bcf36c49ll, 0x32130c6462739094ll, 0x915ff7ea83bb5e2ll, 0xcc03a67084696f4cll, 0xe208ec68ca092003ll, 0xdb6bf75f2140aa60ll, 0xef36c6cc11586b3cll, 0x8afb419de213ac3dll, 0x5e382f8e520c04e1ll, 0x5f92c6bc34ebbb89ll, 0x398db743bfb374fall, 0xe19e0e6e3ceeaa42ll, 0x9e7ea5242816d0d6ll, 0x11dcaa44038fce38ll, 0xc6e3725ac42e00a4ll, 0x71443ce92e90d37cll, 0x23dd572671f8b692ll, 0xde27f7bde1a28bb9ll, 0xe3fc1cc34a75f2ddll, 0x8e5380a90833dfd6ll, 0x4da2b42de56731fdll, 0x36a6009f9eee69bcll, 0xf4db711db3e119ccll, 0x5e8ccc990795a889ll, 0xafd703419dfea9a0ll, 0x30463f3c6ee2b046ll, 0x9e2b31784a512bbll, 0xe94ad634e74bcf82ll, 0x45f3db452cd7f9bbll, 0xde6b6252dd33782bll, 0x193d5f6c04a27310ll, 0x3acb8fa99c329707ll, 0x22d69badb37471f0ll, 0xf1c99f10c56b7cf5ll, 0xfe6110920f406657ll, 0x2258d6aa622458d9ll, 0xd635d70116f7d3fcll, 0x623d3502b380a0fell, 0x967638214e7752d8ll, 0x8ebe4d0ebc1b3a36ll, 0x4693a1c02ec948b3ll, 0xd30995eb3700fd54ll, 0x703135b40eee5981ll, 0x47513b99adac2c47ll, 0x4807b69ef64a62e3ll, 0x351d3b0f620edf57ll, 0xe2d20985d347dddfll, 0xcf1d0224f5f5ba61ll};

unsigned int dx2[] = {0x90a25e6d, 0x266b1fdd, 0xc7e5f9b1, 0x8b331f76, 0x86c26260, 0x11602198, 0x3d449367, 0xe221abb8, 0x20512bec, 0xee5d4140, 0x7a4a63ea, 0x9c9a8137, 0x4518e71d, 0x90bb01b9, 0xc8a0dbb1, 0x1c5859f8, 0xb71d74a2, 0xe627b83d, 0x63ad710f, 0x1b0590aa, 0x47f31b7d, 0x8b247df8, 0x680ceb24, 0x84c82aa9, 0xe16a482, 0x1b303012, 0xa6208757, 0x62842d6f, 0x27ae4f0c, 0x893d6f09, 0xa4687085, 0xa1ffc667, 0xea82a239, 0x17d73135, 0xcb5a9d69, 0x9fb058fc, 0x2998e75c, 0xf0fc9d2, 0xfb1c8ce, 0x79e3bc7c, 0x231366b5, 0x40392f2, 0x3a096a07, 0xd3b975cf, 0xfc49c863, 0xd9b437c6, 0xfea912dc, 0x67224518, 0x92ea1d92, 0x32594898, 0x7436f88d, 0x9a3b2801, 0x54930fd, 0x9273f995, 0x3b9bdecb, 0x44a0665f, 0xde53d2e8, 0xe611f8a1, 0x47dd26d5, 0x62006088, 0x18adef96, 0x40a57445, 0x96d64655, 0xc1c91992, 0xd8d12730, 0x7aeda909, 0x85eaf4c4, 0x60f20c05, 0xc1ed3293, 0xc322ca3f, 0x4c7ed042, 0x73c2b31d, 0xe0396b9d, 0x3020d47f, 0xd4c5d6c6, 0x523f71c0, 0x8ce47a37, 0x7859e90a, 0x7686e532, 0xdb086e00, 0x909ac2c3, 0x574ed015, 0x9e6fa778, 0x54ecf2a9, 0x2db98885, 0xec671188, 0xc0de2005, 0x5ec92a71, 0x78b628e2, 0x9c9816e2, 0x69e13f3d, 0x4fc3827b, 0xae393a9e, 0x6f45648, 0xa85e2113, 0xa2b591bc, 0xaaca0b99, 0x24033483, 0xf90f13b2, 0xa29a99cc};

unsigned int dy2[] = {0x90a25e6d, 0x266b1fdd, 0xc7e5f9b1, 0x8b331f76, 0x86c26260, 0x11602198, 0x3d449367, 0xe221abb8, 0x20512bec, 0xee5d4140, 0x7a4a63ea, 0x9c9a8137, 0x4518e71d, 0x90bb01b9, 0xc8a0dbb1, 0x1c5859f8, 0xb71d74a2, 0xe627b83d, 0x63ad710f, 0x1b0590aa, 0x47f31b7d, 0x8b247df8, 0x680ceb24, 0x84c82aa9, 0xe16a482, 0x1b303012, 0xa6208757, 0x62842d6f, 0x27ae4f0c, 0x893d6f09, 0xa4687085, 0xa1ffc667, 0xea82a239, 0x17d73135, 0xcb5a9d69, 0x9fb058fc, 0x2998e75c, 0xf0fc9d2, 0xfb1c8ce, 0x79e3bc7c, 0x231366b5, 0x40392f2, 0x3a096a07, 0xd3b975cf, 0xfc49c863, 0xd9b437c6, 0xfea912dc, 0x67224518, 0x92ea1d92, 0x32594898, 0x7436f88d, 0x9a3b2801, 0x54930fd, 0x9273f995, 0x3b9bdecb, 0x44a0665f, 0xde53d2e8, 0xe611f8a1, 0x47dd26d5, 0x62006088, 0x18adef96, 0x40a57445, 0x96d64655, 0xc1c91992, 0xd8d12730, 0x7aeda909, 0x85eaf4c4, 0x60f20c05, 0xc1ed3293, 0xc322ca3f, 0x4c7ed042, 0x73c2b31d, 0xe0396b9d, 0x3020d47f, 0xd4c5d6c6, 0x523f71c0, 0x8ce47a37, 0x7859e90a, 0x7686e532, 0xdb086e00, 0x909ac2c3, 0x574ed015, 0x9e6fa778, 0x54ecf2a9, 0x2db98885, 0xec671188, 0xc0de2005, 0x5ec92a71, 0x78b628e2, 0x9c9816e2, 0x69e13f3d, 0x4fc3827b, 0xae393a9e, 0x6f45648, 0xa85e2113, 0xa2b591bc, 0xaaca0b99, 0x24033483, 0xf90f13b2, 0xa29a99cc};


void make_uint288(uint288* out, uint* in, int len) {
    int blocks = len / 9;
    unsigned int rand = 0;
    for (int i = 0; i < blocks; i++) {
        for (int j = 0; j < 9; j++) {
            for (int k = 0; k < 8; k++, rand++) {
                // ATTENTION!! UINT288 is big-endian, so least significant is data[8].
                out[i * 9 + j].data[k + 1] = in[j * 8 + k] + rand;
            }
            out[i * 9 + j].data[0] = 0;
        }
    }
    printf("made %d numbers, %d left\n", blocks * 9, len - blocks * 9);
    for (int start = blocks * 9; start < len; start++) {
        out[start].data[8] = 0x3f3fffff + (rand++); // for intergrity.
    }
    //int start = blocks * 
    printf("check uint288: ");
    for (int i = 0; i < 9; i++) {
        printf("0x%llx ", out[0].data[i]);
    }
    printf("\n");
}

#ifndef dh_mybig_copy
#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}
#endif

__device__ __host__ void my_point_copy(Jpoint* from, Jpoint* to) {
    dh_mybig_copy(to->x, from->x);
    dh_mybig_copy(to->y, from->y);
    dh_mybig_copy(to->z, from->z);
}

void print_big_arr(UINT64* nums,int n){
    for(int i=0;i<n;i++){
        h_mybig_print(nums+i*4);
        printf("\n");
    }
}
void print_jpoint_arr(Jpoint* nums,int n){
    for(int i=0;i<n;i++){
        h_print_pointJ(nums+i);
        printf("\n");
    }
}

void init_Jpoint_toOne(Jpoint* p, int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<4;j++){
            p[i].x[j] = h_Gx[j];
            p[i].y[j] = h_Gy[j];
            p[i].z[j] = h_ONE[j];
        }
    }
}

void make_data(Jpoint *in, int datasize) {
    int block = 31;
    int blocknum = datasize / 31;
    int other = datasize - blocknum * 31;
    printf("datasize = %d\n", datasize);
    unsigned int rand = 1;
    for (int i = 0; i < blocknum; i++) {
        for (int j = 0; j < block; j++, rand++) {
            int idx = i * block + j;
            
            int wtf = (j / 2) * 4; // att: can't simplify this.
            if (j % 2 == 1) {
                dh_mybig_copy(in[idx].x, px+j);
                dh_mybig_copy(in[idx].y, py+j);
            } else {
                dh_mybig_copy(in[idx].x, px3+j);
                dh_mybig_copy(in[idx].y, py3+j);
            }
            //in[idx].y[0] += rand;
        }
    }
    for (int start = blocknum * 31; start < datasize; start++) {
        dh_mybig_copy(in[start].x, px);
        dh_mybig_copy(in[start].y, py);
    }
    printf("makes data\n");
}

void __global__ test_point_double(Jpoint *p1,Jpoint *p2){//only for warm-up.
    UINT64 zinv[4], zinv2[4];
    int tx = threadIdx.x;
    ppoint_double(p1+tx, p2+tx);
    dh_mybig_moninv((p2+tx)->z, zinv);
    dh_mybig_monmult_64(zinv, dc_R2, zinv);
    dh_mybig_monmult_64(zinv, zinv, zinv2);
    dh_mybig_monmult_64((p2+tx)->x, zinv2, (p2+tx)->x);
    dh_mybig_monmult_64(zinv, zinv2, zinv);
    dh_mybig_monmult_64((p2+tx)->y, zinv, (p2+tx)->y);
}

// these functions are not jp1 -> jp2(mont), but jp1 -> jp1(mont), jp2 -> jp2(mont).
void __global__ point_to_monjj(Jpoint* jp1,Jpoint* jp2){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_BIGNUM){
        dh_mybig_monmult_64((jp1+idx)->x,dc_R2,(jp1+idx)->x);
        dh_mybig_monmult_64((jp1+idx)->y,dc_R2,(jp1+idx)->y);
        dh_mybig_monmult_64((jp1+idx)->z,dc_R2,(jp1+idx)->z);

        dh_mybig_monmult_64((jp2+idx)->x,dc_R2,(jp2+idx)->x);
        dh_mybig_monmult_64((jp2+idx)->y,dc_R2,(jp2+idx)->y);
        dh_mybig_monmult_64((jp2+idx)->z,dc_R2,(jp2+idx)->z);
    }

}

void __global__ point_from_monjj(Jpoint* jp1,Jpoint* jp2){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_BIGNUM){
        dh_mybig_monmult_64((jp1+idx)->x,dc_ONE,(jp1+idx)->x);
        dh_mybig_monmult_64((jp1+idx)->y,dc_ONE,(jp1+idx)->y);
        dh_mybig_monmult_64((jp1+idx)->z,dc_ONE,(jp1+idx)->z);

        dh_mybig_monmult_64((jp2+idx)->x,dc_ONE,(jp2+idx)->x);
        dh_mybig_monmult_64((jp2+idx)->y,dc_ONE,(jp2+idx)->y);
        dh_mybig_monmult_64((jp2+idx)->z,dc_ONE,(jp2+idx)->z);
    }

}

__global__ void point_from_mont(Jpoint* from, Jpoint* to) {
    UINT64 zinv[4], zinv2[4];
    int tx = threadIdx.x;
    // jacobian->affine
    dh_mybig_moninv((from+tx)->z, zinv);
    dh_mybig_monmult_64(zinv, dc_R2, zinv);
    dh_mybig_monmult_64(zinv, zinv, zinv2);
    dh_mybig_monmult_64((from+tx)->x, zinv2, (to+tx)->x);
    dh_mybig_monmult_64(zinv, zinv2, zinv);
    dh_mybig_monmult_64((from+tx)->y, zinv, (to+tx)->y);
    // mont->normal
    dh_mybig_monmult_64((to+tx)->x,dc_ONE,(to+tx)->x);
    dh_mybig_monmult_64((to+tx)->y,dc_ONE,(to+tx)->y);
    dh_mybig_monmult_64((to+tx)->z,dc_ONE,(to+tx)->z);
}

__global__ void accumulate_sum_per_block(Jpoint* in) { //in = out[]
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int nthread = blockDim.x;
    in = in + bx * nthread * 2;
    for (int i = nthread, j = 1; i; i /= 2, j++) { // 5 = log2(thread)
#ifdef DEBUG
        if (tx == 0) {
            //printf("tx=%d, point=%llx\n", tx, in[tx].x[0]);
        }
#endif
        if (tx < i) {
            //printf("tx=%d, before add: point1=%llx, point2=%llx\n", tx, in[tx].x[0], in[tx + i].x[0]);
#ifdef VERBOSE_MODE
            dh_ellipticAdd_JJ_verbose(&in[tx], &in[tx + i], &in[tx]);
#else
            dh_ellipticAdd_JJ(&in[tx], &in[tx + i], &in[tx]);
#endif
            //printf("tx=%d, after add: point=%llx\n", tx, in[tx].x[0]);
        }
        __syncthreads();
    }
    // if (tx == 0) {
    //     UINT64 zinv[4], zinv2[4];
    //     dh_mybig_moninv((in+tx)->z, zinv);
    //     dh_mybig_monmult_64(zinv, dc_R2, zinv);
    //     dh_mybig_monmult_64(zinv, zinv, zinv2);
    //     dh_mybig_monmult_64((in+tx)->x, zinv2, (in+tx)->x);
    //     dh_mybig_monmult_64(zinv, zinv2, zinv);
    //     dh_mybig_monmult_64((in+tx)->y, zinv, (in+tx)->y);
    // }// res == in[0]
}

__global__ void accumulate_blocks(Jpoint* in, int blocksize) { //in = out[]
    //int bx = blockIdx.x;
    int tx = threadIdx.x; // tx is last step's block id
    int nthread = blockDim.x;
    in = in + tx * blocksize;
    for (int i = nthread, j = 1; i; i /= 2, j++) { // 5 = log2(thread)
#ifdef DEBUG
        if (tx == 0) {
            //printf("tx=%d, point=%llx\n", tx, in[tx].x[0]);
        }
#endif
        if (tx < i) {
            //printf("tx=%d, before add: point1=%llx, point2=%llx\n", tx, (in + tx * blocksize)->x[0], (in + (tx + i) * blocksize)->x[0]);
#ifdef VERBOSE_MODE
            dh_ellipticAdd_JJ_verbose(&in[tx], &in[tx + i], &in[tx]);
#else
            dh_ellipticAdd_JJ(in + tx * blocksize , in + (tx + i) * blocksize, in + tx * blocksize);
#endif
            //printf("tx=%d, after add: point=%llx\n", tx, (in + tx * blocksize)->x[0]);
        }
        __syncthreads();
    }
}

__global__ void trivial_sum(Jpoint* in, int num) {
    for (int i = 1; i < num; i++) {
        printf("do algorithm: i=%d\n", i);
        dh_ellipticAdd_JJ(in, in + i, in);
        //__syncthreads__();
    }
}

__global__ void trivial_sum_blocks(Jpoint* in, int num, int blocksize) {
    for (int i = 1; i < num; i++) {
        printf("do algorithm: i=%d\n", i);
        dh_ellipticAdd_JJ(in, in + i * blocksize, in);
        //__syncthreads__();
    }
}

void __global__ test_point_add(Jpoint *p1, Jpoint* p2, Jpoint* p3) {
    Jpoint p1_aff, p2_aff, p3_aff;
    UINT64 zinv[4], zinv2[4];
    int tx = 0;
    Jpoint* from = &testG1;
    Jpoint* to = &p1_aff;
    // jacobian->affine
    dh_mybig_moninv((from+tx)->z, zinv);
    dh_mybig_monmult_64(zinv, dc_R2, zinv);
    dh_mybig_monmult_64(zinv, zinv, zinv2);
    dh_mybig_monmult_64((from+tx)->x, zinv2, (to+tx)->x);
    dh_mybig_monmult_64(zinv, zinv2, zinv);
    dh_mybig_monmult_64((from+tx)->y, zinv, (to+tx)->y);
    // mont->normal
    dh_mybig_monmult_64((to+tx)->x,dc_ONE,(to+tx)->x);
    dh_mybig_monmult_64((to+tx)->y,dc_ONE,(to+tx)->y);
    dh_mybig_monmult_64((to+tx)->z,dc_ONE,(to+tx)->z);
    
    from = &testG2;
    to = &p2_aff;
        // jacobian->affine
    dh_mybig_moninv((from+tx)->z, zinv);
    dh_mybig_monmult_64(zinv, dc_R2, zinv);
    dh_mybig_monmult_64(zinv, zinv, zinv2);
    dh_mybig_monmult_64((from+tx)->x, zinv2, (to+tx)->x);
    dh_mybig_monmult_64(zinv, zinv2, zinv);
    dh_mybig_monmult_64((from+tx)->y, zinv, (to+tx)->y);
    // mont->normal
    dh_mybig_monmult_64((to+tx)->x,dc_ONE,(to+tx)->x);
    dh_mybig_monmult_64((to+tx)->y,dc_ONE,(to+tx)->y);
    dh_mybig_monmult_64((to+tx)->z,dc_ONE,(to+tx)->z);
    my_point_copy(&p1_aff, p1);
    my_point_copy(&p2_aff, p2);
    // do main
    dh_ellipticAdd_JJ_verbose(&testG1, &testG2, &p3_aff);

    from = &p3_aff;
    to = p3;
        // jacobian->affine
    dh_mybig_moninv((from+tx)->z, zinv);
    dh_mybig_monmult_64(zinv, dc_R2, zinv);
    dh_mybig_monmult_64(zinv, zinv, zinv2);
    dh_mybig_monmult_64((from+tx)->x, zinv2, (to+tx)->x);
    dh_mybig_monmult_64(zinv, zinv2, zinv);
    dh_mybig_monmult_64((from+tx)->y, zinv, (to+tx)->y);
    // mont->normal
    dh_mybig_monmult_64((to+tx)->x,dc_ONE,(to+tx)->x);
    dh_mybig_monmult_64((to+tx)->y,dc_ONE,(to+tx)->y);
    dh_mybig_monmult_64((to+tx)->z,dc_ONE,(to+tx)->z);
    printf("p1_aff: %llx %llx %llx %llx\n", p1_aff.x[0], p1_aff.x[1], p1_aff.x[2], p1_aff.x[3]);

}


#define from(x, y, z) *(from + (x) * DBC_level1 + (y) * DBC_level2 + (z))
const int DBC_level1 = 3 * DBC_MAXLENGTH;
const int DBC_level2 = 3;
__global__ void check_dbc(uint288* nums, int* dbc_store, int* dbc_value) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int nthread = blockDim.x;
    int dbc_id = nthread * bx + tx;

    int* from = dbc_store + dbc_id * 6 * DBC_MAXLENGTH;
    if (tx == 120) {
        printf("check viability: ");
        for (int i = 0; i < 120; i++) {
            printf("0x%x ", u_pow23[0][i].data[8]);
        }
        printf("\n");
        //printf("step0: checkdata, tx=%d, bx=%d, nth=%d, data=0x%x\n", tx, bx, nthread, (unsigned int)u_pow23);
        //printf("step0: check_dbc, tx=%d, bx=%d, nthread=%d\n", threadIdx, blockIdx, nthread);
    }
    if(nums->data[0] != 0) {
        printf("panic: thread %d has illegal input: %llx, %llx, %llx, %llx\n", tx, nums->data[0], nums->data[1], nums->data[2], nums->data[3]);
    }
    __syncthreads();
    int n = get_DBC(nums + dbc_id, from, dbc_value + dbc_id * 2);
    //printf("thread %d returns %d\n", tx, n);
    //__syncthreads();
    //get_DBC(nums + dbc_id, from, dbc_value + dbc_id * 2);
    for (int i = 0; i < DBC_MAXLENGTH; i++) {
        for (int j = 0; j < 3; j++) {
            from(0, i, j) = from(n, i, j);
        }
    }
    dbc_value[dbc_id * 2] = dbc_value[dbc_id * 2 + n];
}

__device__ void my_check_point(Jpoint *pt) {
    printf("[0x%x, 0x%x, 0x%x]\n", pt->x[0], pt->y[0], pt->z[0]);
}
//#define RELEASE
__global__ void dbc_main(uint288* nums, int* dbc_store, int* dbc_value, Jpoint *in, Jpoint *out) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int nthread = blockDim.x;
    int dbc_id = nthread * bx + tx;
    int n = 0;
    if (bx < CONST_BLOCK - 1) n = get_DBC(nums + dbc_id, dbc_store + dbc_id * 6 * DBC_MAXLENGTH, dbc_value + dbc_id * 2);
    int len = *(dbc_value + dbc_id * 2 + n);
    int* dbc = dbc_store + dbc_id * 6 * DBC_MAXLENGTH + n * 3 * DBC_MAXLENGTH;
#ifndef RELEASE
     __syncthreads();
    if(bx == 30 && tx == 0) {
        printf("check:bx=%d thread %d has input: %llx, %llx, %llx, %llx\n", bx, tx, nums->data[0], nums->data[1], nums->data[2], nums->data[3]);
    }
    if(nums->data[0] != 0 || len < 1 || len > 110) {
        printf("panic:bx=%d thread %d has illegal input: %llx, %llx, %llx, %llx\n", bx, tx, nums->data[0], nums->data[1], nums->data[2], nums->data[3]);
    }
    if (bx == 0 && tx == 3) {
        printf("check point: %llx %llx %llx %llx\n", in[tx].x[0], in[tx].x[1], in[tx].x[2], in[tx].x[3]);
        printf("check dbc value in block %d, len = %d: \n", bx, len);
        for (int j = 0; j < len; j++) {
            int* wtf = dbc + j * 3;
            printf("(%d)*(2**%d)*(3**%d) + ", wtf[0], wtf[1], wtf[2]);
        }
        printf("\n");
    }
    __syncthreads();
#endif
    int cnt = 0;
    if (bx < CONST_BLOCK - 1) cnt = run_DBC_v2(in + dbc_id, out + dbc_id, dbc, len);
    if ((out + dbc_id)->x[0] == 0) {
        printf("bx=%d, tx=%d reports bug:x is 0!!\n", bx, tx);
    }
    // if (tx == 0) {
    //     printf("bx=%d runs %d ops, check point value\n", bx, cnt);
    //     printf("{[%d]2^%d 3^%d} ", dbc[0], dbc[1], dbc[2]);
    //     my_check_point(in+dbc_id);
    //     //my_check_point(out+dbc_id);
    // }
    //__syncthreads();
    //printf("(%d %d) ", bx, tx);
}

unsigned int test_scalars[][8] = {
{205973761u,655691475u,3212678982u,2780673202u,246882742u,3373203617u,841114574u,682807874u,},
{536632485u,1902021725u,3931787030u,1466379785u,3621011277u,3829202438u,3671590951u,1035836490u,},
{0x24f7b145U, 0xb6bbeaf4U, 0x55900d74U, 0x12353d3fU, 0x74d181beU, 0xe21550a1U, 0x2f0f14bbU, 0x2c2a7c86U},
{0x7a4ef51,0x1a2d23c4,0xd75e6793,0x6452e002,0x634fd4b,0x44bdb6c,0xba27475,0xf39e377 },
{0x7a4ef51,0x1a2d23c4,0xd75e6793,0x6452e002,0x634fd4b,0x44bdb6c,0xba27475,0xf39e377 },
{0x912deadb, 0xa1fd83a4, 0x913a6692, 0xc55c504b, 0x3ca1f176, 0x3ae5f26d, 0x982ef124, 0x2067044, },
{0x3f775df8, 0x59b3e897, 0x847d2ba8, 0x83b012d5, 0xf4a7fb0f, 0x81697b01, 0xcd24dd12, 0x270d1ed5, },
{0xa88c61be, 0xb0569c3c, 0x8ce7b4cf, 0xf48393d2, 0x2d8d52a2, 0x81a0496, 0x3d1b1bbf, 0x2705079a, },
};

UINT64 test_P_x[][4] = {
{0x92c962f7cbdc4d93ULL, 0x2a75b8ea6e855c79ULL, 0x73918e61b2923579ULL, 0x23266f61c372b8a2ULL}, 
{0x485682bab86dd257ULL, 0xd1e709213a7dbdcbULL, 0x82f0ca7b221c62f0ULL, 0xc9cd242c3d45a0ULL}, 
{0x841032503305927cULL, 0x47d958a69f9c9fbbULL, 0x379bd2d072516b28ULL, 0x72da969599cd2c8ULL}, 
{0x83becdda8f67d191ULL, 0x9b3d6900d5d66cd6ULL, 0x53a14c3c30be2a04ULL, 0x1e9fb502a52950f6ULL}, 
    {0x9d453953d29b791fULL, 0x3e21372b6d330e94ULL, 0x33f52c13f38afa1fULL, 0x29b073c75fb72771ULL},
    {0x5168b89708f5e1ebULL, 0xd2668b78e993750dULL, 0x2edace5f7678bcf1ULL, 0x5b8fa45091ab9e1ULL},
    {0x25d86e3d59542496ULL, 0xe2a8e448860f425dULL, 0x7d259942ea413630ULL, 0x1be50b1000752ffdULL},
    {0x10b30dbe33895a85ULL, 0xce23b30959e50cfaULL, 0xb9844001735f9518ULL, 0x271e9140aa677758ULL},
};

UINT64 test_P_y[][4] = {
{0x1b791c46c7839836ULL, 0x8110f0c64954f3b0ULL, 0xbf12e221b83a290fULL, 0x19496f827f16dc62ULL}, 
{0xa7fffe84abe23a8aULL, 0x8e3bbc9dbe53ec52ULL, 0x32a84e7242c9d703ULL, 0x20f30a387a9ea65fULL}, 
{0x5aac5bee888c5438ULL, 0x1b701dc8ac369890ULL, 0x41d4c0ef44133497ULL, 0x11c1fb2396ee5f33ULL,}, 
{0x768c40cfa36fc445ULL, 0x4453fd6230d52663ULL, 0x64b7fe22bba218f9ULL, 0x362da0ae65698bfULL}, 
    {0xd6a36c9b5f868b69ULL, 0x9558c02e3a5baeaeULL, 0xb492ae1a12637ff7ULL, 0x217ca8ee989a5d43ULL},
    {0x84cf600640672428ULL, 0x40ca230e51fcc345ULL, 0x523a2d9de4c48d1aULL, 0x2acf4d5231cb7411ULL},
    {0xb2458c27d6900e1cULL, 0xc06d987d52dc3320ULL, 0xcadb27d8f29fd6c9ULL, 0x22779b1d38ce6d7ULL},
    {0xb4e8c2f9c0db3ea3ULL, 0x72b99a23081ae9a4ULL, 0x6688814cde0fcb84ULL, 0x29550fd01950fa3bULL},
};

UINT64 test_P_z[][4] = {
    {1, 0, 0, 0},
    {1, 0, 0, 0},
    {1, 0, 0, 0},
    {1, 0, 0, 0},
    {1, 0, 0, 0},
    {1, 0, 0, 0},
    {1, 0, 0, 0},
    {1, 0, 0, 0},
};

__global__ void test_scalar_multiplication(UINT64* scalar, Jpoint* in, Jpoint* out) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int nthread = blockDim.x;
    int idx = nthread * bx + tx;
    if (tx == 0) {
        printf("Check input data: scalar=%llx, in=%llx\n", scalar[4 * tx], in->x[0]);
    }
    scalar_multiplication(scalar + 4 * idx, in + idx, out + idx);
}

//#define DEBUG
int correctness_test() {
    struct timeval s1,e1;
    long long time_use=1;
    uint288* scalar;
    uint288* d_scalar;
    int* dbc_store_host;
    int* dbc_len_host;
    int* dbc_store_device;
    int* dbc_len_device;

    Jpoint* w_p1;
    Jpoint* w_p2;
    Jpoint* wd_p1;
    Jpoint* wd_p2;
    Jpoint* d_p1;
    Jpoint* d_p2;
    Jpoint* t_p1;
    Jpoint* t_p2;
    Jpoint* td_p1;
    Jpoint* td_p2;
    UINT64* trivial_scalar;
    UINT64* h_trivial_scalar;

    
    
    CUDA_SAFE_CALL(hipMalloc((void**)&dbc_store_device, 6 * DBC_MAXLENGTH * 16 * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dbc_len_device, 2 * 16 * sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(dbc_store_device, 0, sizeof(dbc_store_device)));
    
    //CUDA_SAFE_CALL(hipMemcpy(d_dbc, dbc, sizeof(DBC)*N_BIGNUM, hipMemcpyHostToDevice));
    
    printf("data required\n");
    // point init
    w_p1 = (Jpoint*)malloc(100 * sizeof(Jpoint));
    w_p2 = (Jpoint*)malloc(100 * sizeof(Jpoint));
    for (int i = 0; i < 50; i++) {
        for(int j = 0;j < 4; j++){
            w_p1[i].x[j] = px[j];
            w_p1[i].y[j] = py[j];
            w_p1[i].z[j] = h_ONE[j];
        }
    }
    for (int i = 50; i < 100; i++) {
        for(int j = 0;j < 4; j++){
            w_p1[i].x[j] = px[4 + j];
            w_p1[i].y[j] = py[4 + j];
            w_p1[i].z[j] = h_ONE[j];
        }
    }

    t_p1 = (Jpoint*)malloc(16 * sizeof(Jpoint));
    t_p2 = (Jpoint*)malloc(16 * sizeof(Jpoint));
    for (int i = 0; i < 8; i++) {
        for(int j = 0; j < 4; j++){
            t_p1[i].x[j] = test_P_x[i][j];
            t_p1[i].y[j] = test_P_y[i][j];
            // only for valid z
            if (j != 2) t_p1[i].z[j] = h_ONE[j];
            else t_p1[i].z[j] = test_P_z[i][j];
        }
    }
    scalar = (uint288*)malloc(16*sizeof(uint288));
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 8; j++) {
            scalar[i].data[j + 1] = test_scalars[i][j];
        }
        scalar[i].data[0] = 0;
    }
    for (int i = 4; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            scalar[i].data[8 - j] = test_scalars[i][j];
        }
        scalar[i].data[0] = 0;
    }
    trivial_scalar = (UINT64*)malloc(16 * 4 * sizeof(UINT64));
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            if (j == 0) trivial_scalar[4 * i] = 3;
            else trivial_scalar[4 * i + j] = 0; 
            trivial_scalar[4 * i + j] = test_P_x[0][j];
        }
    }
    printf("data inited\n");
    
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p1,16*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p2,16*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMemcpy(d_p1,t_p1,16*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2,t_p2,16*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_scalar,sizeof(uint288)*16));
    CUDA_SAFE_CALL(hipMemcpy(d_scalar, scalar, 16*sizeof(uint288),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void**)&h_trivial_scalar, 16 * 4 * sizeof(UINT64)));
    CUDA_SAFE_CALL(hipMemcpy(h_trivial_scalar, trivial_scalar, 16 * 4 * sizeof(UINT64), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void**)&wd_p1,100*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&wd_p2,100*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMemcpy(wd_p1,w_p1,100*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(wd_p2,w_p2,100*sizeof(Jpoint),hipMemcpyHostToDevice));
    printf("ready\n");

    // warm-up.
    // attention: even small data, test data needs warm-up. Else the output would be random.
    accumulate_sum_per_block<<<1, 50>>>(wd_p1);
    CUDA_SAFE_CALL(hipMemcpy(t_p1,wd_p1,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    CUDA_SAFE_CALL(hipMemcpy(w_p1,wd_p1,100*sizeof(Jpoint),hipMemcpyDeviceToHost));
    test_point_add<<<1, 1>>>(wd_p1 + 1, wd_p1 + 2, wd_p1 + 3);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    CUDA_SAFE_CALL(hipMemcpy(t_p1,wd_p1,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    printf("special check: checks p1(aff), p2(aff) and ans(jacob)\n ");
    for (int i = 0; i < 4; i++) {
        printf("0x%016llx 0x%016llx 0x%016llx 0x%016llx\n", t_p1[i].x[3], t_p1[i].x[2], t_p1[i].x[1], t_p1[i].x[0]);
        printf("0x%016llx 0x%016llx 0x%016llx 0x%016llx\n", t_p1[i].y[3], t_p1[i].y[2], t_p1[i].y[1], t_p1[i].y[0]);
    }
    //CUDA_SAFE_CALL(hipMemcpy(t_p,wd_p2,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    printf("warm-up ends.\n");


    // go test.
    // step 1: point to montgomery(jacobian is default format).
    point_to_monjj<<<1, 8>>>(d_p1,d_p2);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    
#ifdef DEBUG
    printf("dbc dp1:\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p1,d_p1,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    for (int i = 0; i < 8; i++) {
        printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[i].x[3], t_p1[i].x[2], t_p1[i].x[1], t_p1[i].x[0]);
    }
#endif
    //test_point_double<<<1, 8>>>(d_p1, d_p2);
    dbc_main<<<1,8>>>(d_scalar, dbc_store_device, dbc_len_device, d_p1, d_p2);
    //test_scalar_multiplication<<<1, 8>>>(h_trivial_scalar, d_p1, d_p2);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

#ifdef DEBUG
    printf("dbc ans on mont:\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p2,d_p2,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    for (int i = 0; i < 8; i++) {
        printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p2[i].x[3], t_p2[i].x[2], t_p2[i].x[1], t_p2[i].x[0]);
    }
#endif
    // just check dbc_main
    // point_from_monjj<<<1, 8>>>(d_p1, d_p2);
    point_from_mont<<<1, 8>>>(d_p2, d_p1);
    // hipDeviceSynchronize();
    // CUDA_CHECK_ERROR();

#ifdef DEBUG
    printf("dbc single ans's:\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p2,d_p1,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    for (int i = 0; i < 8; i++) {
        printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p2[i].x[3], t_p2[i].x[2], t_p2[i].x[1], t_p2[i].x[0]);
    }
#endif

    accumulate_sum_per_block<<<1, 4>>>(d_p2);
    //trivial_sum<<<1, 1>>>(d_p2, 8);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

    printf("test sum ans(on Montgomery):\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p1,d_p2,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[0].x[3], t_p1[0].x[2], t_p1[0].x[1], t_p1[0].x[0]);
    printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[0].y[3], t_p1[0].y[2], t_p1[0].y[1], t_p1[0].y[0]);
    printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[0].z[0], t_p1[0].z[1], t_p1[0].z[2], t_p1[0].z[3]);

    point_from_mont<<<1, 8>>>(d_p2, d_p1);
    //point_from_monjj<<<1, 8>>>(d_p1,d_p2);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

    printf("dbc sum ans:\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p1,d_p1,16*sizeof(Jpoint),hipMemcpyDeviceToHost));
    printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[0].x[3], t_p1[0].x[2], t_p1[0].x[1], t_p1[0].x[0]);
    printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[0].z[0], t_p1[0].z[1], t_p1[0].z[2], t_p1[0].z[3]);
}
// Att: 已经改为三倍点实现. 3.28
//#define CORRECTNESS_TEST
int main() {
    hipSetDevice(0);

#ifdef CORRECTNESS_TEST
    correctness_test();
    return 0;
#endif

    struct timeval s1,e1;
    long long time_use=1;
    int nB,nT;

    hipOccupancyMaxPotentialBlockSize(&nB,&nT,dbc_main);
    printf("NB=%d,NT=%d\n",nB,nT);

    uint288* scalar;
    uint288* d_scalar;
    int* dbc_store_host;
    int* dbc_len_host;
    int* dbc_store_device;
    int* dbc_len_device;

    Jpoint* h_p1;
    Jpoint* h_p2;
    Jpoint* d_p1;
    Jpoint* d_p2;
    Jpoint* t_p1;
    Jpoint* t_p2;
    Jpoint* td_p1;
    Jpoint* td_p2;

    // number init.
    scalar = (uint288*)malloc(BATCHSIZE*sizeof(uint288));
    dbc_store_host = (int*)malloc(6 * DBC_MAXLENGTH * BATCHSIZE * sizeof(int)); // dbc_store[2][DBC_MAXSIZE][3];
    dbc_len_host = (int*)malloc(2 * BATCHSIZE * sizeof(int)); // dbc_len[2];
    int dbc_size = 6 * DBC_MAXLENGTH;
    make_uint288(scalar, dx2, BATCHSIZE); // init int288
    for (int i = N_THREAD_PER_BLOCK - 10; i < N_THREAD_PER_BLOCK; i++) {
        int idx = (N_BLOCK - 1) * N_THREAD_PER_BLOCK + i;
        int semi_idx = idx - N_THREAD_PER_BLOCK;
        //printf("CHECK SEMI-LAST BLOCK DATA-thread %d, offset=%d: %llx, %llx, %llx, %llx\n", i, (&scalar[semi_idx]) - scalar, scalar[semi_idx].data[0], scalar[semi_idx].data[1], scalar[semi_idx].data[2], scalar[semi_idx].data[3]);
        //printf("CHECK LAST BLOCK DATA-thread %d, offset=%d: %llx, %llx, %llx, %llx\n", i, (&scalar[idx]) - scalar, scalar[idx].data[0], scalar[idx].data[1], scalar[idx].data[2], scalar[idx].data[3]);
    }

    CUDA_SAFE_CALL(hipMalloc((void**)&d_scalar,sizeof(uint288)*BATCHSIZE));
    CUDA_SAFE_CALL(hipMalloc((void**)&dbc_store_device, 6 * DBC_MAXLENGTH * BATCHSIZE * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dbc_len_device, 2 * BATCHSIZE * sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(dbc_store_device, 0, sizeof(dbc_store_device)));
    //CUDA_SAFE_CALL(hipMemcpy(d_dbc, dbc, sizeof(DBC)*N_BIGNUM, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_scalar, scalar, BATCHSIZE*sizeof(uint288),hipMemcpyHostToDevice));

    // point init
    t_p1 = (Jpoint*)malloc(100 * sizeof(Jpoint));
    t_p2 = (Jpoint*)malloc(100 * sizeof(Jpoint));
    for (int i = 0; i < 100; i++) {
        for(int j = 0;j < 4; j++){
            t_p1[i].x[j] = px[i*4+j];
            t_p1[i].y[j] = py[i*4+j];
            t_p1[i].z[j] = h_ONE[j];
        }
    }
    h_p1 = (Jpoint*)malloc(BATCHSIZE*sizeof(Jpoint));
    h_p2 = (Jpoint*)malloc(BATCHSIZE*sizeof(Jpoint));
    init_Jpoint_toOne(h_p1, BATCHSIZE);
    make_data(h_p1, BATCHSIZE);
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p1,BATCHSIZE*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p2,BATCHSIZE*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMemcpy(d_p1,h_p1,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2,h_p2,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));

    //=== TEST =====
    printf("ready\n");
    gettimeofday(&s1,NULL);
    check_dbc<<<1,128>>>(d_scalar, dbc_store_device, dbc_len_device);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

    // CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    //print_jpoint_arr(h_p1,1);
    CUDA_SAFE_CALL(hipMemcpy(dbc_store_host, dbc_store_device, 6 * DBC_MAXLENGTH * BATCHSIZE * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(dbc_len_host, dbc_len_device,2 * BATCHSIZE * sizeof(int), hipMemcpyDeviceToHost));
    gettimeofday(&e1,NULL);
    time_use=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("test usage is %ld us\n",time_use);
    
    //==== WARM UP =====
    gettimeofday(&s1,NULL);
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    test_point_double<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    gettimeofday(&e1,NULL);
    time_use=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("warm up time usage is %ld us\n",time_use);

    //==== MAIN =====
    gettimeofday(&s1,NULL);

    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

#ifdef PRECOMPUTE
    multi_scalar_multiple<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_dbc, d_p1, d_p2);
#else
    dbc_main<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_scalar, dbc_store_device, dbc_len_device, d_p1, d_p2);
#endif

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    if (N_BLOCK > 2 * N_THREAD_PER_BLOCK) accumulate_sum_per_block<<<1,N_THREAD_PER_BLOCK>>>(d_p2);
    else accumulate_sum_per_block<<<1,N_BLOCK/2>>>(d_p2);
    
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
#ifdef DEBUG
    printf("dbc dp1:\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p1,d_p1,100*sizeof(Jpoint),hipMemcpyDeviceToHost));
    for (int i = 0; i < 10; i++) {
        printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[i].x[0], t_p1[i].x[1], t_p1[i].x[2], t_p1[i].x[3]);
    }
#endif

    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p2,d_p1);
    hipDeviceSynchronize();

    CUDA_CHECK_ERROR();
    gettimeofday(&e1,NULL);
    time_use=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("time_use is %ld us\n",time_use);

    // used in accuracy test.
    // printf("Check DBC ans: \n");
    // for (int i = 0; i < 8; i++) {
    //     printf("\nCheck DBC %d ans: \n", i+1);
    //     for (int j = 0; j < dbc_len_host[2 * i]; j++) {
    //         int* wtf = dbc_store_host + i * 6 * DBC_MAXLENGTH + j * 3;
    //         printf("{[%d]2^%d 3^%d} ", wtf[0], wtf[1], wtf[2]);
    //     }
    // }

    free(scalar);
    free(dbc_store_host);
    //free(h_p1);
    //free(h_p2);
    CUDA_SAFE_CALL(hipFree(d_scalar));
    CUDA_SAFE_CALL(hipFree(dbc_store_device));
    CUDA_SAFE_CALL(hipFree(dbc_len_device));
    CUDA_SAFE_CALL(hipFree(d_p1));
    CUDA_SAFE_CALL(hipFree(d_p2));
}
#include "hip/hip_runtime.h"
#include<stdio.h>
#include "gpuec256.h"
#include "cuda_common.h"
#include<sys/time.h>
#include<random>
// typedef unsigned long long UINT64; //定义64位字类型
// typedef long long INT64;

#define N_BIGNUM 256*256
#define N_POINT N_BIGNUM
#define N_THREAD_PER_BLOCK 256
#define N_BLOCK ((N_BIGNUM+N_THREAD_PER_BLOCK-1)/N_THREAD_PER_BLOCK)
// const UINT64 h_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};
// const UINT64 h_mon_ONE[4]={0x1000003d1L,0x0L,0x0L,0x0L};
// const UINT64 h_ONE[4]={0x1L,0x0L,0x0L,0x0L};
const UINT64 h_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};

const UINT64 h_6Gx[4]={0x252931db128244c9L,0x80ec2e92027d7e6eL,0x32c5ee6d51cb1e89L,0xb89bd74c7352f570L};
const UINT64 h_6Gy[4]={0xd8cbce4f20d0d9e4L,0x5b636389add7cc6eL,0xccd07463f61e7fbeL,0x13fae72c0d3c849bL};
const UINT64 h_6Gz[4]={0xbc5c645f1b1c297dL,0x0ba1469cd0bdd88aL,0x40bad30e143dcdceL,0x4bba49beb75cce43L};



const UINT64 h_Gx[4]={0x59F2815B16F81798L,0x029BFCDB2DCE28D9L,0x55A06295CE870B07L,0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4]={0x9C47D08FFB10D4B8L,0xFD17B448A6855419L,0x5DA4FBFC0E1108A8L,0x483ADA7726A3C465L};

const UINT64 h_3Gx[4]={0x8601f113bce036f9L,0xb531c845836f99b0L,0x49344f85f89d5229L,0xf9308a019258c310L};
const UINT64 h_3Gy[4]={0x6cb9fd7584b8e672L,0x6500a99934c2231bL,0x0fe337e62a37f356L,0x388f7b0f632de814L};
const UINT64 h_3Gz[4]={0x1L,0x0L,0x0L,0x0L};

// __constant__ UINT64 dc_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};
// __constant__ UINT64 dc_ONE[4]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// __constant__ UINT64 dc_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};
unsigned long long px[] = {0xc1291ea73f54ce04ll, 0xbaae63f186512829ll, 0x849ffc311521657all, 0xdb05b8be3bc3ea12ll, 0xec734eb90e60d146ll, 0x47e3a925f4c2f8d3ll, 0xe404a3c34bfc4331ll, 0xa09f73b213ee39abll, 0xdb13ac478ea2128ell, 0x2c53e4e8f64e797dll, 0x39b7a582468b2b26ll, 0x8a7cc04c32f86ce1ll, 0xcd358ec4bf05d09ell, 0x6b2044c68221ce4ll, 0x87a85903221d0748ll, 0x46f323835084b5a9ll, 0x46f97386edeaf803ll, 0xa49b5c87eb315db7ll, 0xe3081f1ad7d6d04all, 0x83059f0ea01a4aeell, 0xee6fba127fec7397ll, 0xaba6a4a20bad9c85ll, 0x7644585b6c958d2cll, 0x84938de81702b2a3ll, 0x1a3ad24da633218ell, 0xec8c6b7a2e2d7fd4ll, 0x5893289ae2b2d9bll, 0xe5ef285c943653aell, 0xa04bd5faef99b5cell, 0x978380dff860c313ll, 0xa993bd51570b133ell, 0xb6d827a6ffe8c506ll, 0xf3fc602ee2294187ll, 0x885972979b942caell, 0xfe36bc732ebf4a28ll, 0x44524be479e97992ll, 0xf46d14dc6fdec9fll, 0x8548cb4caa31b9d7ll, 0xeefb2bfedba2f870ll, 0xee2cf31377c30c02ll, 0x8abcb93f4f13ae56ll, 0x4b03bbfee680553ell, 0x17a359c48f5f772ell, 0x4b8590bc7ad10993ll, 0xd81ef93a1c4bc230ll, 0x3fe5e94a6137a639ll, 0xdf6937fecfe2ad57ll, 0x70c0b2bac6f886f1ll, 0x9f04236a79cf01f6ll, 0x13e1640167007df8ll, 0xf780f2e0d9a1ae0fll, 0xbf7d094c158291cfll, 0x212b875aa42d2b3cll, 0x446ebb199ab8c39fll, 0xdcaf114ec3d14cfdll, 0x2ba87b8b8d3f1f59ll, 0x9bcd364ea3325d90ll, 0x3a47a15979d76791ll, 0x587c8cc21bc825dell, 0x89c87f72dfc370fcll, 0xb0d283bbfa0e29b3ll, 0x26308aa1138b7509ll, 0xaf529fa55238314ll, 0x39816c256e503a7ell, 0x5f22c397ef9fe0all, 0xb9e5bdb683e15a95ll, 0xaaa3fb7ade2d4d88ll, 0x4d7e7ee5951e2398ll, 0x5470acdb8695857ll, 0x5166d1ad1ad03d79ll, 0xf3305c87d6eb6a66ll, 0xc176900a101104fll, 0xce06d34209f8948all, 0xd82193ffdfe2c9e5ll, 0x931075c79ce0d306ll, 0x775fb75a7642d179ll, 0x888cda266c2aad54ll, 0x40fc82795775f1fall, 0x497bad018e821b36ll, 0xa29b1ff55c8448c9ll, 0x69af3b5510579da2ll, 0x2f1fbea9ba3d7984ll, 0x774fd33a1752595cll, 0x56a3a84b5041c8ebll, 0x7cde84ea0461ba1cll, 0x603181bf3283c6d6ll, 0x62f2c535e53dcee8ll, 0x2451e9cc1fdf3d7ll, 0xda232c0afcdc6597ll, 0xeb4ba40875b6a4dall, 0x2d5f9584192f072bll, 0x3198177f0b01bcdcll, 0xacd3d97303bbd12cll, 0x8db5b517cbe942e0ll, 0x51d09eb6d2933d2ll, 0xe6d9c03b36990772ll, 0xb964889dbf78ebd5ll, 0xb9811d6fc0356d18ll, 0xa2410b8e143e5130ll, 0xcc38ce20d9807de7ll};

unsigned long long py[] = {0x8ceceeb28411ba32ll, 0x3e781792a382e71ell, 0xd3fa02dae181fc95ll, 0x2e304597764f60a6ll, 0xfce1486fcc18d692ll, 0x1375c5b6e87f2c3all, 0xb48702fc8a96ba27ll, 0x57e69125d0114f2cll, 0x362d1b3393fa7588ll, 0x78796df78872500dll, 0x9138dfc1bdc15a3dll, 0xa167f3d2414d0205ll, 0x1b9b9a5f34b8afecll, 0x583a0f67a99a3788ll, 0xc4acd28dffe1410cll, 0xf07e21417abbcc3cll, 0x3ade3c45dfbc863fll, 0x9eda6f139f31b018ll, 0xe0adb41adfca8622ll, 0x15ea15417084f16bll, 0xb3b2c55417193756ll, 0x13712c30fdc9e718ll, 0x2952caab2f05eab1ll, 0x358afc9cbd7e5192ll, 0xd536b8e7f4c507d6ll, 0x66c5a5ff171507e0ll, 0x725b2f3fb86bac7bll, 0x9e36795fd946b087ll, 0x357fcb8b68e5b458ll, 0x7766f17d2f3054d7ll, 0x8343812a93422063ll, 0xa1f100198fe5c71dll, 0x68620cb7296393a2ll, 0xd95ea31298a2c21dll, 0x77d391aa12784ca4ll, 0x8fcf1ba16eaf7c5bll, 0xc68cfe2925c1ac51ll, 0x2352bcdd153d436cll, 0xcfeeeb13fb787307ll, 0xac675e68a3bff56bll, 0xd621070cf7495c20ll, 0x31c021898ac69039ll, 0x8f0c823687669c4all, 0x144b3504c3329402ll, 0x10a0a577850a8ac9ll, 0x2a0026f19fabf26ll, 0x8b726c9485375803ll, 0xf2e492176f575e7ell, 0xbba9a542f2b6cf79ll, 0x117cb4563c3d2f96ll, 0x65b88205addecb52ll, 0x615c0a66a7d6a4dell, 0x35f4afe2701217e2ll, 0x15e4a785245be7e3ll, 0x2993bd9756922e8ell, 0x5c2ff49d4f54fda0ll, 0xa82c16071118f14fll, 0x800ca14fc42286e7ll, 0x1e5b6da6f7f886e3ll, 0x37997aecc99fc704ll, 0x6991710fc91f47e3ll, 0x9877646ac9334869ll, 0x61ac8c5c317aba27ll, 0x69b0ede62479a906ll, 0xf1d3bf52ace992d5ll, 0x1c9980570a29208all, 0x53784402059629d6ll, 0x9698f219e1c2443all, 0xf6f31dc5852bdbbbll, 0x261e0d1867a702f8ll, 0xd166e64d5d8829b9ll, 0xc77dd21b6cbdd1b6ll, 0xe69accceb5e2b4e9ll, 0x69baa5bbe515be9dll, 0xb616554bece011cdll, 0xa34be20aa7d9c90cll, 0x5e7cc1b8428e709cll, 0x5c58a6ac2c829e11ll, 0x9a3dce73b43a903ll, 0xa58d08ec4c58bd52ll, 0x7f7751124655e13ll, 0x5424aa4fdbf8b18dll, 0xc982cbae45ccce95ll, 0x9398e8c27258250dll, 0xf17129bfa5b1b510ll, 0x41fe283dfb4327afll, 0x5f2f33e7714c97dall, 0x559cf7242bb8fe8ell, 0x17b3a3bb46dd3e91ll, 0x1af7d0a9f1b04817ll, 0x6d23e0dd841a2672ll, 0x8f6d1a2693629455ll, 0xff20d2520292748dll, 0x24d2beca6756a36dll, 0x7644c94f54b6cb43ll, 0x8b5d5592833e959fll, 0xb0c016d4371e5f9ell, 0xa0acac44e0ca9561ll, 0x9d27f8c2c840e7ell, 0x73b4245bfb56fc89ll};

unsigned long long px3[] = {0x4d3c296b3e8b6988ll, 0xddcc9ca08396d1cbll, 0x7ad77e63364511dfll, 0xa0ac877d59fe43all, 0x4a87562b177eae72ll, 0xf52831b49c6a6112ll, 0xf0b0a46021a488d2ll, 0xd4575222d4cda88bll, 0x9fd2acb1e587292bll, 0xd8c007515b734f4ll, 0x86d785c545c96c0dll, 0x81b2826c2c4d2dd8ll, 0x94001024d35f4e03ll, 0xa7f67326231808c0ll, 0x49e61768c043f864ll, 0xa6bb07c8044cb754ll, 0x3bc5fce63b5c6eb7ll, 0xe975105a35c835b0ll, 0x5449ac7731bf26a6ll, 0x8bb21922e7870e9cll, 0xbf51fc11ed6b7becll, 0x90c2a7dd970b6a7all, 0x87b1d7c096ab0fd1ll, 0xef4efa6172ed7f00ll, 0x70bfa5991f8040b5ll, 0xf63d2f69ab92e571ll, 0x30f4f32599299452ll, 0xc55ac8e19c98b574ll, 0x1e7bd7e1f2a7f00all, 0x9bcc5a675c011ab3ll, 0x1b0ac2245d3f3e60ll, 0x6f91233e31a13e1cll, 0x15c3bbf222eb03b3ll, 0x39c2af9d552948efll, 0x51a6d8ded9469e3bll, 0x162d2cf1cfd90c8ell, 0xc3241eee21e04335ll, 0x1f7ef0ec35bb3a16ll, 0x2082ddcd712cea20ll, 0xea6dd878e1cf83bell, 0xe20708512d57af51ll, 0x53d8c5e73bfd68e7ll, 0xbc06c96708a2d80dll, 0xe5303f14e2c4ed2ll, 0xf2baa21ec43fbf3cll, 0xc66aaf40605089d8ll, 0xb98898d342d1a70all, 0x79eaa5df944cdcc3ll, 0xa4ae33d8a9392101ll, 0xd2bd0d1e685b2895ll, 0xf5dee131682af06all, 0xee9d971db32100a9ll, 0x979d6bc8cba8f21all, 0x627a73d284d1e077ll, 0x3791b8bed2f990ecll, 0xf1b664cf858bf41ll, 0x8bf00b84d5ef3043ll, 0x5290e9b8024f8d15ll, 0x586a32e199bc60afll, 0xb0011ca2bfc88320ll, 0xd246570ad39d881dll, 0xa8d58a8fb5195e07ll, 0x2d81c5167f40a226ll, 0x4d5f609d233f8b61ll, 0xe9c3989a896b89d7ll, 0xeb4f7588bf903347ll, 0x833013f909b93343ll, 0x9a573acb50f4a9ebll, 0x66d01a5810f8d137ll, 0xaa9f929069bcc2dell, 0xa6adb6c784046febll, 0x773fb2dd1fca11d2ll, 0xf8504ef7bbb60baell, 0xb8ff3fd004c40d6dll, 0xa2709481fa5be1f4ll, 0xe3b67f87945b1f0dll, 0xb8c2b550fe024263ll, 0x162da9f5da318c7ell, 0x43bd85c11f0e6f40ll, 0x576e83352bbff70bll, 0x6f532e35310c746ll, 0xde275c4ae8b7c108ll, 0x366542ca116d445dll, 0x192377a9f89ad8f5ll, 0xee8f3b67d22d5d99ll, 0x579cb13009210d66ll, 0xf3873431e3850df3ll, 0x7c69e10852936f22ll, 0x2b3a9bd371dfaba7ll, 0x6be9676d1bbce350ll, 0x7e6ee26c8b0dc356ll, 0x1b3261a0e40f0779ll, 0xa9706717ed456ab8ll, 0x5d765dabbeb674c6ll, 0x7d739283e4f43033ll, 0x8e305790a447736all, 0xe352f5943cf487fbll, 0x64398b5b8cbe1143ll, 0xfc2cb7207d937bd2ll, 0x7753cf89671c8cdell};

unsigned long long py3[] = {0xcacdf749cb8d5f19ll, 0x3a26b17a6f13f006ll, 0x1cde4777eaf1fa11ll, 0xa4414ed881c0e56dll, 0x3b571c5fa5508c0ell, 0x7cdc1849f76f2ec2ll, 0x1a468db56c8568e2ll, 0x8c57ec98bf38fe59ll, 0xd10b947bbcaf4bf5ll, 0x4e64eb670fae53bell, 0x430c61252c4de283ll, 0xab68117b7ff30299ll, 0x26a8f75f7d79ee30ll, 0x1053de0301a4bccbll, 0xbb50c1849212a0f1ll, 0x423862d46548c4bell, 0xc31f86616d9390ecll, 0xfa83bcfd2bc8718fll, 0x22d6538e7ca2b142ll, 0x21ebff421deb94b7ll, 0x8a5128224b2e1e5bll, 0xb518a254c60bd352ll, 0x8c10a7ec1dd632cfll, 0xb8b94fd1a20f561ell, 0xcbf7274e4320fde9ll, 0x7f825189b430ad19ll, 0x7fc5b0de13e6ae42ll, 0xb61ac93784c46e70ll, 0x86e902c618d1d0e2ll, 0xc7ab820d0f5ddfa6ll, 0x2927e90be13fcc69ll, 0x5a7a8539a069fa70ll, 0x8fa0db2c14690950ll, 0xee3365a358ca1d1bll, 0xd2107bae0e0d2c93ll, 0xb19305443f66057ll, 0x505575ab92a3dbd0ll, 0xfc0d0110bab28c74ll, 0x64b5014ed56a5165ll, 0x210a5c2d815d6c7ell, 0x71c2a24bc399717ll, 0x2f7c82a1be3ed40all, 0xa0fe03461542dc9all, 0x9c8175bc3b7aac92ll, 0x52a5d275fa70f38fll, 0x7f280e75ef818498ll, 0x782e3358a54ea194ll, 0xb286ede5b6c69b77ll, 0xb2fae70f5b4cb3e6ll, 0x985669303148fc90ll, 0xba4f880f241bfd5ll, 0x5134a1168ffb9c2bll, 0x88aaa0f440c2fc84ll, 0x732a815a26af6096ll, 0x8ada0e8b66cc7c48ll, 0x85d80c3e63934b97ll, 0xc276948f76dc160cll, 0x83354992d9f8f4ecll, 0xdb89f215877d3fd2ll, 0x607cf4a51e6d4beall, 0x7bc812a269cdb273ll, 0x5e91c14838a6da32ll, 0xeea1beca0c82783cll, 0xb402c8733b869ce8ll, 0x6bb6539a3a626f9fll, 0x65c3c28b9fb0dd7ell, 0xfd31658429a1fd8cll, 0x283fb6056d1dbd54ll, 0x4d1fd3ab9ce96b92ll, 0x59307fca2f3e66d2ll, 0x47899b04717f8830ll, 0x2c04c9c438a97ccell, 0x41a43e034990ccb2ll, 0xac5c62a314ae7fc1ll, 0x2bc9c15f5ce5f21dll, 0x4351c7f4bc543587ll, 0xe714a0633eb54391ll, 0x49a981d8f96d8c83ll, 0xcc039db192259e41ll, 0xb0365ab40ef69a54ll, 0xf7c0aa189d06d5a4ll, 0xe3c9ef6dbcbe0cb3ll, 0x5053f2e2d58472e7ll, 0xeed3a254c43ae758ll, 0x783efea48bc01e51ll, 0xfd450216e34bb240ll, 0xaa59875b491dfe0dll, 0x18c4e66c88aa43a4ll, 0x21ea3d478bc5b59bll, 0x50b61cc4607a4882ll, 0x7c3a24521c8e8bb8ll, 0xdcc23b4a6fae5af9ll, 0x34f723ffe88c492ll, 0x7a1049306af2c39cll, 0xb8fafb10e9e11d70ll, 0x45e280142165ae6ell, 0xd1bc897d2caf1fb5ll, 0x58c41c04b69ddcfdll, 0x3869b0569cc204c5ll, 0xfc0be1c06a7ecb4ell};


unsigned long long dx[] = {0x3e9b58b481a0c1e7ll, 0xd2790f58ad702ac3ll, 0xf3df624ee861f1all, 0x7422dbc25514e317ll, 0xb6c363056b6d570ll, 0x64b8ed462321d1b1ll, 0xfaf242de5b1e7585ll, 0xd4c307f48c33830fll, 0xd418348214808c65ll, 0xf8932fa155f832ebll, 0xd9145a22f78c225cll, 0xac037fc3f8423648ll, 0xa28bafa272ad91a5ll, 0x9795cc3f52dd1fdfll, 0x617bf6ea64012f39ll, 0x15c959cd69b96173ll, 0x79938dacb05340f0ll, 0xb73c17845e5f0655ll, 0x3caed57f4bd41d80ll, 0xa1ce5385c6053c48ll, 0x2970d3e74b38c8a2ll, 0x981e9824714f72f9ll, 0x8d5deba7fe2831cfll, 0x94fc5ff28dd55157ll, 0x9b7d882b71f43546ll, 0x7c6c74ebad9a3f32ll, 0xbebd6b634e1a88eell, 0xeb0ac31cf931b374ll, 0x7861226eaee648bll, 0x3862d67ed71d09f5ll, 0x1505d4e24a270420ll, 0xf8b0088e5648acecll, 0x90c1545404bc4a0ll, 0xe297c7d92264f5fbll, 0x9ab771c2882e07a5ll, 0x30a271fd9414e288ll, 0xc47f35f595e118b5ll, 0x552c5a97b9aa7516ll, 0x144f974df1b2ad09ll, 0x427e96ea27f5846bll, 0x78820a6c772f2688ll, 0x47a88dfd3542f024ll, 0x40c3d261f513e1a8ll, 0xd62b6192ae7bdab9ll, 0x77f9db302dcd5c28ll, 0xba5881b5c7ddd140ll, 0xfe44a6e288bf3851ll, 0x4d1f56e7962c2dell, 0x3c0ca396a6f16a01ll, 0xdea3647470db11b2ll, 0xb1dfb8febcc7541bll, 0x91d5f4ac42502ad1ll, 0x43a3f5dc4363be16ll, 0x60a7186206c08f8bll, 0x4ad258e55da97160ll, 0x510df6e5c3232ec7ll, 0xa2da54efe96f9514ll, 0xe3c1b95dad759e1bll, 0xd03e5645dbea5195ll, 0x2b64bdbb8c1cd0a6ll, 0x3d64efeb284b8ad2ll, 0xed645de57f9311ccll, 0xa9caeb58742da602ll, 0xabb9772059774479ll, 0xa23cae275137c4d1ll, 0x826d0433243e9a9ell, 0x95eba298e227b592ll, 0x3a2ed739fb98c7e9ll, 0x5cfebb35d7f90e1bll, 0xf732aff4ff5a4fb7ll, 0x748112df101e87fcll, 0x834fdc11baab3838ll, 0xd75c9e46d0aee654ll, 0x1745092a6abf8770ll, 0x1362d90dcded199ell, 0x863dfc5e91a7a9b5ll, 0xd18f3cd686e412fll, 0xe04fe216bcc01378ll, 0xb3a1cd3bdce4c117ll, 0x2283efec24582825ll, 0xf744c2c74567e2f5ll, 0x97afc09b7b282e43ll, 0xd198c777f05886bbll, 0x6b23e4c0f51dadb5ll, 0xdae050c76c1eddb3ll, 0xdaab67905a7d0659ll, 0x2b174248a0584645ll, 0xd34e1a4d0bd960b1ll, 0x808d3cc6dd83fc59ll, 0xbf91a51648dd0960ll, 0xf38567e92f48df68ll, 0xacb571f8dd66500ll, 0xecd41f0b86138682ll, 0x21e873127c645bb6ll, 0xb5a0e535ee29b6b1ll, 0xeaeeab56d06c2c0ell, 0x929db043d8f2cdfall, 0xc0d2a2608be97e5ll, 0x426dc138552c4432ll, 0x868a971759b755ccll};

unsigned long long dy[] = {0x3542486d3e8384e3ll, 0x13d051ad6b464dedll, 0x5b42fc37a6293dbdll, 0x6e4b2cbd7e92276fll, 0x9fef0e9c8b15264fll, 0xabb70e6bee9cb2ll, 0xee25466956363179ll, 0xfd11ac5d51563753ll, 0x58b1fc3af2489a29ll, 0xa82aa19eaac8748ll, 0x51f02a42fec69c5fll, 0x725f87d6edcef39ll, 0xb51e0df1a07df532ll, 0x8e08134f4074abd4ll, 0xbfdfdba93252fa31ll, 0x37c5c7692f917865ll, 0xe0ed9cb3d636f6b3ll, 0x5b83a93f64405366ll, 0xf06a4f823a1989d7ll, 0xb9ff31ce194773bfll, 0x1d98a6d47f5270c2ll, 0xb43486e2eb7e136bll, 0x64d788eb8413f93dll, 0x9bb30eb73e93d3b2ll, 0xf309ee64e7d4f92ll, 0xb8dde885b32312all, 0x21118ded95ac6f29ll, 0xc983dab25faec20bll, 0x7b74ac6b8d452c1bll, 0xcaccbe26c46e1ea2ll, 0xfeab89e32a652279ll, 0x81b537578dece23dll, 0x714ece790da7d5ebll, 0x771bbd884024ee48ll, 0x6881642b2437ea69ll, 0x72a353465dbfc234ll, 0x2e7739c8b3284985ll, 0x518b6ce43f74b9fdll, 0x1d11b310f142340cll, 0xb8b5f8a7047cc78bll, 0xc98112a0eccee6aell, 0xd4a12b5796e5b7a0ll, 0x24d6431fd4b6ea05ll, 0xafa380a9e9478faall, 0x98a35a0f382f2211ll, 0x608d3b1db228e853ll, 0xec7ada4b8c2e0277ll, 0x59b28e0ee1d21e1cll, 0xc0a93100d57028call, 0xb2ee8e366d7a78d3ll, 0x409e61311cac105bll, 0x152478a51a53cec0ll, 0xa6a22e64bcf36c49ll, 0x32130c6462739094ll, 0x915ff7ea83bb5e2ll, 0xcc03a67084696f4cll, 0xe208ec68ca092003ll, 0xdb6bf75f2140aa60ll, 0xef36c6cc11586b3cll, 0x8afb419de213ac3dll, 0x5e382f8e520c04e1ll, 0x5f92c6bc34ebbb89ll, 0x398db743bfb374fall, 0xe19e0e6e3ceeaa42ll, 0x9e7ea5242816d0d6ll, 0x11dcaa44038fce38ll, 0xc6e3725ac42e00a4ll, 0x71443ce92e90d37cll, 0x23dd572671f8b692ll, 0xde27f7bde1a28bb9ll, 0xe3fc1cc34a75f2ddll, 0x8e5380a90833dfd6ll, 0x4da2b42de56731fdll, 0x36a6009f9eee69bcll, 0xf4db711db3e119ccll, 0x5e8ccc990795a889ll, 0xafd703419dfea9a0ll, 0x30463f3c6ee2b046ll, 0x9e2b31784a512bbll, 0xe94ad634e74bcf82ll, 0x45f3db452cd7f9bbll, 0xde6b6252dd33782bll, 0x193d5f6c04a27310ll, 0x3acb8fa99c329707ll, 0x22d69badb37471f0ll, 0xf1c99f10c56b7cf5ll, 0xfe6110920f406657ll, 0x2258d6aa622458d9ll, 0xd635d70116f7d3fcll, 0x623d3502b380a0fell, 0x967638214e7752d8ll, 0x8ebe4d0ebc1b3a36ll, 0x4693a1c02ec948b3ll, 0xd30995eb3700fd54ll, 0x703135b40eee5981ll, 0x47513b99adac2c47ll, 0x4807b69ef64a62e3ll, 0x351d3b0f620edf57ll, 0xe2d20985d347dddfll, 0xcf1d0224f5f5ba61ll};

unsigned int dx2[] = {0x90a25e6d, 0x266b1fdd, 0xc7e5f9b1, 0x8b331f76, 0x86c26260, 0x11602198, 0x3d449367, 0xe221abb8, 0x20512bec, 0xee5d4140, 0x7a4a63ea, 0x9c9a8137, 0x4518e71d, 0x90bb01b9, 0xc8a0dbb1, 0x1c5859f8, 0xb71d74a2, 0xe627b83d, 0x63ad710f, 0x1b0590aa, 0x47f31b7d, 0x8b247df8, 0x680ceb24, 0x84c82aa9, 0xe16a482, 0x1b303012, 0xa6208757, 0x62842d6f, 0x27ae4f0c, 0x893d6f09, 0xa4687085, 0xa1ffc667, 0xea82a239, 0x17d73135, 0xcb5a9d69, 0x9fb058fc, 0x2998e75c, 0xf0fc9d2, 0xfb1c8ce, 0x79e3bc7c, 0x231366b5, 0x40392f2, 0x3a096a07, 0xd3b975cf, 0xfc49c863, 0xd9b437c6, 0xfea912dc, 0x67224518, 0x92ea1d92, 0x32594898, 0x7436f88d, 0x9a3b2801, 0x54930fd, 0x9273f995, 0x3b9bdecb, 0x44a0665f, 0xde53d2e8, 0xe611f8a1, 0x47dd26d5, 0x62006088, 0x18adef96, 0x40a57445, 0x96d64655, 0xc1c91992, 0xd8d12730, 0x7aeda909, 0x85eaf4c4, 0x60f20c05, 0xc1ed3293, 0xc322ca3f, 0x4c7ed042, 0x73c2b31d, 0xe0396b9d, 0x3020d47f, 0xd4c5d6c6, 0x523f71c0, 0x8ce47a37, 0x7859e90a, 0x7686e532, 0xdb086e00, 0x909ac2c3, 0x574ed015, 0x9e6fa778, 0x54ecf2a9, 0x2db98885, 0xec671188, 0xc0de2005, 0x5ec92a71, 0x78b628e2, 0x9c9816e2, 0x69e13f3d, 0x4fc3827b, 0xae393a9e, 0x6f45648, 0xa85e2113, 0xa2b591bc, 0xaaca0b99, 0x24033483, 0xf90f13b2, 0xa29a99cc};

unsigned int dy2[] = {0x90a25e6d, 0x266b1fdd, 0xc7e5f9b1, 0x8b331f76, 0x86c26260, 0x11602198, 0x3d449367, 0xe221abb8, 0x20512bec, 0xee5d4140, 0x7a4a63ea, 0x9c9a8137, 0x4518e71d, 0x90bb01b9, 0xc8a0dbb1, 0x1c5859f8, 0xb71d74a2, 0xe627b83d, 0x63ad710f, 0x1b0590aa, 0x47f31b7d, 0x8b247df8, 0x680ceb24, 0x84c82aa9, 0xe16a482, 0x1b303012, 0xa6208757, 0x62842d6f, 0x27ae4f0c, 0x893d6f09, 0xa4687085, 0xa1ffc667, 0xea82a239, 0x17d73135, 0xcb5a9d69, 0x9fb058fc, 0x2998e75c, 0xf0fc9d2, 0xfb1c8ce, 0x79e3bc7c, 0x231366b5, 0x40392f2, 0x3a096a07, 0xd3b975cf, 0xfc49c863, 0xd9b437c6, 0xfea912dc, 0x67224518, 0x92ea1d92, 0x32594898, 0x7436f88d, 0x9a3b2801, 0x54930fd, 0x9273f995, 0x3b9bdecb, 0x44a0665f, 0xde53d2e8, 0xe611f8a1, 0x47dd26d5, 0x62006088, 0x18adef96, 0x40a57445, 0x96d64655, 0xc1c91992, 0xd8d12730, 0x7aeda909, 0x85eaf4c4, 0x60f20c05, 0xc1ed3293, 0xc322ca3f, 0x4c7ed042, 0x73c2b31d, 0xe0396b9d, 0x3020d47f, 0xd4c5d6c6, 0x523f71c0, 0x8ce47a37, 0x7859e90a, 0x7686e532, 0xdb086e00, 0x909ac2c3, 0x574ed015, 0x9e6fa778, 0x54ecf2a9, 0x2db98885, 0xec671188, 0xc0de2005, 0x5ec92a71, 0x78b628e2, 0x9c9816e2, 0x69e13f3d, 0x4fc3827b, 0xae393a9e, 0x6f45648, 0xa85e2113, 0xa2b591bc, 0xaaca0b99, 0x24033483, 0xf90f13b2, 0xa29a99cc};



#ifndef dh_mybig_copy
#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}
#endif

void make_data(Jpoint *in, int datasize) {
    int block = 31;
    int blocknum = datasize / 31;
    int other = datasize - blocknum * 31;
    printf("datasize = %d\n", datasize);
    unsigned int rand = 1;
    for (int i = 0; i < blocknum; i++) {
        for (int j = 0; j < block; j++, rand++) {
            int idx = i * block + j;
            
            int wtf = (j / 2) * 4; // att: can't simplify this.
            if (j % 2 == 1) {
                dh_mybig_copy(in[idx].x, px+j);
                dh_mybig_copy(in[idx].y, py+j);
            } else {
                dh_mybig_copy(in[idx].x, px3+j);
                dh_mybig_copy(in[idx].y, py3+j);
            }
            //in[idx].y[0] += rand;
        }
    }
    for (int start = blocknum * 31; start < datasize; start++) {
        dh_mybig_copy(in[start].x, px);
        dh_mybig_copy(in[start].y, py);
    }
    printf("makes data\n");
}

void make_data2(Jpoint *in, int datasize) {
    int block = 31;
    int blocknum = datasize / 31;
    int other = datasize - blocknum * 31;
    printf("datasize = %d\n", datasize);
    unsigned int rand = 1;
    dh_mybig_copy(in[0].x, px);
    dh_mybig_copy(in[0].y, py);
    for (int i = 0; i < blocknum; i++) {
        for (int j = 0; j < block; j++, rand++) {
            int idx = i * block + j;
            
            int wtf = (j / 2) * 4; // att: can't simplify this.
            if (j % 2 == 1) {
                dh_mybig_copy(in[idx-1].x, dx+j);
                dh_mybig_copy(in[idx-1].y, dy+j);
            } else {
                dh_mybig_copy(in[idx-1].x, dx+j);
                dh_mybig_copy(in[idx-1].y, dy+j);
            }
            //in[idx].y[0] += rand;
        }
    }
    for (int start = blocknum * 31 + 1; start < datasize; start++) {
        dh_mybig_copy(in[start].x, dx);
        dh_mybig_copy(in[start].y, dy);
    }
    printf("makes data\n");
}

void init_Jpoint_toOne(Jpoint* p, int n){
    for(int i=0;i<n;i++){
        for(int j=0;j<4;j++){
            p[i].x[j] = h_Gx[j];
            p[i].y[j] = h_Gy[j];
            p[i].z[j] = h_ONE[j];
        }
    }
}


void __global__ testadd(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_modadd_64(a+tx*4,b+tx*4,c+tx*4);
}
void __global__ testsub(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_modsub_64(a+tx*4,b+tx*4,c+tx*4);
}
void __global__ testmul(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*4,dc_R2,a+tx*4);
    // dh_mybig_monmult_64(b+tx*4,dc_R2,b+tx*4);
    dh_mybig_monmult_64(a+tx*4,b+tx*4,c+tx*4);
    dh_mybig_monmult_64(c+tx*4,dc_ONE,c+tx*4);
}



void __global__ testhalf(UINT64 *a,UINT64 *b){
    int tx = threadIdx.x;
    dh_mybig_half_64(a+tx*4,b+tx*4);
}

__global__ void testinv(UINT64 *a,UINT64 *b){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*4,dc_R2,a+tx*4);
    dh_mybig_moninv(a+tx*4,b+tx*4);
}

__global__ void testexp(UINT64 *a,UINT64 *b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*4,dc_R2,a+tx*4);
    dh_mybig_modexp(a+tx*4,b+tx*4,c+tx*4);
    dh_mybig_monmult_64(c+tx*4,dc_ONE,c+tx*4);
}

void __global__ point_to_monjj(Jpoint* jp1,Jpoint* jp2){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_BIGNUM){
        dh_mybig_monmult_64((jp1+idx)->x,dc_R2,(jp1+idx)->x);
        dh_mybig_monmult_64((jp1+idx)->y,dc_R2,(jp1+idx)->y);
        dh_mybig_monmult_64((jp1+idx)->z,dc_R2,(jp1+idx)->z);

        dh_mybig_monmult_64((jp2+idx)->x,dc_R2,(jp2+idx)->x);
        dh_mybig_monmult_64((jp2+idx)->y,dc_R2,(jp2+idx)->y);
        dh_mybig_monmult_64((jp2+idx)->z,dc_R2,(jp2+idx)->z);
    }
    
}



void __global__ point_from_monjj(Jpoint* jp1,Jpoint* jp2){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_BIGNUM){
        dh_mybig_monmult_64((jp1+idx)->x,dc_ONE,(jp1+idx)->x);
        dh_mybig_monmult_64((jp1+idx)->y,dc_ONE,(jp1+idx)->y);
        dh_mybig_monmult_64((jp1+idx)->z,dc_ONE,(jp1+idx)->z);

        dh_mybig_monmult_64((jp2+idx)->x,dc_ONE,(jp2+idx)->x);
        dh_mybig_monmult_64((jp2+idx)->y,dc_ONE,(jp2+idx)->y);
        dh_mybig_monmult_64((jp2+idx)->z,dc_ONE,(jp2+idx)->z);
    }
    
}


// void __global__ point_to_monaj(Jpoint* jp1,Apoint* jp2){
//     int tx = threadIdx.x;
//     dh_mybig_monmult_64((jp1+tx)->x,dc_R2,(jp1+tx)->x);
//     dh_mybig_monmult_64((jp1+tx)->y,dc_R2,(jp1+tx)->y);
//     dh_mybig_monmult_64((jp1+tx)->z,dc_R2,(jp1+tx)->z);

//     dh_mybig_monmult_64((jp2+tx)->x,dc_R2,(jp2+tx)->x);
//     dh_mybig_monmult_64((jp2+tx)->y,dc_R2,(jp2+tx)->y);
//     // dh_mybig_monmult_64((jp2+tx)->z,dc_R2,(jp2+tx)->z);
// }



// void __global__ point_from_monaj(Jpoint* jp1,Apoint* jp2){
//     int tx = threadIdx.x;
//     dh_mybig_monmult_64((jp1+tx)->x,dc_ONE,(jp1+tx)->x);
//     dh_mybig_monmult_64((jp1+tx)->y,dc_ONE,(jp1+tx)->y);
//     dh_mybig_monmult_64((jp1+tx)->z,dc_ONE,(jp1+tx)->z);

//     dh_mybig_monmult_64((jp2+tx)->x,dc_ONE,(jp2+tx)->x);
//     dh_mybig_monmult_64((jp2+tx)->y,dc_ONE,(jp2+tx)->y);
//     // dh_mybig_monmult_64((jp2+tx)->z,dc_ONE,(jp2+tx)->z);
// }


void __global__ testdouble(UINT64* a ,UINT64* b){
    int tx = threadIdx.x;
    dh_mybig_moddouble_64(a+tx*4,dc_p,b+tx*4);
}

// void __global__ test_point_addaj(Jpoint* jp,Apoint *ap){
//     int tx = threadIdx.x;
//     dh_ellipticSumEqual_AJ(jp+tx,ap+tx);
// }
void __global__ test_point_addjj(Jpoint* p1,Jpoint *p2,Jpoint *p3){
    int tx = threadIdx.x;
    dh_ellipticAdd_JJ(p1+tx,p2+tx,p3+tx);
}

void __global__ test_point_double(Jpoint *p1,Jpoint *p2){
    int tx = threadIdx.x;
    ppoint_double(p1+tx,p2+tx);
}
void __global__ test_point_triple(Jpoint *p1,Jpoint *p2){
    int tx = threadIdx.x;
    ppoint_triple_v2(p1+tx,p2+tx);
}

void __global__ testbasemul(Jpoint *res,UINT64 *k){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_POINT){
        // d_mybig_print(k+idx*4);
        d_base_point_mul(res+idx,k+idx*4);
    }

}

void __global__ test_point_mul_inplace(Jpoint *p1,UINT64 *k){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_inplace(p1+idx,k+idx*4);
}
void __global__ test_point_mul_outofplace(Jpoint *p1,UINT64 *k,Jpoint *p2){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_outofplace(p1+idx,k+idx*4,p2+idx);
}
void __global__ test_point_mul_finalversion(Jpoint *p1,UINT64 *k,Jpoint *p2){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_finalversion(p1+idx,k+idx*4,p2+idx);
}

void __global__ test_point_mul_uint32(Jpoint *p1,int k,Jpoint *p2){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_uint32(p1+idx,k,p2+idx);
}

// void __global__ test_point_mul_apoint(Jpoint *p1,Apoint *p2,UINT64 *k){
//     int tx = threadIdx.x;
//     dh_apoint_mult(p1+tx,p2+tx,k+tx*4);
// }

// void __global__ testmul(UINT64* a,UINT64 *b,UINT64 *c){
//     int tx = threadIdx.x;
//     dh_mybig_monmult_64(a+tx*8,h_R2,a+tx*8);
//     // dh_mybig_monmult_64(a+tx*8,h_R2,a+tx*8);
//     dh_mybig_monmult_64(a+tx*8,a+tx*8,c+tx*8);
//     // dh_mybig_monmult_64(c+tx*8,h_ONE,c+tx*8);
// }

void print_big_arr(UINT64* nums,int n){
    for(int i=0;i<n;i++){
        h_mybig_print(nums+i*4);
        printf("\n");
    }
}
void print_jpoint_arr(Jpoint* nums,int n){
    for(int i=0;i<n;i++){
        h_print_pointJ(nums+i);
        printf("\n");
    }
}
// void print_apoint_arr(Apoint* nums,int n){
//     for(int i=0;i<n;i++){
//         h_print_pointA(nums+i);
//         printf("\n");
//     }
// }

void init_big(UINT64 *nums){
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<4;j++){
            nums[i*4+j]=0;
        }
        nums[i*4] = 0x3;
        nums[i*4+1] = 0x103;
    }
    // nums[0]=25;
    // nums[15]=35;
}
void init_big2(UINT64 *nums, unsigned int mask = 1){
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<4;j++){
            nums[i*4+j]=0xabcdef0123456789 ^ mask;
            mask = mask + 1;
            // nums[i*4+j]=0;
        }
        // nums[i*4] = 8;
    }
    // nums[0]=25;
    // nums[15]=35;
}
void init_random_big(UINT64 *nums){
    timeval start;
    gettimeofday(&start,NULL);
    std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine;
    engine.seed(start.tv_usec);//设定随机数种子
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<4;j++){
            nums[i*4+j]=engine();
        }
    }
}
// void init_Apoint(Apoint* p){
//     for(int i=0;i<N_POINT;i++){
//         for(int j=0;j<4;j++){
//             p[i].x[j] = h_Gx[j];
//             p[i].y[j] = h_Gy[j];
//             // p[i].z[j] = h_ONE[j];
//         }
//     }
// }

void init_Jpoint(Jpoint* p){
    for(int i=0;i<N_POINT;i++){
        for(int j=0;j<4;j++){
            p[i].x[j] = h_Gx[j];
            p[i].y[j] = h_Gy[j];
            p[i].z[j] = h_ONE[j];
        }
    }
}
void init_Jpoint2(Jpoint* p){
    for(int i=0;i<N_POINT;i++){
        for(int j=0;j<4;j++){
            p[i].x[j] = h_6Gx[j];
            p[i].y[j] = h_6Gy[j];
            p[i].z[j] = h_6Gz[j];
        }
    }
}
// void init_Apoint(Apoint* p){
//     for(int i=0;i<N_POINT;i++){
//         for(int j=0;j<4;j++){
//             p[i].x[j] = h_3Gx[j];
//             p[i].y[j] = h_3Gy[j];
//             // p[i].z[j] = h_mon_ONE[j];
//         }
//     }
// }

void test_num_lib() {
        struct timeval s1,e1;
    long long time_use=1;
    int nB,nT;
    // UINT64 tmpbig[4]={0x6903021ca8bd10e,1,0,0};
    // h_mybig_print(tmpbig);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testbasemul);
    // printf("NB=%d,NT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,testadd);
    printf("NB=%d,NT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,testmul);
    printf("NB=%d,NT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,testinv);
    printf("NB=%d,NT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,testexp);
    printf("NB=%d,NT=%d\n",nB,nT);

    UINT64 *h_nums1 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    UINT64 *h_nums2 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    UINT64 *h_nums3 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    init_big2(h_nums1, 0xf0f0f0f0f0f0f0);
    init_big2(h_nums2, 0x0f0f0f0f0f0f0f);

    UINT64 *d_nums1;
    UINT64 *d_nums2;
    UINT64 *d_nums3;



    CUDA_SAFE_CALL(hipMalloc((void**)&d_nums1,sizeof(UINT64)*4*N_BIGNUM));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nums2,sizeof(UINT64)*4*N_BIGNUM));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_nums3,sizeof(UINT64)*4*N_BIGNUM));

    CUDA_SAFE_CALL(hipMemcpy(d_nums1,h_nums1,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_nums2,h_nums2,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));

    // warm 
    gettimeofday(&s1,NULL);

    testmul<<<N_BLOCK/2,N_THREAD_PER_BLOCK*2>>>(d_nums1,d_nums2,d_nums3);
    CUDA_CHECK_ERROR();
    
    gettimeofday(&e1,NULL);
    // warmend

    gettimeofday(&s1,NULL);

    testmul<<<N_BLOCK/2,N_THREAD_PER_BLOCK*2>>>(d_nums1,d_nums2,d_nums3);
    CUDA_CHECK_ERROR();
    
    gettimeofday(&e1,NULL);

    auto time_use1=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    // CUDA_SAFE_CALL(hipMemcpy(h_nums3,d_nums3,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyDeviceToHost));
    //CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // print_big_arr(h_nums1,N_BIGNUM);
    // print_big_arr(h_nums2,N_BIGNUM);
    // print_big_arr(h_nums1,N_BIGNUM);
    // print_big_arr(h_nums2,N_BIGNUM);
    //print_big_arr(h_nums3,N_BIGNUM);
    //printf("\n");

    
    gettimeofday(&s1,NULL);

    testadd<<<N_BLOCK/4,N_THREAD_PER_BLOCK*4>>>(d_nums1,d_nums2,d_nums3);
    CUDA_CHECK_ERROR();
    
    gettimeofday(&e1,NULL);

    auto time_use2=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒

    
    gettimeofday(&s1,NULL);

    testinv<<<N_BLOCK/2,N_THREAD_PER_BLOCK*2>>>(d_nums1,d_nums2);
    CUDA_CHECK_ERROR();
    
    gettimeofday(&e1,NULL);

    auto time_use3=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒

    
    gettimeofday(&s1,NULL);

    testexp<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_nums1,d_nums2,d_nums3);
    CUDA_CHECK_ERROR();
    
    gettimeofday(&e1,NULL);

    auto time_use4=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒

    printf("lib int: scale=%d*%d\nadd: %ld\nmul: %ld\ninv: %ld\nexp: %ld\n", N_BLOCK, N_THREAD_PER_BLOCK, time_use2, time_use1, time_use3, time_use4);

    free(h_nums1);
    free(h_nums2);
    free(h_nums3);
    CUDA_SAFE_CALL(hipFree(d_nums1));
    CUDA_SAFE_CALL(hipFree(d_nums2));
    CUDA_SAFE_CALL(hipFree(d_nums3));
}

//#define TEST_NUM
int main(){
#ifdef TEST_NUM    
    test_num_lib();
    return 0;
#endif
    hipSetDevice(0);

    struct timeval s1,e1;
    long long time_use=1;
    int nB,nT;
    // UINT64 tmpbig[4]={0x6903021ca8bd10e,1,0,0};
    // h_mybig_print(tmpbig);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testbasemul);
    // printf("NB=%d,NT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,test_point_triple);
    printf("NB=%d,NT=%d\n",nB,nT);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testmul);
    // printf("NB=%d,NT=%d\n",nB,nT);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testinv);
    // printf("NB=%d,NT=%d\n",nB,nT);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,point_to_monjj);
    // printf("NB=%d,NT=%d\n",nB,nT);

    // UINT64 *h_nums1 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    // UINT64 *h_nums2 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    // UINT64 *h_nums3 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    // init_big2(h_nums1);
    // init_big2(h_nums2);

    // UINT64 *d_nums1;
    // UINT64 *d_nums2;
    // UINT64 *d_nums3;



    // CUDA_SAFE_CALL(hipMalloc((void**)&d_nums1,sizeof(UINT64)*4*N_BIGNUM));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_nums2,sizeof(UINT64)*4*N_BIGNUM));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_nums3,sizeof(UINT64)*4*N_BIGNUM));

    // CUDA_SAFE_CALL(hipMemcpy(d_nums1,h_nums1,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    // CUDA_SAFE_CALL(hipMemcpy(d_nums2,h_nums2,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));

    // testmul<<<1,N_BIGNUM>>>(d_nums1,d_nums2,d_nums3);

    // CUDA_SAFE_CALL(hipMemcpy(h_nums3,d_nums3,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyDeviceToHost));
    // // print_big_arr(h_nums1,N_BIGNUM);
    // // print_big_arr(h_nums2,N_BIGNUM);
    // // print_big_arr(h_nums1,N_BIGNUM);
    // // print_big_arr(h_nums2,N_BIGNUM);
    // print_big_arr(h_nums3,N_BIGNUM);
    // printf("\n");

    // free(h_nums1);
    // free(h_nums2);
    // free(h_nums3);
    // CUDA_SAFE_CALL(hipFree(d_nums1));
    // CUDA_SAFE_CALL(hipFree(d_nums2));
    // CUDA_SAFE_CALL(hipFree(d_nums3));


// ========================================


    Jpoint* h_p1;
    Jpoint* h_p2;
    Jpoint* h_p3;
    Jpoint* d_p1;
    Jpoint* d_p2;
    Jpoint* d_p3;
    // Apoint* h_Ap;
    // Apoint* d_Ap;
    UINT64* h_num;
    UINT64* d_num;
    // Jpoint* d_result;
    
    h_p1 = (Jpoint*)malloc(N_POINT*sizeof(Jpoint));
    h_p2 = (Jpoint*)malloc(N_POINT*sizeof(Jpoint));
    h_p3 = (Jpoint*)malloc(N_POINT*sizeof(Jpoint));
    // h_Ap = (Apoint*)malloc(N_POINT*sizeof(Apoint));
    h_num = (UINT64*)malloc(4*N_BIGNUM*sizeof(UINT64));

    init_Jpoint(h_p1);
    init_Jpoint(h_p2);
    // init_Apoint(h_Ap);
    init_random_big(h_num);
    // h_mybig_print(h_num);
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p1,N_POINT*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p2,N_POINT*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p3,N_POINT*sizeof(Jpoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_Ap,N_POINT*sizeof(Apoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_result,N_POINT*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_num,sizeof(UINT64)*4*N_BIGNUM));

    init_Jpoint_toOne(h_p1, N_POINT);
    make_data(h_p1, N_POINT);
    init_Jpoint_toOne(h_p2, N_POINT);
    make_data(h_p2, N_POINT);


//===========warm up
    // init_random_big(h_num);
    // h_mybig_print(h_num);
    // init_Jpoint(h_p1);
    // init_Jpoint(h_p2);
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_p1,N_POINT*sizeof(Jpoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_p2,N_POINT*sizeof(Jpoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_num,sizeof(UINT64)*4*N_BIGNUM));

    CUDA_SAFE_CALL(hipMemcpy(d_num,h_num,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p1,h_p1,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2,h_p2,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p3,h_p3,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    


    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    test_point_double<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    print_jpoint_arr(h_p1,1);

//==================warm end









    CUDA_SAFE_CALL(hipMemcpy(d_num,h_num,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    
    CUDA_SAFE_CALL(hipMemcpy(d_p1,h_p1,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2,h_p2,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p3,h_p3,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();    
    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    gettimeofday(&s1,NULL);
    test_point_addjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2,d_p3);
    hipDeviceSynchronize();
    
    CUDA_CHECK_ERROR();
    gettimeofday(&e1,NULL);
    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    
    CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    auto time_use1=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    //printf("time_use is %ld us\n",time_use1);
    

    
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();    
    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    gettimeofday(&s1,NULL);
    test_point_double<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    hipDeviceSynchronize();
    
    CUDA_CHECK_ERROR();
    gettimeofday(&e1,NULL);
    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    
    CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    auto time_use2=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    //printf("time_use is %ld us\n",time_use1);

    
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();    
    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    gettimeofday(&s1,NULL);
    test_point_triple<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    hipDeviceSynchronize();
    
    CUDA_CHECK_ERROR();
    gettimeofday(&e1,NULL);
    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    
    CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    auto time_use3=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("add_cost is %ld us\ndouble_cost is %ld us\ntriple_cost is %ldus\n",time_use1, time_use2, time_use3);


    print_jpoint_arr(h_p1,1);
    // print_jpoint_arr(h_p2,1);
    // print_jpoint_arr(h_p2,N_POINT);



    




    free(h_p1);
    free(h_p2);
    free(h_num);
    CUDA_SAFE_CALL(hipFree(d_p1));
    CUDA_SAFE_CALL(hipFree(d_p2));
    CUDA_SAFE_CALL(hipFree(d_num));

}
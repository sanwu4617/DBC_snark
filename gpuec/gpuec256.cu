#include "hip/hip_runtime.h"
#include "gpuec256.h"

#include "device_arr.h"
// typedef unsigned long long UINT64; //????64��??????
// typedef long long INT64;
// typedef unsigned int UINT32;
// // ???????
// typedef struct Affine_point{
// 	UINT64 x[8];
// 	UINT64 y[8];
// }Apoint;

// // ???????
// typedef struct Jacobi_point{
// 	UINT64 x[8];
// 	UINT64 y[8];
// 	UINT64 z[8];
// }Jpoint;

extern const __device__ double d_pow23[][MAX_3];
extern const __device__ uint288 u_pow23[][MAX_3];

// ??tesla C2050??????????????��??????????????
#define PARAL 64
#define BLOCKNUM (14*8)
#define BLOCKSIZE 32
#define THREADNUM (BLOCKNUM*BLOCKSIZE)


// ????__global__?????????????16???????
#define d_BIN_WINDOW_16 16 //16?????
#define d_ROWS_16 32
#define d_COLS_16 (1L<<d_BIN_WINDOW_16)


#define HANDLE_ERROR( err ) { if (err != hipSuccess) { \
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), __FILE__, __LINE__ );\
	  exit( EXIT_FAILURE ); }  \
}

//?��?512??????????????????????256????��
//??��?????????????????????????
//h_ONE??host????????1
//dc_ONE??gpu??????????1
//h_mon_ONE??dc_mon_ONE?????????host??gpu???????????????????1
//h_p??dc_p??512????????
//Pa0??Pa7???????????????????

// const UINT64 h_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// const UINT64 h_mon_ONE[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000L,0x0000000000000000L,0x0000000000000001L,0x0000000100000000L,0x0000000000000000L};
// const UINT64 h_p[8]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0x0L,0x0L,0x0L,0x0L};

// __constant__ UINT64 dc_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// __constant__ UINT64 dc_mon_ONE[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000L,0x0000000000000000L,0x0000000000000001L,0x0000000100000000L,0x0000000000000000L};
// __constant__ UINT64 dc_p[8]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0x0L,0x0L,0x0L,0x0L};


// const UINT64 h_ONE[4]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// const UINT64 h_mon_ONE[4]={0x1000003d1L,0x0L,0x0L,0x0L};
// const UINT64 h_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};

// __constant__ UINT64 dc_ONE[4]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// __constant__ UINT64 dc_mon_ONE[4]={0x1000003d1L,0x0L,0x0L,0x0L};
// __constant__ UINT64 dc_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};

//__constant__ UINT64 dc_mon_inv_two[4]={0x0L,0x0L,0x0L,0x8000000000000000L};
//const UINT64 h_mon_inv_two[4]={0x0L,0x0L,0x0L,0x8000000000000000L};

#define Pa0 0xFFFFFFFEFFFFFC2FLL //-1
#define Pa1 0xFFFFFFFFFFFFFFFFLL 
#define Pa2 0xFFFFFFFFFFFFFFFFLL //-1
#define Pa3 0xFFFFFFFFFFFFFFFFLL 
// #define Pa4 0x0 //-1
// #define Pa5 0x0 //-2
// #define Pa6 0x0 
// #define Pa7 0x0 //-1
#define Pn0 0xBFD25E8CD0364141L //?????????pn0????dc_N
#define Pn1 0xBAAEDCE6AF48A03BL
#define Pn2 0xFFFFFFFFFFFFFFFEL
#define Pn3 0xFFFFFFFFFFFFFFFFL


/////////////////GPU???????????????////////////////////////////////////////////////////

// #define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];(a)[4]=(b)[4];(a)[5]=(b)[5];(a)[6]=(b)[6];(a)[7]=(b)[7];}
#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}

//?????????????????
__device__ __host__ void dh_mybig_modadd_64(const UINT64 *x, const UINT64 *y, UINT64 *z)//????z=x+y, x=x+y, ???????y=x+y??
{
	int i;
	UINT64 f,g;
		
	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
	#else
		const UINT64 *Pa=h_p;
	#endif	
	
	z[0] = x[0] + y[0]; f = z[0] < y[0];
	z[1] = x[1] + f; g = z[1] < f; z[1] += y[1]; g += z[1] < y[1];
	z[2] = x[2] + g; f = z[2] < g; z[2] += y[2]; f += z[2] < y[2];
	z[3] = x[3] + f; g = z[3] < f; z[3] += y[3]; g += z[3] < y[3];
	
	// z[4] = x[4] + g; f = z[4] < g; z[4] += y[4]; f += z[4] < y[4];
	// z[5] = x[5] + f; g = z[5] < f; z[5] += y[5]; g += z[5] < y[5];
	// z[6] = x[6] + g; f = z[6] < g; z[6] += y[6]; f += z[6] < y[6];
	// z[7] = x[7] + f; g = z[7] < f; z[7] += y[7]; g += z[7] < y[7];
		
	if(g==0)
	{
		for(i=3;i>=0;i--)
		{
			if(z[i]!=Pa[i])
			{
				g=(z[i]>Pa[i]);
				break;
			}
			else if(i==0)//????,??t=P, ??????????0????
			{
				g=1;
			}
		}
	}
	
	if(g)//x+y??????????
	{
		f = z[0] < Pa0; z[0] -= Pa0;
		g = z[1] < f; z[1] -= f; g += z[1] < Pa1; z[1] -= Pa1;
		f = z[2] < g; z[2] -= g; f += z[2] < Pa2; z[2] -= Pa2;
		z[3] -= f; z[3] -= Pa3;		                                                  
		// g = z[3] < f; z[3] -= f; g += z[3] < Pa3; z[3] -= Pa3;		
		// f = z[4] < g; z[4] -= g; f += z[4] < Pa4; z[4] -= Pa4;		                                                  
		// g = z[5] < f; z[5] -= f; g += z[5] < Pa5; z[5] -= Pa5;
		// f = z[6] < g; z[6] -= g; f += z[6] < Pa6; z[6] -= Pa6;				
		// z[7] -= f; z[7] -= Pa7;
	}
}

__device__ __host__ void dh_mybig_modadd_64_modN(const UINT64 *x, const UINT64 *y, UINT64 *z)//????z=x+y, x=x+y, ???????y=x+y??
{
	int i;
	UINT64 f,g;
		
	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_N;
	#else
		const UINT64 *Pa=h_N;
	#endif	
	
	z[0] = x[0] + y[0]; f = z[0] < y[0];
	z[1] = x[1] + f; g = z[1] < f; z[1] += y[1]; g += z[1] < y[1];
	z[2] = x[2] + g; f = z[2] < g; z[2] += y[2]; f += z[2] < y[2];
	z[3] = x[3] + f; g = z[3] < f; z[3] += y[3]; g += z[3] < y[3];
	
	// z[4] = x[4] + g; f = z[4] < g; z[4] += y[4]; f += z[4] < y[4];
	// z[5] = x[5] + f; g = z[5] < f; z[5] += y[5]; g += z[5] < y[5];
	// z[6] = x[6] + g; f = z[6] < g; z[6] += y[6]; f += z[6] < y[6];
	// z[7] = x[7] + f; g = z[7] < f; z[7] += y[7]; g += z[7] < y[7];
		
	if(g==0)
	{
		for(i=3;i>=0;i--)
		{
			if(z[i]!=Pa[i])
			{
				g=(z[i]>Pa[i]);
				break;
			}
			else if(i==0)//????,??t=P, ??????????0????
			{
				g=1;
			}
		}
	}
	
	if(g)//x+y??????????
	{
		f = z[0] < Pa[0]; z[0] -= Pa[0];
		g = z[1] < f; z[1] -= f; g += z[1] < Pa[1]; z[1] -= Pa[1];
		f = z[2] < g; z[2] -= g; f += z[2] < Pa[2]; z[2] -= Pa[2];
		z[3] -= f; z[3] -= Pa[3];		                                                  
		// g = z[3] < f; z[3] -= f; g += z[3] < Pa3; z[3] -= Pa3;		
		// f = z[4] < g; z[4] -= g; f += z[4] < Pa4; z[4] -= Pa4;		                                                  
		// g = z[5] < f; z[5] -= f; g += z[5] < Pa5; z[5] -= Pa5;
		// f = z[6] < g; z[6] -= g; f += z[6] < Pa6; z[6] -= Pa6;				
		// z[7] -= f; z[7] -= Pa7;
	}
}

//??????????????????
__device__ __host__ void dh_mybig_modsub_64(const UINT64 *x, const UINT64 *y, UINT64 *z)//????z=x-y, x=x-y, ???????y=x-y??
{
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(x[0]<y[0]); z[0]=x[0]-y[0];
	g=(x[1]<f); z[1]=x[1]-f; g+=(z[1]<y[1]); z[1]-=y[1];
	f=(x[2]<g); z[2]=x[2]-g; f+=(z[2]<y[2]); z[2]-=y[2];
	g=(x[3]<f); z[3]=x[3]-f; g+=(z[3]<y[3]); z[3]-=y[3];
	
	// f=(x[4]<g); z[4]=x[4]-g; f+=(z[4]<y[4]); z[4]-=y[4];
	// g=(x[5]<f); z[5]=x[5]-f; g+=(z[5]<y[5]); z[5]-=y[5];
	// f=(x[6]<g); z[6]=x[6]-g; f+=(z[6]<y[6]); z[6]-=y[6];
	// g=(x[7]<f); z[7]=x[7]-f; g+=(z[7]<y[7]); z[7]-=y[7];
	//??��?g;

	if(g)//??��??��????????????
	{
		z[0]+=Pa0; f=(z[0]<Pa0);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa1; f+=(z[1]<Pa1);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa2; f+=(z[2]<Pa2);
		z[3]+=f; z[3]+=Pa3;		
		// z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		// z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		// z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		// z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		// z[7]+=f; z[7]+=Pa7;		
	}	
}

//x-y mod N
__device__ __host__ void dh_mybig_modsub_64_modN(const UINT64 *x, const UINT64 *y, UINT64 *z){

	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_N;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_N;
	#endif	
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(x[0]<y[0]); z[0]=x[0]-y[0];
	g=(x[1]<f); z[1]=x[1]-f; g+=(z[1]<y[1]); z[1]-=y[1];
	f=(x[2]<g); z[2]=x[2]-g; f+=(z[2]<y[2]); z[2]-=y[2];
	g=(x[3]<f); z[3]=x[3]-f; g+=(z[3]<y[3]); z[3]-=y[3];
	
	// f=(x[4]<g); z[4]=x[4]-g; f+=(z[4]<y[4]); z[4]-=y[4];
	// g=(x[5]<f); z[5]=x[5]-f; g+=(z[5]<y[5]); z[5]-=y[5];
	// f=(x[6]<g); z[6]=x[6]-g; f+=(z[6]<y[6]); z[6]-=y[6];
	// g=(x[7]<f); z[7]=x[7]-f; g+=(z[7]<y[7]); z[7]-=y[7];
	//??��?g;

	if(g)//??��??��????????????
	{
		z[0]+=Pa[0]; f=(z[0]<Pa[0]);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa[1]; f+=(z[1]<Pa[1]);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa[2]; f+=(z[2]<Pa[2]);
		z[3]+=f; z[3]+=Pa[3];		
		// z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		// z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		// z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		// z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		// z[7]+=f; z[7]+=Pa7;		
	}	
}

__device__ __host__ void dh_mybig_modsub_64_ui32_modN(const UINT64 *x, unsigned int y, UINT64 *z){

	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_N;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_N;
	#endif	
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(x[0]<y); z[0]=x[0]-y;
	g=(x[1]<f); z[1]=x[1]-f; 
	f=(x[2]<g); z[2]=x[2]-g; 
	g=(x[3]<f); z[3]=x[3]-f; 
	
	// f=(x[4]<g); z[4]=x[4]-g; f+=(z[4]<y[4]); z[4]-=y[4];
	// g=(x[5]<f); z[5]=x[5]-f; g+=(z[5]<y[5]); z[5]-=y[5];
	// f=(x[6]<g); z[6]=x[6]-g; f+=(z[6]<y[6]); z[6]-=y[6];
	// g=(x[7]<f); z[7]=x[7]-f; g+=(z[7]<y[7]); z[7]-=y[7];
	//??��?g;

	if(g)//??��??��????????????
	{
		z[0]+=Pa[0]; f=(z[0]<Pa[0]);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa[1]; f+=(z[1]<Pa[1]);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa[2]; f+=(z[2]<Pa[2]);
		z[3]+=f; z[3]+=Pa[3];		
		// z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		// z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		// z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		// z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		// z[7]+=f; z[7]+=Pa7;		
	}	
}

__device__ __host__ void dh_mybig_neg(UINT64 *y,UINT64 *z){
	// #ifdef __CUDA_ARCH__	
	// 	UINT64 *Pa=dc_N;	
	// 	#define h_Hi64 __umul64hi
	// #else
	// 	const UINT64 *Pa=h_N;
	// #endif	
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(0<y[0]); z[0]=0-y[0];
	g=(0<f); z[1]=0-f; g+=(z[1]<y[1]); z[1]-=y[1];
	f=(0<g); z[2]=0-g; f+=(z[2]<y[2]); z[2]-=y[2];
	g=(0<f); z[3]=0-f; g+=(z[3]<y[3]); z[3]-=y[3];
	

	//??��?g;

	if(g)//??��??��????????????
	{
		z[0]+=Pa0; f=(z[0]<Pa0);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa1; f+=(z[1]<Pa1);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa2; f+=(z[2]<Pa2);
		z[3]+=f; z[3]+=Pa3;				
		// z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		// z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		// z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		// z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		// z[7]+=f; z[7]+=Pa7;		
	}	
}

__device__ __host__ void dh_mybig_neg_modN(UINT64 *y,UINT64 *z){
	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_N;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_N;
	#endif	
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(0<y[0]); z[0]=0-y[0];
	g=(0<f); z[1]=0-f; g+=(z[1]<y[1]); z[1]-=y[1];
	f=(0<g); z[2]=0-g; f+=(z[2]<y[2]); z[2]-=y[2];
	g=(0<f); z[3]=0-f; g+=(z[3]<y[3]); z[3]-=y[3];
	

	//??��?g;

	if(g)//??��??��????????????
	{
		z[0]+=Pa[0]; f=(z[0]<Pa[0]);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa[1]; f+=(z[1]<Pa[1]);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa[2]; f+=(z[2]<Pa[2]);
		z[3]+=f; z[3]+=Pa[3];		
		// z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		// z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		// z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		// z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		// z[7]+=f; z[7]+=Pa7;		
	}	
}

//????????????????64��?????��
static inline UINT64 h_Hi64(const UINT64 x, const UINT64 y)
{
   UINT64 z;   
   __asm__ __volatile__ (
	  "movq  %0,%%rax\n"   
	  "mulq %1\n"
	  "movq %%rdx,%2\n"
	  : 
	  : "m"(x),"m"(y),"m"(z)
	  : "rax","rdx","memory"
	  );      	
	return z;
}

//?????????????C=a*b
/////////////////////////////////////////
//Montgomery???
//C=A*B*2^-512 mod P
//????CIOS??
//////////////////////////////////////////
__device__ __host__  void dh_mybig_monmult_64(const UINT64 *Aa, const UINT64 *Ba, UINT64 *Ca)//pass, c=a*b
{
	UINT64 t[4+2]={0};//8??64????????,???512
	
	//minv*P[0] mod 2^wordlen = -1. ????��????????64?????P[0]=0xffffffffffffffff=-1??????minv=1, minv*P[0]=-1 mod 2^64=-1
	
	//?????????P?????????????????��???????????????????????????
	// UINT64 minv=1;//minv????????????P?????��P[0]????2^64=-1(???P[0]??64????????????32?????????????2^32=-1??????? ????????????????????????2^wordlen?????????
	UINT64 minv = 0xd838091dd2253531;
	UINT64 m;	
	UINT64 c,s,cin;
	int i,j;
	
	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_p;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_p;
	#endif	
	
	for(i=0;i<4;i++)
	{
		c=0;
		m=Ba[i];
		
		s=m*Aa[0]; c+=t[0];	cin=(c<t[0]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[0])+cin; 	t[0]=s;
		s=m*Aa[1]; c+=t[1];	cin=(c<t[1]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[1])+cin; 	t[1]=s;
		s=m*Aa[2]; c+=t[2];	cin=(c<t[2]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[2])+cin; 	t[2]=s;
		s=m*Aa[3]; c+=t[3];	cin=(c<t[3]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[3])+cin; 	t[3]=s;
		// s=m*Aa[4]; c+=t[4];	cin=(c<t[4]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[4])+cin; 	t[4]=s;
		// s=m*Aa[5]; c+=t[5];	cin=(c<t[5]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[5])+cin; 	t[5]=s;
		// s=m*Aa[6]; c+=t[6];	cin=(c<t[6]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[6])+cin; 	t[6]=s;
		// s=m*Aa[7]; c+=t[7];	cin=(c<t[7]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[7])+cin; 	t[7]=s;		
								
		s=t[4]+c;
		c=(s<c);
		t[4]=s;
		t[4+1]=c;
				
		m=minv*t[0];//????p??minv=1???????m=t[0]
		c=h_Hi64(m,Pa0);
		s=m*Pa0+t[0];   //??????????????????Pa0=-1, minv=1??????s=0;		
		c+=(s<t[0]);
		
		s=m*Pa1; c+=t[1]; cin=(c<t[1]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa1)+cin; t[0]=s;		
		s=m*Pa2; c+=t[2]; cin=(c<t[2]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa2)+cin; t[1]=s;
		s=m*Pa3; c+=t[3]; cin=(c<t[3]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa3)+cin; t[2]=s;		
	    // s=m*Pa4; c+=t[4]; cin=(c<t[4]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa4)+cin; t[3]=s;		
		// s=m*Pa5; c+=t[5]; cin=(c<t[5]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa5)+cin; t[4]=s;
		// s=m*Pa6; c+=t[6]; cin=(c<t[6]); s+=c;	cin+=(s<c); c=h_Hi64(m,Pa6)+cin; t[5]=s;		
		// s=m*Pa7; c+=t[7]; cin=(c<t[7]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa7)+cin; t[6]=s;
		
		s=t[4]+c;
		c=(s<c);
		t[4-1]=s;
		t[4]=t[4+1]+c;
	}
	
	j=(t[4]!=0);
	if(j==0)
	{
		for(i=4-1;i>=0;i--)
		{
			if(t[i]!=Pa[i])
			{				
				j=(t[i]>Pa[i]);
				break;
			}
			else if(i==0)//????,??t=P, ??????????0????
			{
				j=1;
			}
		}
	}
	
	//????	
	if(j)
	{
		cin=1;	
		for(i=0;i<4;i++)
		{
			m=cin+(~Pa[i]);
			cin=(m<cin);
			m+=t[i];
			cin+=(m<t[i]);
			Ca[i]=m;
		}
	}
	else
	{
		for(i=0;i<4;i++)
		{
			Ca[i]=t[i];
		}
	}
	
	#ifdef __CUDA_ARCH__	
		#undef h_Hi64 
	#endif	
}

__device__ __host__  void dh_mybig_monmult_64_modN(const UINT64 *Aa, const UINT64 *Ba ,UINT64 *Ca)//pass, c=a*b
{
	UINT64 t[4+2]={0};//8??64????????,???512
	
	//minv*P[0] mod 2^wordlen = -1. ????��????????64?????P[0]=0xffffffffffffffff=-1??????minv=1, minv*P[0]=-1 mod 2^64=-1
	
	//?????????P?????????????????��???????????????????????????
	// UINT64 minv=1;//minv????????????P?????��P[0]????2^64=-1(???P[0]??64????????????32?????????????2^32=-1??????? ????????????????????????2^wordlen?????????
	UINT64 minv = 0x4b0dff665588b13f;
	UINT64 m;	
	UINT64 c,s,cin;
	int i,j;
	
	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_N;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_N;
	#endif	
	// const UINT64 *Pa=P; 
	
	for(i=0;i<4;i++)
	{
		c=0;
		m=Ba[i];
		
		s=m*Aa[0]; c+=t[0];	cin=(c<t[0]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[0])+cin; 	t[0]=s;
		s=m*Aa[1]; c+=t[1];	cin=(c<t[1]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[1])+cin; 	t[1]=s;
		s=m*Aa[2]; c+=t[2];	cin=(c<t[2]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[2])+cin; 	t[2]=s;
		s=m*Aa[3]; c+=t[3];	cin=(c<t[3]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[3])+cin; 	t[3]=s;
		// s=m*Aa[4]; c+=t[4];	cin=(c<t[4]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[4])+cin; 	t[4]=s;
		// s=m*Aa[5]; c+=t[5];	cin=(c<t[5]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[5])+cin; 	t[5]=s;
		// s=m*Aa[6]; c+=t[6];	cin=(c<t[6]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[6])+cin; 	t[6]=s;
		// s=m*Aa[7]; c+=t[7];	cin=(c<t[7]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[7])+cin; 	t[7]=s;		
								
		s=t[4]+c;
		c=(s<c);
		t[4]=s;
		t[4+1]=c;
				
		m=minv*t[0];//????p??minv=1???????m=t[0]
		c=h_Hi64(m,Pa[0]);
		s=m*Pa[0]+t[0];   //??????????????????Pa0=-1, minv=1??????s=0;		
		c+=(s<t[0]);
		
		s=m*Pa[1]; c+=t[1]; cin=(c<t[1]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa[1])+cin; t[0]=s;		
		s=m*Pa[2]; c+=t[2]; cin=(c<t[2]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa[2])+cin; t[1]=s;
		s=m*Pa[3]; c+=t[3]; cin=(c<t[3]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa[3])+cin; t[2]=s;		
	    // s=m*Pa4; c+=t[4]; cin=(c<t[4]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa4)+cin; t[3]=s;		
		// s=m*Pa5; c+=t[5]; cin=(c<t[5]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa5)+cin; t[4]=s;
		// s=m*Pa6; c+=t[6]; cin=(c<t[6]); s+=c;	cin+=(s<c); c=h_Hi64(m,Pa6)+cin; t[5]=s;		
		// s=m*Pa7; c+=t[7]; cin=(c<t[7]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa7)+cin; t[6]=s;
		
		s=t[4]+c;
		c=(s<c);
		t[4-1]=s;
		t[4]=t[4+1]+c;
	}
	
	j=(t[4]!=0);
	if(j==0)
	{
		for(i=4-1;i>=0;i--)
		{
			if(t[i]!=Pa[i])
			{				
				j=(t[i]>Pa[i]);
				break;
			}
			else if(i==0)//????,??t=P, ??????????0????
			{
				j=1;
			}
		}
	}
	
	//????	
	if(j)
	{
		cin=1;	
		for(i=0;i<4;i++)
		{
			m=cin+(~Pa[i]);
			cin=(m<cin);
			m+=t[i];
			cin+=(m<t[i]);
			Ca[i]=m;
		}
	}
	else
	{
		for(i=0;i<4;i++)
		{
			Ca[i]=t[i];
		}
	}
	
	#ifdef __CUDA_ARCH__	
		#undef h_Hi64 
	#endif	
}




//////////////////////////////////////test inv//////////////////////////////////////
//A!=0 return 0
//A=0 return 1
__device__ __host__ int dh_mybig_iszero_64(const UINT64 *A)
{
	int i;
	for(i=0;i<4;i++)
	{
		if(A[i]!=0)	return 0;
	}
	return 1;	
}


//????��
//return 1	A>B
//return 0	A=B
//return -1 A<B
__device__ __host__ int dh_mybig_compare_64(const UINT64 *A, const UINT64 *B)
{
	int i;
	int flag=0;

	for(i=3;i>=0;i--)
	{
		if(A[i]>B[i])
		{
			flag=1;
			break;
		}
		else if(A[i]<B[i])
		{
			flag=-1;
			break;
		}
	}
	return (flag);
}

//C=A/2
__device__ __host__ void dh_mybig_half_64(const UINT64 *A, UINT64 *C)
{
	int i;
	UINT64 c,c1;

	c=0;
	for(i=4-1;i>=0;i--)
	{
		c1=A[i]&0x1;
		C[i]=c<<(63)|A[i]>>1;
		c=c1;
	}
}

//C=2A
__device__ __host__ UINT64 dh_mybig_double_64(const UINT64 *A, UINT64 *C)//pass
{
	int i;
	UINT64 c,c1;

	c=0;
	for(i=0;i<4;i++)
	{
		c1=A[i]>>63;
		C[i]=(C[i]<<1)|c;
		c=c1;
	}
	return c;
}

//C=A-B
__device__ __host__ void dh_mybig_sub_64(const UINT64 *A, const UINT64 *B, UINT64 *C)
{
	int i;
	UINT64 c,l,h;

	c=1;
	for(i=0;i<4;i++)
	{
		l=(~B[i])+c;
		h=(l<c);
		l+=A[i];
		c=h+(l<A[i]);
		C[i]=l;	
	}
}

//C=A+B
__device__ __host__ UINT64 dh_mybig_add_64(const UINT64 *A, const UINT64 *B, UINT64 *C)
{
	int i;
	UINT64 c,l,h;
	
	c=0;
	for(i=0;i<4;i++)
	{
		l=A[i]+c;
		h=(l<c);
		l+=B[i];
		c=h+(l<B[i]);
		C[i]=l;
	}
	return c;
}

/////////////////////////////////////////
//??2?
//????:		A P
//???:		C
//C=2*A mod P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moddouble_64(const UINT64 *A, const UINT64 *P, UINT64 *C)
{
	int i,sub_en=0;
	UINT64 cin,c,temp64;

	//??��
	cin=(A[0]>>63)&0x1;//????????��,?????2
	C[0]=A[0]<<1;
	for(i=1;i<4;i++)
	{
		c=(A[i]>>63)&0x1;
		C[i]=(A[i]<<1)|cin;
		cin=c;
	}

	//????��
	if(cin==1)//??????????��?1,????????????????��??2*A?????2*A?????p??
	{
		sub_en=1;
	}
	else
	{
		for(i=3;i>=0;i--)
		{
			if(C[i]!=P[i])
			{
				if(C[i]>P[i]) sub_en=1;
				break;
			}
			else if(i==0) sub_en=1;//????????
		}
	}
	
	//????
	if(sub_en)//???????��?if(sub_en!=0)???????????
	{
		cin=1;
		for(i=0;i<4;i++)
		{			
			temp64=(~P[i])+cin;
			c=(temp64<cin);
			temp64+=C[i];
			cin=c+(temp64<C[i]);
			C[i]=temp64;
		}
	}
}


__device__ __host__  int dh_mybig_iszero(const UINT64 * A)
{
	// if( A[0]|A[1]|A[2]|A[3]|A[4]|A[5]|A[6]|A[7])
	// 	return 0;
	// else return 1;
	if( A[0]|A[1]|A[2]|A[3])
		return 0;
	else return 1;
}

void h_mybig_print(const UINT64 *a)
{
	int i;
	unsigned char *t=(unsigned char*) a;
	for(i=32-1;i>=0;i--) if(*(t+i)) break;//??0???????
	if(i<0) printf("0");
	else
	{
		printf("%x",*(t+i)&0xff);//?????0?????
		for(i=i-1;i>=0;i--)printf("%02x",*(t+i)&0xff);
	}
	printf("\n");	
}
__device__ void d_mybig_print(const UINT64 *a)
{
	int i;
	unsigned char *t=(unsigned char*) a;
	for(i=32-1;i>=0;i--) if(*(t+i)) break;//??0???????
	if(i<0) printf("0");
	else
	{
		printf("%x",*(t+i)&0xff);//?????0?????
		for(i=i-1;i>=0;i--)printf("%02x",*(t+i)&0xff);
	}
	printf("\n");	
}

/////////////////////////////////////////
//??????: ???????C=A^-512 * 2^512 mod P
//????:		A C l(l???????????512)
//???:		C (C=A^-1 * 2^512 mod P)
//???: A<P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moninv(const UINT64 *A,UINT64 *C)//test
{
	int i,k;
	UINT64 U[4],V[4],R[4],S[4];
	int z,cp,cs,cr,sh;
	/*
	#ifdef __CUDA_ARCH__	
		UINT64 *P=dc_p;	
	#else
		const UINT64 *P=h_p;
	#endif	
	//*/

	/************************??********************************
	Preferences: E. Savas, C.K.Koc, The Montgomery Modular
	Inverse-Revisited, IEEE TRANSACTIONS ON COMPUTERS,
	VOL. 49, NO.7, JULY 2000

	--Phase I
	U=P V=A R=0 S=1
	k=0
	while(V>0)
		if U is even then U=U/2 S=2S
		else if V is even then V=V/2 R=2R
		else if U>V then U=(U-V)/2 R=R+S S=2S
		else if V>=U then V=(V-U)/2 S=S+R R=2R
		k=k+1
	if R>=P then R=R-P
	R=P-R

	--Phase II
  R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P 
  = A^-1 * 2^l mod P
	
	return R
	*************************************************************/

	////0. initial
	//U=P V=A R=0 S=1
	// U[0]=0xffffffffffffffffL; //U=P??P???????????????
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = Pa0;
	U[1] = Pa1;
	U[2] = Pa2;
	U[3] = Pa3;


	for(i=0;i<4;i++) V[i]=A[i];
	for(i=0;i<4;i++) R[i]=0;
	for(i=1;i<4;i++) S[i]=0;
	S[0]=0x1UL; 

	////1. phase I
	//get R=A^-1*2^k mod P
	k=0;
	z=dh_mybig_iszero_64(V);
	while(z==0)
	{	
		//printf("here k=%d  ",k);//getchar();
		cp=dh_mybig_compare_64(U,V);
		if((U[0]&0x1)==0)
		{
			dh_mybig_half_64(U,U);
			cs=dh_mybig_double_64(S,S);
		}
		else if((V[0]&0x1)==0)
		{			
			dh_mybig_half_64(V,V);
			cr=dh_mybig_double_64(R,R);
		}
		else if(cp==1)
		{
			dh_mybig_sub_64(U,V,U);
			dh_mybig_half_64(U,U);
			cr=dh_mybig_add_64(R,S,R);
			cs=dh_mybig_double_64(S,S);
		}
		else if(cp<1) //if(cp==0 || cp==-1)
		{
			dh_mybig_sub_64(V,U,V);
			dh_mybig_half_64(V,V);
			cs=dh_mybig_add_64(S,R,S);
			cr=dh_mybig_double_64(R,R);
		}
		k++;
		z=dh_mybig_iszero_64(V);
		//if(cr==1) printf("\nr\n");
		//if(cs==1) printf("\ns\n");		
	}
	
	// U[0]=0xffffffffffffffffL;
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = Pa0;
	U[1] = Pa1;
	U[2] = Pa2;
	U[3] = Pa3;

	cp=dh_mybig_compare_64(R,U);
	if(cp==1 || cp==0 || cr==1)
	{
		dh_mybig_sub_64(R,U,R);
	}
	dh_mybig_sub_64(U,R,R);
	////////////////////
	//the result of phase I is R
	//R=A^-1 * 2^k mod P

    
	////2. phaseII
	//get R=A^-1 * 2^l mod P
 	// R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P = A^-1 * 2^l mod P
 	 if(k>256)
	 {	
		// printf("case 1\n");
	    for(i=0;i<4;i++) V[i]=0;
	    V[(int)((512-k)/64)]=(((UINT64)1)<<((int)((512-k)%64)));
	    dh_mybig_monmult_64(R,V,C);
	 }	
	 else if(k==256) 
	 {
			// printf("case 2\n");
			// V[0]=0x0000000000000001L;
			// V[1]=0x0000000100000000L;
			// V[2]=0x0000000000000000L;
			// V[3]=0x0000000100000000L;
			// V[4]=0x0000000000000000L;
			// V[5]=0x0000000000000001L;
			// V[6]=0x0000000100000000L;
			// V[7]=0x0000000000000000L;
			// dh_mybig_monmult_64(R,V,C);		
			for(i=0;i<4;i++) C[i]=R[i];	
		}
		else if(k<256) 
		{
		//   printf("case 3\n");
		  for(i=1;i<=(256-k);i++)
			{		
				dh_mybig_moddouble_64(R,U,R);			
			}
		  for(i=0;i<4;i++) C[i]=R[i];
		}

}

/////////////////////////////////////////
//??????: ???????C=A^-512 * 2^512 mod P ???????P
//????:		A C l(l???????????512)
//???:		C (C=A^-1 * 2^512 mod P)
//???: A<P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moninv_modN(const UINT64 *A,UINT64 *C)//test
{
	int i,k;
	UINT64 U[4],V[4],R[4],S[4];
	int z,cp,cs,cr,sh;
	/*
	#ifdef __CUDA_ARCH__	
		UINT64 *P=dc_p;	
	#else
		const UINT64 *P=h_p;
	#endif	
	//*/

	/************************??********************************
	Preferences: E. Savas, C.K.Koc, The Montgomery Modular
	Inverse-Revisited, IEEE TRANSACTIONS ON COMPUTERS,
	VOL. 49, NO.7, JULY 2000

	--Phase I
	U=P V=A R=0 S=1
	k=0
	while(V>0)
		if U is even then U=U/2 S=2S
		else if V is even then V=V/2 R=2R
		else if U>V then U=(U-V)/2 R=R+S S=2S
		else if V>=U then V=(V-U)/2 S=S+R R=2R
		k=k+1
	if R>=P then R=R-P
	R=P-R

	--Phase II
  R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P 
  = A^-1 * 2^l mod P
	
	return R
	*************************************************************/

	////0. initial
	//U=P V=A R=0 S=1
	// U[0]=0xffffffffffffffffL; //U=P??P???????????????
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = 0xBFD25E8CD0364141L;
	U[1] = 0xBAAEDCE6AF48A03BL;
	U[2] = 0xFFFFFFFFFFFFFFFEL;
	U[3] = 0xFFFFFFFFFFFFFFFFL;


	for(i=0;i<4;i++) V[i]=A[i];
	for(i=0;i<4;i++) R[i]=0;
	for(i=1;i<4;i++) S[i]=0;
	S[0]=0x1UL; 

	////1. phase I
	//get R=A^-1*2^k mod P
	k=0;
	z=dh_mybig_iszero_64(V);
	while(z==0)
	{	
		//printf("here k=%d  ",k);//getchar();
		cp=dh_mybig_compare_64(U,V);
		if((U[0]&0x1)==0)
		{
			dh_mybig_half_64(U,U);
			cs=dh_mybig_double_64(S,S);
		}
		else if((V[0]&0x1)==0)
		{			
			dh_mybig_half_64(V,V);
			cr=dh_mybig_double_64(R,R);
		}
		else if(cp==1)
		{
			dh_mybig_sub_64(U,V,U);
			dh_mybig_half_64(U,U);
			cr=dh_mybig_add_64(R,S,R);
			cs=dh_mybig_double_64(S,S);
		}
		else if(cp<1) //if(cp==0 || cp==-1)
		{
			dh_mybig_sub_64(V,U,V);
			dh_mybig_half_64(V,V);
			cs=dh_mybig_add_64(S,R,S);
			cr=dh_mybig_double_64(R,R);
		}
		k++;
		z=dh_mybig_iszero_64(V);
		//if(cr==1) printf("\nr\n");
		//if(cs==1) printf("\ns\n");		
	}
	
	// U[0]=0xffffffffffffffffL;
	// U[1]=0xfffffffeffffffffL;
	// U[2]=0xffffffffffffffffL;
	// U[3]=0xfffffffeffffffffL;
	// U[4]=0xffffffffffffffffL;
	// U[5]=0xfffffffffffffffeL;
	// U[6]=0xfffffffeffffffffL;
	// U[7]=0xffffffffffffffffL;
	U[0] = 0xBFD25E8CD0364141L;
	U[1] = 0xBAAEDCE6AF48A03BL;
	U[2] = 0xFFFFFFFFFFFFFFFEL;
	U[3] = 0xFFFFFFFFFFFFFFFFL;

	cp=dh_mybig_compare_64(R,U);
	if(cp==1 || cp==0 || cr==1)
	{
		dh_mybig_sub_64(R,U,R);
	}
	dh_mybig_sub_64(U,R,R);
	////////////////////
	//the result of phase I is R
	//R=A^-1 * 2^k mod P

    
	////2. phaseII
	//get R=A^-1 * 2^l mod P
 	// R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P = A^-1 * 2^l mod P
 	 if(k>256)
	 {	
		// printf("case 1\n");
	    for(i=0;i<4;i++) V[i]=0;
	    V[(int)((512-k)/64)]=(((UINT64)1)<<((int)((512-k)%64)));
	    dh_mybig_monmult_64_modN(R,V,C);
	 }	
	 else if(k==256) 
	 {
			// printf("case 2\n");
			// V[0]=0x0000000000000001L;
			// V[1]=0x0000000100000000L;
			// V[2]=0x0000000000000000L;
			// V[3]=0x0000000100000000L;
			// V[4]=0x0000000000000000L;
			// V[5]=0x0000000000000001L;
			// V[6]=0x0000000100000000L;
			// V[7]=0x0000000000000000L;
			// dh_mybig_monmult_64(R,V,C);		
			for(i=0;i<4;i++) C[i]=R[i];	
		}
		else if(k<256) 
		{
		//   printf("case 3\n");
		  for(i=1;i<=(256-k);i++)
			{		
				dh_mybig_moddouble_64(R,U,R);			
			}
		  for(i=0;i<4;i++) C[i]=R[i];
		}

}

//GAO: mod exp
//C = A^k mod P
__device__ __host__ void dh_mybig_modexp(UINT64* a,UINT64 *k,UINT64* c){
	//gyy
	int i,j;
	UINT64 tbn[4];
	// Jpoint t2;
	

	// find first 1
	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("mod exp:k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if(((k[i]>>j)&0x01)!=0) break;
	}
	
	dh_mybig_copy(tbn,a);

	j--;


		
		for(;j>=0;j--){
				// printf("double\n");
			dh_mybig_monmult_64(tbn,tbn,tbn);
			// ppoint_double(pt1,pt1);
			if(((k[i]>>j)&0x01)==1){
					// printf("add\n");
				dh_mybig_monmult_64(tbn,a,tbn);
				// dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				// ppoint_double(pt1,pt1);
				dh_mybig_monmult_64(tbn,tbn,tbn);
				if(((k[i]>>j)&0x01)==1){
						// printf("add\n");
					// dh_ellipticAdd_JJ(pt1,&tp,pt1);
					dh_mybig_monmult_64(tbn,a,tbn);
				}
			}
		}

		dh_mybig_copy(c,tbn);
	
	// printf("j=%d,i=%d\n",j,i);
	
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

__device__ __host__ void dh_mybig_modexp_modN(UINT64* a,UINT64 *k,UINT64* c){
	//gyy
	int i,j;
	UINT64 tbn[4];
	// Jpoint t2;
	

	// find first 1
	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("mod exp:k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if(((k[i]>>j)&0x01)!=0) break;
	}
	
	dh_mybig_copy(tbn,a);

	j--;


		
		for(;j>=0;j--){
				// printf("double\n");
			dh_mybig_monmult_64_modN(tbn,tbn,tbn);
			// ppoint_double(pt1,pt1);
			if(((k[i]>>j)&0x01)==1){
					// printf("add\n");
				dh_mybig_monmult_64_modN(tbn,a,tbn);
				// dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				// ppoint_double(pt1,pt1);
				dh_mybig_monmult_64_modN(tbn,tbn,tbn);
				if(((k[i]>>j)&0x01)==1){
						// printf("add\n");
					// dh_ellipticAdd_JJ(pt1,&tp,pt1);
					dh_mybig_monmult_64_modN(tbn,a,tbn);
				}
			}
		}

		dh_mybig_copy(c,tbn);
	
	// printf("j=%d,i=%d\n",j,i);
	
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

__device__  void dh_mybig_modexp_ui32_modN(UINT64* a,unsigned int k,UINT64* c){
	//gyy
	if(k==0){
		dh_mybig_copy(c,dc_mon_ONE_modN);
		return;
	}
	int i;
	UINT64 tbn[4];
	// Jpoint t2;
	
	for(i=31;i>=0;i--){
		if(((k>>i)&0x01)!=0) break;
	}
	// find first 1
	
	// if(i<0){
	// 	printf("mod exp:k==0!!!!!!!!!\n");
	// }
	// for(j=63;j>=0;j--){
	// 	if((k[i]>>j)&0x01!=0) break;
	// }
	
	dh_mybig_copy(tbn,a);

	i--;

	for(;i>=0;i--){
		dh_mybig_monmult_64_modN(tbn,tbn,tbn);
		if(((k>>i)&0x01)==1){
			dh_mybig_monmult_64_modN(tbn,a,tbn);
		}
	}


	dh_mybig_copy(c,tbn);
	
	// printf("j=%d,i=%d\n",j,i);
	
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

//GAO: mod exp
//C = 2^k1 * 3^k2 mod P
__device__ __host__ void dh_mybig_modexp23_modN(unsigned int k2, unsigned int k3, UINT64* c){
    //occulticplus
    int i,j;
    UINT64 tbn[4] = {0};

    UINT64 pow2[34] = {0x1L, 0x2L, 0x4L, 0x8L, 0x10L, 0x20L, 0x40L, 0x80L, 0x100L, 0x200L, 0x400L, 0x800L,
                   0x1000L, 0x2000L, 0x4000L, 0x8000L, 0x10000L, 0x20000L, 0x40000L, 0x80000L,
                   0x100000L, 0x200000L, 0x400000L, 0x800000L, 0x10000000L, 0x2000000L, 0x4000000L, 0x8000000L,
                   0x10000000L, 0x20000000L, 0x40000000L, 0x80000000L, 0x1000000000L};
    UINT64 pow3[41] = {1L, 3L, 9L, 27L, 81L, 243L, 729L, 2187L, 6561L, 19683L, 59049L, 177147L, 531441L, 1594323L,
                       4782969L, 14348907L, 43046721L, 129140163L, 387420489L, 1162261467L, 3486784401L, 10460353203L,
                       31381059609L, 94143178827L, 282429536481L, 847288609443L, 2541865828329L, 7625597484987L,
                       22876792454961L, 68630377364883L, 205891132094649L, 617673396283947L, 1853020188851841L,
                       5559060566555523L, 16677181699666569L, 50031545098999707L, 150094635296999121L,
                       450283905890997363L, 1350851717672992089L, 4052555153018976267L};
    // Jpoint t2;

    int base2 = k2 && ((1 << 6) - 1);
    int id2 = k2 >> 6;
    assert(id2 <= 3);
    tbn[id2] |= 1 << base2;
    int base3 = k3 && ((1 << 6) - 1);
    int id3 = k3 >> 6;
    UINT64 dot[4] = {pow3[32]};
    for (int i = 0; i < base3; i++) {
        dh_mybig_monmult_64(tbn, tbn, dot); // no definition;
    }
    dot[0] = pow3[id3];
    dh_mybig_monmult_64(tbn, tbn, dot);

    dh_mybig_copy(c,tbn);

    // printf("j=%d,i=%d\n",j,i);

    // printf("copy\n");
    // dh_mybig_copy(pt1->x,tp.x);
    // dh_mybig_copy(pt1->y,tp.y);
    // dh_mybig_copy(pt1->z,tp.z);
}

//////////////////////////////////////test inv end//////////////////////////////////////

/////////////////GPU????????????????////////////////////////////////////////////////////


/////////////////GPU?????????????????��?????????????????????????????????????????????///////////////////////////////////////
//GAO:��???
__device__ __host__  void dh_setzero_J(Jpoint *pt)
{
	int i;
	for(i=0;i<8;i++) pt->z[i]=0UL;
}
//GAO:��???
__device__ __host__  void dh_setzero_A(Apoint *pt)
{
	int i;
	for(i=0;i<8;i++) pt->x[i]=0UL;	
}

__device__ __host__  int dh_iszero_J(const Jpoint *pt)
{
	// if((pt->z[0]|pt->z[1]|pt->z[2]|pt->z[3]|pt->z[4]|pt->z[5]|pt->z[6]|pt->z[7]) == 0UL)	return 1;
	// return 0;	
	if((pt->z[0]|pt->z[1]|pt->z[2]|pt->z[3]) == 0UL)	return 1;
	return 0;
}


__device__ __host__  int dh_iszero_A(const Apoint *pt)
{	
	// if((pt->x[0]|pt->x[1]|pt->x[2]|pt->x[3]|pt->x[4]|pt->x[5]|pt->x[6]|pt->x[7]) == 0UL)	return 1;
	// return 0;
	if((pt->x[0]|pt->x[1]|pt->x[2]|pt->x[3]) == 0UL)	return 1;
	return 0;
}

//GAO:��???
//???????????????????????
__device__ __host__  void dh_normlize_J(Jpoint *pt)
{	
	UINT64 tmp[8],invtmp[8];
	
	if(dh_iszero_J(pt))	return;
	
	tmp[0]=0x1L; tmp[1]=0x0L; tmp[2]=0x0L; tmp[3]=0x0L; tmp[4]=0x0L; tmp[5]=0x0L; tmp[6]=0x0L; tmp[7]=0x0L;
	
	
	dh_mybig_monmult_64(pt->z,tmp, invtmp);
	dh_mybig_moninv(invtmp, invtmp);
	//invMod(invtmp,pt->z,mod);//invtmp=1/z
	
	//squareMod(tmp,invtmp,mod);//tmp=1/zz
	dh_mybig_monmult_64(invtmp,invtmp, tmp);
	
	//productMod(pt->x,pt->x,tmp,mod);//x/zz
	dh_mybig_monmult_64(pt->x,tmp, pt->x);		
	
	//productMod(tmp,tmp,invtmp,mod);//1/zzz
	dh_mybig_monmult_64(tmp,invtmp, tmp);
	
	//productMod(pt->y,pt->y,tmp,mod);//y/zzz
	dh_mybig_monmult_64(pt->y,tmp, pt->y);
	
	//z=mon_one
	memcpy(pt->z, dc_mon_ONE, sizeof(pt->z));
	// pt->z[0]=0x1000003d1L;
	// pt->z[1]=0x0000000100000000L;
	// pt->z[2]=0x0000000000000000L;
	// pt->z[3]=0x0000000100000000L;
	// pt->z[4]=0x0000000000000000L;
	// pt->z[5]=0x0000000000000001L;
	// pt->z[6]=0x0000000100000000L;
	// pt->z[7]=0x0000000000000000L;	
	
}
//GAO:��???
__device__ void d_mon2normal_J(Jpoint *pt)
{
	dh_mybig_monmult_64(pt->x, dc_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, dc_ONE, pt->y);
	dh_mybig_monmult_64(pt->z, dc_ONE, pt->z);	
}
//GAO:��???
__device__ __host__  void dh_mon2normal_A(Apoint *pt)
{
	#ifdef __CUDA_ARCH__
	dh_mybig_monmult_64(pt->x, dc_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, dc_ONE, pt->y);
	#else
	dh_mybig_monmult_64(pt->x, h_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, h_ONE, pt->y);
	#endif
}

void h_print_pointA(const Apoint *pt)
{
	if(dh_iszero_A(pt))
	{ 
		printf("(Infinity)\n");
		return;
	}	
	printf("x: ");h_mybig_print(pt->x);
	printf("y: ");h_mybig_print(pt->y);
	
}
 
void h_print_pointJ(const Jpoint *pt)
{
	/*
	if(dh_iszero_J(pt))
	{ 
		printf("(Infinity)\n");
		return;
	}*/
	printf("x: ");h_mybig_print(pt->x);
	printf("y: ");h_mybig_print(pt->y);
	printf("z: ");h_mybig_print(pt->z);
}

void __device__ d_print_pointJ(const Jpoint *pt)
{
	d_mybig_print(pt->x);printf("\n");
	d_mybig_print(pt->y);printf("\n");
	d_mybig_print(pt->z);printf("\n");
}

__device__ __host__ void ppoint_double(Jpoint *pt1,Jpoint* pt2){
	UINT64 u1[4],u2[4],u3[4];

	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
	#else
		const UINT64 *Pa=h_p;
	#endif	

	//?????��??????????
	//secp256k1??a?0??????????????????????????
	dh_mybig_moddouble_64(pt1->y,Pa,u1); 	//u1=2y
	dh_mybig_monmult_64(pt1->z,u1,pt2->z);		//z=u1*z=2yz

	dh_mybig_monmult_64(pt1->x,pt1->x,u2);	//u2=x^2
	dh_mybig_moddouble_64(u2,Pa,u3);		//u3=2*u2=2x^2
	dh_mybig_modadd_64(u3,u2,u3);			//u3=u3+u2 = 3x^2 = lambda_1
	dh_mybig_monmult_64(u1,pt1->y,u1);		//u1 = u1*y = 2y^2
	dh_mybig_monmult_64(u1,pt1->x,u2);		//u2 = u1*x = 2xy^2 
	dh_mybig_moddouble_64(u2,Pa,u2);		//u2 = 2*u2 = 4xy^2= lambda_2
	dh_mybig_monmult_64(u3,u3,pt2->x);		//pt1x = lambda_1^2
	dh_mybig_moddouble_64(u2,Pa,pt2->y);		//pt1y = 2*u2 = 2*lambda_2
	dh_mybig_modsub_64(pt2->x,pt2->y,pt2->x);	//x = pt1x-pt1y = lambda_1^2-2*labmda_2

	dh_mybig_monmult_64(u1,u1,u1);			//u1 = u1*u1 = 4y^4;
	dh_mybig_moddouble_64(u1,Pa,u1);		//u1 = 2u1 = 8y^4 = lambda_3

	dh_mybig_modsub_64(u2,pt2->x,u2);		//u2 = u2-pt1x = lambda2-pt1x
	dh_mybig_monmult_64(u3,u2,pt2->y);		//pt1y = u2*u3 = Lambda_1 * (lambda2-pt1x)
	dh_mybig_modsub_64(pt2->y,u1,pt2->y);	//y = pt1y - labmda_3;
}
//???????
__device__ __host__ void ppoint_double_V2(Jpoint *pt1,Jpoint* pt2){
	UINT64 u1[4],u2[4],u3[4];

	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
	#else
		const UINT64 *Pa=h_p;
	#endif	

	dh_mybig_monmult_64(pt1->x,pt1->x,u2); 	//u2 = x^2
	dh_mybig_moddouble_64(u2,Pa,pt2->y);		//u1 = 2u2 = 2*x^2
	dh_mybig_modadd_64(pt2->y,u2,pt2->y);		//u1 = u1+u2 = 3*x^3 = lamb1
	dh_mybig_moddouble_64(u3,Pa,u3);		//p2y = 2y
	dh_mybig_monmult_64(pt2->z,u3,pt2->z); 	//p2y = 2yz
	
	dh_mybig_monmult_64(u3,u3,u3);  //p2y = 4y^2
	dh_mybig_monmult_64(pt1->x,u3,u2);		//u2 = 4xy^2

	dh_mybig_monmult_64(u3,u3,u3);  //p2y = 2py2 = 16y^4
	dh_mybig_monmult_64(u3,dc_mon_inv_two,u3);//p2y = p2y/2 = 8y^4

	dh_mybig_monmult_64(pt2->y,pt2->y,pt2->x);			//p2x = u1*u1 = lamb1^2
	dh_mybig_modsub_64(pt2->x,pt2->y,pt2->x);		//p2x = p2x-u2
	dh_mybig_modsub_64(pt2->x,pt2->y,pt2->x);		//p2x = p2x-u2 = lamb1^2-2lamb2

	dh_mybig_modsub_64(u2,pt2->x,u2);			//u2 = u2-p2x=lamb2-x
	dh_mybig_monmult_64(pt2->y,u2,pt2->y);

	dh_mybig_modsub_64(pt2->y,u3,pt2->y);
	//?????��??????????3
	//secp256k1??a?0??????????????????????????
	// dh_mybig_moddouble_64(pt1->y,Pa,u1); 	//u1=2y
	// dh_mybig_monmult_64(pt1->z,u1,pt2->z);		//z=u1*z=2yz

	// dh_mybig_monmult_64(pt1->x,pt1->x,u2);	//u2=x^2
	// dh_mybig_moddouble_64(u2,Pa,u3);		//u3=2*u2=2x^2
	// dh_mybig_modadd_64(u3,u2,u3);			//u3=u3+u2 = 3x^2 = lambda_1
	// dh_mybig_monmult_64(u1,pt1->y,u1);		//u1 = u1*y = 2y^2
	// dh_mybig_monmult_64(u1,pt1->x,u2);		//u2 = u1*x = 2xy^2 
	// dh_mybig_moddouble_64(u2,Pa,u2);		//u2 = 2*u2 = 4xy^2= lambda_2
	// dh_mybig_monmult_64(u3,u3,pt2->x);		//pt1x = lambda_1^2
	// dh_mybig_moddouble_64(u2,Pa,pt2->y);		//pt1y = 2*u2 = 2*lambda_2
	// dh_mybig_modsub_64(pt2->x,pt2->y,pt2->x);	//x = pt1x-pt1y = lambda_1^2-2*labmda_2

	// dh_mybig_monmult_64(u1,u1,u1);			//u1 = u1*u1 = 4y^4;
	// dh_mybig_moddouble_64(u1,Pa,u1);		//u1 = 2u1 = 8y^4 = lambda_3

	// dh_mybig_modsub_64(u2,pt2->x,u2);		//u2 = u2-pt1x = lambda2-pt1x
	// dh_mybig_monmult_64(u3,u2,pt2->y);		//pt1y = u2*u3 = Lambda_1 * (lambda2-pt1x)
	// dh_mybig_modsub_64(pt2->y,u1,pt2->y);	//y = pt1y - labmda_3;
}

/*
 * Point Triple Function.
 * function: pt2 = pt1^3 = 3 \dots pt1.
 * requirements: pt1, pt2 coordinates are represented in Montgomery Field.
 * Attention: pt2 can't be pt1.
 * */
__device__ __host__ void ppoint_triple(Jpoint *pt1,Jpoint* pt2){
    UINT64 h_mon_a[4]={0xfffffffffffffffcL,0xfffffffc00000003L,0xffffffffffffffffL,0xfffffffbffffffffL};
    UINT64 c[4], m[4], e[4], t[4], u[4], wtf[4], foo[4];
#ifdef __CUDA_ARCH__
    const UINT64 *Pa=dc_p;
#else
    const UINT64 *Pa=h_p;
#endif
    dh_mybig_monmult_64(pt1->z, pt1->z, pt2->z); //zz = z1^2
    dh_mybig_monmult_64(pt1->y, pt1->y, pt2->y); //yy = y1^2
    dh_mybig_moddouble_64(pt2->y, Pa, c); //c = 2 * yy

    dh_mybig_monmult_64(pt1->x, pt1->x, pt2->x);
    dh_mybig_moddouble_64(pt2->x, Pa, m);
    dh_mybig_modadd_64(m, pt2->x, m);
    //dh_mybig_monmult_64(h_mon_a, pt2->z, wtf); // a must be in montgomery field.
    //dh_mybig_modadd_64(m, wtf, m);// m = 3 * x1^2 + a * zz^2

    dh_mybig_monmult_64(pt1->x, c, foo);
    dh_mybig_monmult_64(m, m, wtf);
    dh_mybig_moddouble_64(foo, Pa, e);
    dh_mybig_modadd_64(foo, e, foo);
    dh_mybig_moddouble_64(foo, Pa, e);
    dh_mybig_modsub_64(e, wtf, e);//e = 6 * x1 * c - M^2

    dh_mybig_monmult_64(c, c, wtf);
    dh_mybig_moddouble_64(wtf, Pa, t);//t = 2 * c^2

    dh_mybig_monmult_64(m, e, u);
    dh_mybig_modsub_64(u, t, u);//u = m*e - t

    dh_mybig_monmult_64(e, e, wtf);//ee = e^2, use wtf to store.

    dh_mybig_moddouble_64(u, Pa, foo);
    dh_mybig_moddouble_64(foo, Pa, foo);//u4 = 4 * u, use foo to store.

    dh_mybig_monmult_64(pt1->x, wtf, pt2->x);
    dh_mybig_monmult_64(c, foo, c);
    dh_mybig_modsub_64(pt2->x, c, pt2->x);//x3 = x1*ee - c * u4

    dh_mybig_modsub_64(t, u, t);
    dh_mybig_monmult_64(foo, t, u); // foo = u4 out of life, u = u4(t - u).
    dh_mybig_monmult_64(e, wtf, foo);
    dh_mybig_modsub_64(u, foo, wtf);
    dh_mybig_monmult_64(pt1->y, wtf, pt2->y);//y3 = y1*(u4*(t-u) - e*ee)

    dh_mybig_monmult_64(pt1->z, e, pt2->z);//z3 = z1*e

}

// 7m + 7s solution
__device__ __host__ void ppoint_triple_v2(Jpoint *pt1,Jpoint* pt2) {
    UINT64 h_mon_a[4] = {0xfffffffffffffffcL, 0xfffffffc00000003L, 0xffffffffffffffffL, 0xfffffffbffffffffL};
    UINT64 dy[4], dz[4], dm[4], de[4], qy[4], t[4], m[4], e[4], u[4], wtf[4];

#ifdef __CUDA_ARCH__
    const UINT64 *Pa=dc_p;
#else
    const UINT64 *Pa=h_p;
#endif
    dh_mybig_monmult_64(pt1->y, pt1->y, dy); // dy = y1^2
    dh_mybig_monmult_64(pt1->z, pt1->z, dz); // dz = z1^2
    dh_mybig_monmult_64(dy, dy, qy); // qy = dy^2

	dh_mybig_monmult_64(pt1->x, pt1->x, m);
	dh_mybig_moddouble_64(m, Pa, wtf); // wtf = 2 * dx1
	dh_mybig_modadd_64(m, wtf, m); // m = 3 * dx1
	dh_mybig_monmult_64(dz, dz, wtf);
	dh_mybig_monmult_64(wtf, h_mon_a, wtf);
    dh_mybig_modadd_64(m, wtf, m);
	// error! this is relevant to a. m = 3*x1^2 + a * dz^2

    dh_mybig_monmult_64(m, m, dm); // dm = m^2

    dh_mybig_monmult_64(pt1->x, dy, wtf);
    dh_mybig_moddouble_64(wtf, Pa, e);
    dh_mybig_modadd_64(e, wtf, e);
    dh_mybig_moddouble_64(e, Pa, wtf);
    dh_mybig_moddouble_64(wtf, Pa, e);
    dh_mybig_modsub_64(e, dm, e); // e = 12 * x1 * dy - dm

    dh_mybig_monmult_64(e, e, de); // de = e^2

    dh_mybig_moddouble_64(qy, Pa, wtf);
    dh_mybig_moddouble_64(wtf, Pa, t);
    dh_mybig_moddouble_64(t, Pa, wtf);
    dh_mybig_moddouble_64(wtf, Pa, t); //t = 16 * qy

    dh_mybig_modadd_64(m, e, wtf);
    dh_mybig_monmult_64(wtf, wtf, u);
    dh_mybig_modadd_64(t, dm, t);
    dh_mybig_modadd_64(t, de, t);
    dh_mybig_modsub_64(u, t, u); // u = (m + e)^2 - dm - de - t

    dh_mybig_monmult_64(dy, u, wtf);
    dh_mybig_moddouble_64(wtf, Pa, pt2->x);
    dh_mybig_moddouble_64(pt2->x, Pa, wtf); // wtf = 4 * dy * u
    dh_mybig_monmult_64(pt1->x, de, pt2->x);
    dh_mybig_modsub_64(pt2->x, wtf, pt2->x);
    dh_mybig_moddouble_64(pt2->x, Pa, wtf);
    dh_mybig_moddouble_64(wtf, Pa, pt2->x); // x3 = 4(x1 * de - 4 * dy * u)


    dh_mybig_modsub_64(t, u, wtf);
    dh_mybig_monmult_64(wtf, u, pt2->y); // store u * (t - u)
    dh_mybig_monmult_64(e, de, wtf);
    dh_mybig_modsub_64(pt2->y, wtf, pt2->y);
    dh_mybig_monmult_64(pt2->y, pt1->y, wtf); // wtf = y1 * (u * (t - u) - e * de)
    dh_mybig_moddouble_64(wtf, Pa, pt2->y);
    dh_mybig_moddouble_64(pt2->y, Pa, wtf);
    dh_mybig_moddouble_64(wtf, Pa, pt2->y);// y3 = 8 * y1 * (u * (t - u) - e * de)

    dh_mybig_modadd_64(pt1->z, e, wtf);
    dh_mybig_monmult_64(wtf, wtf, pt2->z);
    dh_mybig_modadd_64(de, dz, de);
    dh_mybig_modsub_64(pt2->z, de, pt2->z); // z3 = (z1 + e)^2 - dz - de


}

__device__ __host__ void dh_ellipticAdd_JJ(Jpoint *pt1, Jpoint* pt2,Jpoint* pt3){
	UINT64 u1[4],u2[4],u3[4],u4[4];
	//u1=z2^2

	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
		const UINT64 *mon_inv_two = dc_mon_inv_two;

	#else
		const UINT64 *Pa=h_p;
		const UINT64 *mon_inv_two = h_mon_inv_two;

	#endif	

	dh_mybig_monmult_64(pt2->z,pt2->z,u1);

	

	//u2=z1^2
	dh_mybig_monmult_64(pt1->z,pt1->z,u2);


	//u3 = u1*x1=x1*z2^2=lam1
	dh_mybig_monmult_64(pt1->x,u1,u3);
	
	//u4 = u2*x2 = x2*z1^2=lam2
	dh_mybig_monmult_64(pt2->x,u2,u4);
	
	

	//u1 = z2^3
	dh_mybig_monmult_64(u1,pt2->z,u1);
	//u2 = z1^3
	dh_mybig_monmult_64(u2,pt1->z,u2);
	
	//u1 = u1*y1 = lam4
	dh_mybig_monmult_64(u1,pt1->y,u1);
	//u2 = u2*y2 = lam5
	dh_mybig_monmult_64(u2,pt2->y,u2);


	//pt3y = u3-u4 = lam3
	dh_mybig_modsub_64(u3,u4,pt3->y);

	

	//u3+=u4 = lam1+lam2 = lam7
	dh_mybig_modadd_64(u3,u4,u3);
	//U4=u1-u2=lam4-lam5=lam6
	dh_mybig_modsub_64(u1,u2,u4);

	
	//u1+=u2 = lam4+lam5=lam8
	dh_mybig_modadd_64(u1,u2,u1);

	

	//pt3z = z1*z2*lam3
	dh_mybig_monmult_64(pt1->z,pt2->z,pt3->z);
	dh_mybig_monmult_64(pt3->z,pt3->y,pt3->z);

	
	//pt3x = u4^2 = lam6^2
	dh_mybig_monmult_64(u4,u4,pt3->x);
	//u2 = pt3y^2 = lam3^2
	dh_mybig_monmult_64(pt3->y,pt3->y,u2);

	

	//pt3y = pt3y*u2 = lam3^3
	dh_mybig_monmult_64(pt3->y,u2,pt3->y);

	//u2 = u2*u3 = lam7*lam3^2
	dh_mybig_monmult_64(u2,u3,u2);
	
	

	
	//pt3x -= u2 = lam6^2-lam7*lam3^2
	dh_mybig_modsub_64(pt3->x,u2,pt3->x);

	

	//u3 = 2pt3x
	dh_mybig_moddouble_64(pt3->x,Pa,u3);
	//u2-=u3=lam7lam3^2-2pt3x=lam9
	dh_mybig_modsub_64(u2,u3,u2);

	

	//u1 *= pt3y = lam8*lam3^3
	dh_mybig_monmult_64(u1,pt3->y,u1);

	//pt3y = u2*u4 = lam9*lam6
	dh_mybig_monmult_64(u2,u4,pt3->y);

	// dh_mybig_copy(pt1->x,u1);
	// // dh_mybig_copy(pt1->y,u2);
	// return;

	//pt3y-=u1
	dh_mybig_modsub_64(pt3->y,u1,pt3->y);

	

	//pt3y/=2
	dh_mybig_monmult_64(pt3->y,mon_inv_two,pt3->y);
	// dh_mybig_half_64(pt3->y,pt3->y);

}
//?????pt1???????pt2, pt1+=pt2
__device__ __host__ void dh_ellipticSumEqual_AJ2(Jpoint *pt1, Apoint* pt2)//pt1,pt2????????????????????????��?
{
	UINT64 u1[4],u2[4];
	if(dh_iszero_A(pt2))	return;	
	if(dh_iszero_J(pt1))
	{
		dh_mybig_copy(pt1->x, pt2->x);
		dh_mybig_copy(pt1->y, pt2->y);
		//Z????mon_ONE?????????????p????????
		//0x1000003d1L,0x0L,0x0L,0x0L
		pt1->z[0]=0x1000003d1L;		pt1->z[1]=0x0L;		pt1->z[2]=0x0L;		pt1->z[3]=0x0L;
		// pt1->z[4]=0x0000000000000000L;		pt1->z[5]=0x0000000000000001L;		pt1->z[6]=0x0000000100000000L;		pt1->z[7]=0x0000000000000000L;				
		return;
	}	
	//??????????��??????????????????????
	
	
	//3.????u1=(pt1->z)^2.
	dh_mybig_monmult_64(pt1->z, pt1->z, u1);
	
	//4.????u2=(pt1->z)*u1.
	dh_mybig_monmult_64(pt1->z, u1, u2);
	
	//5.????u1=(pt2->x)*u1.
	dh_mybig_monmult_64(pt2->x, u1, u1);
	
	//6.????u2=(pt2->y)*u2.
	dh_mybig_monmult_64(pt2->y, u2, u2);
	
	//7.????u1=u1-pt1->x.		
	dh_mybig_modsub_64(u1, pt1->x,u1);
	
	//8.????u2=u2-pt1->y.	
	dh_mybig_modsub_64(u2,pt1->y,u2);
	
	///*	
	//9.?��???,???????????????.
	if(dh_mybig_iszero(u1))
	{
		if(dh_mybig_iszero(u2))
		{
			//GAO:????????????????????
			ppoint_double(pt1,pt1);//y???????????????????
			printf("here! use ppoint double!\n");
			
			return;
		}
		else//?????????????????????
		{
			dh_setzero_J(pt1);
			return ;
		}
	}
	//*/
	//10.pt1->z=pt1->z*u1.
	dh_mybig_monmult_64(pt1->z, u1, pt1->z);
	
	//11.????pt2->x=u1^2.
	dh_mybig_monmult_64(u1, u1, pt2->x);
	
	//12.????pt2->y=pt2->x*u1.
	dh_mybig_monmult_64(u1, pt2->x, pt2->y);
	
	//13.????pt2->x=pt1->x*pt2->x.
	dh_mybig_monmult_64(pt1->x, pt2->x, pt2->x);
	
	//14.????u1=2*pt2->x.
	dh_mybig_modadd_64(pt2->x,pt2->x,u1);
	
	//15.x1=u2^2.
	dh_mybig_monmult_64(u2, u2, pt1->x);
	
	//16.x1=pt2->x
	dh_mybig_modsub_64(pt1->x,u1,pt1->x);
	
	//17.x1=x1-pt2->y
	dh_mybig_modsub_64(pt1->x,pt2->y,pt1->x);
	
	//18.????pt2->x=pt2->x-x1.
	dh_mybig_modsub_64(pt2->x,pt1->x,pt2->x);
	
	//19.pt2->x=pt2->x*u2
	dh_mybig_monmult_64(pt2->x, u2, pt2->x);
	
	//20.pt2->y=pt2->y*y1
	dh_mybig_monmult_64(pt2->y, pt1->y, pt2->y);
	
	//21.y1=pt2->x-pt2->y
	dh_mybig_modsub_64(pt2->x,pt2->y,pt1->y);
}
__device__ __host__ void dh_ellipticSumEqual_AJ(Jpoint *pt1, Apoint* pt2)//pt1,pt2????????????????????????��?
{
	UINT64 u1[4],u2[4],u3[4];
	if(dh_iszero_A(pt2))	return;	
	if(dh_iszero_J(pt1))
	{
		dh_mybig_copy(pt1->x, pt2->x);
		dh_mybig_copy(pt1->y, pt2->y);
		//Z????mon_ONE?????????????p????????
		//0x1000003d1L,0x0L,0x0L,0x0L
		pt1->z[0]=0x1000003d1L;		pt1->z[1]=0x0L;		pt1->z[2]=0x0L;		pt1->z[3]=0x0L;
		// pt1->z[4]=0x0000000000000000L;		pt1->z[5]=0x0000000000000001L;		pt1->z[6]=0x0000000100000000L;		pt1->z[7]=0x0000000000000000L;				
		return;
	}	
	//??????????��??????????????????????
	
	
	//3.????u1=(pt1->z)^2.
	dh_mybig_monmult_64(pt1->z, pt1->z, u1);
	
	//4.????u2=(pt1->z)*u1.
	dh_mybig_monmult_64(pt1->z, u1, u2);
	
	//5.????u1=(pt2->x)*u1.
	dh_mybig_monmult_64(pt2->x, u1, u1);
	
	//6.????u2=(pt2->y)*u2.
	dh_mybig_monmult_64(pt2->y, u2, u2);
	
	//7.????u1=u1-pt1->x.		
	dh_mybig_modsub_64(u1, pt1->x,u1);
	
	//8.????u2=u2-pt1->y.	
	dh_mybig_modsub_64(u2,pt1->y,u2);
	
	///*	
	//9.?��???,???????????????.
	if(dh_mybig_iszero(u1))
	{
		if(dh_mybig_iszero(u2))
		{
			//GAO:????????????????????
			ppoint_double(pt1,pt1);//y???????????????????
			printf("here! use ppoint double!\n");
			
			return;
		}
		else//?????????????????????
		{
			dh_setzero_J(pt1);
			return ;
		}
	}
	//*/
	//10.pt1->z=pt1->z*u1.
	dh_mybig_monmult_64(pt1->z, u1, pt1->z);

	// //11.????pt2->x=u1^2.
	// dh_mybig_monmult_64(u1, u1, pt2->x);
	//11.u3 = u1^2
	dh_mybig_monmult_64(u1,u1,u3);

	
	// //12.????pt2->y=pt2->x*u1.
	// dh_mybig_monmult_64(u1, pt2->x, pt2->y);
	//12.u1 = u1*u3
	dh_mybig_monmult_64(u1,u3,u1);
	
	//13.????pt2->x=pt1->x*pt2->x.
	// dh_mybig_monmult_64(pt1->x, pt2->x, pt2->x);
	//13.u3 = u3*pt1->x
	dh_mybig_monmult_64(u3, pt1->x, u3);
	
	//14.????u1=2*pt2->x.
	// dh_mybig_modadd_64(pt2->x,pt2->x,u1);
	//14.x1 = u2^2
	dh_mybig_monmult_64(u2, u2, pt1->x);
	
	//15.x1=u2^2.
	// dh_mybig_monmult_64(u2, u2, pt1->x);
	//15.x1 = x1-u3
	dh_mybig_modsub_64(pt1->x,u3,pt1->x);

	
	//16.x1=pt2->x
	// dh_mybig_modsub_64(pt1->x,u1,pt1->x);
	//16.x1=x1-u3
	dh_mybig_modsub_64(pt1->x,u3,pt1->x);
	
	//17.x1=x1-pt2->y
	// dh_mybig_modsub_64(pt1->x,pt2->y,pt1->x);
	//17.x1 = x1-u1
	dh_mybig_modsub_64(pt1->x,u1,pt1->x);
	

	//18.????pt2->x=pt2->x-x1.
	// dh_mybig_modsub_64(pt2->x,pt1->x,pt2->x);
	//18.u3 = u3 - x1
	dh_mybig_modsub_64(u3,pt1->x,u3);

	
	//19.pt2->x=pt2->x*u2
	// dh_mybig_monmult_64(pt2->x, u2, pt2->x);
	//19.u3 = u3*u2
	dh_mybig_monmult_64(u3, u2, u3);
	
	//20.pt2->y=pt2->y*y1
	// dh_mybig_monmult_64(pt2->y, pt1->y, pt2->y);
	//20.u1 = u1 * pt1->y
	dh_mybig_monmult_64(u1, pt1->y, u1);

	
	//21.y1=pt2->x-pt2->y
	// dh_mybig_modsub_64(pt2->x,pt2->y,pt1->y);
	//21.y1=u3-u1
	dh_mybig_modsub_64(u3,u1,pt1->y);
}

__device__ __host__ void dh_point_mult_inplace(Jpoint* pt1,UINT64 *k){
	//gyy
	int i,j;
	Jpoint tp;
	Jpoint t2;
	//????????1??��??

	//testcode
	// dh_mybig_copy(tp.x,pt1->x);
	// dh_mybig_copy(tp.y,pt1->y);
	// dh_mybig_copy(tp.z,pt1->z);



	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);

	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("pm_in:k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if(((k[i]>>j)&0x01)!=0) break;
	}
	
	dh_mybig_copy(tp.x,pt1->x);
	dh_mybig_copy(tp.y,pt1->y);
	dh_mybig_copy(tp.z,pt1->z);
	j--;
	// printf("j=%d,i=%d\n",j,i);
	for(;j>=0;j--){
			// printf("double\n");
		ppoint_double(pt1,pt1);
		if(((k[i]>>j)&0x01)==1){
				// printf("add\n");
			dh_ellipticAdd_JJ(pt1,&tp,pt1);
		}
	}
	i--;
	for(;i>=0;i--){
		for(j=63;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt1,pt1);
			if(((k[i]>>j)&0x01)==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
	}
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

__device__ __host__ void dh_point_mult_finalversion(Jpoint* pt1,UINT64 *k,Jpoint* pt2){
	//gyy
	int i,j;
	Jpoint tp;
	// Jpoint t2;
	
	//????????1??��??

	//testcode
	// dh_mybig_copy(tp.x,pt1->x);
	// dh_mybig_copy(tp.y,pt1->y);
	// dh_mybig_copy(tp.z,pt1->z);

	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// ppoint_double(pt1,pt1);
	// dh_ellipticAdd_JJ(pt1,&tp,pt1);
	
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);

	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("pm:k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if(((k[i]>>j)&0x01)!=0) break;
	}
	
	dh_mybig_copy(tp.x,pt1->x);
	dh_mybig_copy(tp.y,pt1->y);
	dh_mybig_copy(tp.z,pt1->z);
	j--;

	if(pt1==pt2){
		
		for(;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt1,pt1);
			if(((k[i]>>j)&0x01)==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				ppoint_double(pt1,pt1);
				if(((k[i]>>j)&0x01)==1){
						// printf("add\n");
					dh_ellipticAdd_JJ(pt1,&tp,pt1);
				}
			}
		}
	}else{
		
		// h_print_pointJ(pt1);
		// h_print_pointJ(&tp);
		for(;j>=0;j--){
				// printf("double\n");
			ppoint_double(&tp,&tp);
			if(((k[i]>>j)&0x01)==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(&tp,pt1,&tp);
			}
		}
		i--;
		for(;i>=0;i--){
			for(j=63;j>=0;j--){
					// printf("double\n");
				ppoint_double(&tp,&tp);
				if(((k[i]>>j)&0x01)==1){
						// printf("add\n");
					dh_ellipticAdd_JJ(&tp,pt1,&tp);
				}
			}
		}
		// h_print_pointJ(&tp);
		dh_mybig_copy(pt2->x,tp.x);
		dh_mybig_copy(pt2->y,tp.y);
		dh_mybig_copy(pt2->z,tp.z);
	}
	// printf("j=%d,i=%d\n",j,i);
	
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}

__device__ __host__ void dh_point_mult_uint32(Jpoint* pt1, int k,Jpoint* pt2){
	int i;
	Jpoint tp;
	for(i=31;i>=0;i--){
		if((k>>i)&0x01!=0) break;
	}
	dh_mybig_copy(tp.x,pt1->x);
	dh_mybig_copy(tp.y,pt1->y);
	dh_mybig_copy(tp.z,pt1->z);
	i--;

	if(pt1==pt2){
		for(;i>=0;i--){
			// printf("double\n");
			ppoint_double(pt1,pt1);
			if(((k>>i)&0x01)==1){
				// printf("add\n");
				dh_ellipticAdd_JJ(pt1,&tp,pt1);
			}
		}
	}else{
		for(;i>=0;i--){
			// printf("double\n");
			ppoint_double(&tp,&tp);
			if(((k>>i)&0x01)==1){
				// printf("add\n");
				dh_ellipticAdd_JJ(&tp,pt1,&tp);
			}
		}
		dh_mybig_copy(pt2->x,tp.x);
		dh_mybig_copy(pt2->y,tp.y);
		dh_mybig_copy(pt2->z,tp.z);
	}

}

__device__ void d_base_point_mul(Jpoint *res,UINT64 *k){
	int i,j;
	Jpoint t;
	for(i=0;i<4;i++){
		// printf("k[%d]=%llx\n",i,k[i]);
		if(k[i]!=0) break;
	}
	// printf("i=%d\n",i);
	if(i==4){
		printf("basepm:k==0!!!!!!!!!\n");
	}
	for(j=0;j<64;j++){
		if(((k[i]>>j)&0x01)!=0) break;
	}
	// printf("j=%d\n",j);
	dh_mybig_copy(t.x,basePointMulPCT[i*64+j].x);
	dh_mybig_copy(t.y,basePointMulPCT[i*64+j].y);
	dh_mybig_copy(t.z,dc_mon_ONE);
	j++;
	for(;j<64;j++){

		if(((k[i]>>j)&0x01)==1){
			// printf("i=%d,j=%d\n",i,j);
			dh_ellipticSumEqual_AJ(&t,&basePointMulPCT[i*64+j]);
		}
	}
	i++;
	for(;i<4;i++){
		for(j=0;j<64;j++){
			if(((k[i]>>j)&0x01)==1){

				// printf("i=%d,j=%d\n",i,j); 
				dh_ellipticSumEqual_AJ(&t,&basePointMulPCT[i*64+j]);
				

			}
		}
	}
	dh_mybig_copy(res->x,t.x);
	dh_mybig_copy(res->y,t.y);
	dh_mybig_copy(res->z,t.z);

	
}


__device__ __host__ void dh_point_mult_outofplace(Jpoint* pt1,UINT64 *k,Jpoint* pt2){
	//gyy
	int i,j;

	//????????1??��??

	//testcode
	dh_mybig_copy(pt2->x,pt1->x);
	dh_mybig_copy(pt2->y,pt1->y);
	dh_mybig_copy(pt2->z,pt1->z);



	for(i=3;i>=0;i--){
		if(k[i]!=0)	break;
	}
	if(i<0){
		printf("pm_out:k==0!!!!!!!!!\n");
	}
	for(j=63;j>=0;j--){
		if(((k[i]>>j)&0x01)!=0) break;
	}
	

	j--;
	// printf("j=%d,i=%d\n",j,i);
	for(;j>=0;j--){
			// printf("double\n");
		ppoint_double(pt2,pt2);
		if(((k[i]>>j)&0x01)==1){
				// printf("add\n");
			dh_ellipticAdd_JJ(pt2,pt1,pt2);
		}
	}
	i--;
	for(;i>=0;i--){
		for(j=63;j>=0;j--){
				// printf("double\n");
			ppoint_double(pt2,pt2);
			if(((k[i]>>j)&0x01)==1){
					// printf("add\n");
				dh_ellipticAdd_JJ(pt2,pt1,pt2);
			}
		}
	}
	// printf("copy\n");
	// dh_mybig_copy(pt1->x,tp.x);
	// dh_mybig_copy(pt1->y,tp.y);
	// dh_mybig_copy(pt1->z,tp.z);
}


//GAO:��???
//????????
__device__ void d_multi_inverse(UINT64 *x)
{
	int i,j;
	UINT64 invlj[PARAL*8];//????????��??????4KB????4096=64*256,??????256??512??????
	UINT64 lmd[8];
	
	dh_mybig_copy(invlj,x);//	for(i=0;i<8;i++) invlj[0][i]=x[0][i];
		
	for(i=1;i<PARAL;i++)
	{			    					
		dh_mybig_monmult_64(invlj+(i-1)*8,x+i*8,invlj+i*8);
	}
		
	dh_mybig_monmult_64(invlj+(PARAL-1)*8,dc_ONE,lmd);//z=Z mod P //????????????
	dh_mybig_moninv(lmd,lmd); //?????????????2^n??
		
	for(i=PARAL-1;i>0;i--)
	{
		dh_mybig_monmult_64(invlj+(i-1)*8,lmd,invlj+i*8);
		dh_mybig_monmult_64(x+i*8,lmd,lmd);
		dh_mybig_copy(x+i*8,invlj+i*8);
		//for(j=0;j<8;j++) x[i*8+j]=invlj[i*8+j];	
	}	
	dh_mybig_copy(x,lmd);
	//for(j=0;j<8;j++) x[j]=lmd[j];	
}
//GAO:��???
//???????��?????
__global__ void d_multi_normlize_J(Jpoint *A, int n)
{	
	//????n?????PARAL????????
	
	int i,j,k;
	UINT64 tmp[8];
	int mytid=threadIdx.x+blockDim.x*blockIdx.x;
	int threadnum=blockDim.x*gridDim.x;
	UINT64 z[8*PARAL];//???z????��????????
			
	for(i=mytid*PARAL;i<n;i+=threadnum*PARAL) //n????PARAL??????????��????��??????��?????????
	{
		for(j=0;j<PARAL;j++)
		{
			for(k=0;k<8;k++)
				z[j*8+k]=A[i+j].z[k];
		}
				
		d_multi_inverse(z);
		
		for(j=0;j<PARAL;j++)
		{
			dh_mybig_monmult_64(z+j*8,z+j*8, tmp);
			dh_mybig_monmult_64(A[i+j].x,tmp, A[i+j].x);		
			dh_mybig_monmult_64(tmp,z+j*8, tmp);
			dh_mybig_monmult_64(A[i+j].y,tmp, A[i+j].y);

			//z??????mon_ONE?????mon_ONE??????��?
			A[i+j].z[0]=0x0000000000000001L;	A[i+j].z[1]=0x0000000100000000L;	A[i+j].z[2]=0x0000000000000000L;	A[i+j].z[3]=0x0000000100000000L;
			A[i+j].z[4]=0x0000000000000000L;	A[i+j].z[5]=0x0000000000000001L;	A[i+j].z[6]=0x0000000100000000L;	A[i+j].z[7]=0x0000000000000000L;	
			
			d_mon2normal_J(A+i+j);
		}
	}
	
}

/////////////////GPU?????????????////////////////////////////////////////////////////

__device__ __host__ void dh_my_point_copy(Jpoint* from, Jpoint* to) {
    dh_mybig_copy(to->x, from->x);
    dh_mybig_copy(to->y, from->y);
    dh_mybig_copy(to->x, from->z);
}

/**
 * run_DBC:
 * use n (represented by DBC) calculate res = n * point, the (*) calculator means scalar multiple.
 *
 * @param dbc double-base chain's pointer.
 * @param point jacobian-montgomery base point.
 * @param res jacobian-montogomery result point. res can't be point.
 */
// __device__ __host__ void run_DBC(DBCv2* dbc, Jpoint* point, Jpoint* res) {
// #ifdef __CUDA_ARCH__
//     const UINT64 *Pa=dc_p;
// #else
//     const UINT64 *Pa=h_p;
// #endif

//     Jpoint base, wtf;
//     dh_my_point_copy(point, &base);
//     //dh_my_point_copy(point, &wtf);
//     int now_dbl = 0, now_tpl = 0;
//     if (dbc && dbc->length) {
//         for (int i = 0; i < dbc->length; i++) {
//             while (1) {
//                 if (now_dbl < dbc->data[i].dbl) {
//                     ppoint_double(&base, &wtf);
//                     dh_my_point_copy(&wtf, &base);
//                     now_dbl++;
//                 } else if (now_tpl < dbc->data[i].tpl) {
//                     ppoint_triple(&base, &wtf);
//                     dh_my_point_copy(&wtf, &base);
//                     now_tpl++;
//                 } else
//                     break;
//             } //here wtf = base = new dbl value.

//             if (dbc->data[i].minus) {
//                 // this actually goes for P - a*R, not P - a. but in montgomery field,
//                 // we actually want (P - a) * R, while (P - a)*R === P - a*R mod P.
//                 dh_mybig_sub_64(Pa, base.y, wtf.y);
//             }
//             if (i) { // i > 0
//                 dh_ellipticAdd_JJ(res, &wtf, res);
//             } else { // i == 0, just copy the first value.
//                 dh_my_point_copy(res, &wtf);
//             }
//         }
//     } else {
//         // dbc is null or dbc.length == 0. Now just do nothing.
//     }
// }

/**
 * run_DBC_v2:
 * use n (represented by DBC) calculate res = n * point, the (*) calculator means scalar multiple.
 *
 * @param dbc double-base chain's pointer.
 * @param point jacobian-montgomery base point.
 * @param res jacobian-montogomery result point. res can't be point.
 */
__device__ int run_DBC_v2(Jpoint* pt1, Jpoint* res, int*DBC, int len) {
	// DBC is a int[len][3], DBC[i] = {stmp, alpha, beta}
#ifdef __CUDA_ARCH__
    const UINT64 *Pa=dc_p;
#else
    const UINT64 *Pa=h_p;
#endif
	int now_dbl = 0, now_tpl = 0;
	bool first = true;
	Jpoint base, tmp;
	dh_my_point_copy(pt1, &base);
	int cnt = 0;
	for (int i = len - 1; i >= 0; i--)
	{
		// if (!i && threadIdx.x % 4 == 0) {
		// 	printf("bx=%d, tx=%d: DBC is %d %d\n", blockIdx.x, threadIdx.x, DBC[1], DBC[2]);
		// 	__syncthreads();
		// }
		while (1)
		{
			if (now_dbl < DBC[i*3 + 1])
			{
				ppoint_double(&base, &tmp);
				dh_my_point_copy(&tmp, &base);
				//EC_POINT_dbl(group, mult_points, mult_points, ctx);
				now_dbl++;
			}
			else if (now_tpl < DBC[i*3 + 2])
			{
				// we dont have a good triple version.
				ppoint_double(&base, &tmp);
				dh_ellipticAdd_JJ(&base, &tmp, &base);
				
				//EC_POINT_tpl(group, mult_points, mult_points, ctx);
				now_tpl++;
			}
			else break;
			cnt++;
		}
		dh_my_point_copy(&base, &tmp);
		if (DBC[i * 3] == -1)
		{
			//EC_POINT_invert(group, mult_points, ctx);
			dh_mybig_sub_64(Pa, base.y, tmp.y);
		}

		if (first)
		{
			//EC_POINT_copy(r, mult_points);
			dh_my_point_copy(&tmp, res);
			first = false;
		} else {
			dh_ellipticAdd_JJ(res, &tmp, res);
			//EC_POINT_add(group, r, r, mult_points, ctx);
		}
		cnt++;
		// if (cnt % 10 == 0 && blockIdx.x == 0 && threadIdx.x < 10) {
		// 	printf("tx=%d run %d cnts\n",  threadIdx.x, cnt);
		// }
	}
	return cnt;
}

/**
 * get_DBC:
 * use n (represented by DBC) calculate res = n * point, the (*) calculator means scalar multiple.
 *
 * @param dbc double-base chain's pointer.
 * @param point jacobian-montgomery base point.
 * @param res jacobian-montogomery result point. res can't be point.
 */
#define DBC_store(x, y, z) *(DBC_store + (x) * DBC_level1 + (y) * DBC_level2 + (z))
__device__ int get_DBC(uint288* n, int*DBC_store, int* DBC_len) {
	int tx = threadIdx.x;
    int bx = blockIdx.x;
    int nthread = blockDim.x;
    int dbc_id = nthread * bx + tx;
	uint288 B;
	uint288 six_n;
	uint288 record_outer;
	uint288 temp_outer;
	uint64 n0;
	//int DBC_store[2][DBC_MAXLENGTH][3] = {0}; //��1ά����ͬ��DBC����2ά��һ��DBC�Ĳ�ͬ���3ά�����ţ�2�Ĵ�����3�Ĵ���
	//int DBC_len[2] = {0};
	const int DBC_level1 = 3 * DBC_MAXLENGTH;
	const int DBC_level2 = 3;
	
	//int bBound[MAX_2] = {0};

	//��nתΪ˫��������
	double dbl_n = n->to_double();
	//����B1,B2
	double B1 = 0.9091372900969896 * dbl_n; // 9*n/(7*sqrt(2))
	double B2 = 1.0774960475223583 * dbl_n; // 16*sqrt(2)*n/21
	//����LBound,RBound
	int LBound[MAX_3];
	int RBound[MAX_3];
	int DBC_index = 0;
	DBC_len[1] = 1 << 20; //��ʼ��Ϊ�㹻���������֤��ֻ�����1��DBCʱ��DBC��ʶ��ΪΪ��̵��Ǹ������������ж�
	for (int z = 0; z < DBC_COEF; z++)
	{
		int b = b_try[z];
		LBound[b] = log2(B1 / d_pow23[0][b]) + 1;
		RBound[b] = log2(B2 / d_pow23[0][b]);
		if (LBound[b] == RBound[b])
		{
			int a = RBound[b];
			int i = 0;
			int b_temp = b;
			uint288 t;
			// OCCULTICPLUS: for many reason we need to cancel the copy function for uint288, so explicit deep copy.
			for (int i = 0; i < 9; i++) {
				t.data[i] = n->data[i];
			}
			int s = 1;
			while (!t.iszero())
			{
				//����alpha,beta
				double dbl_t = t.to_double();
				int alpha = a, beta = b_temp;
				double logt = log2(dbl_t);
				const double log3 = log2(3.0);
				for (int j = b_temp; j >= max(0, b_temp - 6); j--)
				{
					int alpha_j;
					if (d_pow23[0][j] >= dbl_t) {
						alpha_j = 0;
					} else {
						int k_j = int(logt - j * log3);
						if (k_j >= a)
							alpha_j = a;
						else
						{
							if (abs(dbl_t - d_pow23[k_j][j]) <= abs(d_pow23[k_j + 1][j] - dbl_t))
								alpha_j = k_j;
							else
								alpha_j = k_j + 1;
						}
					}
					if (abs(dbl_t - d_pow23[alpha_j][j]) <= abs(d_pow23[alpha][beta] - dbl_t))
					{
						alpha = alpha_j;
						beta = j;
					}
				}
				int stmp = s;
				if (!(t >= u_pow23[alpha][beta]))
				 	s = -s;
				DBC_store(DBC_index, i, 0) = stmp;
				DBC_store(DBC_index, i, 1) = alpha;
				DBC_store(DBC_index, i, 2) = beta;
				// in case t not be 0.
				//if (i % 10 == 0) t.data[i / 10] = 0;
				i++;
				if (t >= u_pow23[alpha][beta])
				 	t = t - u_pow23[alpha][beta];
				 else
				 	t = u_pow23[alpha][beta] - t;
				a = alpha;
				b_temp = beta;
	#ifdef DEBUG
				if (tx == 1 && i % 10 == 0) {
					printf("check tx1 data, iteration=%d: ", i);
					for (int i = 0; i < 9; i++) {
						printf("0x%x ", t.data[i]);
					}
					printf("\ncheckminus data %d %d:\n", alpha, beta);
					for (int i = 0; i < 9; i++) {
						printf("0x%x ", u_pow23[alpha][beta].data[i]);
					}
					printf("\n");
				}
				if (i > 100) {
					printf("tx = %d exception: infinite loop!");
					for (int i = 0; i < 9; i++) {
						printf("0x%x ", t.data[i]);
					}
					printf("\ncheckminus data %d %d:\n", alpha, beta);
					for (int i = 0; i < 9; i++) {
						printf("0x%x ", u_pow23[alpha][beta].data[i]);
					}
				}
	#endif
			}
			DBC_len[DBC_index] = i;
			int temp0 = DBC_len[0] * ADD_COST + DBC_store(0, 0, 1) * DBL_COST + DBC_store(0, 0, 2) * TPL_COST;
			int temp1 = DBC_len[1] * ADD_COST + DBC_store(1, 0, 1) * DBL_COST + DBC_store(1, 0, 2) * TPL_COST;
			if (temp0 < temp1)
				DBC_index = 1;
			else
				DBC_index = 0;
		}
	}
	int min_index = 1 - DBC_index;
	return min_index;
}

/////////////////occultic plus's work: use DBC to calculate ended.////////////////////////////////////////////////////
__global__ void d_get_para(UINT64 *para)
{
	int i;
	UINT64 *pt64;
	
	if(threadIdx.x==0 && blockIdx.x==blockDim.x-1)
	{
		pt64=para+  0; for(i=0;i<8;i++) pt64[i]=dc_p[i];
		pt64=para+ 56; for(i=0;i<8;i++) pt64[i]=dc_ONE[i];
		pt64=para+ 64; for(i=0;i<8;i++) pt64[i]=dc_mon_ONE[i];						
	}	
}

void h_print_para()
{
	int groups=11;
	UINT64 *testdata=(UINT64 *)malloc(groups*8*8);	
	UINT64 *d_testdata;
	
	HANDLE_ERROR( hipMalloc((void**)&d_testdata, groups*8*8) );
	
	d_get_para<<<BLOCKNUM,BLOCKSIZE>>>(d_testdata);
	
	HANDLE_ERROR( hipMemcpy( testdata, d_testdata,	64*groups, hipMemcpyDeviceToHost));
	printf("\n__const__ UINT64 testdc_p[8]=");h_mybig_print(testdata);
	printf("\n__const__ UINT64 testdc_ONE[8]=");h_mybig_print(testdata+56);
	printf("\n__const__ UINT64 testdc_mon_ONE[8]=");h_mybig_print(testdata+64);
	
	free(testdata);
	HANDLE_ERROR(hipFree(d_testdata));
}



int h_get_gpu_info()
{
	  hipDeviceProp_t  prop;

    int count;
    HANDLE_ERROR( hipGetDeviceCount( &count ) );
    
    for (int i=0; i< count; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
    }

	return 1;
}

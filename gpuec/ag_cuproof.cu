#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <random>
#include "gmp.h"
#include "ag_gpuec256.h"
#include "openssl/sha.h"
#include "cuda_common.h"

#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}
// #define NCOMMIT 2


mpz_t n;
const UINT64 h_Gx[4]={0x59F2815B16F81798L,0x029BFCDB2DCE28D9L,0x55A06295CE870B07L,0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4]={0x9C47D08FFB10D4B8L,0xFD17B448A6855419L,0x5DA4FBFC0E1108A8L,0x483ADA7726A3C465L};
const UINT64 h_Gz[4]={0x1L,0x0L,0x0L,0x0L};

const UINT64 h_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};
const UINT64 h_R2modN[4]={0x896cf21467d7d140L,0x741496c20e7cf878L,0xe697f5e45bcd07c6L,0x9d671cd581c69bc5L};




char Vx[] =     "4e37ee0ff806bc2a90adb4a9fbc2bcac4853e688f96074c27d4f8504067bb821" ;
char Vy[] =     "35d0be4c081d5886d3b537be233a0523a03e065281f0f6fb7824d7d3407428cb" ;

char Ax[] =     "ccd393ca0432f633be28af8f9418e1b7a6c04a561470ad31eedae90014213b2c" ;
char Ay[] =     "777eb3cc1e68c7810c299273e98f0843975fe13d3666666f2d82c3195ccb282d" ;

char Sx[] =     "b978a8f312d3dc589ea2ec1a13b6297ce143977d0580abd4fdf3e3f0ac757bd6" ;
char Sy[] =     "fccf48f6190ff1ace5627770148e0ef4775f55995bd2beb9e34e4f3e06462953" ;

char T1x[] =    "8e5a60f5c2783a9c12afa44e4409e9e2e9009695f534763075bb9494c67e9089" ;
char T1y[] =    "96d689b27346008aa41ba218d8edc64054dece65c70041034d14e4012fcac3d3" ;

char T2x[] =    "e0aa38bed979fe0f86b7b60ba0f93f821258bf80cf2384106ed0cb89ca9febcb" ;
char T2y[] =    "1050d4fe521403a511914adc58c7541dfd77d8c4a5e18ef02337e4de536fb0c8" ;

char Taux[] =   "ebdff4501a8f6d887c15093e03386b286e86d447471ede1b54c9afee39f844db" ;
char Tprime[] = "d19a68ebf69769536b13014e30bcd92f8279713159e336ca0400ecb309498868" ;
char Mu[] =     "fdf10871a351b0caca73a33fff99f3ca0959a3d29898cbc89eba716e178b675b" ;

char BL[32][65] ={
    "a63f67f8c305d78d5d63a0dc22cf7a46f72e580d5b48c0a8a564d514b049a489",
    "75aa7e628479e74e1921e316a80306f417f08204b8647092a31ca30c9a07ba80",
    "ab7c4b468d630427e70a842e3450c2e8954c6210692203164dd9a118b4eff097",
    "9e003ab6ebfe627b0500885c65546f8a92ec638bf08ad8d4a9c060c31ba22b19",
    "a489b858f486035383cb2a4dd4397404f4ae960e2c80f6023b896ee8f40be392",
    "f13dd205d75902942e9f2503d3b278fb0ce4b23bb3a1d8e82f51e772ba2c2707",
    "802ae424198adc330d7893ef93a5912b0ca10686ba82b61eddf22c19b4920249",
    "2918c4f35508dd5921886086347b573a6bbfbd1582c65407e99a6978a61ddca5",
    "baeb30102adbf71149654cd29e245f21ba5668508abc84e187014b4ee1975ffd",
    "dd1bbd8a79403d4ddae4040623e693f780f3a33b62bae25e12a150a727083180",
    "3fa3a1b24e13cf04eb9b7b0b1b1a629de1c7955132b1f5ed208db54cb0696ab8",
    "99ad9f71d29d78b28923d52e9a0bc3f300e6d0ee9bc0a211da1e3e4fb0e90f2c",
    "a5ca12879aa8fbb11ea7dfcfb8d810e1e553425855fa730529dd747cc4e1f0e5",
    "b8e5e3cdb7c4ecdd9a2e5cd1c537c3b3db5245a39496a1235c2f5d6da3bfd787",
    "334711005ac9f6e4e8fdd4ccfe5c5389a359a7013e68cb64cfde5ef9f4b4fd71",
    "68a9c0de5b56bb519fc6b3c2a93147e9cd0d5c954adcca9a4d803cf830975968",
    "75855d128957fb656264623899be1f91605e9e6a607b0ffad5db560d05228ed2",
    "61f665e4ebdc33bf6a4394d6dc0d48408ab7db1cba7952db74cd721c0022d219",
    "54b1bbe429e4edc28fe53c955f995b7618735e99645e273b30ecef9b48b199f4",
    "92da1cb50b015f025296771f93cf62d8138ccd53d8340a216c1cbeef3ededf0f",
    "3e5bfad7c9312e45a2df4674b39a1c105d6adab51f0c809ddc2c700f655660c8",
    "7d2a6b6be39bedc1112229d9f087c6456c0f53ed38454e0c11e5e4c5d66d6b93",
    "a0c52f47311d9e1b9c7c131f1879bee3730276699f138fd6148b78ce7d647c3e",
    "4792e9c21deb13eaa8d9d5e0326512ee25ba4b731fc60a0ee191758be1bc1937",
    "e3d3a4799d187242a84f1e61b18a422f0a0a58469833e4dc34d6519d61b48b59",
    "1d91fbb378f8e048034d617e9fb8c5817b7ba020b5e76793ab6cd89237638fce",
    "27acf8951a83690b973e505695cec74fc13c80b2b731b361765d130046c78f58",
    "2368d16270296445b1110d894b34939c3d74597797c3f823f83312a72d39f8bb",
    "80044e678d25cd67da108b7a6fcc3d85087d5fddb464f55e64d22ae367a8e2fc",
    "cce1f9fe480a684174530d14db3845efd1126895c8198f7ab42751b44e772103",
    "429dd79ee498b3a0bf61c7706abb7a3ec67400ccd897b538d6398caad423fb95",
    "a299d46ab118a13a92c81662b03068fdb5e1a702525eedbdf242ea7774ca6420"
};
char BR[32][65] ={
    "ba70ff7c351ea26397ad0c3bcf094db05dfdd77f4236fa4c54ec61782047a375",
    "db7be64a9783e9dfccaf553c193fdb8b0c82a0f388df52b983358d47aa3d4a8d",
    "889e7236abebd33b0c5602890977cdb7371a575833119962e57a507f36913ba0",
    "b22443d202834ccaac65a621759ac51eb70f44488345a81368222f53bf8dfc40",
    "7a743ac6ade7756a22ed646375ab2747542d490855eb7c373c6c8a99f383b412",
    "235136ac18f4a5839c8259614e80fa8c7a415fd5329c844f4f63256df688f81d",
    "331ec69f64499b469f5510407fff0674d9802c9e536415a854391353ad3375a4",
    "3aa30e5f6abd82a7d9f8dc8d810f1608d059e3e075d687058ae2394e6f899e06",
    "f22ce0f73c75e10a27e9132ec8ce57cbb66456fbc85d5016df29b48d720bab7e",
    "56270145a12fcb44aa2e7376c7beb2ae7a5053e2231d2501f03d098d677f0b6c",
    "360b24e9be4cc4535c8118958ecc6de833ffb3c1387ee06875edead1a10c5533",
    "4b9f11fc705eb1e5e8eb7e69b6f610c95804511fc660ffcefafef354c08c66b3",
    "e71a62131167e21bc47a9ca02277647d35dc1a0d3fbb693d8cfbc4e91a0fc57",
    "19dd683b98e85ef1cffd6ed48d66e731e931614e60c527ad8e436308670ae588",
    "453c4bf9a850f93848f48e816683ecfc7da730106ffeb8da8f88585bbe26774e",
    "3c0c9e3cfa57639890b0266295d9d63ab54e80a9289a5cc62ff758582f848755",
    "db0acf3dd412f5f97cfaee7bb7e6bf104beb844f284d4e965fe5470460c5c526",
    "29ed05fe749ca34f9e27904988788a844f355c55b959cf031287c6a9eae99ee6",
    "968382aeb9271aba91de083bc664b1de7418b504c980bb115637b40bfc8217ff",
    "f4059c9d3cd7a26c728fe976852eafd5f9ae22900653f14e84f8bece990dab1c",
    "160a3d1ac5455223f01add67962fcf0cbf71c5c753fcddec55b3a51f33365f40",
    "b58b41193b25c80e1a044c2d14f32ed35bd728e5e0033b72a88287cc00630029",
    "e876e725b1066b42732c81689787714e89b3ee02ba46fbc69fbf9594e7a4758d",
    "e840a0bacff2daf8ebe9790443252247423ba0a007d171f8ae9d1de6459ada1a",
    "23cc706144607e87d91162f6929119fbd1c03f198c8d74b66aa940671b494af8",
    "b769031887f39a5236dc4228064a1eb15e3a9183d9e140356dde404c8e4cd4be",
    "47173fbecb64e907f178e763bbba94e3960ed161e9c4bfd01f9ffbf03f99c95",
    "34e41609df5e815bdbcad7572393d58bd29408340d50d7e14ad6a7f16d54c51c",
    "c02381ffaee0de970e798fadeef5854288f695ae0acf9dd4112dd1573d504c30",
    "7c268c9b3a57e017979ed640b44ede50354b7e07588e31cc69945d8b4722bb1c",
    "a8dcf4ab9445368c2f1c7de581d8606435d24d992d8b2d16aa04d85656494079",
    "14c706871dce0dcc8a2080e9a6d98ceec8586ae89f85d2cefa62e34332f84472"
    
};



BPSetupParams h_params;
initParamRandom h_ranParams;
std::string SEED="gyy hello world";
BPProve h_prove;
UINT64 h_bLR[256];

BPSetupParams *d_params;
initParamRandom *d_ranParams;

BPProve *d_prove;
UINT64 *d_bLR;
void str2uint64(char *s,UINT64* x){
    std::string tmps(s);
    int len = tmps.size();
    // std::cout<<"len="<<len<<std::endl;
    // std::cout<<"size="<<tmps.size()<<std::endl;
    // std::cout<<"tmps="<<tmps<<std::endl;
    // std::cout<<"s="<<s<<std::endl;
    // std::cout<<tmps<<std::endl;
    // std::cout<<"0="<<tmps.substr(tmps.size()-16,16).c_str()<<std::endl;
    // std::cout<<"1="<<tmps.substr(tmps.size()-32,16).c_str()<<std::endl;
    // std::cout<<"2="<<tmps.substr(tmps.size()-48,16).c_str()<<std::endl;
    // std::cout<<"3="<<tmps.substr(0,16-(64-tmps.size())).c_str()<<std::endl;
    if(len<16){
        x[0]=strtoull(tmps.c_str(),NULL,16);
        x[1]=0;
        x[2]=0;
        x[3]=0;
        return ;
    }
    if(len<32){
        x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
        x[1]=strtoull(tmps.substr(0,16-(32-tmps.size())).c_str(),NULL,16);
        printf("%s\n",tmps.substr(0,16-(32-tmps.size())).c_str());
        x[2]=0;
        x[3]=0;
        return ;
    }
    if(len<48){
        x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
        x[1]=strtoull(tmps.substr(tmps.size()-32,16).c_str(),NULL,16);
        x[2]=strtoull(tmps.substr(0,16-(48-tmps.size())).c_str(),NULL,16);
        x[3]=0;
        return ;
    }
    x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
    x[1]=strtoull(tmps.substr(tmps.size()-32,16).c_str(),NULL,16);
    x[2]=strtoull(tmps.substr(tmps.size()-48,16).c_str(),NULL,16);
    x[3]=strtoull(tmps.substr(0,16-(64-tmps.size())).c_str(),NULL,16);
    return ;
    

    // h_mybig_print(x);
    // std::cout<<std::endl;
}
void setJpoint(char* x,char* y,Jpoint* jp){
    str2uint64(x,jp->x);
    str2uint64(y,jp->y);
    str2uint64("1",jp->z);
}

void sha256(char *string, char *outputBuffer)
{
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    SHA256_Update(&sha256, string, strlen(string));
    SHA256_Final(hash, &sha256);
    int i = 0;
    for(i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(outputBuffer + (i * 2), "%02x", hash[i]);
    }
    outputBuffer[64] = 0;
}
void uint642str(UINT64* x,char *s){
    int cur=0;
    for(int i=3;i>=0;i--){
        for(int j=0;j<16;j++){
            // std::cout<<((x[i]>>((15-j)*4))&0xf==0)<<std::endl;
            // printf("%d\n",(x[i]>>((15-j)*4))&0xf);
            // printf("%d\n",(int)(x[i]>>((15-j)*4))&0xf == (int)0);
            if(cur==0 && (((x[i]>>((15-j)*4))&0xf) ==0)){
                continue;
            }
            sprintf(s + cur, "%x", (x[i]>>((15-j)*4))&0xf);
            cur++;
            // sprintf(s + (3-i)*16+j*2, "%02x", (x[i]>>((7-j)*8))&0xff);
            // printf("%x\n",(x[i]>>((7-j)*8))&0xff);
        }
        
    }
    s[cur]=0;
}
void uint642bin(UINT64* x,unsigned char *s){
    for(int i=3;i>=0;i--){
        for(int j=0;j<8;j++){
            s[(3-i)*8+j] = (x[i]>>((7-j)*8))&0xff;
            // printf("%02x\n",(x[i]>>((7-j)*8))&0xff);
        }
        
    }
    s[64]=0;
}
void HashBP(Jpoint* A,Jpoint* S,UINT64 *o1,UINT64 *o2){
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    // unsigned char tmp[32];

    char tmp[65];
    char o1str[65];
    uint642str(A->x,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(A->y,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(S->x,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(S->y,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    SHA256_Final(hash, &sha256);
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(o1str + (i * 2), "%02x", hash[i]);
    }
    o1str[64] = 0;
    str2uint64(o1str,o1);
    // printf("tmp=%s\n",o1str);
    // h_mybig_print(o1);
    if(o2==NULL) return;
    SHA256_CTX sha2562;
    SHA256_Init(&sha2562);
    // unsigned char tmp[32];

    uint642str(A->x,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(A->y,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(S->x,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(S->y,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    SHA256_Update(&sha2562, o1str, strlen(o1str));
    SHA256_Final(hash, &sha2562);
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(tmp + (i * 2), "%02x", hash[i]);
    }
    str2uint64(tmp,o2);
    // printf("tmp=%s\n",tmp);
    // h_mybig_print(o2);
    // o2[64] = 0;
}
// void init(){
//     setJpoint(Vx,Vy,&h_prove.V);
//     setJpoint(Ax,Ay,&h_prove.A); 
//     setJpoint(Sx,Sy,&h_prove.S);
//     setJpoint(T1x,T1y,&h_prove.T1);
//     setJpoint(T2x,T2y,&h_prove.T2);

//     str2uint64(Taux,h_prove.Taux);
//     str2uint64(Mu,h_prove.Mu);
//     str2uint64(Tprime,h_prove.Tprime);

//     for(int i=0;i<32;i++){
//         str2uint64(BL[i],&h_bLR[4*i]); 
//         str2uint64(BR[i],&h_bLR[(32+i)*4]); 
//     }
// }
    

void calInvy(UINT64 *yinv,char *stry){
    mpz_init(n);
    mpz_set_str(n,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141",16);
    mpz_t bny,bnyinv;
    mpz_init(bny);
    mpz_set_str(bny,stry,16);
    mpz_invert(bnyinv,bny,n);
    mpz_mod(bnyinv,bnyinv,n);
    char tmp[65]={0};
    mpz_get_str(tmp,16,bnyinv);
    str2uint64(tmp,yinv);
}

void inline JpointCpyFromXYZ(Jpoint *jp,const UINT64 *x,const UINT64 *y,const UINT64 *z){
    jp->x[0] = x[0];
    jp->x[1] = x[1];
    jp->x[2] = x[2];
    jp->x[3] = x[3];

    jp->y[0] = y[0];
    jp->y[1] = y[1];
    jp->y[2] = y[2];
    jp->y[3] = y[3];

    jp->z[0] = z[0];
    jp->z[1] = z[1];
    jp->z[2] = z[2];
    jp->z[3] = z[3];
}

void inline JpointCpy(Jpoint *des,Jpoint *src){
    des->x[0]=src->x[0];
    des->x[1]=src->x[1];
    des->x[2]=src->x[2];
    des->x[3]=src->x[3];

    des->y[0]=src->y[0];
    des->y[1]=src->y[1];
    des->y[2]=src->y[2];
    des->y[3]=src->y[3];

    des->z[0]=src->z[0];
    des->z[1]=src->z[1];
    des->z[2]=src->z[2];
    des->z[3]=src->z[3];
}

void sha256(const std::string &srcStr, std::string &encodedHexStr)  
{  
    // 调用sha256哈希    
    unsigned char mdStr[33] = {0};  
    SHA256((const unsigned char *)srcStr.c_str(), srcStr.length(), mdStr);  
  
    // 哈希后的字符串    
    // 哈希后的十六进制串 32字节    
    char buf[65] = {0};  
    char tmp[3] = {0};  
    for (int i = 0; i < 32; i++)  
    {  
        sprintf(tmp, "%02x", mdStr[i]);  
        strcat(buf, tmp);  
    }  
    buf[64] = '\0';   
    encodedHexStr = std::string(buf);  
} 
int check_quadratic_residue(mpz_t num){
    mpz_t t1,t2,d;
    mpz_init(t1);
    mpz_init(t2);
    mpz_init(d);
    mpz_set_str(d,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F",16);

    mpz_sub_ui(t1,d,1);
    mpz_tdiv_q_ui(t2,t1,2);
    // gmp_printf("%#Zx\n",t1);
    mpz_powm(t2,num,t2,d);
    // gmp_printf("%#Zx\n",t2);
    if(mpz_cmp_ui(t2,1)==0){
        // gmp_printf("right\n");
        return 1;
    }
    if(mpz_cmp(t2,t1)==0){
        // gmp_printf("not\n");
        return -1;
    }
    return 0;
}

void mapToGroup(const std::string &s,Jpoint* jp){
    std::string tmphex;
    sha256(s,tmphex);
    // std::cout<<tmphex<<std::endl;
    mpz_t hexr,d;
    mpz_t t1,t2;
    mpz_t rx,ry;
    mpz_init(hexr);
    mpz_init(d);
    mpz_set_str(d,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F",16);
    mpz_init(t1);
    mpz_init(t2);
    mpz_init(rx);
    mpz_init(ry);
    mpz_set_str(hexr,tmphex.c_str(),16);


    for(int i=0;i<2048;i+=1){
        mpz_add_ui(rx,hexr,i);
        mpz_powm_ui(hexr,rx,3,d);
        mpz_add_ui(hexr,hexr,7);
        mpz_mod(hexr,hexr,d);


        // mpz_sub_ui(t1,d,1);
        // mpz_tdiv_q_ui(t2,t1,2);
        // gmp_printf("%#Zx\n",t1);
        // mpz_powm(t2,hexr,t2,d);
        if(check_quadratic_residue(hexr)==1){
            // gmp_printf("right\n");
            break;
        }
        if(check_quadratic_residue(hexr)==-1){
            // gmp_printf("not\n");
        }
        // gmp_printf("%#Zx\n",t2);
    }
    //这里hexr就是满足条件的二次剩余，现在要解二次剩余,rx里存的是x值
    //由于secp256k1曲线素数的特殊性，所以直接采用Tonelli-Shanks算法
    mpz_add_ui(t1,d,1);

    mpz_tdiv_qr_ui(t1,t2,t1,4);
    mpz_powm(ry,hexr,t1,d);

    // gmp_printf("%#Zx\n",rx);
    // gmp_printf("%#Zx\n",ry);
    char jx[65]={0};
    char jy[65]={0};
    mpz_get_str(jx,16,rx);
    mpz_get_str(jy,16,ry);
    str2uint64(jx,jp->x);
    str2uint64(jy,jp->y);
    jp->z[0]=0x1L;
    // h_print_pointJ(jp);
    //验证

    // mpz_powm_ui(t1,rx,3,d);
    // mpz_add_ui(t1,t1,7);
    // mpz_mod(t1,t1,d);
    // mpz_powm_ui(t2,ry,2,d);
    // mpz_mod(t2,t2,d);
    // if(mpz_cmp(t1,t2)==0){
    //     gmp_printf("x and y right\n");
    // }else{
    //     gmp_printf("WRONG!!!\n");
    // }
    
    //下面是Cipolla算法第一步找a^2-n为非二次剩余（由于最终采用了别的方法，所以注释掉）
    /*
    while(1){
        mpz_add_ui(t2,hexr,i);
        mpz_powm_ui(t1,t2,2,d);
        mpz_sub(t1,t1,hexr);
        mpz_mod(t1,t1,d);
        if(check_quadratic_residue(t1)==-1){
            break;
        }
        i++;
    }
    gmp_printf("i=%d\n",i);
    gmp_printf("%#Zx\n",t2);
    gmp_printf("%#Zx\n",hexr);
    */


}
void gen_random_uint64(std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> &engine,UINT64 s[4]){
    // std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine(clock());
    for(int i=0;i<4;i++){
        s[i] = engine();
    }
    if(s[0]==0xFFFFFFFFFFFFFFFF&&s[1]==0xFFFFFFFFFFFFFFFF&&
        s[2]==0xFFFFFFFFFFFFFFFF&&s[3]>0xFFFFFFFEFFFFFC2F){
            s[3]-=0xFFFFFFFEFFFFFC2F;
    }
}

void init_random_param(){
    std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine(19970504);
    // std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine;
    // gen_random_uint64(engine,h_ranParams.gamma);
    gen_random_uint64(engine,h_ranParams.alpha);
    gen_random_uint64(engine,h_ranParams.rho);
    gen_random_uint64(engine,h_ranParams.tau1);
    gen_random_uint64(engine,h_ranParams.tau2);
    
    for(int i=0;i<NCOMMIT;i++){
        gen_random_uint64(engine,h_ranParams.gamma+i*4);
    }

    for(int i=0;i<32*NCOMMIT;i++){

        gen_random_uint64(engine,&(h_ranParams.SL[i*4]));
        gen_random_uint64(engine,&(h_ranParams.SR[i*4]));         
        
        
    }
    // h_mybig_print(ranParams.gamma);
    // h_mybig_print(ranParams.alpha);
}

void compute_al_ar(int v,int *al,UINT64 *ar,int n){
    for(int i=0;i<n;i++){
        al[i] = (v>>i) &0x1;
        if(al[i]){
            ar[i*4]=0;
            ar[i*4+1]=0;
            ar[i*4+2]=0;
            ar[i*4+3]=0;
        }else{
            ar[i*4  ]=0xBFD25E8CD0364141-1;
            ar[i*4+1]=0xBAAEDCE6AF48A03B;
            ar[i*4+2]=0xFFFFFFFFFFFFFFFE;
            ar[i*4+3]=0xFFFFFFFFFFFFFFFF;
            //  
        }
        
    }
}

void updateGen(Jpoint *hprime,Jpoint *Hh,UINT64 *y,int N){
    UINT64 yinv[4],expy[4],mony[4];
    JpointCpy(&hprime[0],&Hh[0]);
    dh_mybig_monmult_64_modN(y,h_R2modN,mony);
    // h_mybig_print(mony);
    dh_mybig_copy(expy,mony);
    h_mybig_print(expy);
    dh_mybig_moninv_modN(expy,yinv);
    
    printf("yinv=\n");
    h_mybig_print(yinv);
    // dh_mybig_monmult_64(expy,h_R2,expy);
    for(int i=1;i<N;i++){
        dh_mybig_moninv_modN(expy,yinv);
        dh_mybig_monmult_64(Hh[i].x,h_R2,hprime[i].x);
        dh_mybig_monmult_64(Hh[i].y,h_R2,hprime[i].y);
        dh_mybig_monmult_64(Hh[i].z,h_R2,hprime[i].z);
        dh_point_mult_finalversion(&hprime[i],yinv,&hprime[i]);
        dh_mybig_monmult_64(hprime[i].x,h_ONE,hprime[i].x);
        dh_mybig_monmult_64(hprime[i].y,h_ONE,hprime[i].y);
        dh_mybig_monmult_64(hprime[i].z,h_ONE,hprime[i].z);
        dh_mybig_monmult_64_modN(expy,mony,expy);
    }
}

__device__ d_value[NCOMMIT];

int main(){
    // init();


    int al[32]={0};
    UINT64 ar[32*4]={0};
    int value[NCOMMIT];
    for(int i=0;i<NCOMMIT;i++){
        value[i]= rand();
        printf("value[%d]=%d\n",i,value);
    }
    
    compute_al_ar(value,al,ar,32);
    printf("value=%d\n",value);
    // for(int i=31;i>=0;i--){
    //     printf("%d",al[i]);
    // }
    // printf("\n");
    // for(int i=31;i>=0;i--){
    //     printf("%d",ar[i]);
    // }
    // printf("\n");
    
    // std::string s = "gyy hello world";
    JpointCpyFromXYZ(&h_params.G,h_Gx,h_Gy,h_Gz);
    mapToGroup(SEED,&h_params.H);
    printf("param.H=\n");
    h_print_pointJ(&h_params.H);
    h_params.N=32;
    // cout<<SEED
    for(int i=0;i<h_params.N;i++){
        
        char tmp[3];
        sprintf(tmp,"%u",i);
        mapToGroup(SEED+"h"+tmp,&h_params.Hh[i]);
        mapToGroup(SEED+"g"+tmp,&h_params.Gg[i]);
    }
  

 
    init_random_param();


    
    printf("rho=\n");
    h_mybig_print(h_ranParams.rho);
    printf("gamma=\n");
    h_mybig_print(h_ranParams.gamma);
    printf("alpha=\n");
    h_mybig_print(h_ranParams.alpha);
    printf("tau1=\n");
    h_mybig_print(h_ranParams.tau1);
    printf("tau2=\n");
    h_mybig_print(h_ranParams.tau2);

    unsigned char sd[32];
    UINT64 x[4];
    UINT64 y[4];
    UINT64 z[4];
    UINT64 yinv[4];
    HashBP(&h_prove.A,&h_prove.S,y,z);
    // h_mybig_print(y);
    // h_mybig_print(z);
    Jpoint* hprime = (Jpoint*)malloc(sizeof(Jpoint)*h_params.N);
    updateGen(hprime,h_params.Hh,y,h_params.N);

    // h_print_pointJ(&hprime[0]);
    // h_print_pointJ(&hprime[1]);
    h_print_pointJ(&hprime[31]);
    // std::cout<<o1<<std::endl;
    // calInvy(yinv,o1);

    // std::cout<<o2<<std::endl;
    // HashBP(&h_prove.T1,&h_prove.T2,o1,NULL);
    // std::cout<<o1<<std::endl;
    // h_mybig_print(b);
    // h_mybig_print(c);
    // h_mybig_print(d);

    

    // printf("V=\n");
    // h_print_pointJ(&h_prove.V);
    // printf("A=\n");
    // h_print_pointJ(&h_prove.A);
    // printf("S=\n");
    // h_print_pointJ(&h_prove.S);
    // printf("T1=\n");
    // h_print_pointJ(&h_prove.T1);
    // printf("T2=\n");
    // h_print_pointJ(&h_prove.T2);
    // printf("Taux=\n");
    // h_mybig_print(h_prove.Taux);
    // printf("Tprime=\n");
    // h_mybig_print(h_prove.Tprime);
    // printf("Mu=\n");
    // h_mybig_print(h_prove.Mu);
    // printf("BL=\n");
    // for(int i=0;i<32;i++){
    //     h_mybig_print(h_bLR+4*i);
    // }
    // printf("BR=\n");
    // for(int i=0;i<32;i++){
    //     h_mybig_print(h_bLR+4*(i+32));
    // }

    std::cout<<"Hello world!"<<std::endl;
}
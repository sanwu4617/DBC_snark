#include "hip/hip_runtime.h"
#include<stdio.h>
#include "gpuec256.h"
#include "cuda_common.h"
#include<sys/time.h>
#include<random>
// typedef unsigned long long UINT64; //定义64位字类型
// typedef long long INT64;

#define N_BIGNUM 1024*1024
#define N_POINT N_BIGNUM
#define N_THREAD_PER_BLOCK 384
#define N_BLOCK ((N_BIGNUM+N_THREAD_PER_BLOCK-1)/N_THREAD_PER_BLOCK)
// const UINT64 h_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};
// const UINT64 h_mon_ONE[4]={0x1000003d1L,0x0L,0x0L,0x0L};
// const UINT64 h_ONE[4]={0x1L,0x0L,0x0L,0x0L};
const UINT64 h_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};

const UINT64 h_6Gx[4]={0x252931db128244c9L,0x80ec2e92027d7e6eL,0x32c5ee6d51cb1e89L,0xb89bd74c7352f570L};
const UINT64 h_6Gy[4]={0xd8cbce4f20d0d9e4L,0x5b636389add7cc6eL,0xccd07463f61e7fbeL,0x13fae72c0d3c849bL};
const UINT64 h_6Gz[4]={0xbc5c645f1b1c297dL,0x0ba1469cd0bdd88aL,0x40bad30e143dcdceL,0x4bba49beb75cce43L};



const UINT64 h_Gx[4]={0x59F2815B16F81798L,0x029BFCDB2DCE28D9L,0x55A06295CE870B07L,0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4]={0x9C47D08FFB10D4B8L,0xFD17B448A6855419L,0x5DA4FBFC0E1108A8L,0x483ADA7726A3C465L};

const UINT64 h_3Gx[4]={0x8601f113bce036f9L,0xb531c845836f99b0L,0x49344f85f89d5229L,0xf9308a019258c310L};
const UINT64 h_3Gy[4]={0x6cb9fd7584b8e672L,0x6500a99934c2231bL,0x0fe337e62a37f356L,0x388f7b0f632de814L};
const UINT64 h_3Gz[4]={0x1L,0x0L,0x0L,0x0L};

// __constant__ UINT64 dc_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};
// __constant__ UINT64 dc_ONE[4]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
// __constant__ UINT64 dc_p[4]={0xFFFFFFFEFFFFFC2FL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL,0xFFFFFFFFFFFFFFFFL};



void __global__ testadd(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_modadd_64(a+tx*4,b+tx*4,c+tx*4);
}
void __global__ testsub(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_modsub_64(a+tx*4,b+tx*4,c+tx*4);
}
void __global__ testmul(UINT64* a ,UINT64* b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*4,dc_R2,a+tx*4);
    // dh_mybig_monmult_64(b+tx*4,dc_R2,b+tx*4);
    dh_mybig_monmult_64(a+tx*4,b+tx*4,c+tx*4);
    dh_mybig_monmult_64(c+tx*4,dc_ONE,c+tx*4);
}



void __global__ testhalf(UINT64 *a,UINT64 *b){
    int tx = threadIdx.x;
    dh_mybig_half_64(a+tx*4,b+tx*4);
}

__global__ void testinv(UINT64 *a,UINT64 *b){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*4,dc_R2,a+tx*4);
    dh_mybig_moninv(a+tx*4,b+tx*4);
}

__global__ void testexp(UINT64 *a,UINT64 *b,UINT64 *c){
    int tx = threadIdx.x;
    dh_mybig_monmult_64(a+tx*4,dc_R2,a+tx*4);
    dh_mybig_modexp(a+tx*4,b+tx*4,c+tx*4);
    dh_mybig_monmult_64(c+tx*4,dc_ONE,c+tx*4);
}

void __global__ point_to_monjj(Jpoint* jp1,Jpoint* jp2){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_BIGNUM){
        dh_mybig_monmult_64((jp1+idx)->x,dc_R2,(jp1+idx)->x);
        dh_mybig_monmult_64((jp1+idx)->y,dc_R2,(jp1+idx)->y);
        dh_mybig_monmult_64((jp1+idx)->z,dc_R2,(jp1+idx)->z);

        dh_mybig_monmult_64((jp2+idx)->x,dc_R2,(jp2+idx)->x);
        dh_mybig_monmult_64((jp2+idx)->y,dc_R2,(jp2+idx)->y);
        dh_mybig_monmult_64((jp2+idx)->z,dc_R2,(jp2+idx)->z);
    }
    
}



void __global__ point_from_monjj(Jpoint* jp1,Jpoint* jp2){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_BIGNUM){
        dh_mybig_monmult_64((jp1+idx)->x,dc_ONE,(jp1+idx)->x);
        dh_mybig_monmult_64((jp1+idx)->y,dc_ONE,(jp1+idx)->y);
        dh_mybig_monmult_64((jp1+idx)->z,dc_ONE,(jp1+idx)->z);

        dh_mybig_monmult_64((jp2+idx)->x,dc_ONE,(jp2+idx)->x);
        dh_mybig_monmult_64((jp2+idx)->y,dc_ONE,(jp2+idx)->y);
        dh_mybig_monmult_64((jp2+idx)->z,dc_ONE,(jp2+idx)->z);
    }
    
}


// void __global__ point_to_monaj(Jpoint* jp1,Apoint* jp2){
//     int tx = threadIdx.x;
//     dh_mybig_monmult_64((jp1+tx)->x,dc_R2,(jp1+tx)->x);
//     dh_mybig_monmult_64((jp1+tx)->y,dc_R2,(jp1+tx)->y);
//     dh_mybig_monmult_64((jp1+tx)->z,dc_R2,(jp1+tx)->z);

//     dh_mybig_monmult_64((jp2+tx)->x,dc_R2,(jp2+tx)->x);
//     dh_mybig_monmult_64((jp2+tx)->y,dc_R2,(jp2+tx)->y);
//     // dh_mybig_monmult_64((jp2+tx)->z,dc_R2,(jp2+tx)->z);
// }



// void __global__ point_from_monaj(Jpoint* jp1,Apoint* jp2){
//     int tx = threadIdx.x;
//     dh_mybig_monmult_64((jp1+tx)->x,dc_ONE,(jp1+tx)->x);
//     dh_mybig_monmult_64((jp1+tx)->y,dc_ONE,(jp1+tx)->y);
//     dh_mybig_monmult_64((jp1+tx)->z,dc_ONE,(jp1+tx)->z);

//     dh_mybig_monmult_64((jp2+tx)->x,dc_ONE,(jp2+tx)->x);
//     dh_mybig_monmult_64((jp2+tx)->y,dc_ONE,(jp2+tx)->y);
//     // dh_mybig_monmult_64((jp2+tx)->z,dc_ONE,(jp2+tx)->z);
// }


void __global__ testdouble(UINT64* a ,UINT64* b){
    int tx = threadIdx.x;
    dh_mybig_moddouble_64(a+tx*4,dc_p,b+tx*4);
}

// void __global__ test_point_addaj(Jpoint* jp,Apoint *ap){
//     int tx = threadIdx.x;
//     dh_ellipticSumEqual_AJ(jp+tx,ap+tx);
// }
void __global__ test_point_addjj(Jpoint* p1,Jpoint *p2,Jpoint *p3){
    int tx = threadIdx.x;
    dh_ellipticAdd_JJ(p1+tx,p2+tx,p3+tx);
}

void __global__ test_point_double(Jpoint *p1,Jpoint *p2){
    int tx = threadIdx.x;
    ppoint_double(p1+tx,p2+tx);
}

void __global__ testbasemul(Jpoint *res,UINT64 *k){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<N_POINT){
        // d_mybig_print(k+idx*4);
        d_base_point_mul(res+idx,k+idx*4);
    }

}

void __global__ test_point_mul_inplace(Jpoint *p1,UINT64 *k){
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_inplace(p1+idx,k+idx*4);
}
void __global__ test_point_mul_outofplace(Jpoint *p1,UINT64 *k,Jpoint *p2){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_outofplace(p1+idx,k+idx*4,p2+idx);
}
void __global__ test_point_mul_finalversion(Jpoint *p1,UINT64 *k,Jpoint *p2){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_finalversion(p1+idx,k+idx*4,p2+idx);
}

void __global__ test_point_mul_uint32(Jpoint *p1,int k,Jpoint *p2){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    dh_point_mult_uint32(p1+idx,k,p2+idx);
}

// void __global__ test_point_mul_apoint(Jpoint *p1,Apoint *p2,UINT64 *k){
//     int tx = threadIdx.x;
//     dh_apoint_mult(p1+tx,p2+tx,k+tx*4);
// }

// void __global__ testmul(UINT64* a,UINT64 *b,UINT64 *c){
//     int tx = threadIdx.x;
//     dh_mybig_monmult_64(a+tx*8,h_R2,a+tx*8);
//     // dh_mybig_monmult_64(a+tx*8,h_R2,a+tx*8);
//     dh_mybig_monmult_64(a+tx*8,a+tx*8,c+tx*8);
//     // dh_mybig_monmult_64(c+tx*8,h_ONE,c+tx*8);
// }

void print_big_arr(UINT64* nums,int n){
    for(int i=0;i<n;i++){
        h_mybig_print(nums+i*4);
        printf("\n");
    }
}
void print_jpoint_arr(Jpoint* nums,int n){
    for(int i=0;i<n;i++){
        h_print_pointJ(nums+i);
        printf("\n");
    }
}
// void print_apoint_arr(Apoint* nums,int n){
//     for(int i=0;i<n;i++){
//         h_print_pointA(nums+i);
//         printf("\n");
//     }
// }

void init_big(UINT64 *nums){
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<4;j++){
            nums[i*4+j]=0;
        }
        nums[i*4] = 0x3;
        nums[i*4+1] = 0x103;
    }
    // nums[0]=25;
    // nums[15]=35;
}
void init_big2(UINT64 *nums){
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<4;j++){
            nums[i*4+j]=0xabcdef0123456789;
            // nums[i*4+j]=0;
        }
        // nums[i*4] = 8;
    }
    // nums[0]=25;
    // nums[15]=35;
}
void init_random_big(UINT64 *nums){
    timeval start;
    gettimeofday(&start,NULL);
    std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine;
    engine.seed(start.tv_usec);//设定随机数种子
    for(int i=0;i<N_BIGNUM;i++){
        for(int j=0;j<4;j++){
            nums[i*4+j]=engine();
        }
    }
}
// void init_Apoint(Apoint* p){
//     for(int i=0;i<N_POINT;i++){
//         for(int j=0;j<4;j++){
//             p[i].x[j] = h_Gx[j];
//             p[i].y[j] = h_Gy[j];
//             // p[i].z[j] = h_ONE[j];
//         }
//     }
// }

void init_Jpoint(Jpoint* p){
    for(int i=0;i<N_POINT;i++){
        for(int j=0;j<4;j++){
            p[i].x[j] = h_Gx[j];
            p[i].y[j] = h_Gy[j];
            p[i].z[j] = h_ONE[j];
        }
    }
}
void init_Jpoint2(Jpoint* p){
    for(int i=0;i<N_POINT;i++){
        for(int j=0;j<4;j++){
            p[i].x[j] = h_6Gx[j];
            p[i].y[j] = h_6Gy[j];
            p[i].z[j] = h_6Gz[j];
        }
    }
}
// void init_Apoint(Apoint* p){
//     for(int i=0;i<N_POINT;i++){
//         for(int j=0;j<4;j++){
//             p[i].x[j] = h_3Gx[j];
//             p[i].y[j] = h_3Gy[j];
//             // p[i].z[j] = h_mon_ONE[j];
//         }
//     }
// }

int main(){
    struct timeval s1,e1;
    long long time_use=1;
    int nB,nT;
    // UINT64 tmpbig[4]={0x6903021ca8bd10e,1,0,0};
    // h_mybig_print(tmpbig);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testbasemul);
    // printf("NB=%d,NT=%d\n",nB,nT);
    hipOccupancyMaxPotentialBlockSize(&nB,&nT,test_point_double);
    printf("NB=%d,NT=%d\n",nB,nT);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testmul);
    // printf("NB=%d,NT=%d\n",nB,nT);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,testinv);
    // printf("NB=%d,NT=%d\n",nB,nT);
    // hipOccupancyMaxPotentialBlockSize(&nB,&nT,point_to_monjj);
    // printf("NB=%d,NT=%d\n",nB,nT);

    // UINT64 *h_nums1 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    // UINT64 *h_nums2 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    // UINT64 *h_nums3 = (UINT64*)malloc(sizeof(UINT64)*4*N_BIGNUM);
    // init_big2(h_nums1);
    // init_big2(h_nums2);

    // UINT64 *d_nums1;
    // UINT64 *d_nums2;
    // UINT64 *d_nums3;



    // CUDA_SAFE_CALL(hipMalloc((void**)&d_nums1,sizeof(UINT64)*4*N_BIGNUM));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_nums2,sizeof(UINT64)*4*N_BIGNUM));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_nums3,sizeof(UINT64)*4*N_BIGNUM));

    // CUDA_SAFE_CALL(hipMemcpy(d_nums1,h_nums1,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    // CUDA_SAFE_CALL(hipMemcpy(d_nums2,h_nums2,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));

    // testmul<<<1,N_BIGNUM>>>(d_nums1,d_nums2,d_nums3);

    // CUDA_SAFE_CALL(hipMemcpy(h_nums3,d_nums3,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyDeviceToHost));
    // // print_big_arr(h_nums1,N_BIGNUM);
    // // print_big_arr(h_nums2,N_BIGNUM);
    // // print_big_arr(h_nums1,N_BIGNUM);
    // // print_big_arr(h_nums2,N_BIGNUM);
    // print_big_arr(h_nums3,N_BIGNUM);
    // printf("\n");

    // free(h_nums1);
    // free(h_nums2);
    // free(h_nums3);
    // CUDA_SAFE_CALL(hipFree(d_nums1));
    // CUDA_SAFE_CALL(hipFree(d_nums2));
    // CUDA_SAFE_CALL(hipFree(d_nums3));


// ========================================


    Jpoint* h_p1;
    Jpoint* h_p2;
    Jpoint* d_p1;
    Jpoint* d_p2;
    // Apoint* h_Ap;
    // Apoint* d_Ap;
    UINT64* h_num;
    UINT64* d_num;
    // Jpoint* d_result;
    
    h_p1 = (Jpoint*)malloc(N_POINT*sizeof(Jpoint));
    h_p2 = (Jpoint*)malloc(N_POINT*sizeof(Jpoint));
    // h_Ap = (Apoint*)malloc(N_POINT*sizeof(Apoint));
    h_num = (UINT64*)malloc(4*N_BIGNUM*sizeof(UINT64));

    init_Jpoint(h_p1);
    init_Jpoint(h_p2);
    // init_Apoint(h_Ap);
    init_random_big(h_num);
    // h_mybig_print(h_num);
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p1,N_POINT*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_p2,N_POINT*sizeof(Jpoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_Ap,N_POINT*sizeof(Apoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_result,N_POINT*sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_num,sizeof(UINT64)*4*N_BIGNUM));


//===========warm up
    // init_random_big(h_num);
    // h_mybig_print(h_num);
    // init_Jpoint(h_p1);
    // init_Jpoint(h_p2);
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_p1,N_POINT*sizeof(Jpoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_p2,N_POINT*sizeof(Jpoint)));
    // CUDA_SAFE_CALL(hipMalloc((void**)&d_num,sizeof(UINT64)*4*N_BIGNUM));

    CUDA_SAFE_CALL(hipMemcpy(d_num,h_num,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p1,h_p1,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2,h_p2,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    


    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    test_point_double<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);

    CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    print_jpoint_arr(h_p1,1);

//==================warm end









    CUDA_SAFE_CALL(hipMemcpy(d_num,h_num,sizeof(UINT64)*4*N_BIGNUM,hipMemcpyHostToDevice));
    
    CUDA_SAFE_CALL(hipMemcpy(d_p1,h_p1,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2,h_p2,N_POINT*sizeof(Jpoint),hipMemcpyHostToDevice));
    point_to_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();

    gettimeofday(&s1,NULL);
    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    test_point_double<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    hipDeviceSynchronize();
    gettimeofday(&e1,NULL);
    CUDA_CHECK_ERROR();
    point_from_monjj<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_p2);
    CUDA_CHECK_ERROR();
    CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    time_use=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("time_use is %ld us\n",time_use);
    print_jpoint_arr(h_p1,1);
    // print_jpoint_arr(h_p2,1);
    // print_jpoint_arr(h_p2,N_POINT);



    




    free(h_p1);
    free(h_p2);
    free(h_num);
    CUDA_SAFE_CALL(hipFree(d_p1));
    CUDA_SAFE_CALL(hipFree(d_p2));
    CUDA_SAFE_CALL(hipFree(d_num));

}
#include "hip/hip_runtime.h"
#include <stdio.h>
typedef unsigned long long UINT64; //����64λ������
typedef long long INT64;

// ����㹹��
typedef struct Affine_point{
	UINT64 x[8];
	UINT64 y[8];
}Apoint;

// ��Ӱ�㹹��
typedef struct Jacobi_point{
	UINT64 x[8];
	UINT64 y[8];
	UINT64 z[8];
}Jpoint;


// ��tesla C2050��Ŀǰ����������Ч����ߣ������޸ġ�
#define PARAL 64
#define BLOCKNUM (14*8)
#define BLOCKSIZE 32
#define THREADNUM (BLOCKNUM*BLOCKSIZE)


// ����__global__���͵ı��������16���ر���
#define d_BIN_WINDOW_16 16 //16���ر�
#define d_ROWS_16 32
#define d_COLS_16 (1L<<d_BIN_WINDOW_16)


#define HANDLE_ERROR( err ) { if (err != hipSuccess) { \
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), __FILE__, __LINE__ );\
	  exit( EXIT_FAILURE ); }  \
}

//�汾Ϊ512�������߹�ģ���������޸�Ϊ256���ذ汾
//�ð汾����������������Ϊ��������
//h_ONE��host�õ���Ԫ��1
//dc_ONE��gpu���õ���Ԫ��1
//h_mon_ONE��dc_mon_ONE��ȣ��ֱ���host��gpu�����õ��ɸ��������ϵ�1
//h_p��dc_p��512��������
//Pa0��Pa7�ǽ������ֿ�����Ϊ�궨��

const UINT64 h_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
const UINT64 h_mon_ONE[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000L,0x0000000000000000L,0x0000000000000001L,0x0000000100000000L,0x0000000000000000L};
const UINT64 h_p[8]={0xffffffffffffffffL,0xfffffffeffffffffL,0xffffffffffffffffL,0xfffffffeffffffffL,0xffffffffffffffffL,0xfffffffffffffffeL,0xfffffffeffffffffL,0xffffffffffffffffL};

__constant__ UINT64 dc_ONE[8]={0x0000000000000001L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L,0x0000000000000000L};
__constant__ UINT64 dc_mon_ONE[8]={0x0000000000000001L,0x0000000100000000L,0x0000000000000000L,0x0000000100000000L,0x0000000000000000L,0x0000000000000001L,0x0000000100000000L,0x0000000000000000L};
__constant__ UINT64 dc_p[8]={0xffffffffffffffffL,0xfffffffeffffffffL,0xffffffffffffffffL,0xfffffffeffffffffL,0xffffffffffffffffL,0xfffffffffffffffeL,0xfffffffeffffffffL,0xffffffffffffffffL};

#define Pa0 0xffffffffffffffffLL //-1
#define Pa1 0xfffffffeffffffffLL 
#define Pa2 0xffffffffffffffffLL //-1
#define Pa3 0xfffffffeffffffffLL 
#define Pa4 0xffffffffffffffffLL //-1
#define Pa5 0xfffffffffffffffeLL //-2
#define Pa6 0xfffffffeffffffffLL 
#define Pa7 0xffffffffffffffffLL //-1


/////////////////GPU���������㺯����ʼ////////////////////////////////////////////////////

#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];(a)[4]=(b)[4];(a)[5]=(b)[5];(a)[6]=(b)[6];(a)[7]=(b)[7];}

//ʵ����������ģ�Ӻ���
__device__ __host__ void dh_mybig_modadd_64(const UINT64 *x, const UINT64 *y, UINT64 *z)//����z=x+y, x=x+y, ����ʵ��y=x+y��
{
	int i;
	UINT64 f,g;
		
	#ifdef __CUDA_ARCH__
		const UINT64 *Pa=dc_p;
	#else
		const UINT64 *Pa=h_p;
	#endif	
	
	z[0] = x[0] + y[0]; f = z[0] < y[0];
	z[1] = x[1] + f; g = z[1] < f; z[1] += y[1]; g += z[1] < y[1];
	z[2] = x[2] + g; f = z[2] < g; z[2] += y[2]; f += z[2] < y[2];
	z[3] = x[3] + f; g = z[3] < f; z[3] += y[3]; g += z[3] < y[3];
	
	z[4] = x[4] + g; f = z[4] < g; z[4] += y[4]; f += z[4] < y[4];
	z[5] = x[5] + f; g = z[5] < f; z[5] += y[5]; g += z[5] < y[5];
	z[6] = x[6] + g; f = z[6] < g; z[6] += y[6]; f += z[6] < y[6];
	z[7] = x[7] + f; g = z[7] < f; z[7] += y[7]; g += z[7] < y[7];
		
	if(g==0)
	{
		for(i=7;i>=0;i--)
		{
			if(z[i]!=Pa[i])
			{
				g=(z[i]>Pa[i]);
				break;
			}
			else if(i==0)//ȫ���,��t=P, ��ʵ��ʱ��ֵΪ0����
			{
				g=1;
			}
		}
	}
	
	if(g)//x+y���ܵ���ģ��
	{
		f = z[0] < Pa0; z[0] -= Pa0;
		g = z[1] < f; z[1] -= f; g += z[1] < Pa1; z[1] -= Pa1;
		f = z[2] < g; z[2] -= g; f += z[2] < Pa2; z[2] -= Pa2;		                                                  
		g = z[3] < f; z[3] -= f; g += z[3] < Pa3; z[3] -= Pa3;		
		f = z[4] < g; z[4] -= g; f += z[4] < Pa4; z[4] -= Pa4;		                                                  
		g = z[5] < f; z[5] -= f; g += z[5] < Pa5; z[5] -= Pa5;
		f = z[6] < g; z[6] -= g; f += z[6] < Pa6; z[6] -= Pa6;				
		z[7] -= f; z[7] -= Pa7;
	}
}

//ʵ����������ģ������
__device__ __host__ void dh_mybig_modsub_64(const UINT64 *x, const UINT64 *y, UINT64 *z)//����z=x-y, x=x-y, ����ʵ��y=x-y��
{
	UINT64 f,g;
	//UINT64 z0,z1,z2,z3,z4,z5,z6,z7;
	f=(x[0]<y[0]); z[0]=x[0]-y[0];
	g=(x[1]<f); z[1]=x[1]-f; g+=(z[1]<y[1]); z[1]-=y[1];
	f=(x[2]<g); z[2]=x[2]-g; f+=(z[2]<y[2]); z[2]-=y[2];
	g=(x[3]<f); z[3]=x[3]-f; g+=(z[3]<y[3]); z[3]-=y[3];
	
	f=(x[4]<g); z[4]=x[4]-g; f+=(z[4]<y[4]); z[4]-=y[4];
	g=(x[5]<f); z[5]=x[5]-f; g+=(z[5]<y[5]); z[5]-=y[5];
	f=(x[6]<g); z[6]=x[6]-g; f+=(z[6]<y[6]); z[6]-=y[6];
	g=(x[7]<f); z[7]=x[7]-f; g+=(z[7]<y[7]); z[7]-=y[7];
	//��λΪg;

	if(g)//ֻ�жϽ�λ���ٶ�Ӧ�ÿ�һ��
	{
		z[0]+=Pa0; f=(z[0]<Pa0);
		z[1]+=f; f=(z[1]<f); z[1]+=Pa1; f+=(z[1]<Pa1);
		z[2]+=f; f=(z[2]<f); z[2]+=Pa2; f+=(z[2]<Pa2);
		z[3]+=f; f=(z[3]<f); z[3]+=Pa3; f+=(z[3]<Pa3);
		z[4]+=f; f=(z[4]<f); z[4]+=Pa4; f+=(z[4]<Pa4);
		z[5]+=f; f=(z[5]<f); z[5]+=Pa5; f+=(z[5]<Pa5);
		z[6]+=f; f=(z[6]<f); z[6]+=Pa6; f+=(z[6]<Pa6);
		z[7]+=f; z[7]+=Pa7;		
	}	
}

//����������Գ˷���ȡ64λ�˷���λ
static inline UINT64 h_Hi64(const UINT64 x, const UINT64 y)
{
   UINT64 z;   
   __asm__ __volatile__ (
	  "movq  %0,%%rax\n"   
	  "mulq %1\n"
	  "movq %%rdx,%2\n"
	  : 
	  : "m"(x),"m"(y),"m"(z)
	  : "rax","rdx","memory"
	  );      	
	return z;
}

//ʵ���ɸ������˷�C=a*b
/////////////////////////////////////////
//Montgomeryģ��
//C=A*B*2^-512 mod P
//����CIOS�㷨
//////////////////////////////////////////
__device__ __host__  void dh_mybig_monmult_64(const UINT64 *Aa, const UINT64 *Ba, UINT64 *Ca)//pass, c=a*b
{
	UINT64 t[8+2]={0};//8��64��������,�޶�512
	
	//minv*P[0] mod 2^wordlen = -1. ��Ϊ�ó����ֳ���64���أ�P[0]=0xffffffffffffffff=-1������minv=1, minv*P[0]=-1 mod 2^64=-1
	
	//���������P������Ҫ���¼������á����ֵ����Ǵ��룬���߶���Ϊȫ�ֱ���
	UINT64 minv=1;//minvֵ��Ҫ��������P�����λP[0]�˻�ģ2^64=-1(��ΪP[0]��64�����֣������32�����֣����ҵ�ģ2^32=-1�Ǹ����� ���õ�����������������ֵ�ģ2^wordlen����ĸ�ֵ��

	UINT64 m;	
	UINT64 c,s,cin;
	int i,j;
	
	#ifdef __CUDA_ARCH__	
		UINT64 *Pa=dc_p;	
		#define h_Hi64 __umul64hi
	#else
		const UINT64 *Pa=h_p;
	#endif	
	
	for(i=0;i<8;i++)
	{
		c=0;
		m=Ba[i];
		
		s=m*Aa[0]; c+=t[0];	cin=(c<t[0]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[0])+cin; 	t[0]=s;
		s=m*Aa[1]; c+=t[1];	cin=(c<t[1]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[1])+cin; 	t[1]=s;
		s=m*Aa[2]; c+=t[2];	cin=(c<t[2]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[2])+cin; 	t[2]=s;
		s=m*Aa[3]; c+=t[3];	cin=(c<t[3]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[3])+cin; 	t[3]=s;
		s=m*Aa[4]; c+=t[4];	cin=(c<t[4]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[4])+cin; 	t[4]=s;
		s=m*Aa[5]; c+=t[5];	cin=(c<t[5]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[5])+cin; 	t[5]=s;
		s=m*Aa[6]; c+=t[6];	cin=(c<t[6]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[6])+cin; 	t[6]=s;
		s=m*Aa[7]; c+=t[7];	cin=(c<t[7]);		s+=c;		cin+=(s<c);	c=h_Hi64(m,Aa[7])+cin; 	t[7]=s;		
								
		s=t[8]+c;
		c=(s<c);
		t[8]=s;
		t[8+1]=c;
				
		m=minv*t[0];//����p��minv=1���ȼ���m=t[0]
		c=h_Hi64(m,Pa0);
		s=m*Pa0+t[0];   //��Ϊ���������������Pa0=-1, minv=1������s=0;		
		c+=(s<t[0]);
		
		s=m*Pa1; c+=t[1]; cin=(c<t[1]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa1)+cin; t[0]=s;		
		s=m*Pa2; c+=t[2]; cin=(c<t[2]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa2)+cin; t[1]=s;
		s=m*Pa3; c+=t[3]; cin=(c<t[3]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa3)+cin; t[2]=s;		
	    s=m*Pa4; c+=t[4]; cin=(c<t[4]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa4)+cin; t[3]=s;		
		s=m*Pa5; c+=t[5]; cin=(c<t[5]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa5)+cin; t[4]=s;
		s=m*Pa6; c+=t[6]; cin=(c<t[6]); s+=c;	cin+=(s<c); c=h_Hi64(m,Pa6)+cin; t[5]=s;		
		s=m*Pa7; c+=t[7]; cin=(c<t[7]); s+=c;   cin+=(s<c); c=h_Hi64(m,Pa7)+cin; t[6]=s;
		
		s=t[8]+c;
		c=(s<c);
		t[8-1]=s;
		t[8]=t[8+1]+c;
	}
	
	j=(t[8]!=0);
	if(j==0)
	{
		for(i=8-1;i>=0;i--)
		{
			if(t[i]!=Pa[i])
			{				
				j=(t[i]>Pa[i]);
				break;
			}
			else if(i==0)//ȫ���,��t=P, ��ʵ��ʱ��ֵΪ0����
			{
				j=1;
			}
		}
	}
	
	//����	
	if(j)
	{
		cin=1;	
		for(i=0;i<8;i++)
		{
			m=cin+(~Pa[i]);
			cin=(m<cin);
			m+=t[i];
			cin+=(m<t[i]);
			Ca[i]=m;
		}
	}
	else
	{
		for(i=0;i<8;i++)
		{
			Ca[i]=t[i];
		}
	}
	
	#ifdef __CUDA_ARCH__	
		#undef h_Hi64 
	#endif	
}



//////////////////////////////////////test inv//////////////////////////////////////
//A!=0 return 0
//A=0 return 1
__device__ __host__ int dh_mybig_iszero_64(const UINT64 *A)
{
	int i;
	for(i=0;i<8;i++)
	{
		if(A[i]!=0)	return 0;
	}
	return 1;	
}

//�Ƚϴ�С
//return 1	A>B
//return 0	A=B
//return -1 A<B
__device__ __host__ int dh_mybig_compare_64(const UINT64 *A, const UINT64 *B)
{
	int i;
	int flag=0;

	for(i=7;i>=0;i--)
	{
		if(A[i]>B[i])
		{
			flag=1;
			break;
		}
		else if(A[i]<B[i])
		{
			flag=-1;
			break;
		}
	}
	return (flag);
}

//C=A/2
__device__ __host__ void dh_mybig_half_64(const UINT64 *A, UINT64 *C)
{
	int i;
	UINT64 c,c1;

	c=0;
	for(i=8-1;i>=0;i--)
	{
		c1=A[i]&0x1;
		C[i]=c<<(63)|A[i]>>1;
		c=c1;
	}
}

//C=2A
__device__ __host__ UINT64 dh_mybig_double_64(const UINT64 *A, UINT64 *C)//pass
{
	int i;
	UINT64 c,c1;

	c=0;
	for(i=0;i<8;i++)
	{
		c1=A[i]>>63;
		C[i]=(C[i]<<1)|c;
		c=c1;
	}
	return c;
}

//C=A-B
__device__ __host__ void dh_mybig_sub_64(const UINT64 *A, const UINT64 *B, UINT64 *C)
{
	int i;
	UINT64 c,l,h;

	c=1;
	for(i=0;i<8;i++)
	{
		l=(~B[i])+c;
		h=(l<c);
		l+=A[i];
		c=h+(l<A[i]);
		C[i]=l;	
	}
}

//C=A+B
__device__ __host__ UINT64 dh_mybig_add_64(const UINT64 *A, const UINT64 *B, UINT64 *C)
{
	int i;
	UINT64 c,l,h;
	
	c=0;
	for(i=0;i<8;i++)
	{
		l=A[i]+c;
		h=(l<c);
		l+=B[i];
		c=h+(l<B[i]);
		C[i]=l;
	}
	return c;
}

/////////////////////////////////////////
//��2ģ
//����:		A P
//���:		C
//C=2*A mod P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moddouble_64(const UINT64 *A, const UINT64 *P, UINT64 *C)
{
	int i,sub_en=0;
	UINT64 cin,c,temp64;

	//��λ
	cin=(A[0]>>63)&0x1;//ȫ������һλ,�൱�ڳ�2
	C[0]=A[0]<<1;
	for(i=1;i<8;i++)
	{
		c=(A[i]>>63)&0x1;
		C[i]=(A[i]<<1)|cin;
		cin=c;
	}

	//�Ƚϴ�С
	if(cin==1)//�����һ����λΪ1,������ô���ֽ��Ѿ��治��2*A�����2*A�϶���p��
	{
		sub_en=1;
	}
	else
	{
		for(i=7;i>=0;i--)
		{
			if(C[i]!=P[i])
			{
				if(C[i]>P[i]) sub_en=1;
				break;
			}
			else if(i==0) sub_en=1;//����ȫ���
		}
	}
	
	//����
	if(sub_en)//����Ӧ��дΪif(sub_en!=0)���ɱ�������
	{
		cin=1;
		for(i=0;i<8;i++)
		{			
			temp64=(~P[i])+cin;
			c=(temp64<cin);
			temp64+=C[i];
			cin=c+(temp64<C[i]);
			C[i]=temp64;
		}
	}
}


__device__ __host__  int dh_mybig_iszero(const UINT64 * A)
{
	if( A[0]|A[1]|A[2]|A[3]|A[4]|A[5]|A[6]|A[7])
		return 0;
	else return 1;
}

void h_mybig_print(const UINT64 *a)
{
	int i;
	unsigned char *t=(unsigned char*) a;
	for(i=64-1;i>=0;i--) if(*(t+i)) break;//��0������ȥ
	if(i<0) printf("0");
	else
	{
		printf("%x",*(t+i)&0xff);//��һ��0����ӡ
		for(i=i-1;i>=0;i--)printf("%02x",*(t+i)&0xff);
	}	
}
__device__ void d_mybig_print(const UINT64 *a)
{
	int i;
	unsigned char *t=(unsigned char*) a;
	for(i=32-1;i>=0;i--) if(*(t+i)) break;//��0������ȥ
	if(i<0) printf("0");
	else
	{
		printf("%x",*(t+i)&0xff);//��һ��0����ӡ
		for(i=i-1;i>=0;i--)printf("%02x",*(t+i)&0xff);
	}
	printf("\n");	
}
/////////////////////////////////////////
//������: ����ģ��C=A^-512 * 2^512 mod P
//����:		A C l(lΪ���ı��س���512)
//���:		C (C=A^-1 * 2^512 mod P)
//˵��: A<P
//////////////////////////////////////////
__device__ __host__ void dh_mybig_moninv(const UINT64 *A,UINT64 *C)//test
{
	int i,k;
	UINT64 U[8],V[8],R[8],S[8];
	int z,cp,cs,cr,sh;
	/*
	#ifdef __CUDA_ARCH__	
		UINT64 *P=dc_p;	
	#else
		const UINT64 *P=h_p;
	#endif	
	//*/

	/************************�㷨********************************
	Preferences: E. Savas, C.K.Koc, The Montgomery Modular
	Inverse-Revisited, IEEE TRANSACTIONS ON COMPUTERS,
	VOL. 49, NO.7, JULY 2000

	--Phase I
	U=P V=A R=0 S=1
	k=0
	while(V>0)
		if U is even then U=U/2 S=2S
		else if V is even then V=V/2 R=2R
		else if U>V then U=(U-V)/2 R=R+S S=2S
		else if V>=U then V=(V-U)/2 S=S+R R=2R
		k=k+1
	if R>=P then R=R-P
	R=P-R

	--Phase II
  R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P 
  = A^-1 * 2^l mod P
	
	return R
	*************************************************************/

	////0. initial
	//U=P V=A R=0 S=1
	U[0]=0xffffffffffffffffL; //U=P��P��ֵ������Ҫȥ����
	U[1]=0xfffffffeffffffffL;
	U[2]=0xffffffffffffffffL;
	U[3]=0xfffffffeffffffffL;
	U[4]=0xffffffffffffffffL;
	U[5]=0xfffffffffffffffeL;
	U[6]=0xfffffffeffffffffL;
	U[7]=0xffffffffffffffffL;


	for(i=0;i<8;i++) V[i]=A[i];
	for(i=0;i<8;i++) R[i]=0;
	for(i=1;i<8;i++) S[i]=0;
	S[0]=0x1UL; 

	////1. phase I
	//get R=A^-1*2^k mod P
	k=0;
	z=dh_mybig_iszero_64(V);
	while(z==0)
	{	
		//printf("here k=%d  ",k);//getchar();
		cp=dh_mybig_compare_64(U,V);
		if((U[0]&0x1)==0)
		{
			dh_mybig_half_64(U,U);
			cs=dh_mybig_double_64(S,S);
		}
		else if((V[0]&0x1)==0)
		{			
			dh_mybig_half_64(V,V);
			cr=dh_mybig_double_64(R,R);
		}
		else if(cp==1)
		{
			dh_mybig_sub_64(U,V,U);
			dh_mybig_half_64(U,U);
			cr=dh_mybig_add_64(R,S,R);
			cs=dh_mybig_double_64(S,S);
		}
		else if(cp<1) //if(cp==0 || cp==-1)
		{
			dh_mybig_sub_64(V,U,V);
			dh_mybig_half_64(V,V);
			cs=dh_mybig_add_64(S,R,S);
			cr=dh_mybig_double_64(R,R);
		}
		k++;
		z=dh_mybig_iszero_64(V);
		//if(cr==1) printf("\nr\n");
		//if(cs==1) printf("\ns\n");		
	}
	
	U[0]=0xffffffffffffffffL;
	U[1]=0xfffffffeffffffffL;
	U[2]=0xffffffffffffffffL;
	U[3]=0xfffffffeffffffffL;
	U[4]=0xffffffffffffffffL;
	U[5]=0xfffffffffffffffeL;
	U[6]=0xfffffffeffffffffL;
	U[7]=0xffffffffffffffffL;

	cp=dh_mybig_compare_64(R,U);
	if(cp==1 || cp==0 || cr==1)
	{
		dh_mybig_sub_64(R,U,R);
	}
	dh_mybig_sub_64(U,R,R);
	////////////////////
	//the result of phase I is R
	//R=A^-1 * 2^k mod P

    
////2. phaseII
//get R=A^-1 * 2^l mod P
 // R=Mont(R*mR)=A^-1 * 2^k *  2^(2l-k) * 2^(-l) mod P = A^-1 * 2^l mod P
 	 if(k>512)
	 {	
	    for(i=0;i<8;i++) V[i]=0;
	    V[(int)((1024-k)/64)]=(((UINT64)1)<<((int)((1024-k)%64)));
	    dh_mybig_monmult_64(R,V,C);
	 }	
	 else if(k==512) 
	 {
			V[0]=0x0000000000000001L;
			V[1]=0x0000000100000000L;
			V[2]=0x0000000000000000L;
			V[3]=0x0000000100000000L;
			V[4]=0x0000000000000000L;
			V[5]=0x0000000000000001L;
			V[6]=0x0000000100000000L;
			V[7]=0x0000000000000000L;
			dh_mybig_monmult_64(R,V,C);		
		}
		else if(k<512) 
		{
		  //printf("2\n");
		  for(i=1;i<=(512-k);i++)
			{		
				dh_mybig_moddouble_64(R,U,R);			
			}
		  for(i=0;i<8;i++) C[i]=R[i];
		}

}

//////////////////////////////////////test inv end//////////////////////////////////////

/////////////////GPU���������㺯������////////////////////////////////////////////////////


/////////////////GPU��Ӻͱ��㺯����ʼ��û��дȫ�ĵ�Ӻ������������������������㣬�Լ����ݵ�ӹ�ʽ����///////////////////////////////////////
__device__ __host__  void dh_setzero_J(Jpoint *pt)
{
	int i;
	for(i=0;i<8;i++) pt->z[i]=0UL;
}

__device__ __host__  void dh_setzero_A(Apoint *pt)
{
	int i;
	for(i=0;i<8;i++) pt->x[i]=0UL;	
}

__device__ __host__  int dh_iszero_J(const Jpoint *pt)
{
	if((pt->z[0]|pt->z[1]|pt->z[2]|pt->z[3]|pt->z[4]|pt->z[5]|pt->z[6]|pt->z[7]) == 0UL)	return 1;
	return 0;	
}


__device__ __host__  int dh_iszero_A(const Apoint *pt)
{	
	if((pt->x[0]|pt->x[1]|pt->x[2]|pt->x[3]|pt->x[4]|pt->x[5]|pt->x[6]|pt->x[7]) == 0UL)	return 1;
	return 0;
}


//����Ӱ������ת��Ϊ��������
__device__ __host__  void dh_normlize_J(Jpoint *pt)
{	
	UINT64 tmp[8],invtmp[8];
	
	if(dh_iszero_J(pt))	return;
	
	tmp[0]=0x1L; tmp[1]=0x0L; tmp[2]=0x0L; tmp[3]=0x0L; tmp[4]=0x0L; tmp[5]=0x0L; tmp[6]=0x0L; tmp[7]=0x0L;
	
	
	dh_mybig_monmult_64(pt->z,tmp, invtmp);
	dh_mybig_moninv(invtmp, invtmp);
	//invMod(invtmp,pt->z,mod);//invtmp=1/z
	
	//squareMod(tmp,invtmp,mod);//tmp=1/zz
	dh_mybig_monmult_64(invtmp,invtmp, tmp);
	
	//productMod(pt->x,pt->x,tmp,mod);//x/zz
	dh_mybig_monmult_64(pt->x,tmp, pt->x);		
	
	//productMod(tmp,tmp,invtmp,mod);//1/zzz
	dh_mybig_monmult_64(tmp,invtmp, tmp);
	
	//productMod(pt->y,pt->y,tmp,mod);//y/zzz
	dh_mybig_monmult_64(pt->y,tmp, pt->y);
	
	//z=mon_one
	pt->z[0]=0x0000000000000001L;
	pt->z[1]=0x0000000100000000L;
	pt->z[2]=0x0000000000000000L;
	pt->z[3]=0x0000000100000000L;
	pt->z[4]=0x0000000000000000L;
	pt->z[5]=0x0000000000000001L;
	pt->z[6]=0x0000000100000000L;
	pt->z[7]=0x0000000000000000L;	
	
}

__device__ void d_mon2normal_J(Jpoint *pt)
{
	dh_mybig_monmult_64(pt->x, dc_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, dc_ONE, pt->y);
	dh_mybig_monmult_64(pt->z, dc_ONE, pt->z);	
}
__device__ __host__  void dh_mon2normal_A(Apoint *pt)
{
	#ifdef __CUDA_ARCH__
	dh_mybig_monmult_64(pt->x, dc_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, dc_ONE, pt->y);
	#else
	dh_mybig_monmult_64(pt->x, h_ONE, pt->x);
	dh_mybig_monmult_64(pt->y, h_ONE, pt->y);
	#endif
}

void h_print_pointA(const Apoint *pt)
{
	if(dh_iszero_A(pt))
	{ 
		printf("(Infinity)\n");
		return;
	}	
	h_mybig_print(pt->x);printf("\n");
	h_mybig_print(pt->y);printf("\n");
	
}

void h_print_pointJ(const Jpoint *pt)
{
	/*
	if(dh_iszero_J(pt))
	{ 
		printf("(Infinity)\n");
		return;
	}*/
	h_mybig_print(pt->x);printf("\n");
	h_mybig_print(pt->y);printf("\n");
	h_mybig_print(pt->z);printf("\n");
}


//��Ӱ��pt1�ӵȷ����pt2, pt1+=pt2
__device__ __host__ void dh_ellipticSumEqual_AJ(Jpoint *pt1, Apoint* pt2)//pt1,pt2���뱣֤������Զ�㣬�ں��������ж�
{
	UINT64 u1[8],u2[8];
	if(dh_iszero_A(pt2))	return;	
	if(dh_iszero_J(pt1))
	{
		dh_mybig_copy(pt1->x, pt2->x);
		dh_mybig_copy(pt1->y, pt2->y);
		//Z��ֵΪmon_ONE�����ֵ��Ҫ����p��������
		pt1->z[0]=0x0000000000000001L;		pt1->z[1]=0x0000000100000000L;		pt1->z[2]=0x0000000000000000L;		pt1->z[3]=0x0000000100000000L;
		pt1->z[4]=0x0000000000000000L;		pt1->z[5]=0x0000000000000001L;		pt1->z[6]=0x0000000100000000L;		pt1->z[7]=0x0000000000000000L;				
		return;
	}	
	//���ڳ������жϣ���ֹ�ӷ������������Զ��
	
	
	//3.����u1=(pt1->z)^2.
	dh_mybig_monmult_64(pt1->z, pt1->z, u1);
	
	//4.����u2=(pt1->z)*u1.
	dh_mybig_monmult_64(pt1->z, u1, u2);
	
	//5.����u1=(pt2->x)*u1.
	dh_mybig_monmult_64(pt2->x, u1, u1);
	
	//6.����u2=(pt2->y)*u2.
	dh_mybig_monmult_64(pt2->y, u2, u2);
	
	//7.����u1=u1-pt1->x.		
	dh_mybig_modsub_64(u1, pt1->x,u1);
	
	//8.����u2=u2-pt1->y.	
	dh_mybig_modsub_64(u2,pt1->y,u2);
	
	///*	
	//9.�жϵȵ�,��Ҫ���ö��������.
	if(dh_mybig_iszero(u1))
	{
		if(dh_mybig_iszero(u2))
		{
			//ppoint_double(pt1,pt3);//y����Ҳ��ͬ�����ض�����
			printf("here! use ppoint double!\n");
			return;
		}
		else//��������ӣ���������Զ��
		{
			dh_setzero_J(pt1);
			return ;
		}
	}
	//*/
	//10.pt1->z=pt1->z*u1.
	dh_mybig_monmult_64(pt1->z, u1, pt1->z);
	
	//11.����pt2->x=u1^2.
	dh_mybig_monmult_64(u1, u1, pt2->x);
	
	//12.����pt2->y=pt2->x*u1.
	dh_mybig_monmult_64(u1, pt2->x, pt2->y);
	
	//13.����pt2->x=pt1->x*pt2->x.
	dh_mybig_monmult_64(pt1->x, pt2->x, pt2->x);
	
	//14.����u1=2*pt2->x.
	dh_mybig_modadd_64(pt2->x,pt2->x,u1);
	
	//15.x1=u2^2.
	dh_mybig_monmult_64(u2, u2, pt1->x);
	
	//16.x1=pt2->x
	dh_mybig_modsub_64(pt1->x,u1,pt1->x);
	
	//17.x1=x1-pt2->y
	dh_mybig_modsub_64(pt1->x,pt2->y,pt1->x);
	
	//18.����pt2->x=pt2->x-x1.
	dh_mybig_modsub_64(pt2->x,pt1->x,pt2->x);
	
	//19.pt2->x=pt2->x*u2
	dh_mybig_monmult_64(pt2->x, u2, pt2->x);
	
	//20.pt2->y=pt2->y*y1
	dh_mybig_monmult_64(pt2->y, pt1->y, pt2->y);
	
	//21.y1=pt2->x-pt2->y
	dh_mybig_modsub_64(pt2->x,pt2->y,pt1->y);
}

//��������
__device__ void d_multi_inverse(UINT64 *x)
{
	int i,j;
	UINT64 invlj[PARAL*8];//�Ĵ���ÿ��С�˿��Է�4KB�ֽڣ�4096=64*256,������256��512������
	UINT64 lmd[8];
	
	dh_mybig_copy(invlj,x);//	for(i=0;i<8;i++) invlj[0][i]=x[0][i];
		
	for(i=1;i<PARAL;i++)
	{			    					
		dh_mybig_monmult_64(invlj+(i-1)*8,x+i*8,invlj+i*8);
	}
		
	dh_mybig_monmult_64(invlj+(PARAL-1)*8,dc_ONE,lmd);//z=Z mod P //��ȥ���ɸ�����
	dh_mybig_moninv(lmd,lmd); //�����ʱ���ִ���2^n��
		
	for(i=PARAL-1;i>0;i--)
	{
		dh_mybig_monmult_64(invlj+(i-1)*8,lmd,invlj+i*8);
		dh_mybig_monmult_64(x+i*8,lmd,lmd);
		dh_mybig_copy(x+i*8,invlj+i*8);
		//for(j=0;j<8;j++) x[i*8+j]=invlj[i*8+j];	
	}	
	dh_mybig_copy(x,lmd);
	//for(j=0;j<8;j++) x[j]=lmd[j];	
}

//�������滯��Ӱ��
__global__ void d_multi_normlize_J(Jpoint *A, int n)
{	
	//����n����ΪPARAL��������
	
	int i,j,k;
	UINT64 tmp[8];
	int mytid=threadIdx.x+blockDim.x*blockIdx.x;
	int threadnum=blockDim.x*gridDim.x;
	UINT64 z[8*PARAL];//�˴�z����α��������
			
	for(i=mytid*PARAL;i<n;i+=threadnum*PARAL) //n����PARAL��������һ�Σ�ÿ��С����һ�Σ�ѭ������ȥ
	{
		for(j=0;j<PARAL;j++)
		{
			for(k=0;k<8;k++)
				z[j*8+k]=A[i+j].z[k];
		}
				
		d_multi_inverse(z);
		
		for(j=0;j<PARAL;j++)
		{
			dh_mybig_monmult_64(z+j*8,z+j*8, tmp);
			dh_mybig_monmult_64(A[i+j].x,tmp, A[i+j].x);		
			dh_mybig_monmult_64(tmp,z+j*8, tmp);
			dh_mybig_monmult_64(A[i+j].y,tmp, A[i+j].y);

			//z��ʼ��Ϊmon_ONE�����mon_ONE���ֳ��й�
			A[i+j].z[0]=0x0000000000000001L;	A[i+j].z[1]=0x0000000100000000L;	A[i+j].z[2]=0x0000000000000000L;	A[i+j].z[3]=0x0000000100000000L;
			A[i+j].z[4]=0x0000000000000000L;	A[i+j].z[5]=0x0000000000000001L;	A[i+j].z[6]=0x0000000100000000L;	A[i+j].z[7]=0x0000000000000000L;	
			
			d_mon2normal_J(A+i+j);
		}
	}
	
}

/////////////////GPU��Ӻͱ��㺯������////////////////////////////////////////////////////

__global__ void d_get_para(UINT64 *para)
{
	int i;
	UINT64 *pt64;
	
	if(threadIdx.x==0 && blockIdx.x==blockDim.x-1)
	{
		pt64=para+  0; for(i=0;i<8;i++) pt64[i]=dc_p[i];
		pt64=para+ 56; for(i=0;i<8;i++) pt64[i]=dc_ONE[i];
		pt64=para+ 64; for(i=0;i<8;i++) pt64[i]=dc_mon_ONE[i];						
	}	
}

void h_print_para()
{
	int groups=11;
	UINT64 *testdata=(UINT64 *)malloc(groups*8*8);	
	UINT64 *d_testdata;
	
	HANDLE_ERROR( hipMalloc((void**)&d_testdata, groups*8*8) );
	
	d_get_para<<<BLOCKNUM,BLOCKSIZE>>>(d_testdata);
	
	HANDLE_ERROR( hipMemcpy( testdata, d_testdata,	64*groups, hipMemcpyDeviceToHost));
	printf("\n__const__ UINT64 testdc_p[8]=");h_mybig_print(testdata);
	printf("\n__const__ UINT64 testdc_ONE[8]=");h_mybig_print(testdata+56);
	printf("\n__const__ UINT64 testdc_mon_ONE[8]=");h_mybig_print(testdata+64);
	
	free(testdata);
	HANDLE_ERROR(hipFree(d_testdata));
}



int h_get_gpu_info()
{
	  hipDeviceProp_t  prop;

    int count;
    HANDLE_ERROR( hipGetDeviceCount( &count ) );
    
    for (int i=0; i< count; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
    }

	return 1;
}

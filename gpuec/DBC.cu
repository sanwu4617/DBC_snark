#include "hip/hip_runtime.h"
//
// Created by occul on 2021/10/15.
//

#include "DBC.h"
#include "constants.h"
#include "variables.h"

#define DBC_COEF 10      //次优DBC参数设置，本参数越小计算DBC越快，但DBC质量越好。不过参数过小可能会引起bug，建议不要小于10
int bBound[MAX_2] = { 0 };
//extern int MAX_T;
// int DBC_store[DBC_COEF][MAX_2][3]={0};   //第1维：不同的DBC，第2维：一个DBC的不同项，第3维：符号，2的次数，3的次数
int DBC_store[1][MAX_2][3]={0}; 
int DBC_len[MAX_2]={0};

// __host__ __device__ Chain::Chain() {
//     dbl = 0;
//     tpl = 0;
//     minus = 0;
// }

// __host__ __device__ void Chain::setdata(byte dbl, byte tpl, bool minus)
// {
// 	this->dbl = dbl;
// 	this->tpl = tpl;
// 	this->minus = minus;
// }
// __host__ __device__ int min_index(int a, int b, int c)
// {
// 	if (a < b)
// 	{
// 		if (a < c)
// 			return 0;
// 		return 2;
// 	}
// 	if (b < c)
// 		return 1;
// 	return 2;
// }
// __host__ __device__ int min_index(int a, int b)
// {
// 	if (a < b)
// 	{
// 		return 0;
// 	}
// 	return 1;
// }



// __host__ DBC::DBC() // runs in host.
// {
//     length = 0;
//     memset(w_min0, 0, sizeof(w_min0));
//     memset(w_rec, 0, sizeof(w_rec));
//     w_rec[1][1][1] = 1024;   //��Ϊ�ܴ������ʾNULL��ʹ����minL����ʱ���ᱻȡ��
//     for (int i = 0; i < MAX_2; i++)
//     {
//         w_rec[i][0][0] = 1024;
//         w_rec[i][0][1] = 1024;
//     }
//     for (int t = 0; t < MAX_3; t++)
//     {
//         w_rec[0][t][0] = 1024;
//         w_rec[0][t][1] = 1024;
//     }
//     for (int b = 0; b < MAX_2 - 1; b++)
//     {
//         for (int t = 0; t < MAX_3 - 1; t++)
//         {
//             if (t > t_right_cpu[b])
//             {
//                 w_rec[b + 1][t + 1][0] = 1024;
//                 w_rec[b + 1][t + 1][1] = 1024;
//             }
//             if (t < t_left_cpu[b])
//             {
//                 w_rec[b + 1][t + 1][0] = 1024;
//                 w_rec[b + 1][t + 1][1] = 1024;
//             }
//         }
//     }
// }

// __host__ __device__ inline int getDBC(uint288 n)
// {
// 	int b_try[130]={
// 	72,71,73,74,70,69,75,68,76,67,
// 	77,66,65,78,64,79,63,80,62,61,
// 	81,60,82,59,58,83,57,84,56,55,
// 	85,54,53,86,52,51,87,50,88,49,
// 	48,89,47,46,90,45,44,91,43,42,
// 	92,41,40,93,39,38,94,37,36,95,
// 	35,34,96,33,32,97,31,30,98,29,
// 	28,27,99,26,25,100,24,23,101,22,
// 	21,102,20,19,103,18,17,104,16,15,
// 	105,14,13,106,12,11,107,10,9,108,
// 	8,7,6,109,5,4,110,3,2,111,
// 	1,112,0,113,114,115,116,117,118,119
// 	};
	
// 	//将n转为双精度类型
// 	double dbl_n=n.to_double();
// 	//计算B1,B2
// 	double B1=0.9091372900969896*dbl_n;    //9*n/(7*sqrt(2))
// 	double B2=1.0774960475223583*dbl_n;    //16*sqrt(2)*n/21
// 	//计算LBound,RBound
// 	int LBound[MAX_3];
// 	int RBound[MAX_3];
// 	int DBC_index=0;

// 	for(int z=0;z<DBC_COEF;z++)
// 	{
// 		int b=b_try[z];
// 		LBound[b]=log(B1/d_pow23[0][b])/log(2)+1;
// 		RBound[b]=log(B2/d_pow23[0][b])/log(2);
// 		if(LBound[b]==RBound[b])
// 		{
// 			int a=RBound[b];
// 			int i=0;
// 			int b_temp=b;
// 			uint288 t=n;
// 			int s=1;
// 			while(!t.iszero())
// 			{
// 				//计算alpha,beta
// 				double dbl_t=t.to_double();
// 				int alpha=a,beta=b_temp;
// 				double logt=log(dbl_t)/log(2);
// 				double log3=log(3)/log(2);
// 				for(int j=b_temp;j>=max(0,b_temp-6);j--)
// 				{
// 					int alpha_j;
// 					if(d_pow23[0][j]>=dbl_t)
// 						alpha_j=0;
// 					else
// 					{
// 						int k_j=int(logt-j*log3);
// 						if(k_j>=a)
// 							alpha_j=a;
// 						else
// 						{
// 							if(abs(dbl_t-d_pow23[k_j][j])<=abs(d_pow23[k_j+1][j]-dbl_t))
// 								alpha_j=k_j;
// 							else
// 								alpha_j=k_j+1;
// 						}
// 					}
// 					if(abs(dbl_t-d_pow23[alpha_j][j])<=abs(d_pow23[alpha][beta]-dbl_t))
// 					{
// 						alpha=alpha_j;
// 						beta=j;
// 					}
// 				}
				
// 				int stmp=s;
// 				if(!(t>=u_pow23[alpha][beta]))
// 					s=-s;
// 				DBC_store[DBC_index][i][0]=stmp;
// 				DBC_store[DBC_index][i][1]=alpha;
// 				DBC_store[DBC_index][i][2]=beta;
// 				i++;
// 				if(t>=u_pow23[alpha][beta])
// 					t=t-u_pow23[alpha][beta];
// 				else
// 					t=u_pow23[alpha][beta]-t;
// 				a=alpha;
// 				b_temp=beta;

// 			}
// 			DBC_len[DBC_index]=i;
// 			DBC_index++;
			
// 		}
		
// 	}
// 	int V=9999999;
// 	int min_index=-1;
// 	for(int i=0;i<DBC_index;i++)
// 	{
// 		int temp=DBC_len[i]*150+DBC_store[i][0][1]*70+DBC_store[i][0][2]*126;
// 		if(temp<V)
// 		{
// 			V=temp;
// 			min_index=i;
// 		}
// 	}

// 	return min_index;
// }

// __host__ __device__ void DBC::get(uint288* np, int monitor) // runs in device.
// {
//     uint288 n = *np; 
//     uint288 B;
//     uint288 six_n;
//     uint288 record_outer;
//     uint288 temp_outer;
//     uint64 n0;
//     int bBound[MAX_2] = { 0 };
//     //w_min = n;
//     w_min0[2] = getV(np);
//     B = n.mul_2();
//     six_n = B.mul_3();
//     record_outer = B;
//     int log2B = 0;
//     int log3B = 0;
//     int t = 0, b = 0;
//     for (int i = 257; i >= 0; i--)
//     {
//         if (B >= pow23_256[i][0])
//         {
//             log2B = i;
//             break;
//         }
//     }
//     for (int i = 162; i >= 0; i--)
//     {
//         if (B >= pow23_256[0][i])
//         {
//             log3B = i;
//             break;
//         }
//     }
//     bBound[0] = log2B;
//     int i = 256;
//     for (t = 1; t <= log3B; t++)
//     {
//         for (; i >= 0; i--)
//         {
//             if (B >= pow23_256[i][t])
//             {
//                 bBound[t] = i;
//                 break;
//             }
//         }
//         i--;
//     }

//     if (monitor) {
//         //printf("DBC::get(): DBC_len is %d\n, wrec 0, 1 is %d, %d", DBC_len);
//         printf("w_rec[]:\n");
//         for (int i = 0; i < 20; i++) {
//             for (int j = 0; j < 16; j++) {
//                 printf("(%d %d %d)\t", w_rec[i][j][0], w_rec[i][j][1], w_rec[i][j][2]);
//             }
//             printf("\n\n");
//         }
//     }

//     //alpha=32,beta=18
//     int count = 0;
//     record_outer = six_n;
//     int j_bound = log3B / 18 + 1;
//     int i_outer = 0, j_outer = 0, i_inner = 0, j_inner = 0;
//     int min_index_0 = -1;
//     for (j_outer = 0; j_outer <= j_bound; j_outer++)
//     {
//         temp_outer = record_outer;
//         int i_bound = (bBound[j_outer * 18] >> 5) + 1;
//         for (i_outer = 0; i_outer <= i_bound; i_outer++)
//         {
//             n0 = temp_outer.mod_2_33_3_19();
//             for (j_inner = 0; j_inner <= 17; j_inner++)
//             {
//                 t = j_outer * 18 + j_inner;
//                 if (t > log3B)    //���������������������б���ѭ����������t>log3B�����ֱ��break
//                     break;
//                 if (t > MAX_T)     //����DBC�������Ž⣺triple���ʹ��t��
//                     break;
//                 for (i_inner = 0; i_inner <= 31; i_inner++)
//                 {
//                     count++;
//                     b = (i_outer << 5) + i_inner;
//                     if (t > t_right[b])
//                     {
//                         continue;
//                     }
//                     if (t < t_left[b])
//                     {
//                         continue;
//                     }
//                     if (b > 256)
//                         break;
//                     if (b + t > 0 && b <= bBound[t])
//                     {
//                         int quot = (n0 / pow23_1[i_inner][j_inner]) % 6;
//                         if (quot < 2)
//                         {
//                             //min_index_0 = min_index(w0[b][t + 1][1], w0[b + 1][t][1], w_0[b + 1][t][1] + 1);
//                             min_index_0 = min_index(w_rec[b][t + 1][0], w_rec[b + 1][t][0], w_rec[b + 1][t][1] + 1);
//                             //min_index_0=2;
//                             if (min_index_0 == 0)     //w(b,t)=w(b-1,t), w_(b,t)=w_(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0];
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1] + 1;
//                                 //mode:4λ16���������ֱ�Ϊw(b,t)���w(b,t)���w_(b,t)���w_(b,t)����
//                                 w_rec[b + 1][t + 1][2] = 0x0123;
//                             }
//                             else if (min_index_0 == 1)    //w(b,t)=w(b,t-1), w_(b,t)=w_(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b + 1][t][0];
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x0223;
//                             }
//                             else      //w(b,t)=w_(b,t-1)+2^b*3^(t-1), w_(b,t)=w_(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b + 1][t][1] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x1423;
//                             }
//                         }
//                         else if (quot == 2)
//                         {
//                             //min_index_0 = min_index(w0[b][t + 1][1], w_0[b][t + 1][1] + 1, w_0[b + 1][t][1] + 1);
//                             min_index_0 = min_index(w_rec[b][t + 1][0], w_rec[b][t + 1][1] + 1, w_rec[b + 1][t][1] + 1);
//                             if (min_index_0 == 0)     //w(b,t)=w(b-1,t), w_(b,t)=w_(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0];
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x0123;
//                             }
//                             else if (min_index_0 == 1)    //w(b,t)=w_(b-1,t)+2^(b-1)*3^t, w_(b,t)=w_(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][1] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x2323;
//                             }
//                             else      //w(b,t)=w_(b,t-1)+2^b*3^(t-1), w_(b,t)=w_(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b + 1][t][1] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x1423;
//                             }
//                         }
//                         else if (quot == 3)
//                         {
//                             //w(b,t)=w(b-1,t)+2^(b-1)*3^t
//                             min_index_0 = min_index(w_rec[b][t + 1][0] + 1, w_rec[b][t + 1][1], w_rec[b + 1][t][1] + 1);
//                             if (min_index_0 == 0)    //w(b-1,t)-2^(b-1)*3^t
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x2121;
//                             }
//                             else if (min_index_0 == 1)   //w_(b-1,t)
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1];
//                                 w_rec[b + 1][t + 1][2] = 0x2103;
//                             }
//                             else     //w_(b,t-1)-2^b*3^(t-1)
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b + 1][t][1] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x2114;
//                             }
//                         }
//                         else
//                         {
//                             //w(b,t)=w(b-1,t)+2^(b-1)*3^t
//                             min_index_0 = min_index(w_rec[b][t + 1][0] + 1, w_rec[b][t + 1][1], w_rec[b + 1][t][1] + 1);
//                             if (min_index_0 == 0)    //w(b,t-1)-2^b*3^(t-1)
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b + 1][t][0] + 1;
//                                 w_rec[b + 1][t + 1][2] = 0x2112;
//                             }
//                             else if (min_index_0 == 1)   //w_(b-1,t)
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b][t + 1][1];
//                                 w_rec[b + 1][t + 1][2] = 0x2103;
//                             }
//                             else     //w_(b,t-1)
//                             {
//                                 w_rec[b + 1][t + 1][0] = w_rec[b][t + 1][0] + 1;
//                                 w_rec[b + 1][t + 1][1] = w_rec[b + 1][t][1];
//                                 w_rec[b + 1][t + 1][2] = 0x2104;
//                             }
//                         }
//                     }
//                     if (b == bBound[t])
//                     {
//                         if (n >= pow23_256[b][t])
//                         {    //n>=pow23[b][t]  ->  n>nbt
//                             int V1 = w_rec[b + 1][t + 1][0] * 15 + b * 7 + t * 22;
//                             if (V1 < w_min0[2])
//                             {
//                                 w_min0[0] = b;
//                                 w_min0[1] = t;
//                                 w_min0[2] = V1;
//                                 w_min0[3] = 1;
//                             }
//                         }
//                         else {           //n=nbt
//                             int V1 = (w_rec[b + 1][t + 1][0] - 1) * 15 + (b - 1) * 7 + t * 22;
//                             int V2 = w_rec[b + 1][t + 1][1] * 15 + b * 7 + t * 22;
//                             if (V1 < w_min0[2])
//                             {
//                                 if (V2 < V1)
//                                 {
//                                     w_min0[0] = b;
//                                     w_min0[1] = t;
//                                     w_min0[2] = V2;
//                                     w_min0[3] = 3;
//                                 }
//                                 else
//                                 {
//                                     w_min0[0] = b;
//                                     w_min0[1] = t;
//                                     w_min0[2] = V1;
//                                     w_min0[3] = 2;
//                                 }
//                             }
//                             else if (V2 < w_min0[2])
//                             {
//                                 w_min0[0] = b;
//                                 w_min0[1] = t;
//                                 w_min0[2] = V2;
//                                 w_min0[3] = 3;
//                             }
//                         }
//                     }
//                 }
//             }
//             temp_outer.rshift_32(temp_outer);
//         }
//         record_outer.div_3_18();
//     }
//     //д��data
//     int DBC_len = 0;
//     int out_mode = 0;
//     int flag = 0;
//     int bit = 0;
//     b = w_min0[0];
//     t = w_min0[1];
//     switch (w_min0[3])
//     {
//         case 0:
//             for (int i = 8; i >= 0; i--)
//             {
//                 for (uint64 j = 1; j <= ((uint64)1 << 31); j <<= 1)
//                 {
//                     if ((n.data[i] & j) != 0)
//                     {
//                         data[flag].dbl = bit;
//                         data[flag].tpl = 0;
//                         data[flag].minus = false;
//                         flag++;
//                     }
//                     bit++;
//                 }
//             }
//             length = flag;
//             break;
//         case 1:
//             DBC_len = w_rec[b + 1][t + 1][0];
//             data[DBC_len].dbl = b;
//             data[DBC_len].tpl = t;
//             data[DBC_len].minus = 0;
//             length = DBC_len + 1;
//             break;
//         case 2:
//             DBC_len = w_rec[b + 1][t + 1][0];
//             length = DBC_len;
//             break;
//         case 3:
//             DBC_len = w_rec[b + 1][t + 1][1];
//             data[DBC_len].dbl = b;
//             data[DBC_len].tpl = t;
//             data[DBC_len].minus = 0;
//             out_mode = 1;
//             length = DBC_len + 1;
//             break;
//     }
//     /* occulticplus: check for 3^0 bug */
//     if (monitor) {
//         printf("DBC::get(): DBC_len is %d\n", DBC_len);
//         printf("w_rec[]:\n");
//         for (int i = 0; i < 20; i++) {
//             for (int j = 0; j < 16; j++) {
//                 printf("(%d %d %d)\t", w_rec[i][j][0], w_rec[i][j][1], w_rec[i][j][2]);
//             }
//             printf("\n\n");
//         }
//     }
//     int place = DBC_len - 1;
//     while (1)
//     {
//         int mode1 = (w_rec[b + 1][t + 1][2] & (15 << 12)) >> 12;  //w(b,t)����
//         int mode2 = (w_rec[b + 1][t + 1][2] & (15 << 8)) >> 8;    //w(b,t)����
//         int mode3 = (w_rec[b + 1][t + 1][2] & (15 << 4)) >> 4;    //w_(b,t)����
//         int mode4 = w_rec[b + 1][t + 1][2] & 15;    //w_(b,t)����
//         if (out_mode == 0)
//         {
//             if (mode1 == 1)
//             {
//                 data[place].dbl = b;
//                 data[place].tpl = t - 1;
//                 data[place].minus = 0;
//                 place--;
//             }
//             else if (mode1 == 2)
//             {
//                 data[place].dbl = b - 1;
//                 data[place].tpl = t;
//                 data[place].minus = 0;
//                 place--;
//             }
//             switch (mode2)
//             {
//                 case 0:
//                     break;
//                 case 1:
//                     b--;
//                     break;
//                 case 2:
//                     t--;
//                     break;
//                 case 3:
//                     b--;
//                     out_mode = 1;
//                     break;
//                 case 4:
//                     t--;
//                     out_mode = 1;
//                     break;
//             }
//         }
//         else
//         {
//             if (mode3 == 1)
//             {
//                 data[place].dbl = b;
//                 data[place].tpl = t - 1;
//                 data[place].minus = 1;
//                 place--;
//             }
//             else if (mode3 == 2)
//             {
//                 data[place].dbl = b - 1;
//                 data[place].tpl = t;
//                 data[place].minus = 1;
//                 place--;
//             }
//             switch (mode4)
//             {
//                 case 0:
//                     break;
//                 case 1:
//                     b--;
//                     out_mode = 0;
//                     break;
//                 case 2:
//                     t--;
//                     out_mode = 0;
//                     break;
//                 case 3:
//                     b--;
//                     break;
//                 case 4:
//                     t--;
//                     break;
//             }
//         }
//         if (place < 0)
//             break;
//     }
//     return;
// }


// __host__ __device__  void DBC::setNULL()
// {
// 	isNULL = true;
// 	length = 0;
// }
// __host__ __device__  int DBC::getL() const
// {
// 	if (isNULL)
// 		return 99999999;
// 	return length;
// }
// __host__ __device__  int DBC::getV() const
// {
// 	if (isBasic)
// 		return basic_value;
// 	if (isNULL)
// 		return 99999999;
// 	return dbl_cost * addNode.dbl + tpl_cost * addNode.tpl + add_cost * (length - 1);
// }



// __host__ __device__ DBC DBC::add(int dbl, int tpl, int coef)
// {
// 	if (isNULL)
// 		return *this;
// 	DBC ret;
// 	ret.parent = this;
// 	ret.addNode.dbl = dbl;
// 	ret.addNode.tpl = tpl;
// 	ret.length = this->length + 1;
// 	ret.addNode.minus = (coef != 1);
// 	return ret;
// }


// __host__ __device__ void DBC::simDBC()
// {
// 	if (isBasic)
// 		return;
// 	DBC* present = this;
// 	for (int i = length - 1; i >= 0; i--)
// 	{
// 		now_DBC[i].dbl = present->addNode.dbl;
// 		now_DBC[i].tpl = present->addNode.tpl;
// 		now_DBC[i].minus = present->addNode.minus;
// 		present = present->parent;
// 	}
// }

// __host__ __device__ void DBC::print()
// {
// 	if (isNULL && (!isBasic))
// 	{
// 		cout << isNULL << endl;
// 		cout << isBasic << endl;
// 		cout << "NULL" << endl;
// 		return;
// 	}
// 	//simDBC();
// 	for (int i = 0; i < length; i++)
// 	{
// 		if (now_DBC[i].minus)
// 			cout << "-";
// 		else
// 			cout << "+";
// 		cout << "2^" << int(now_DBC[i].dbl) << "*3^" << int(now_DBC[i].tpl);
// 	}
// 	cout << endl;
// 	return;
// }


// __host__ __device__ DBC& DBC::operator =(int n)
// {
// 	if (n == 0)
// 	{
// 		isNULL = false;
// 		length = 0;
// 	}
// 	else
// 	{
// 		int flag = 0;
// 		for (int i = 0; n > 0; i++)
// 		{
// 			if ((n & 1) == 1)
// 			{
// 				now_DBC[flag++].setdata(i, 0, false);
// 			}
// 		}
// 		length = flag;
// 		basic_value = now_DBC[length - 1].dbl * dbl_cost + add_cost * (length - 1);
// 	}
// 	isBasic = true;
// 	return *this;
// }

// __host__ __device__ DBC& DBC::operator =(uint288 n)
// {
// 	int flag = 0;
// 	int bit = 0;
// 	for (int i = 8; i >= 0; i--)
// 	{
// 		for (uint64 j = 1; j <= ((uint64)1 << 31); j <<= 1)
// 		{
// 			if ((n.data[i] & j) != 0)
// 			{
// 				now_DBC[flag++].setdata(bit, 0, false);
// 			}
// 			bit++;
// 		}
// 	}
// 	length = flag;
// 	basic_value = now_DBC[length - 1].dbl * dbl_cost + add_cost * (length - 1);
// 	isBasic = true;

// 	return *this;
// }


__host__ __device__ int DBCv2::getDBC(uint288 *n) {
	int b_try[130]={
	72,71,73,74,70,69,75,68,76,67,
	77,66,65,78,64,79,63,80,62,61,
	81,60,82,59,58,83,57,84,56,55,
	85,54,53,86,52,51,87,50,88,49,
	48,89,47,46,90,45,44,91,43,42,
	92,41,40,93,39,38,94,37,36,95,
	35,34,96,33,32,97,31,30,98,29,
	28,27,99,26,25,100,24,23,101,22,
	21,102,20,19,103,18,17,104,16,15,
	105,14,13,106,12,11,107,10,9,108,
	8,7,6,109,5,4,110,3,2,111,
	1,112,0,113,114,115,116,117,118,119
	};
	
	//将n转为双精度类型
	double dbl_n=n->to_double();
	//计算B1,B2
	double B1=0.9091372900969896*dbl_n;    //9*n/(7*sqrt(2))
	double B2=1.0774960475223583*dbl_n;    //16*sqrt(2)*n/21
	//计算LBound,RBound
	int LBound[MAX_3];
	int RBound[MAX_3];
	int DBC_index=0;

	for(int z=0;z<DBC_COEF;z++)
	{
		int b=b_try[z];
		LBound[b]=log(B1/d_pow23[0][b])/log(2)+1;
		RBound[b]=log(B2/d_pow23[0][b])/log(2);
		if(LBound[b]==RBound[b])
		{
			int a=RBound[b];
			int i=0;
			int b_temp=b;
			uint288 t;
            for (int i = 0; i < 9; i++) {
                t.data[i] = n->data[i];
            }
			int s=1;
			while(!t.iszero())
			{
				//计算alpha,beta
				double dbl_t=t.to_double();
				int alpha=a,beta=b_temp;
				double logt=log(dbl_t)/log(2);
				double log3=log(3)/log(2);
				for(int j=b_temp;j>=max(0,b_temp-6);j--)
				{
					int alpha_j;
					if(d_pow23[0][j]>=dbl_t)
						alpha_j=0;
					else
					{
						int k_j=int(logt-j*log3);
						if(k_j>=a)
							alpha_j=a;
						else
						{
							if(abs(dbl_t-d_pow23[k_j][j])<=abs(d_pow23[k_j+1][j]-dbl_t))
								alpha_j=k_j;
							else
								alpha_j=k_j+1;
						}
					}
					if(abs(dbl_t-d_pow23[alpha_j][j])<=abs(d_pow23[alpha][beta]-dbl_t))
					{
						alpha=alpha_j;
						beta=j;
					}
				}
				
				int stmp=s;
				if(!(t>=u_pow23[alpha][beta]))
					s=-s;
				DBC_store[DBC_index][i][0]=stmp;
				DBC_store[DBC_index][i][1]=alpha;
				DBC_store[DBC_index][i][2]=beta;
				i++;
				if(t>=u_pow23[alpha][beta])
					t=t-u_pow23[alpha][beta];
				else
					t=u_pow23[alpha][beta]-t;
				a=alpha;
				b_temp=beta;

			}
			DBC_len[DBC_index]=i;
			DBC_index++;
			break;
		}
		
	}
	// int V=9999999;
	// int min_index=-1;
	// for(int i=0;i<DBC_index;i++)
	// {
	// 	int temp=DBC_len[i]*150+DBC_store[i][0][1]*70+DBC_store[i][0][2]*126;
	// 	if(temp<V)
	// 	{
	// 		V=temp;
	// 		min_index=i;
	// 	}
	// }

	return 0;//min_index;
}

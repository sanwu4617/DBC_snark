#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <random>
#include <cstring>
#include <vector>
#include "gpuec256.h"
#include "cuda_common.h"
using namespace std;

#define CONST_SCALE 13
const int N_BIGNUM = 1 << CONST_SCALE;  // maxsize = 2^20, for not run out of memory.
const int BATCHSIZE = 1 << CONST_SCALE; //
const int N_POINT = BATCHSIZE;
#define CONST_THREAD_PER_BLOCK 256
const int N_THREAD_PER_BLOCK = CONST_THREAD_PER_BLOCK;
const int N_BLOCK = ((BATCHSIZE + N_THREAD_PER_BLOCK - 1) / N_THREAD_PER_BLOCK);
#define CONST_BIGNUM (1 << CONST_SCALE)
#define CONST_BATCHSIZE CONST_BIGNUM
#define CONST_BLOCK ((CONST_BATCHSIZE + CONST_THREAD_PER_BLOCK - 1) / CONST_THREAD_PER_BLOCK)

const UINT64 h_R2[4] = {0x000007a2000e90a1L, 0x1L, 0x0L, 0x0L};

const UINT64 h_Gx[4] = {0x59F2815B16F81798L, 0x029BFCDB2DCE28D9L, 0x55A06295CE870B07L, 0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4] = {0x9C47D08FFB10D4B8L, 0xFD17B448A6855419L, 0x5DA4FBFC0E1108A8L, 0x483ADA7726A3C465L};

const UINT64 testGx[4] = {0x252931db128244c9L, 0x80ec2e92027d7e6eL, 0x32c5ee6d51cb1e89L, 0xb89bd74c7352f570L};
const UINT64 testGy[4] = {0xd8cbce4f20d0d9e4L, 0x5b636389add7cc6eL, 0xccd07463f61e7fbeL, 0x13fae72c0d3c849bL};
const UINT64 testGz[4] = {0xbc5c645f1b1c297dL, 0x0ba1469cd0bdd88aL, 0x40bad30e143dcdceL, 0x4bba49beb75cce43L};

Jpoint testG = {{0x252931db128244c9L, 0x80ec2e92027d7e6eL, 0x32c5ee6d51cb1e89L, 0xb89bd74c7352f570L}, {0xd8cbce4f20d0d9e4L, 0x5b636389add7cc6eL, 0xccd07463f61e7fbeL, 0x13fae72c0d3c849bL}, {0xbc5c645f1b1c297dL, 0x0ba1469cd0bdd88aL, 0x40bad30e143dcdceL, 0x4bba49beb75cce43L}};

// #define PRECOMPUTE // precompute test.
#define CUDA_VISIBLE_DEVICES 1

// 32 dbcs for test. to long dbc are very slow to compile.
const int SMALL_DBC_NUM = 32;

unsigned long long px[] = {0xc1291ea73f54ce04ll, 0xbaae63f186512829ll, 0x849ffc311521657all, 0xdb05b8be3bc3ea12ll, 0xec734eb90e60d146ll, 0x47e3a925f4c2f8d3ll, 0xe404a3c34bfc4331ll, 0xa09f73b213ee39abll, 0xdb13ac478ea2128ell, 0x2c53e4e8f64e797dll, 0x39b7a582468b2b26ll, 0x8a7cc04c32f86ce1ll, 0xcd358ec4bf05d09ell, 0x6b2044c68221ce4ll, 0x87a85903221d0748ll, 0x46f323835084b5a9ll, 0x46f97386edeaf803ll, 0xa49b5c87eb315db7ll, 0xe3081f1ad7d6d04all, 0x83059f0ea01a4aeell, 0xee6fba127fec7397ll, 0xaba6a4a20bad9c85ll, 0x7644585b6c958d2cll, 0x84938de81702b2a3ll, 0x1a3ad24da633218ell, 0xec8c6b7a2e2d7fd4ll, 0x5893289ae2b2d9bll, 0xe5ef285c943653aell, 0xa04bd5faef99b5cell, 0x978380dff860c313ll, 0xa993bd51570b133ell, 0xb6d827a6ffe8c506ll, 0xf3fc602ee2294187ll, 0x885972979b942caell, 0xfe36bc732ebf4a28ll, 0x44524be479e97992ll, 0xf46d14dc6fdec9fll, 0x8548cb4caa31b9d7ll, 0xeefb2bfedba2f870ll, 0xee2cf31377c30c02ll, 0x8abcb93f4f13ae56ll, 0x4b03bbfee680553ell, 0x17a359c48f5f772ell, 0x4b8590bc7ad10993ll, 0xd81ef93a1c4bc230ll, 0x3fe5e94a6137a639ll, 0xdf6937fecfe2ad57ll, 0x70c0b2bac6f886f1ll, 0x9f04236a79cf01f6ll, 0x13e1640167007df8ll, 0xf780f2e0d9a1ae0fll, 0xbf7d094c158291cfll, 0x212b875aa42d2b3cll, 0x446ebb199ab8c39fll, 0xdcaf114ec3d14cfdll, 0x2ba87b8b8d3f1f59ll, 0x9bcd364ea3325d90ll, 0x3a47a15979d76791ll, 0x587c8cc21bc825dell, 0x89c87f72dfc370fcll, 0xb0d283bbfa0e29b3ll, 0x26308aa1138b7509ll, 0xaf529fa55238314ll, 0x39816c256e503a7ell, 0x5f22c397ef9fe0all, 0xb9e5bdb683e15a95ll, 0xaaa3fb7ade2d4d88ll, 0x4d7e7ee5951e2398ll, 0x5470acdb8695857ll, 0x5166d1ad1ad03d79ll, 0xf3305c87d6eb6a66ll, 0xc176900a101104fll, 0xce06d34209f8948all, 0xd82193ffdfe2c9e5ll, 0x931075c79ce0d306ll, 0x775fb75a7642d179ll, 0x888cda266c2aad54ll, 0x40fc82795775f1fall, 0x497bad018e821b36ll, 0xa29b1ff55c8448c9ll, 0x69af3b5510579da2ll, 0x2f1fbea9ba3d7984ll, 0x774fd33a1752595cll, 0x56a3a84b5041c8ebll, 0x7cde84ea0461ba1cll, 0x603181bf3283c6d6ll, 0x62f2c535e53dcee8ll, 0x2451e9cc1fdf3d7ll, 0xda232c0afcdc6597ll, 0xeb4ba40875b6a4dall, 0x2d5f9584192f072bll, 0x3198177f0b01bcdcll, 0xacd3d97303bbd12cll, 0x8db5b517cbe942e0ll, 0x51d09eb6d2933d2ll, 0xe6d9c03b36990772ll, 0xb964889dbf78ebd5ll, 0xb9811d6fc0356d18ll, 0xa2410b8e143e5130ll, 0xcc38ce20d9807de7ll};

unsigned long long py[] = {0x8ceceeb28411ba32ll, 0x3e781792a382e71ell, 0xd3fa02dae181fc95ll, 0x2e304597764f60a6ll, 0xfce1486fcc18d692ll, 0x1375c5b6e87f2c3all, 0xb48702fc8a96ba27ll, 0x57e69125d0114f2cll, 0x362d1b3393fa7588ll, 0x78796df78872500dll, 0x9138dfc1bdc15a3dll, 0xa167f3d2414d0205ll, 0x1b9b9a5f34b8afecll, 0x583a0f67a99a3788ll, 0xc4acd28dffe1410cll, 0xf07e21417abbcc3cll, 0x3ade3c45dfbc863fll, 0x9eda6f139f31b018ll, 0xe0adb41adfca8622ll, 0x15ea15417084f16bll, 0xb3b2c55417193756ll, 0x13712c30fdc9e718ll, 0x2952caab2f05eab1ll, 0x358afc9cbd7e5192ll, 0xd536b8e7f4c507d6ll, 0x66c5a5ff171507e0ll, 0x725b2f3fb86bac7bll, 0x9e36795fd946b087ll, 0x357fcb8b68e5b458ll, 0x7766f17d2f3054d7ll, 0x8343812a93422063ll, 0xa1f100198fe5c71dll, 0x68620cb7296393a2ll, 0xd95ea31298a2c21dll, 0x77d391aa12784ca4ll, 0x8fcf1ba16eaf7c5bll, 0xc68cfe2925c1ac51ll, 0x2352bcdd153d436cll, 0xcfeeeb13fb787307ll, 0xac675e68a3bff56bll, 0xd621070cf7495c20ll, 0x31c021898ac69039ll, 0x8f0c823687669c4all, 0x144b3504c3329402ll, 0x10a0a577850a8ac9ll, 0x2a0026f19fabf26ll, 0x8b726c9485375803ll, 0xf2e492176f575e7ell, 0xbba9a542f2b6cf79ll, 0x117cb4563c3d2f96ll, 0x65b88205addecb52ll, 0x615c0a66a7d6a4dell, 0x35f4afe2701217e2ll, 0x15e4a785245be7e3ll, 0x2993bd9756922e8ell, 0x5c2ff49d4f54fda0ll, 0xa82c16071118f14fll, 0x800ca14fc42286e7ll, 0x1e5b6da6f7f886e3ll, 0x37997aecc99fc704ll, 0x6991710fc91f47e3ll, 0x9877646ac9334869ll, 0x61ac8c5c317aba27ll, 0x69b0ede62479a906ll, 0xf1d3bf52ace992d5ll, 0x1c9980570a29208all, 0x53784402059629d6ll, 0x9698f219e1c2443all, 0xf6f31dc5852bdbbbll, 0x261e0d1867a702f8ll, 0xd166e64d5d8829b9ll, 0xc77dd21b6cbdd1b6ll, 0xe69accceb5e2b4e9ll, 0x69baa5bbe515be9dll, 0xb616554bece011cdll, 0xa34be20aa7d9c90cll, 0x5e7cc1b8428e709cll, 0x5c58a6ac2c829e11ll, 0x9a3dce73b43a903ll, 0xa58d08ec4c58bd52ll, 0x7f7751124655e13ll, 0x5424aa4fdbf8b18dll, 0xc982cbae45ccce95ll, 0x9398e8c27258250dll, 0xf17129bfa5b1b510ll, 0x41fe283dfb4327afll, 0x5f2f33e7714c97dall, 0x559cf7242bb8fe8ell, 0x17b3a3bb46dd3e91ll, 0x1af7d0a9f1b04817ll, 0x6d23e0dd841a2672ll, 0x8f6d1a2693629455ll, 0xff20d2520292748dll, 0x24d2beca6756a36dll, 0x7644c94f54b6cb43ll, 0x8b5d5592833e959fll, 0xb0c016d4371e5f9ell, 0xa0acac44e0ca9561ll, 0x9d27f8c2c840e7ell, 0x73b4245bfb56fc89ll};

unsigned long long px3[] = {0x4d3c296b3e8b6988ll, 0xddcc9ca08396d1cbll, 0x7ad77e63364511dfll, 0xa0ac877d59fe43all, 0x4a87562b177eae72ll, 0xf52831b49c6a6112ll, 0xf0b0a46021a488d2ll, 0xd4575222d4cda88bll, 0x9fd2acb1e587292bll, 0xd8c007515b734f4ll, 0x86d785c545c96c0dll, 0x81b2826c2c4d2dd8ll, 0x94001024d35f4e03ll, 0xa7f67326231808c0ll, 0x49e61768c043f864ll, 0xa6bb07c8044cb754ll, 0x3bc5fce63b5c6eb7ll, 0xe975105a35c835b0ll, 0x5449ac7731bf26a6ll, 0x8bb21922e7870e9cll, 0xbf51fc11ed6b7becll, 0x90c2a7dd970b6a7all, 0x87b1d7c096ab0fd1ll, 0xef4efa6172ed7f00ll, 0x70bfa5991f8040b5ll, 0xf63d2f69ab92e571ll, 0x30f4f32599299452ll, 0xc55ac8e19c98b574ll, 0x1e7bd7e1f2a7f00all, 0x9bcc5a675c011ab3ll, 0x1b0ac2245d3f3e60ll, 0x6f91233e31a13e1cll, 0x15c3bbf222eb03b3ll, 0x39c2af9d552948efll, 0x51a6d8ded9469e3bll, 0x162d2cf1cfd90c8ell, 0xc3241eee21e04335ll, 0x1f7ef0ec35bb3a16ll, 0x2082ddcd712cea20ll, 0xea6dd878e1cf83bell, 0xe20708512d57af51ll, 0x53d8c5e73bfd68e7ll, 0xbc06c96708a2d80dll, 0xe5303f14e2c4ed2ll, 0xf2baa21ec43fbf3cll, 0xc66aaf40605089d8ll, 0xb98898d342d1a70all, 0x79eaa5df944cdcc3ll, 0xa4ae33d8a9392101ll, 0xd2bd0d1e685b2895ll, 0xf5dee131682af06all, 0xee9d971db32100a9ll, 0x979d6bc8cba8f21all, 0x627a73d284d1e077ll, 0x3791b8bed2f990ecll, 0xf1b664cf858bf41ll, 0x8bf00b84d5ef3043ll, 0x5290e9b8024f8d15ll, 0x586a32e199bc60afll, 0xb0011ca2bfc88320ll, 0xd246570ad39d881dll, 0xa8d58a8fb5195e07ll, 0x2d81c5167f40a226ll, 0x4d5f609d233f8b61ll, 0xe9c3989a896b89d7ll, 0xeb4f7588bf903347ll, 0x833013f909b93343ll, 0x9a573acb50f4a9ebll, 0x66d01a5810f8d137ll, 0xaa9f929069bcc2dell, 0xa6adb6c784046febll, 0x773fb2dd1fca11d2ll, 0xf8504ef7bbb60baell, 0xb8ff3fd004c40d6dll, 0xa2709481fa5be1f4ll, 0xe3b67f87945b1f0dll, 0xb8c2b550fe024263ll, 0x162da9f5da318c7ell, 0x43bd85c11f0e6f40ll, 0x576e83352bbff70bll, 0x6f532e35310c746ll, 0xde275c4ae8b7c108ll, 0x366542ca116d445dll, 0x192377a9f89ad8f5ll, 0xee8f3b67d22d5d99ll, 0x579cb13009210d66ll, 0xf3873431e3850df3ll, 0x7c69e10852936f22ll, 0x2b3a9bd371dfaba7ll, 0x6be9676d1bbce350ll, 0x7e6ee26c8b0dc356ll, 0x1b3261a0e40f0779ll, 0xa9706717ed456ab8ll, 0x5d765dabbeb674c6ll, 0x7d739283e4f43033ll, 0x8e305790a447736all, 0xe352f5943cf487fbll, 0x64398b5b8cbe1143ll, 0xfc2cb7207d937bd2ll, 0x7753cf89671c8cdell};

unsigned long long py3[] = {0xcacdf749cb8d5f19ll, 0x3a26b17a6f13f006ll, 0x1cde4777eaf1fa11ll, 0xa4414ed881c0e56dll, 0x3b571c5fa5508c0ell, 0x7cdc1849f76f2ec2ll, 0x1a468db56c8568e2ll, 0x8c57ec98bf38fe59ll, 0xd10b947bbcaf4bf5ll, 0x4e64eb670fae53bell, 0x430c61252c4de283ll, 0xab68117b7ff30299ll, 0x26a8f75f7d79ee30ll, 0x1053de0301a4bccbll, 0xbb50c1849212a0f1ll, 0x423862d46548c4bell, 0xc31f86616d9390ecll, 0xfa83bcfd2bc8718fll, 0x22d6538e7ca2b142ll, 0x21ebff421deb94b7ll, 0x8a5128224b2e1e5bll, 0xb518a254c60bd352ll, 0x8c10a7ec1dd632cfll, 0xb8b94fd1a20f561ell, 0xcbf7274e4320fde9ll, 0x7f825189b430ad19ll, 0x7fc5b0de13e6ae42ll, 0xb61ac93784c46e70ll, 0x86e902c618d1d0e2ll, 0xc7ab820d0f5ddfa6ll, 0x2927e90be13fcc69ll, 0x5a7a8539a069fa70ll, 0x8fa0db2c14690950ll, 0xee3365a358ca1d1bll, 0xd2107bae0e0d2c93ll, 0xb19305443f66057ll, 0x505575ab92a3dbd0ll, 0xfc0d0110bab28c74ll, 0x64b5014ed56a5165ll, 0x210a5c2d815d6c7ell, 0x71c2a24bc399717ll, 0x2f7c82a1be3ed40all, 0xa0fe03461542dc9all, 0x9c8175bc3b7aac92ll, 0x52a5d275fa70f38fll, 0x7f280e75ef818498ll, 0x782e3358a54ea194ll, 0xb286ede5b6c69b77ll, 0xb2fae70f5b4cb3e6ll, 0x985669303148fc90ll, 0xba4f880f241bfd5ll, 0x5134a1168ffb9c2bll, 0x88aaa0f440c2fc84ll, 0x732a815a26af6096ll, 0x8ada0e8b66cc7c48ll, 0x85d80c3e63934b97ll, 0xc276948f76dc160cll, 0x83354992d9f8f4ecll, 0xdb89f215877d3fd2ll, 0x607cf4a51e6d4beall, 0x7bc812a269cdb273ll, 0x5e91c14838a6da32ll, 0xeea1beca0c82783cll, 0xb402c8733b869ce8ll, 0x6bb6539a3a626f9fll, 0x65c3c28b9fb0dd7ell, 0xfd31658429a1fd8cll, 0x283fb6056d1dbd54ll, 0x4d1fd3ab9ce96b92ll, 0x59307fca2f3e66d2ll, 0x47899b04717f8830ll, 0x2c04c9c438a97ccell, 0x41a43e034990ccb2ll, 0xac5c62a314ae7fc1ll, 0x2bc9c15f5ce5f21dll, 0x4351c7f4bc543587ll, 0xe714a0633eb54391ll, 0x49a981d8f96d8c83ll, 0xcc039db192259e41ll, 0xb0365ab40ef69a54ll, 0xf7c0aa189d06d5a4ll, 0xe3c9ef6dbcbe0cb3ll, 0x5053f2e2d58472e7ll, 0xeed3a254c43ae758ll, 0x783efea48bc01e51ll, 0xfd450216e34bb240ll, 0xaa59875b491dfe0dll, 0x18c4e66c88aa43a4ll, 0x21ea3d478bc5b59bll, 0x50b61cc4607a4882ll, 0x7c3a24521c8e8bb8ll, 0xdcc23b4a6fae5af9ll, 0x34f723ffe88c492ll, 0x7a1049306af2c39cll, 0xb8fafb10e9e11d70ll, 0x45e280142165ae6ell, 0xd1bc897d2caf1fb5ll, 0x58c41c04b69ddcfdll, 0x3869b0569cc204c5ll, 0xfc0be1c06a7ecb4ell};

unsigned long long dx[] = {0x3e9b58b481a0c1e7ll, 0xd2790f58ad702ac3ll, 0xf3df624ee861f1all, 0x7422dbc25514e317ll, 0xb6c363056b6d570ll, 0x64b8ed462321d1b1ll, 0xfaf242de5b1e7585ll, 0xd4c307f48c33830fll, 0xd418348214808c65ll, 0xf8932fa155f832ebll, 0xd9145a22f78c225cll, 0xac037fc3f8423648ll, 0xa28bafa272ad91a5ll, 0x9795cc3f52dd1fdfll, 0x617bf6ea64012f39ll, 0x15c959cd69b96173ll, 0x79938dacb05340f0ll, 0xb73c17845e5f0655ll, 0x3caed57f4bd41d80ll, 0xa1ce5385c6053c48ll, 0x2970d3e74b38c8a2ll, 0x981e9824714f72f9ll, 0x8d5deba7fe2831cfll, 0x94fc5ff28dd55157ll, 0x9b7d882b71f43546ll, 0x7c6c74ebad9a3f32ll, 0xbebd6b634e1a88eell, 0xeb0ac31cf931b374ll, 0x7861226eaee648bll, 0x3862d67ed71d09f5ll, 0x1505d4e24a270420ll, 0xf8b0088e5648acecll, 0x90c1545404bc4a0ll, 0xe297c7d92264f5fbll, 0x9ab771c2882e07a5ll, 0x30a271fd9414e288ll, 0xc47f35f595e118b5ll, 0x552c5a97b9aa7516ll, 0x144f974df1b2ad09ll, 0x427e96ea27f5846bll, 0x78820a6c772f2688ll, 0x47a88dfd3542f024ll, 0x40c3d261f513e1a8ll, 0xd62b6192ae7bdab9ll, 0x77f9db302dcd5c28ll, 0xba5881b5c7ddd140ll, 0xfe44a6e288bf3851ll, 0x4d1f56e7962c2dell, 0x3c0ca396a6f16a01ll, 0xdea3647470db11b2ll, 0xb1dfb8febcc7541bll, 0x91d5f4ac42502ad1ll, 0x43a3f5dc4363be16ll, 0x60a7186206c08f8bll, 0x4ad258e55da97160ll, 0x510df6e5c3232ec7ll, 0xa2da54efe96f9514ll, 0xe3c1b95dad759e1bll, 0xd03e5645dbea5195ll, 0x2b64bdbb8c1cd0a6ll, 0x3d64efeb284b8ad2ll, 0xed645de57f9311ccll, 0xa9caeb58742da602ll, 0xabb9772059774479ll, 0xa23cae275137c4d1ll, 0x826d0433243e9a9ell, 0x95eba298e227b592ll, 0x3a2ed739fb98c7e9ll, 0x5cfebb35d7f90e1bll, 0xf732aff4ff5a4fb7ll, 0x748112df101e87fcll, 0x834fdc11baab3838ll, 0xd75c9e46d0aee654ll, 0x1745092a6abf8770ll, 0x1362d90dcded199ell, 0x863dfc5e91a7a9b5ll, 0xd18f3cd686e412fll, 0xe04fe216bcc01378ll, 0xb3a1cd3bdce4c117ll, 0x2283efec24582825ll, 0xf744c2c74567e2f5ll, 0x97afc09b7b282e43ll, 0xd198c777f05886bbll, 0x6b23e4c0f51dadb5ll, 0xdae050c76c1eddb3ll, 0xdaab67905a7d0659ll, 0x2b174248a0584645ll, 0xd34e1a4d0bd960b1ll, 0x808d3cc6dd83fc59ll, 0xbf91a51648dd0960ll, 0xf38567e92f48df68ll, 0xacb571f8dd66500ll, 0xecd41f0b86138682ll, 0x21e873127c645bb6ll, 0xb5a0e535ee29b6b1ll, 0xeaeeab56d06c2c0ell, 0x929db043d8f2cdfall, 0xc0d2a2608be97e5ll, 0x426dc138552c4432ll, 0x868a971759b755ccll};

unsigned long long dy[] = {0x3542486d3e8384e3ll, 0x13d051ad6b464dedll, 0x5b42fc37a6293dbdll, 0x6e4b2cbd7e92276fll, 0x9fef0e9c8b15264fll, 0xabb70e6bee9cb2ll, 0xee25466956363179ll, 0xfd11ac5d51563753ll, 0x58b1fc3af2489a29ll, 0xa82aa19eaac8748ll, 0x51f02a42fec69c5fll, 0x725f87d6edcef39ll, 0xb51e0df1a07df532ll, 0x8e08134f4074abd4ll, 0xbfdfdba93252fa31ll, 0x37c5c7692f917865ll, 0xe0ed9cb3d636f6b3ll, 0x5b83a93f64405366ll, 0xf06a4f823a1989d7ll, 0xb9ff31ce194773bfll, 0x1d98a6d47f5270c2ll, 0xb43486e2eb7e136bll, 0x64d788eb8413f93dll, 0x9bb30eb73e93d3b2ll, 0xf309ee64e7d4f92ll, 0xb8dde885b32312all, 0x21118ded95ac6f29ll, 0xc983dab25faec20bll, 0x7b74ac6b8d452c1bll, 0xcaccbe26c46e1ea2ll, 0xfeab89e32a652279ll, 0x81b537578dece23dll, 0x714ece790da7d5ebll, 0x771bbd884024ee48ll, 0x6881642b2437ea69ll, 0x72a353465dbfc234ll, 0x2e7739c8b3284985ll, 0x518b6ce43f74b9fdll, 0x1d11b310f142340cll, 0xb8b5f8a7047cc78bll, 0xc98112a0eccee6aell, 0xd4a12b5796e5b7a0ll, 0x24d6431fd4b6ea05ll, 0xafa380a9e9478faall, 0x98a35a0f382f2211ll, 0x608d3b1db228e853ll, 0xec7ada4b8c2e0277ll, 0x59b28e0ee1d21e1cll, 0xc0a93100d57028call, 0xb2ee8e366d7a78d3ll, 0x409e61311cac105bll, 0x152478a51a53cec0ll, 0xa6a22e64bcf36c49ll, 0x32130c6462739094ll, 0x915ff7ea83bb5e2ll, 0xcc03a67084696f4cll, 0xe208ec68ca092003ll, 0xdb6bf75f2140aa60ll, 0xef36c6cc11586b3cll, 0x8afb419de213ac3dll, 0x5e382f8e520c04e1ll, 0x5f92c6bc34ebbb89ll, 0x398db743bfb374fall, 0xe19e0e6e3ceeaa42ll, 0x9e7ea5242816d0d6ll, 0x11dcaa44038fce38ll, 0xc6e3725ac42e00a4ll, 0x71443ce92e90d37cll, 0x23dd572671f8b692ll, 0xde27f7bde1a28bb9ll, 0xe3fc1cc34a75f2ddll, 0x8e5380a90833dfd6ll, 0x4da2b42de56731fdll, 0x36a6009f9eee69bcll, 0xf4db711db3e119ccll, 0x5e8ccc990795a889ll, 0xafd703419dfea9a0ll, 0x30463f3c6ee2b046ll, 0x9e2b31784a512bbll, 0xe94ad634e74bcf82ll, 0x45f3db452cd7f9bbll, 0xde6b6252dd33782bll, 0x193d5f6c04a27310ll, 0x3acb8fa99c329707ll, 0x22d69badb37471f0ll, 0xf1c99f10c56b7cf5ll, 0xfe6110920f406657ll, 0x2258d6aa622458d9ll, 0xd635d70116f7d3fcll, 0x623d3502b380a0fell, 0x967638214e7752d8ll, 0x8ebe4d0ebc1b3a36ll, 0x4693a1c02ec948b3ll, 0xd30995eb3700fd54ll, 0x703135b40eee5981ll, 0x47513b99adac2c47ll, 0x4807b69ef64a62e3ll, 0x351d3b0f620edf57ll, 0xe2d20985d347dddfll, 0xcf1d0224f5f5ba61ll};

unsigned int dx2[] = {0x90a25e6d, 0x266b1fdd, 0xc7e5f9b1, 0x8b331f76, 0x86c26260, 0x11602198, 0x3d449367, 0xe221abb8, 0x20512bec, 0xee5d4140, 0x7a4a63ea, 0x9c9a8137, 0x4518e71d, 0x90bb01b9, 0xc8a0dbb1, 0x1c5859f8, 0xb71d74a2, 0xe627b83d, 0x63ad710f, 0x1b0590aa, 0x47f31b7d, 0x8b247df8, 0x680ceb24, 0x84c82aa9, 0xe16a482, 0x1b303012, 0xa6208757, 0x62842d6f, 0x27ae4f0c, 0x893d6f09, 0xa4687085, 0xa1ffc667, 0xea82a239, 0x17d73135, 0xcb5a9d69, 0x9fb058fc, 0x2998e75c, 0xf0fc9d2, 0xfb1c8ce, 0x79e3bc7c, 0x231366b5, 0x40392f2, 0x3a096a07, 0xd3b975cf, 0xfc49c863, 0xd9b437c6, 0xfea912dc, 0x67224518, 0x92ea1d92, 0x32594898, 0x7436f88d, 0x9a3b2801, 0x54930fd, 0x9273f995, 0x3b9bdecb, 0x44a0665f, 0xde53d2e8, 0xe611f8a1, 0x47dd26d5, 0x62006088, 0x18adef96, 0x40a57445, 0x96d64655, 0xc1c91992, 0xd8d12730, 0x7aeda909, 0x85eaf4c4, 0x60f20c05, 0xc1ed3293, 0xc322ca3f, 0x4c7ed042, 0x73c2b31d, 0xe0396b9d, 0x3020d47f, 0xd4c5d6c6, 0x523f71c0, 0x8ce47a37, 0x7859e90a, 0x7686e532, 0xdb086e00, 0x909ac2c3, 0x574ed015, 0x9e6fa778, 0x54ecf2a9, 0x2db98885, 0xec671188, 0xc0de2005, 0x5ec92a71, 0x78b628e2, 0x9c9816e2, 0x69e13f3d, 0x4fc3827b, 0xae393a9e, 0x6f45648, 0xa85e2113, 0xa2b591bc, 0xaaca0b99, 0x24033483, 0xf90f13b2, 0xa29a99cc};

unsigned int dy2[] = {0x90a25e6d, 0x266b1fdd, 0xc7e5f9b1, 0x8b331f76, 0x86c26260, 0x11602198, 0x3d449367, 0xe221abb8, 0x20512bec, 0xee5d4140, 0x7a4a63ea, 0x9c9a8137, 0x4518e71d, 0x90bb01b9, 0xc8a0dbb1, 0x1c5859f8, 0xb71d74a2, 0xe627b83d, 0x63ad710f, 0x1b0590aa, 0x47f31b7d, 0x8b247df8, 0x680ceb24, 0x84c82aa9, 0xe16a482, 0x1b303012, 0xa6208757, 0x62842d6f, 0x27ae4f0c, 0x893d6f09, 0xa4687085, 0xa1ffc667, 0xea82a239, 0x17d73135, 0xcb5a9d69, 0x9fb058fc, 0x2998e75c, 0xf0fc9d2, 0xfb1c8ce, 0x79e3bc7c, 0x231366b5, 0x40392f2, 0x3a096a07, 0xd3b975cf, 0xfc49c863, 0xd9b437c6, 0xfea912dc, 0x67224518, 0x92ea1d92, 0x32594898, 0x7436f88d, 0x9a3b2801, 0x54930fd, 0x9273f995, 0x3b9bdecb, 0x44a0665f, 0xde53d2e8, 0xe611f8a1, 0x47dd26d5, 0x62006088, 0x18adef96, 0x40a57445, 0x96d64655, 0xc1c91992, 0xd8d12730, 0x7aeda909, 0x85eaf4c4, 0x60f20c05, 0xc1ed3293, 0xc322ca3f, 0x4c7ed042, 0x73c2b31d, 0xe0396b9d, 0x3020d47f, 0xd4c5d6c6, 0x523f71c0, 0x8ce47a37, 0x7859e90a, 0x7686e532, 0xdb086e00, 0x909ac2c3, 0x574ed015, 0x9e6fa778, 0x54ecf2a9, 0x2db98885, 0xec671188, 0xc0de2005, 0x5ec92a71, 0x78b628e2, 0x9c9816e2, 0x69e13f3d, 0x4fc3827b, 0xae393a9e, 0x6f45648, 0xa85e2113, 0xa2b591bc, 0xaaca0b99, 0x24033483, 0xf90f13b2, 0xa29a99cc};

void make_uint288(uint288 *out, uint *in, int len)
{
    int blocks = len / 9;
    unsigned int rand = 0;
    for (int i = 0; i < blocks; i++)
    {
        for (int j = 0; j < 9; j++)
        {
            for (int k = 0; k < 8; k++, rand++)
            {
                // ATTENTION!! UINT288 is big-endian, so least significant is data[8].

                // 1.稠密数据
                out[i * 9 + j].data[k + 1] = in[j * 8 + k] + rand;

                // 2.稀疏数据
                if (i%101 != 7)
                {
                    out[i * 9 + j].data[k + 1] = 0;
                }
            }
            out[i * 9 + j].data[0] = 0;
        }
    }
    for (int start = blocks * 9; start < len; start++)
    {
        out[start].data[8] = 0x3f3fffff + (rand++); // for intergrity.
    }
    // int start = blocks *
    printf("check uint288: ");
    for (int i = 0; i < 9; i++)
    {
        printf("0x%llx ", out[0].data[i]);
    }
    printf("\n");
}

#ifndef dh_mybig_copy
#define dh_mybig_copy(a, b) \
    {                       \
        (a)[0] = (b)[0];    \
        (a)[1] = (b)[1];    \
        (a)[2] = (b)[2];    \
        (a)[3] = (b)[3];    \
    }
#endif

void my_point_copy(Jpoint *from, Jpoint *to)
{
    dh_mybig_copy(to->x, from->x);
    dh_mybig_copy(to->y, from->y);
    dh_mybig_copy(to->x, from->z);
}

void print_big_arr(UINT64 *nums, int n)
{
    for (int i = 0; i < n; i++)
    {
        h_mybig_print(nums + i * 4);
        printf("\n");
    }
}
void print_jpoint_arr(Jpoint *nums, int n)
{
    for (int i = 0; i < n; i++)
    {
        h_print_pointJ(nums + i);
        printf("\n");
    }
}

void init_Jpoint_toOne(Jpoint *p, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            p[i].x[j] = h_Gx[j];
            p[i].y[j] = h_Gy[j];
            p[i].z[j] = h_ONE[j];
        }
    }
}

void make_data(Jpoint *in, int datasize)
{
    int block = 31;
    int blocknum = datasize / 31;
    int other = datasize - blocknum * 31;
    printf("datasize = %d\n", datasize);
    unsigned int rand = 1;
    for (int i = 0; i < blocknum; i++)
    {
        for (int j = 0; j < block; j++, rand++)
        {
            int idx = i * block + j;

            int wtf = (j / 2) * 4; // att: can't simplify this.
            if (j % 2 == 1)
            {
                dh_mybig_copy(in[idx].x, px + j);
                dh_mybig_copy(in[idx].y, py + j);
            }
            else
            {
                dh_mybig_copy(in[idx].x, px3 + j);
                dh_mybig_copy(in[idx].y, py3 + j);
            }
            // in[idx].y[0] += rand;
        }
    }
    for (int start = blocknum * 31; start < datasize; start++)
    {
        dh_mybig_copy(in[start].x, px);
        dh_mybig_copy(in[start].y, py);
    }
    printf("makes data\n");
}

void __global__ test_point_double(Jpoint *p1, Jpoint *p2)
{ // only for warm-up.
    UINT64 zinv[4], zinv2[4];
    int tx = threadIdx.x;
    ppoint_double(p1 + tx, p2 + tx);
    dh_mybig_moninv((p2 + tx)->z, zinv);
    dh_mybig_monmult_64(zinv, dc_R2, zinv);
    dh_mybig_monmult_64(zinv, zinv, zinv2);
    dh_mybig_monmult_64((p2 + tx)->x, zinv2, (p2 + tx)->x);
    dh_mybig_monmult_64(zinv, zinv2, zinv);
    dh_mybig_monmult_64((p2 + tx)->y, zinv, (p2 + tx)->y);
}

// these functions are not jp1 -> jp2(mont), but jp1 -> jp1(mont), jp2 -> jp2(mont).
void __global__ point_to_monjj(Jpoint *jp1, Jpoint *jp2)
{
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N_BIGNUM)
    {
        dh_mybig_monmult_64((jp1 + idx)->x, dc_R2, (jp1 + idx)->x);
        dh_mybig_monmult_64((jp1 + idx)->y, dc_R2, (jp1 + idx)->y);
        dh_mybig_monmult_64((jp1 + idx)->z, dc_R2, (jp1 + idx)->z);

        dh_mybig_monmult_64((jp2 + idx)->x, dc_R2, (jp2 + idx)->x);
        dh_mybig_monmult_64((jp2 + idx)->y, dc_R2, (jp2 + idx)->y);
        dh_mybig_monmult_64((jp2 + idx)->z, dc_R2, (jp2 + idx)->z);
    }
}

void __global__ point_from_monjj(Jpoint *jp1, Jpoint *jp2)
{
    // int tx = threadIdx.x;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N_BIGNUM)
    {
        dh_mybig_monmult_64((jp1 + idx)->x, dc_ONE, (jp1 + idx)->x);
        dh_mybig_monmult_64((jp1 + idx)->y, dc_ONE, (jp1 + idx)->y);
        dh_mybig_monmult_64((jp1 + idx)->z, dc_ONE, (jp1 + idx)->z);

        dh_mybig_monmult_64((jp2 + idx)->x, dc_ONE, (jp2 + idx)->x);
        dh_mybig_monmult_64((jp2 + idx)->y, dc_ONE, (jp2 + idx)->y);
        dh_mybig_monmult_64((jp2 + idx)->z, dc_ONE, (jp2 + idx)->z);
    }
}

__global__ void accumulate_sum(Jpoint *in)
{ // in = out[]
    int tx = threadIdx.x;
    int nthread = blockDim.x;
#if CONST_SCALE >= 20
    dh_ellipticAdd_JJ(&in[tx], &in[tx + 2048], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 256], &in[tx + 2048 + 256], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 512], &in[tx + 2048 + 512], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 768], &in[tx + 2048 + 768], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 1024], &in[tx + 2048 + 1024], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 1280], &in[tx + 2048 + 1280], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 1536], &in[tx + 2048 + 1536], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 1792], &in[tx + 2048 + 1792], &in[tx]);
#endif
#if CONST_SCALE >= 19
    dh_ellipticAdd_JJ(&in[tx], &in[tx + 1024], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 256], &in[tx + 1024 + 256], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 512], &in[tx + 1024 + 512], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 768], &in[tx + 1024 + 768], &in[tx]);
#endif
#if CONST_SCALE >= 18
    dh_ellipticAdd_JJ(&in[tx], &in[tx + 512], &in[tx]);
    dh_ellipticAdd_JJ(&in[tx + 256], &in[tx + 512 + 256], &in[tx]);
#endif
    for (int i = nthread, j = 1; i; i >>= 1)
    { // 5 = log2(thread)
        if (tx < i)
        {
            dh_ellipticAdd_JJ(&in[tx], &in[tx + i], &in[tx]);
        }
        __syncthreads();
    }
    // res == in[0]
}

#define from(x, y, z) *(from + (x) * DBC_level1 + (y) * DBC_level2 + (z))
const int DBC_level1 = 3 * DBC_MAXLENGTH;
const int DBC_level2 = 3;
__global__ void check_dbc(uint288 *nums, int *dbc_store, int *dbc_value)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int nthread = blockDim.x;
    int dbc_id = nthread * bx + tx;

    int *from = dbc_store + dbc_id * 6 * DBC_MAXLENGTH;
    if (tx == 120)
    {
        printf("check viability: ");
        for (int i = 0; i < 120; i++)
        {
            printf("0x%x ", u_pow23[0][i].data[8]);
        }
        printf("\n");
        // printf("step0: checkdata, tx=%d, bx=%d, nth=%d, data=0x%x\n", tx, bx, nthread, (unsigned int)u_pow23);
        // printf("step0: check_dbc, tx=%d, bx=%d, nthread=%d\n", threadIdx, blockIdx, nthread);
    }
    if (nums->data[0] != 0)
    {
        printf("panic: thread %d has illegal input\n", tx);
    }
    __syncthreads();
    int n = get_DBC(nums + dbc_id, from, dbc_value + dbc_id * 2);
    // printf("thread %d returns %d\n", tx, n);
    //__syncthreads();
    // get_DBC(nums + dbc_id, from, dbc_value + dbc_id * 2);
    for (int i = 0; i < DBC_MAXLENGTH; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            from(0, i, j) = from(n, i, j);
        }
    }
    dbc_value[dbc_id * 2] = dbc_value[dbc_id * 2 + n];
}

__device__ void my_check_point(Jpoint *pt)
{
    printf("[0x%x, 0x%x, 0x%x]\n", pt->x[0], pt->y[0], pt->z[0]);
}
#define RELEASE
__global__ void dbc_main(uint288 *nums, int *dbc_store, int *dbc_value, Jpoint *in, Jpoint *out)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int nthread = blockDim.x;
    int dbc_id = nthread * bx + tx;
    int n = 0;
    if (bx < CONST_BLOCK - 1)
        n = get_DBC(nums + dbc_id, dbc_store + dbc_id * 6 * DBC_MAXLENGTH, dbc_value + dbc_id * 2);
    int len = *(dbc_value + dbc_id * 2 + n);
    int *dbc = dbc_store + dbc_id * 6 * DBC_MAXLENGTH + n * 3 * DBC_MAXLENGTH;
#ifndef RELEASE
    __syncthreads();
    if (nums->data[0] != 0 || len < 1 || len > 110)
    {
        printf("panic: block %d thread %d has illegal input\n", bx, tx);
    }
    if (tx == 0 && bx == 15)
    {
        printf("check dbc value in block %d, len = %d: \n", bx, len);
        for (int j = 0; j < min(len, 10); j++)
        {
            int *wtf = dbc + j * 3;
            printf("{[%d]2^%d 3^%d} ", wtf[0], wtf[1], wtf[2]);
        }
        printf("\n");
    }
    __syncthreads();
#endif
    int cnt = 0;
    if (bx < CONST_BLOCK - 1)
        cnt = run_DBC_v2(in + dbc_id, out + dbc_id, dbc, len);
    // if (tx == 0) {
    //     printf("bx=%d runs %d ops, check point value\n", bx, cnt);
    //     printf("{[%d]2^%d 3^%d} ", dbc[0], dbc[1], dbc[2]);
    //     my_check_point(in+dbc_id);
    //     //my_check_point(out+dbc_id);
    // }
    //__syncthreads();
    // printf("(%d %d) ", bx, tx);
}

int main()
{
    hipSetDevice(0);

    struct timeval s1, e1;
    long long time_use = 1;
    int nB, nT;

    hipOccupancyMaxPotentialBlockSize(&nB, &nT, dbc_main);
    printf("NB=%d,NT=%d\n", nB, nT);

    uint288 *scalar;
    uint288 *d_scalar;
    int *dbc_store_host;
    int *dbc_len_host;
    int *dbc_store_device;
    int *dbc_len_device;

    Jpoint *h_p1;
    Jpoint *h_p2;
    Jpoint *d_p1;
    Jpoint *d_p2;
    Jpoint *t_p1;
    Jpoint *t_p2;
    Jpoint *td_p1;
    Jpoint *td_p2;

    // number init.
    scalar = (uint288 *)malloc(BATCHSIZE * sizeof(uint288));
    dbc_store_host = (int *)malloc(6 * DBC_MAXLENGTH * BATCHSIZE * sizeof(int)); // dbc_store[2][DBC_MAXSIZE][3];
    dbc_len_host = (int *)malloc(2 * BATCHSIZE * sizeof(int));                   // dbc_len[2];
    int dbc_size = 6 * DBC_MAXLENGTH;
    make_uint288(scalar, dx2, BATCHSIZE); // init int288

    CUDA_SAFE_CALL(hipMalloc((void **)&d_scalar, sizeof(uint288) * BATCHSIZE));
    CUDA_SAFE_CALL(hipMalloc((void **)&dbc_store_device, 6 * DBC_MAXLENGTH * BATCHSIZE * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void **)&dbc_len_device, 2 * BATCHSIZE * sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(dbc_store_device, 0, sizeof(dbc_store_device)));
    // CUDA_SAFE_CALL(hipMemcpy(d_dbc, dbc, sizeof(DBC)*N_BIGNUM, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_scalar, scalar, BATCHSIZE * sizeof(uint288), hipMemcpyHostToDevice));

    // point init
    t_p1 = (Jpoint *)malloc(100 * sizeof(Jpoint));
    t_p2 = (Jpoint *)malloc(100 * sizeof(Jpoint));
    for (int i = 0; i < 100; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            t_p1[i].x[j] = px[i * 4 + j];
            t_p1[i].y[j] = py[i * 4 + j];
            t_p1[i].z[j] = h_ONE[j];
        }
    }
    h_p1 = (Jpoint *)malloc(BATCHSIZE * sizeof(Jpoint));
    h_p2 = (Jpoint *)malloc(BATCHSIZE * sizeof(Jpoint));
    init_Jpoint_toOne(h_p1, BATCHSIZE);
    make_data(h_p1, BATCHSIZE);
    CUDA_SAFE_CALL(hipMalloc((void **)&d_p1, BATCHSIZE * sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_p2, BATCHSIZE * sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMemcpy(d_p1, h_p1, N_POINT * sizeof(Jpoint), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_p2, h_p2, N_POINT * sizeof(Jpoint), hipMemcpyHostToDevice));

    //=== TEST =====
    printf("ready\n");
    gettimeofday(&s1, NULL);
    check_dbc<<<1, 128>>>(d_scalar, dbc_store_device, dbc_len_device);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

    // CUDA_SAFE_CALL(hipMemcpy(h_p1,d_p1,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(hipMemcpy(h_p2,d_p2,N_POINT*sizeof(Jpoint),hipMemcpyDeviceToHost));
    // print_jpoint_arr(h_p1,1);
    CUDA_SAFE_CALL(hipMemcpy(dbc_store_host, dbc_store_device, 6 * DBC_MAXLENGTH * BATCHSIZE * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(dbc_len_host, dbc_len_device, 2 * BATCHSIZE * sizeof(int), hipMemcpyDeviceToHost));
    gettimeofday(&e1, NULL);
    time_use = (e1.tv_sec - s1.tv_sec) * 1000000 + (e1.tv_usec - s1.tv_usec); // 微秒
    printf("test usage is %ld us\n", time_use);

    //==== WARM UP =====
    gettimeofday(&s1, NULL);
    point_to_monjj<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_p1, d_p2);

    // testbasemul<<<N_BLOCK,N_THREAD_PER_BLOCK>>>(d_p1,d_num);
    test_point_double<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_p1, d_p2);

    point_from_monjj<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_p1, d_p2);

    gettimeofday(&e1, NULL);
    time_use = (e1.tv_sec - s1.tv_sec) * 1000000 + (e1.tv_usec - s1.tv_usec); // 微秒
    printf("warm up time usage is %ld us\n", time_use);

    //==== MAIN =====
    gettimeofday(&s1, NULL);

    point_to_monjj<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_p1, d_p2);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

#ifdef PRECOMPUTE
    multi_scalar_multiple<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_dbc, d_p1, d_p2);
#else
    dbc_main<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_scalar, dbc_store_device, dbc_len_device, d_p1, d_p2);
#endif

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();

#if CONST_SCALE <= 17
    accumulate_sum<<<1, N_BLOCK / 2>>>(d_p2);
#else
    accumulate_sum<<<1, N_THREAD_PER_BLOCK>>>(d_p2);
#endif

    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
#ifdef DEBUG
    printf("dbc dp1:\n");
    CUDA_SAFE_CALL(hipMemcpy(t_p1, d_p1, 100 * sizeof(Jpoint), hipMemcpyDeviceToHost));
    for (int i = 0; i < 10; i++)
    {
        printf("0x%llx 0x%llx 0x%llx 0x%llx\n", t_p1[i].x[0], t_p1[i].x[1], t_p1[i].x[2], t_p1[i].x[3]);
    }
#endif

    point_from_monjj<<<N_BLOCK, N_THREAD_PER_BLOCK>>>(d_p2, d_p1);
    hipDeviceSynchronize();

    CUDA_CHECK_ERROR();
    gettimeofday(&e1, NULL);
    time_use = (e1.tv_sec - s1.tv_sec) * 1000000 + (e1.tv_usec - s1.tv_usec); // 微秒
    printf("time_use is %ld us\n", time_use);

    // used in accuracy test.
    // printf("Check DBC ans: \n");
    // for (int i = 0; i < 8; i++) {
    //     printf("\nCheck DBC %d ans: \n", i+1);
    //     for (int j = 0; j < dbc_len_host[2 * i]; j++) {
    //         int* wtf = dbc_store_host + i * 6 * DBC_MAXLENGTH + j * 3;
    //         printf("{[%d]2^%d 3^%d} ", wtf[0], wtf[1], wtf[2]);
    //     }
    // }

    free(scalar);
    free(dbc_store_host);
    // free(h_p1);
    // free(h_p2);
    CUDA_SAFE_CALL(hipFree(d_scalar));
    CUDA_SAFE_CALL(hipFree(dbc_store_device));
    CUDA_SAFE_CALL(hipFree(dbc_len_device));
    CUDA_SAFE_CALL(hipFree(d_p1));
    CUDA_SAFE_CALL(hipFree(d_p2));
}
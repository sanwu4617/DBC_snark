#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <random>
#include "gmp.h"
#include "gpuec256.h"
#include "openssl/sha.h"
#include "cuda_common.h"
#include "sha256.cuh"
#include<sys/time.h>

#define dh_mybig_copy(a,b) {(a)[0]=(b)[0];(a)[1]=(b)[1];(a)[2]=(b)[2];(a)[3]=(b)[3];}

typedef struct IntermediateVar{		
	UINT64 t1[4];
    UINT64 t2[4];
}InterVar;


const UINT64 h_Gx[4]={0x59F2815B16F81798L,0x029BFCDB2DCE28D9L,0x55A06295CE870B07L,0x79BE667EF9DCBBACL};
const UINT64 h_Gy[4]={0x9C47D08FFB10D4B8L,0xFD17B448A6855419L,0x5DA4FBFC0E1108A8L,0x483ADA7726A3C465L};
const UINT64 h_Gz[4]={0x1L,0x0L,0x0L,0x0L};

const UINT64 h_Gx_mon[4]={0xd7362e5a487e2097L,0x231e295329bc66dbL,0x979f48c033fd129cL,0x9981e643e9089f48L};
const UINT64 h_Gy_mon[4]={0xb15ea6d2d3dbabe2L,0x8dfc5d5d1f1dc64dL,0x70b6b59aac19c136L,0xcf3f851fd4a582d6L};
const UINT64 h_Gz_mon[4]={0x1000003d1L,0x0L,0x0L,0x0L};
const UINT64 h_R2[4]={0x000007a2000e90a1L,0x1L,0x0L,0x0L};
Jpoint pG_mon={{0xd7362e5a487e2097L,0x231e295329bc66dbL,0x979f48c033fd129cL,0x9981e643e9089f48L},
                {0xb15ea6d2d3dbabe2L,0x8dfc5d5d1f1dc64dL,0x70b6b59aac19c136L,0xcf3f851fd4a582d6L},
                {0x1000003d1L,0x0L,0x0L,0x0L}};

BPSetupParams h_params;
initParamRandom h_ranParams;
std::string SEED="gyy hello world";

BPSetupParams *d_params;
initParamRandom *d_ranParams;

BPProve *d_prove;

UINT64 *d_xyz;
UINT64 h_xyz[12];

InterVar *d_iv;

UINT64 *d_aLR;
UINT64 h_aLR[4*64];
UINT64 *d_VLR;
UINT64 *d_z22nyn;

// void h_mybig_print(const UINT64 *a)
// {
// 	int i;
// 	unsigned char *t=(unsigned char*) a;
// 	for(i=32-1;i>=0;i--) if(*(t+i)) break;//把0都跳过去
// 	if(i<0) printf("0");
// 	else
// 	{
// 		printf("%x",*(t+i)&0xff);//第一个0不打印
// 		for(i=i-1;i>=0;i--)printf("%02x",*(t+i)&0xff);
//     }
//     printf("\n");	
// }
// void h_print_pointJ(const Jpoint *pt)
// {
// 	/*
// 	if(dh_iszero_J(pt))
// 	{ 
// 		printf("(Infinity)\n");
// 		return;
// 	}*/
// 	printf("x: ");h_mybig_print(pt->x);printf("\n");
// 	printf("y: ");h_mybig_print(pt->y);printf("\n");
// 	printf("z: ");h_mybig_print(pt->z);printf("\n");
// }

 void uint642str(UINT64* x,char *s){
    int cur=0;
    for(int i=3;i>=0;i--){
        for(int j=0;j<16;j++){
            // std::cout<<((x[i]>>((15-j)*4))&0xf==0)<<std::endl;
            // printf("%d\n",(x[i]>>((15-j)*4))&0xf);
            // printf("%d\n",(int)(x[i]>>((15-j)*4))&0xf == (int)0);
            if(cur==0 && (((x[i]>>((15-j)*4))&0xf) ==0)){
                continue;
            }
            sprintf(s + cur, "%x", (x[i]>>((15-j)*4))&0xf);
            cur++;
            // sprintf(s + (3-i)*16+j*2, "%02x", (x[i]>>((7-j)*8))&0xff);
            // printf("%x\n",(x[i]>>((7-j)*8))&0xff);
        }
        
    }
    s[cur]=0;
}
// __host__  void uint642byte(UINT64 *x,char *h){
//     for(int i=0;i<4;i++){
//         for(int j=0;j<8;j++){
//             h[i*8+j]=(x[3-i]>>((7-j)*8))&0xff;
//         }
//     }
// }
__device__ __host__ void uint642byte(UINT64 *x, unsigned char *h){
    for(int i=0;i<4;i++){
        for(int j=0;j<8;j++){
            h[i*8+j]=(x[3-i]>>((7-j)*8))&0xff;
        }
    }
}

void inline JpointCpy(Jpoint *jp,const UINT64 *x,const UINT64 *y,const UINT64 *z){
    jp->x[0] = x[0];
    jp->x[1] = x[1];
    jp->x[2] = x[2];
    jp->x[3] = x[3];

    jp->y[0] = y[0];
    jp->y[1] = y[1];
    jp->y[2] = y[2];
    jp->y[3] = y[3];

    jp->z[0] = z[0];
    jp->z[1] = z[1];
    jp->z[2] = z[2];
    jp->z[3] = z[3];
}

void str2uint64(char *s,UINT64* x){
    std::string tmps(s);
    // std::cout<<"size="<<tmps.size()<<std::endl;
    // std::cout<<"tmps="<<tmps<<std::endl;
    // std::cout<<"s="<<s<<std::endl;
    // std::cout<<tmps<<std::endl;
    // std::cout<<"0="<<tmps.substr(tmps.size()-16,16).c_str()<<std::endl;
    // std::cout<<"1="<<tmps.substr(tmps.size()-32,16).c_str()<<std::endl;
    // std::cout<<"2="<<tmps.substr(tmps.size()-48,16).c_str()<<std::endl;
    // std::cout<<"3="<<tmps.substr(0,16-(64-tmps.size())).c_str()<<std::endl;
    x[0]=strtoull(tmps.substr(tmps.size()-16,16).c_str(),NULL,16);
    x[1]=strtoull(tmps.substr(tmps.size()-32,16).c_str(),NULL,16);
    x[2]=strtoull(tmps.substr(tmps.size()-48,16).c_str(),NULL,16);
    x[3]=strtoull(tmps.substr(0,16-(64-tmps.size())).c_str(),NULL,16);
    // h_mybig_print(x);
    // std::cout<<std::endl;
}

void sha256(const std::string &srcStr, std::string &encodedHexStr)  
{  
    // 调用sha256哈希    
    unsigned char mdStr[33] = {0};  
    SHA256((const unsigned char *)srcStr.c_str(), srcStr.length(), mdStr);  
  
    // 哈希后的字符串    
    // 哈希后的十六进制串 32字节    
    char buf[65] = {0};  
    char tmp[3] = {0};  
    for (int i = 0; i < 32; i++)  
    {  
        sprintf(tmp, "%02x", mdStr[i]);  
        strcat(buf, tmp);  
    }  
    buf[64] = '\0';   
    encodedHexStr = std::string(buf);  
} 
int check_quadratic_residue(mpz_t num){
    mpz_t t1,t2,d;
    mpz_init(t1);
    mpz_init(t2);
    mpz_init(d);
    mpz_set_str(d,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F",16);

    mpz_sub_ui(t1,d,1);
    mpz_tdiv_q_ui(t2,t1,2);
    // gmp_printf("%#Zx\n",t1);
    mpz_powm(t2,num,t2,d);
    // gmp_printf("%#Zx\n",t2);
    if(mpz_cmp_ui(t2,1)==0){
        // gmp_printf("right\n");
        return 1;
    }
    if(mpz_cmp(t2,t1)==0){
        // gmp_printf("not\n");
        return -1;
    }
    return 0;
}

void mapToGroup(const std::string &s,Jpoint* jp){
    std::string tmphex;
    sha256(s,tmphex);
    // std::cout<<tmphex<<std::endl;
    mpz_t hexr,d;
    mpz_t t1,t2;
    mpz_t rx,ry;
    mpz_init(hexr);
    mpz_init(d);
    mpz_set_str(d,"FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F",16);
    mpz_init(t1);
    mpz_init(t2);
    mpz_init(rx);
    mpz_init(ry);
    mpz_set_str(hexr,tmphex.c_str(),16);


    for(int i=0;i<2048;i+=1){
        mpz_add_ui(rx,hexr,i);
        mpz_powm_ui(hexr,rx,3,d);
        mpz_add_ui(hexr,hexr,7);
        mpz_mod(hexr,hexr,d);


        // mpz_sub_ui(t1,d,1);
        // mpz_tdiv_q_ui(t2,t1,2);
        // gmp_printf("%#Zx\n",t1);
        // mpz_powm(t2,hexr,t2,d);
        if(check_quadratic_residue(hexr)==1){
            // gmp_printf("right\n");
            break;
        }
        if(check_quadratic_residue(hexr)==-1){
            // gmp_printf("not\n");
        }
        // gmp_printf("%#Zx\n",t2);
    }
    //这里hexr就是满足条件的二次剩余，现在要解二次剩余,rx里存的是x值
    //由于secp256k1曲线素数的特殊性，所以直接采用Tonelli-Shanks算法
    mpz_add_ui(t1,d,1);

    mpz_tdiv_qr_ui(t1,t2,t1,4);
    mpz_powm(ry,hexr,t1,d);

    // gmp_printf("%#Zx\n",rx);
    // gmp_printf("%#Zx\n",ry);
    char jx[65]={0};
    char jy[65]={0};
    mpz_get_str(jx,16,rx);
    mpz_get_str(jy,16,ry);
    str2uint64(jx,jp->x);
    str2uint64(jy,jp->y);
    jp->z[0]=0x1L;
    // h_print_pointJ(jp);
    //验证

    // mpz_powm_ui(t1,rx,3,d);
    // mpz_add_ui(t1,t1,7);
    // mpz_mod(t1,t1,d);
    // mpz_powm_ui(t2,ry,2,d);
    // mpz_mod(t2,t2,d);
    // if(mpz_cmp(t1,t2)==0){
    //     gmp_printf("x and y right\n");
    // }else{
    //     gmp_printf("WRONG!!!\n");
    // }
    
    //下面是Cipolla算法第一步找a^2-n为非二次剩余（由于最终采用了别的方法，所以注释掉）
    /*
    while(1){
        mpz_add_ui(t2,hexr,i);
        mpz_powm_ui(t1,t2,2,d);
        mpz_sub(t1,t1,hexr);
        mpz_mod(t1,t1,d);
        if(check_quadratic_residue(t1)==-1){
            break;
        }
        i++;
    }
    gmp_printf("i=%d\n",i);
    gmp_printf("%#Zx\n",t2);
    gmp_printf("%#Zx\n",hexr);
    */


}
void gen_random_uint64(std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> &engine,UINT64 s[4]){
    // std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine(clock());
    for(int i=0;i<4;i++){
        s[i] = engine();
    }
    if(s[0]==0xFFFFFFFFFFFFFFFF&&s[1]==0xFFFFFFFFFFFFFFFF&&
        s[2]==0xFFFFFFFFFFFFFFFF&&s[3]>0xFFFFFFFEFFFFFC2F){
            s[3]-=0xFFFFFFFEFFFFFC2F;
    }
}

void init_random_param(){
    std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine(19970504);
    // std::independent_bits_engine<std::default_random_engine,64,unsigned long long int> engine;
    gen_random_uint64(engine,h_ranParams.gamma);
    gen_random_uint64(engine,h_ranParams.alpha);
    gen_random_uint64(engine,h_ranParams.rho);
    gen_random_uint64(engine,h_ranParams.tau1);
    gen_random_uint64(engine,h_ranParams.tau2);
    
    

    for(int i=0;i<32;i++){

        gen_random_uint64(engine,&(h_ranParams.SL[i*4]));
        gen_random_uint64(engine,&(h_ranParams.SR[i*4]));         
        
        
    }
    // h_mybig_print(ranParams.gamma);
    // h_mybig_print(ranParams.alpha);
}

__host__ __device__ void Jpoint2Apoint(Jpoint *A,Jpoint *ret){
    #ifdef __CUDA_ARCH__
		const UINT64 *R2=dc_R2;
        const UINT64 *ONE=dc_ONE;
	#else
		const UINT64 *R2=h_R2;
        const UINT64 *ONE=h_ONE;
	#endif
    UINT64 t1[4],t2[4];
    dh_mybig_moninv(A->z,t1);
    dh_mybig_monmult_64(t1,t1,t2);
    dh_mybig_monmult_64(t1,t2,t1);
    dh_mybig_monmult_64(A->x,R2,ret->x);
    dh_mybig_monmult_64(A->y,R2,ret->y);
    dh_mybig_monmult_64(ret->x,t2,ret->x);
    dh_mybig_monmult_64(ret->y,t1,ret->y);

    dh_mybig_monmult_64(ret->x,ONE,ret->x);
    dh_mybig_monmult_64(ret->y,ONE,ret->y);
    ret->z[0]=1;
    ret->z[1]=0;
    ret->z[2]=0;
    ret->z[3]=0;
}

void HashBP(Jpoint* A,Jpoint* S,UINT64 *o1,UINT64 *o2){
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    // unsigned char tmp[32];

    char tmp[65];
    char o1str[65];
    uint642str(A->x,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(A->y,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(S->x,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    uint642str(S->y,tmp);
    SHA256_Update(&sha256, tmp, strlen(tmp));
    SHA256_Final(hash, &sha256);
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(o1str + (i * 2), "%02x", hash[i]);
    }
    o1str[64] = 0;
    str2uint64(o1str,o1);
    // printf("tmp=%s\n",o1str);
    // h_mybig_print(o1);
    if(o2==NULL) return;
    SHA256_CTX sha2562;
    SHA256_Init(&sha2562);
    // unsigned char tmp[32];

    uint642str(A->x,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(A->y,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(S->x,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    uint642str(S->y,tmp);
    SHA256_Update(&sha2562, tmp, strlen(tmp));
    SHA256_Update(&sha2562, o1str, strlen(o1str));
    SHA256_Final(hash, &sha2562);
    for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    {
        sprintf(tmp + (i * 2), "%02x", hash[i]);
    }
    str2uint64(tmp,o2);
    // printf("tmp=%s\n",tmp);
    // h_mybig_print(o2);
    // o2[64] = 0;
}

//不转成str
void HashBP_V2(Jpoint* A,Jpoint* S,UINT64 *o1,UINT64 *o2){
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    // unsigned char tmp[32];

    unsigned char tmp[65];
    char o1str[65];
    uint642byte(A->x,tmp);
    SHA256_Update(&sha256, tmp, 32);
    uint642byte(A->y,tmp);
    SHA256_Update(&sha256, tmp, 32);
    uint642byte(S->x,tmp);
    SHA256_Update(&sha256, tmp, 32);
    uint642byte(S->y,tmp);
    SHA256_Update(&sha256, tmp, 32);
    SHA256_Final(hash, &sha256);
    // for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    // {
    //     sprintf(o1str + (i * 2), "%02x", hash[i]);
    // }
    // for(int i=0;i<32;i++){
    //     printf("%x ",hash[i]);
    // }
    // printf("\n");
    // o1str[64] = 0;
    // str2uint64(o1str,o1);
    o1[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
    o1[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
            |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
    o1[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
            |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
    o1[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
            |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
    // printf("tmp=%s\n",o1str);
    // printf("myO1=\n");
    // h_mybig_print(o1);
    if(o2==NULL) return;
    SHA256_CTX sha2562;
    SHA256_Init(&sha2562);
    // unsigned char tmp[32];

    uint642byte(A->x,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(A->y,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(S->x,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(S->y,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    uint642byte(o1,tmp);
    SHA256_Update(&sha2562, tmp, 32);
    SHA256_Final(hash, &sha2562);
    // for(int i = 0; i < SHA256_DIGEST_LENGTH; i++)
    // {
    //     sprintf(tmp + (i * 2), "%02x", hash[i]);
    // }
    // str2uint64(tmp,o2);
    o2[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
    o2[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
            |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
    o2[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
            |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
    o2[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
            |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
    
    // printf("myO2=\n");
    // h_mybig_print(o2);
    // printf("tmp=%s\n",tmp);
    // h_mybig_print(o2);
    // o2[64] = 0;
}


void compute_al_ar(int v,UINT64 *al,UINT64 *ar,int n){
    for(int i=0;i<n;i++){
        al[i*4]=(v>>i) &0x1;
        al[i*4+1]=0;
        al[i*4+2]=0;
        al[i*4+3]=0;
        if(al[i*4]){
            ar[i*4]=0;
            ar[i*4+1]=0;
            ar[i*4+2]=0;
            ar[i*4+3]=0;
        }else{
            ar[i*4  ]=0xBFD25E8CD0364141-1;
            ar[i*4+1]=0xBAAEDCE6AF48A03B;
            ar[i*4+2]=0xFFFFFFFFFFFFFFFE;
            ar[i*4+3]=0xFFFFFFFFFFFFFFFF;
            //  
        }
        
    }
}

__host__ void commitG1(Jpoint *ret,int v,UINT64 *r,Jpoint* h){
    UINT64 value[4]={(UINT64)v,0,0,0};
    Jpoint tp;
    // h_mybig_print(value);
    // printf("h=\n");
    // h_print_pointJ(h);
    dh_point_mult_finalversion(&pG_mon,value,ret);
    // dh_mybig_monmult_64(h->x,h_R2,h->x);
    // dh_mybig_monmult_64(h->y,h_R2,h->y);
    // dh_mybig_monmult_64(h->z,h_R2,h->z);
    dh_point_mult_finalversion(h,r,&tp);
    
    dh_ellipticAdd_JJ(ret,&tp,ret);

    
    // printf("r=\n");
    // h_mybig_print(r);
    dh_mybig_monmult_64(ret->x,h_ONE,ret->x);
    dh_mybig_monmult_64(ret->y,h_ONE,ret->y);
    dh_mybig_monmult_64(ret->z,h_ONE,ret->z);
    // h_print_pointJ(ret);
}

void jpoint_to_mon(Jpoint *p){
    dh_mybig_monmult_64(p->x,h_R2,p->x);
    dh_mybig_monmult_64(p->y,h_R2,p->y);
    dh_mybig_monmult_64(p->z,h_R2,p->z);
}
void jpoint_from_mon(Jpoint *p){
    dh_mybig_monmult_64(p->x,h_ONE,p->x);
    dh_mybig_monmult_64(p->y,h_ONE,p->y);
    dh_mybig_monmult_64(p->z,h_ONE,p->z);
}
void trans_to_mon(){
    jpoint_to_mon(&(h_params.G));
    jpoint_to_mon(&(h_params.H));
    for(int i=0;i<32;i++){
        jpoint_to_mon(&(h_params.Gg[i]));
        jpoint_to_mon(&(h_params.Hh[i]));
    }
}
void trans_to_mon_N(){
    //SL SR AL AR y z
    // dh_mybig_monmult_64_modN(h_xyz+4,h_R2modN,h_xyz+4);
    // dh_mybig_monmult_64_modN(h_xyz+8,h_R2modN,h_xyz+8);

    // for(int i=0;i<32;i++){
    //     dh_mybig_monmult_64_modN(h_ranParams->SL+i*4,h_R2modN,h_ranParams->SL+i*4);
    //     dh_mybig_monmult_64_modN(h_ranParams->SR+i*4,h_R2modN,h_ranParams->SR+i*4);

        
    // }
    for(int i=0;i<32*2;i++){
        dh_mybig_monmult_64_modN(h_aLR+i*4,h_R2modN,h_aLR+i*4);
        
    }

}

void trans_from_mon(){

}

void cal_minus_Jpoint(Jpoint *ret,Jpoint *src){
    ret->x[0] = src->x[0];
    ret->x[1] = src->x[1];
    ret->x[2] = src->x[2];
    ret->x[3] = src->x[3];

    ret->z[0] = src->z[0];
    ret->z[1] = src->z[1];
    ret->z[2] = src->z[2];
    ret->z[3] = src->z[3];

    // ret->y[0] = src->y[0];
    // ret->y[1] = src->y[1];
    // ret->y[2] = src->y[2];
    // ret->y[3] = src->y[3];

    dh_mybig_modsub_64(h_p,src->y,ret->y);

}

void cal_A(UINT64* al,UINT64* ar, Jpoint* ret){
    dh_point_mult_finalversion(&(h_params.H),h_ranParams.alpha,ret);
    Jpoint tp;
    for(int i=0;i<32;i++){
        if(al[i*4]==1){
            dh_ellipticAdd_JJ(ret,&(h_params.Gg[i]),ret);
        }
        if(ar[i*4]!=0){
            // dh_point_mult_finalversion(&(h_params.Hh[i]),&ar[i*4],&tp);
            // dh_ellipticAdd_JJ(ret,&tp,ret);
            cal_minus_Jpoint(&tp,&(h_params.Hh[i]));
            dh_ellipticAdd_JJ(ret,&tp,ret);
        }
    }
    jpoint_from_mon(ret);
}



__global__ void kernel_hashBP(Jpoint *A,Jpoint *B,UINT64 *x,UINT64 *y,int N){
    int tx = threadIdx.x;
    int data_len = 64;
    
    if(tx==0){
        unsigned char tmp[32];
        unsigned char hash[32];
        // char tmp2[65];
        // char o1str[65];
        MYSHA256_CTX ctx;
		sha256_init(&ctx);
        uint642byte(A->x,tmp);
		sha256_update(&ctx, tmp, 32);
        uint642byte(A->y,tmp);
		sha256_update(&ctx, tmp, 32);
        uint642byte(B->x,tmp);
		sha256_update(&ctx, tmp, 32);
        uint642byte(B->y,tmp);
		sha256_update(&ctx, tmp, 32);
		sha256_final(&ctx, hash);
        // tmp2[64] = '\0';
        // printf("str = %s\n",tmp2);
        // d_mybig_print(x);

        x[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
        x[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
                |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
        x[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
                |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
        x[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
                |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;

        // d_mybig_print(x);

        if(y==NULL) return ;
        MYSHA256_CTX ctx2;
		sha256_init(&ctx2);
        uint642byte(A->x,tmp);
		sha256_update(&ctx2, tmp, 32);
        uint642byte(A->y,tmp);
		sha256_update(&ctx2, tmp, 32);
        uint642byte(B->x,tmp);
		sha256_update(&ctx2, tmp, 32);
        uint642byte(B->y,tmp);
		sha256_update(&ctx2, tmp, 32);
        sha256_update(&ctx2, hash, 32);
		sha256_final(&ctx2, hash);
        y[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
        y[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
                |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
        y[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
                |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
        y[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
                |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
        // d_mybig_print(y);
    }
}

__device__ void device_hashBP(Jpoint *A,Jpoint *B,UINT64 *x,UINT64 *y){
    
    
    
        unsigned char tmp[32];
        unsigned char hash[32];
        // char tmp2[65];
        // char o1str[65];
        MYSHA256_CTX ctx;
		sha256_init(&ctx);
        uint642byte(A->x,tmp);
		sha256_update(&ctx, tmp, 32);
        uint642byte(A->y,tmp);
		sha256_update(&ctx, tmp, 32);
        uint642byte(B->x,tmp);
		sha256_update(&ctx, tmp, 32);
        uint642byte(B->y,tmp);
		sha256_update(&ctx, tmp, 32);
		sha256_final(&ctx, hash);
        // tmp2[64] = '\0';
        // printf("str = %s\n",tmp2);
        // d_mybig_print(x);
        // for(int i=0;i<32;i++){
        //     printf("%x ",hash[i]);
        // }
        // printf("\n");

        x[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
        x[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
                |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
        x[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
                |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
        x[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
                |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;

        // d_mybig_print(x);

        if(y==NULL) return ;
        MYSHA256_CTX ctx2;
		sha256_init(&ctx2);
        uint642byte(A->x,tmp);
		sha256_update(&ctx2, tmp, 32);
        uint642byte(A->y,tmp);
		sha256_update(&ctx2, tmp, 32);
        uint642byte(B->x,tmp);
		sha256_update(&ctx2, tmp, 32);
        uint642byte(B->y,tmp);
		sha256_update(&ctx2, tmp, 32);
        sha256_update(&ctx2, hash, 32);
		sha256_final(&ctx2, hash);
        y[3]=(UINT64)hash[7]|((UINT64)hash[6])<<8|((UINT64)hash[5])<<16|((UINT64)hash[4])<<24
            |((UINT64)hash[3])<<32|((UINT64)hash[2])<<40|((UINT64)hash[1])<<48 |((UINT64)hash[0])<<56;
        y[2]=(UINT64)hash[15]|((UINT64)hash[14])<<8|((UINT64)hash[13])<<16|((UINT64)hash[12])<<24
                |((UINT64)hash[11])<<32|((UINT64)hash[10])<<40|((UINT64)hash[9])<<48 |((UINT64)hash[8])<<56;
        y[1]=(UINT64)hash[23]|((UINT64)hash[22])<<8|((UINT64)hash[21])<<16|((UINT64)hash[20])<<24
                |((UINT64)hash[19])<<32|((UINT64)hash[18])<<40|((UINT64)hash[17])<<48 |((UINT64)hash[16])<<56;
        y[0]=(UINT64)hash[31]|((UINT64)hash[30])<<8|((UINT64)hash[29])<<16|((UINT64)hash[28])<<24
                |((UINT64)hash[27])<<32|((UINT64)hash[26])<<40|((UINT64)hash[25])<<48 |((UINT64)hash[24])<<56;
        // d_mybig_print(y);
    
}

__global__ void kernel_cal_S(Jpoint *ret,UINT64* rho,UINT64* sl,UINT64* sr,Jpoint *H,Jpoint *Gg,Jpoint *Hh,UINT64 *xyz,Jpoint *A){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    
    __shared__ Jpoint tp[64];
    if(bx==0){
        
        if(tx<32){
            dh_point_mult_finalversion(Gg+tx,sl+(tx*4),tp+tx);
            // if(tx==0)
            // d_mybig_print(sl+(tx*4));
            // printf("\n");
        }else if(tx<64){
            dh_point_mult_finalversion(Hh+tx,sr+((tx-32)*4),tp+tx);
            // if(tx==0)
            // d_mybig_print(sr+(tx*4));
            // printf("\n");
        }
        
        __syncthreads();
        for(int i=1;i<=32;i*=2){
            if(tx%(i*2)==0)
            dh_ellipticAdd_JJ(tp+tx,tp+tx+i,tp+tx);
            
        }
        
        // if(tx==0){
        //     dh_ellipticAdd_JJ(tp,tp+16,ret);
        // }if(tx==32){
        //     dh_ellipticAdd_JJ(tp+32,tp+48,ret);
        // }
        if(tx==0){
            dh_point_mult_finalversion(H,rho,&tp[1]);
            dh_ellipticAdd_JJ(&tp[0],&tp[1],&tp[0]);
        }
        
        
        if(tx==0){
            dh_mybig_monmult_64(tp[0].x,dc_ONE,tp[0].x);
            dh_mybig_monmult_64(tp[0].y,dc_ONE,tp[0].y);
            dh_mybig_monmult_64(tp[0].z,dc_ONE,tp[0].z);
            // printf("S='\n");
            // d_mybig_print(tp[0].x);
            // d_mybig_print(tp[0].y);
            // d_mybig_print(tp[0].z);

            dh_mybig_copy(ret->x,tp[0].x);
            dh_mybig_copy(ret->y,tp[0].y);
            dh_mybig_copy(ret->z,tp[0].z);

            Jpoint2Apoint(&tp[0],&tp[0]);
            Jpoint2Apoint(A,&tp[1]);
            UINT64 y[4],z[4];
            device_hashBP(&tp[1],&tp[0],y,z);
            // printf("device HashBP y=\n");
            // d_mybig_print(y);
            // printf("device HashBP z=\n");
            // d_mybig_print(z);
            dh_mybig_monmult_64_modN(y,dc_R2modN,xyz+4);
            dh_mybig_monmult_64_modN(z,dc_R2modN,xyz+8);
        }
        // dh_mybig_copy(ret->x,tp[0].x);
        // dh_mybig_copy(ret->y,tp[0].y);
        // dh_mybig_copy(ret->z,tp[0].z);
    }
    
}

__global__ void kernel_cal_t12(Jpoint *T12,Jpoint *H,UINT64 *xyz,UINT64 *al,UINT64 *ar,UINT64 *sl,UINT64 *sr,UINT64 *tau12,UINT64 *z22nyn){
    int tx = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(tx>=32) return;
    __shared__ UINT64 sh_ip[4*96];
    UINT64 lh[4],rh[4];
    dh_mybig_monmult_64_modN(sl+tx*4,dc_R2modN,sl+tx*4);
    dh_mybig_monmult_64_modN(sr+tx*4,dc_R2modN,sr+tx*4);



    dh_mybig_modsub_64_modN(al+tx*4,xyz+8,sh_ip+tx*4);
    dh_mybig_modexp_ui32_modN(xyz+4,tx,sh_ip+(tx+64)*4);
    dh_mybig_monmult_64_modN(sh_ip+(tx+64)*4,sr+tx*4,sh_ip+(tx+32)*4);

    dh_mybig_monmult_64_modN(sh_ip+(tx)*4,sh_ip+(tx+32)*4,sh_ip+(tx)*4);
    __syncthreads();
    for(int i=16;i>0;i>>=1){
        if(tx<i){
            dh_mybig_modadd_64_modN(sh_ip+(tx)*4,sh_ip+(tx+i)*4,sh_ip+(tx)*4);
        }
        __syncthreads();
    }
    
    if(tx==0){
        dh_mybig_copy(lh,sh_ip);
        // printf("<al-z,ynsr>[0]=\n");
        // dh_mybig_monmult_64_modN(sh_ip,dc_ONE,sh_ip);
        // d_mybig_print(sh_ip);
        // printf("ynsr[0]=\n");
        // dh_mybig_monmult_64_modN(sh_ip+32*4,dc_ONE,sh_ip+32*4);
        // d_mybig_print(sh_ip+32*4);
    }


    dh_mybig_modadd_64_modN(ar+tx*4,xyz+8,sh_ip+tx*4);
    dh_mybig_monmult_64_modN(sh_ip+tx*4,sh_ip+(tx+64)*4,sh_ip+tx*4);
    //先把2_modN放在rh里
    dh_mybig_copy(rh,dc_mon_TWO_modN);    
       

    dh_mybig_modexp_ui32_modN(rh,tx,sh_ip+(tx+32)*4);
    //再把z2放在rh里
    dh_mybig_monmult_64_modN(xyz+8,xyz+8,rh);
    dh_mybig_monmult_64_modN(sh_ip+(tx+32)*4,rh,sh_ip+(tx+32)*4);
    dh_mybig_copy(z22nyn+tx*4,sh_ip+(tx+32)*4);
    dh_mybig_copy(z22nyn+(tx+32)*4,sh_ip+(tx+64)*4);

    dh_mybig_modadd_64_modN(sh_ip+tx*4,sh_ip+(tx+32)*4,sh_ip+tx*4);

    dh_mybig_monmult_64_modN(sh_ip+tx*4,sl+tx*4,sh_ip+tx*4);

    for(int i=16;i>0;i>>=1){
        if(tx<i){
            dh_mybig_modadd_64_modN(sh_ip+(tx)*4,sh_ip+(tx+i)*4,sh_ip+(tx)*4);
        }
        __syncthreads();
    }
    if(tx==0){
        dh_mybig_copy(rh,sh_ip);
        dh_mybig_modadd_64_modN(lh,rh,sh_ip+32*4);
        // printf("t1=\n");
        dh_mybig_monmult_64_modN(sh_ip+32*4,dc_ONE,sh_ip+32*4);
        // d_mybig_print(sh_ip+32*4);
        // dh_mybig_monmult_64_modN(sl,dc_ONE,sl);
        // d_mybig_print(sl);
        // printf("ynsr[0]=\n");
        // dh_mybig_monmult_64_modN(sh_ip+32*4,dc_ONE,sh_ip+32*4);
        // d_mybig_print(sh_ip+32*4);
    }

    //t2
    dh_mybig_monmult_64_modN(sh_ip+(tx+64)*4,sr+tx*4,sh_ip+(tx)*4);
    dh_mybig_monmult_64_modN(sh_ip+(tx)*4,sl+tx*4,sh_ip+(tx)*4);

    for(int i=16;i>0;i>>=1){
        if(tx<i){
            dh_mybig_modadd_64_modN(sh_ip+(tx)*4,sh_ip+(tx+i)*4,sh_ip+(tx)*4);
        }
        __syncthreads();
    }

    if(tx==0){
        dh_mybig_copy(sh_ip+33*4,sh_ip);
        // printf("t2=\n");
        dh_mybig_monmult_64_modN(sh_ip+33*4,dc_ONE,sh_ip+33*4);
        // d_mybig_print(sh_ip+33*4);
    }

    if(tx<2){
        Jpoint tmpx,tmpy;
        d_base_point_mul(&tmpx,sh_ip+(32+tx)*4);
        dh_point_mult_finalversion(H,tau12+tx*4,&tmpy);
        dh_ellipticAdd_JJ(&tmpx,&tmpy,&tmpx);
        dh_mybig_monmult_64(tmpx.x,dc_ONE,tmpx.x);
        dh_mybig_monmult_64(tmpx.y,dc_ONE,tmpx.y);
        dh_mybig_monmult_64(tmpx.z,dc_ONE,tmpx.z);
        dh_mybig_copy((T12+tx)->x,tmpx.x);
        dh_mybig_copy((T12+tx)->y,tmpx.y);
        dh_mybig_copy((T12+tx)->z,tmpx.z);
    }
    if(tx==0){
        Jpoint t1,t2;
        Jpoint2Apoint(T12,&t1);
        Jpoint2Apoint(T12+1,&t2);
        // printf("device T1=\n");
        // d_mybig_print(t1.x);
        // d_mybig_print(t1.y);
        // d_mybig_print(t1.z);
        // printf("device T2=\n");
        // d_mybig_print(t2.x);
        // d_mybig_print(t2.y);
        // d_mybig_print(t2.z);
        // printf("\n");
        // printf("\n");
        device_hashBP(&t1,&t2,lh,NULL);
        // printf("device HashBP x=\n");
        // d_mybig_print(lh);
        dh_mybig_monmult_64_modN(lh,dc_R2modN,xyz);
    }

}

__global__ void kernel_cal_vlr(UINT64 *tprime,UINT64 *VLR,UINT64 *taux,UINT64 *mu,UINT64 *xyz,UINT64 *al,UINT64 *ar,UINT64 *sl,UINT64 *sr,
                                UINT64 *z22nyn,UINT64 *gamma,UINT64 *alpha,UINT64 *rho,UINT64 *tau1,UINT64 *tau2,int N){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int idx = bx*blockDim.x+tx;
    if(tx>=N) return ;
    __shared__ UINT64 sh_ip[4*64];
    dh_mybig_modsub_64_modN(al+tx*4,xyz+8,sh_ip+tx*4);
    dh_mybig_monmult_64_modN(sl+tx*4,xyz,sh_ip+(tx+32)*4);
    dh_mybig_modadd_64_modN(sh_ip+tx*4,sh_ip+(tx+32)*4,sh_ip+tx*4);

    dh_mybig_monmult_64_modN(sr+tx*4,xyz,sh_ip+(tx+32)*4);
    dh_mybig_modadd_64_modN(sh_ip+(tx+32)*4,xyz+8,sh_ip+(tx+32)*4);
    dh_mybig_modadd_64_modN(sh_ip+(tx+32)*4,ar+tx*4,sh_ip+(tx+32)*4);

    dh_mybig_monmult_64_modN(sh_ip+(tx+32)*4,z22nyn+(tx+32)*4,sh_ip+(tx+32)*4);
    dh_mybig_modadd_64_modN(sh_ip+(tx+32)*4,z22nyn+tx*4,sh_ip+(tx+32)*4);

    dh_mybig_copy(VLR+tx*4,sh_ip+(tx)*4);
    dh_mybig_copy(VLR+(tx+32)*4,sh_ip+(tx+32)*4);

    dh_mybig_monmult_64_modN(sh_ip+(tx)*4,sh_ip+(tx+32)*4,sh_ip+(tx)*4);
    for(int i=16;i>0;i>>=1){
        if(tx<i){
            dh_mybig_modadd_64_modN(sh_ip+(tx)*4,sh_ip+(tx+i)*4,sh_ip+(tx)*4);
        }
        __syncthreads();
    }

    if(tx==0){
        
        dh_mybig_monmult_64_modN(sh_ip,dc_ONE,tprime);
        // printf("Tprime=\n");
        // d_mybig_print(tprime);

        dh_mybig_monmult_64_modN(xyz,xyz,sh_ip);
        dh_mybig_monmult_64_modN(tau2,dc_R2modN,sh_ip+4);
        dh_mybig_monmult_64_modN(sh_ip,sh_ip+4,sh_ip);

        dh_mybig_monmult_64_modN(xyz+8,xyz+8,sh_ip+8);


        dh_mybig_monmult_64_modN(tau1,dc_R2modN,sh_ip+4);
        dh_mybig_monmult_64_modN(sh_ip+4,xyz,sh_ip+4);

        dh_mybig_monmult_64_modN(gamma,dc_R2modN,sh_ip+12);
        dh_mybig_monmult_64_modN(sh_ip+8,sh_ip+12,sh_ip+8);

        dh_mybig_modadd_64_modN(sh_ip+4,sh_ip+8,sh_ip+4);
        dh_mybig_modadd_64_modN(sh_ip,sh_ip+4,sh_ip);
        dh_mybig_monmult_64_modN(sh_ip,dc_ONE,taux);

        // printf("Taux=\n");
        // d_mybig_print(taux);

        dh_mybig_monmult_64_modN(rho,dc_R2modN,sh_ip);
        dh_mybig_monmult_64_modN(alpha,dc_R2modN,sh_ip+4);
        dh_mybig_monmult_64_modN(sh_ip,xyz,sh_ip);
        dh_mybig_modadd_64_modN(sh_ip,sh_ip+4,sh_ip);
        dh_mybig_monmult_64_modN(sh_ip,dc_ONE,mu);

        // printf("Mu=\n");
        // d_mybig_print(mu);

        



    }



}

void gpu_cal_S(){
    int nB = 1;
    int nT = 64;
    kernel_cal_S<<<nB,nT>>>(&(d_prove->S),d_ranParams->rho,d_ranParams->SL,d_ranParams->SR,&(d_params->H),d_params->Gg,d_params->Gg,d_xyz,&(d_prove->A));
}
//寻找cuda sprintf，待定
void gpu_hashBP(Jpoint *A,Jpoint *B,UINT64 *x,UINT64 *y){
    int nB = 1;
    int nT = 32;
    kernel_hashBP<<<nB,nT>>>(A,B,x,y,1);
}

void gpu_cal_t12(){
    int nB = 1;
    int nT = 32;
    kernel_cal_t12<<<nB,nT>>>(&(d_prove->T1),&(d_params->H),d_xyz,d_aLR,d_aLR+4*32,d_ranParams->SL,d_ranParams->SR,d_ranParams->tau1,d_z22nyn);
}
void gpu_calvlvr(){
    int nB = 1;
    int nT = 32;
    kernel_cal_vlr<<<nB,nT>>>(d_prove->Tprime,d_VLR,d_prove->Taux,d_prove->Mu,d_xyz,d_aLR,d_aLR+32*4,d_ranParams->SL,d_ranParams->SR,d_z22nyn,
                                d_ranParams->gamma,d_ranParams->alpha,d_ranParams->rho,d_ranParams->tau1,d_ranParams->tau2 ,32);
}

int main(){
    hipSetDevice(0);
    UINT64 *al = h_aLR;
    UINT64 *ar = h_aLR+32*4;
    // UINT64 al[32*4]={0};
    // UINT64 ar[32*4]={0};
    int value = rand();
    compute_al_ar(value,al,ar,32);
    printf("value=%d\n",value);
    // printf("AL=\n");
    // for(int i=0;i<32;i++){
    //     h_mybig_print(al+4*i);
    // }
    // printf("AR=\n");
    // for(int i=0;i<32;i++){
    //     h_mybig_print(ar+4*i);
    // }
    printf("\n");
    
    // std::string s = "gyy hello world";
    JpointCpy(&h_params.G,h_Gx,h_Gy,h_Gz);
    mapToGroup(SEED,&h_params.H);
    printf("param.H=\n");
    h_print_pointJ(&h_params.H);
    h_params.N=32;
    // cout<<SEED
    for(int i=0;i<h_params.N;i++){
        
        char tmp[3];
        sprintf(tmp,"%u",i);
        mapToGroup(SEED+"h"+tmp,&h_params.Hh[i]);
        mapToGroup(SEED+"g"+tmp,&h_params.Gg[i]);
    }
  

 
    init_random_param();
    // printf("SL=\n");
    // for(int i=0;i<32;i++){
    //     h_mybig_print(&(h_ranParams.SL[i*4]));
    // }
    // printf("SR=\n");
    // for(int i=0;i<32;i++){
    //     h_mybig_print(&(h_ranParams.SR[i*4]));
    // }
    // printf("Gg=\n");
    // for(int i=0;i<32;i++){
    //     h_print_pointJ(&h_params.Gg[i]);
    // }
    // printf("Hh=\n");
    // for(int i=0;i<32;i++){
    //     h_print_pointJ(&(h_params.Hh[i]));
    // }
    
    printf("rho=\n");
    h_mybig_print(h_ranParams.rho);
    printf("gamma=\n");
    h_mybig_print(h_ranParams.gamma);
    printf("alpha=\n");
    h_mybig_print(h_ranParams.alpha);
    printf("tau1=\n");
    h_mybig_print(h_ranParams.tau1);
    printf("tau2=\n");
    h_mybig_print(h_ranParams.tau2);
    // return 0;
    // h_mybig_print(h_ranParams.alpha);
    // h_mybig_print(h_ranParams.rho);
    // h_mybig_print(h_ranParams.tau1);
    // h_mybig_print(h_ranParams.tau2);

    CUDA_SAFE_CALL(hipMalloc((void**)&d_params,sizeof(BPSetupParams)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_ranParams,sizeof(initParamRandom)));
    // CUDA_SAFE_CALL(hipMalloc((void**)d_V,sizeof(Jpoint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_prove,sizeof(BPProve)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_xyz,sizeof(UINT64)*4*3));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_iv,sizeof(InterVar)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_aLR,sizeof(UINT64)*4*64));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_VLR,sizeof(UINT64)*4*h_params.N*2));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_z22nyn,sizeof(UINT64)*4*h_params.N*2));

    Jpoint V,A,S;
    trans_to_mon();

    struct timeval s1,e1;
    gettimeofday(&s1,NULL);
    commitG1(&V,value,h_ranParams.gamma,&h_params.H);
    // printf("V=\n");
    // h_print_pointJ(&V);
    cal_A(al,ar,&A);
    // printf("A=\n");
    // h_print_pointJ(&A);
  
    

    CUDA_SAFE_CALL(hipMemcpy(d_params,&h_params,sizeof(BPSetupParams),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_ranParams,&h_ranParams,sizeof(initParamRandom),hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(&(d_prove->V),&V,sizeof(Jpoint),hipMemcpyHostToDevice));
    
    CUDA_SAFE_CALL(hipMemcpy(&(d_prove->A),&A,sizeof(Jpoint),hipMemcpyHostToDevice));

    gpu_cal_S();
    // cal_S_dev<<<1,64>>>(&(d_prove->S),d_ranParams->rho,d_ranParams->SL,d_ranParams->SR,&(d_params->H),d_params->Gg,d_params->Gg);
    // gpu_hashBP(&(d_prove->A),&(d_prove->S),d_xyz+4,d_xyz+8);
    CUDA_SAFE_CALL(hipMemcpy(&S,&(d_prove->S),sizeof(Jpoint),hipMemcpyDeviceToHost));
    
    // h_print_pointJ(&A);
    // Jpoint2Apoint(&A,&A);
    // printf("A=\n");
    // h_print_pointJ(&A);
    // Jpoint2Apoint(&S,&S);
    // printf("S=\n");
    // h_print_pointJ(&S);

    // HashBP_V2(&A,&S,h_xyz+4,h_xyz+8);
    // printf("y==\n");
    // h_mybig_print(h_xyz+4);
    // printf("z==\n");
    // h_mybig_print(h_xyz+8);
    
    trans_to_mon_N();
    CUDA_SAFE_CALL(hipMemcpy(d_aLR,h_aLR,sizeof(UINT64)*4*64,hipMemcpyHostToDevice));
    // CUDA_SAFE_CALL(hipMemcpy(d_xyz,h_xyz,sizeof(UINT64)*12,hipMemcpyHostToDevice));

    gpu_cal_t12();

    Jpoint h_T12[2];
    CUDA_SAFE_CALL(hipMemcpy(h_T12,&(d_prove->T1),sizeof(Jpoint)*2,hipMemcpyDeviceToHost));
    
    // printf("T1\n");
    // Jpoint2Apoint(h_T12,h_T12);
    // h_print_pointJ(h_T12);
    // printf("T2\n");
    // Jpoint2Apoint(h_T12+1,h_T12+1);
    // h_print_pointJ(h_T12+1);
    
    // HashBP_V2(h_T12,h_T12+1,h_xyz,NULL);
    // printf("x==\n");
    // h_mybig_print(h_xyz);


    // dh_mybig_monmult_64_modN(h_xyz,h_R2modN,h_xyz);
    // CUDA_SAFE_CALL(hipMemcpy(d_xyz,h_xyz,sizeof(UINT64)*12,hipMemcpyHostToDevice));
    gpu_calvlvr();
    gettimeofday(&e1,NULL);

    long long time_use;
    time_use=(e1.tv_sec-s1.tv_sec)*1000000+(e1.tv_usec-s1.tv_usec);//微秒
    printf("time_use is %llu\n",time_use);

    CUDA_SAFE_CALL(hipMemcpy(d_xyz,h_xyz,sizeof(UINT64)*12,hipMemcpyHostToDevice));
    // h_print_pointJ(&S);
    printf("hello world\n");
    return 0;
    // h_mybig_print(ranParams.alpha);
    
}